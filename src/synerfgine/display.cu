#include "hip/hip_runtime.h"
#include <memory>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/marching_cubes.h>

#include <tiny-cuda-nn/common.h>

#include <synerfgine/camera.h>
#include <synerfgine/display.h>
#include <synerfgine/file.h>
#include <synerfgine/virtual_object.h>

namespace sng {

bool Display::m_is_init = false;

template <typename T>
void resize_transfer_bind(hipStream_t stream, T* gpu_buffer, std::vector<T>& cpu_texture, 
		std::shared_ptr<GLTexture> gl_texture_obj, GLint internal_fmt, GLenum fmt) {
	auto resolution = gl_texture_obj->resolution();
	auto n_elements = resolution.x * resolution.y;
	if (n_elements != cpu_texture.size()) cpu_texture.resize(n_elements);
	CUDA_CHECK_THROW(hipMemcpyAsync(cpu_texture.data(), gpu_buffer, n_elements * sizeof(T), hipMemcpyDeviceToHost, stream));
	glBindTexture(GL_TEXTURE_2D, gl_texture_obj->texture());
	glTexImage2D(GL_TEXTURE_2D, 0, internal_fmt, resolution.x, resolution.y, 0, fmt, GL_FLOAT, cpu_texture.data());
}

void glfw_error_callback(int error, const char* description) {
	tlog::error() << "GLFW error #" << error << ": " << description;
}

GLFWwindow* Display::init_window(int resw, int resh, bool hidden) {
    m_window_res = {resw, resh};

#ifdef NGP_GUI
	m_glfw_window = renderer.create_glfw_window(m_window_res);
    ui.init_imgui(m_glfw_window);
#endif
	init_buffers();
	Display::m_is_init = true;
	return m_glfw_window;
}

GLFWwindow* Renderer::create_glfw_window(const ivec2& m_window_res) {
	this->m_window_res = m_window_res;
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit()) {
        throw std::runtime_error{"GLFW could not be initialized."};
    }
    
    std::string title = "Synthetic Object NeRF Engine";
    m_glfw_window = glfwCreateWindow(m_window_res.x, m_window_res.y, title.c_str(), NULL, NULL);
    if (m_glfw_window == NULL) {
        throw std::runtime_error{"GLFW window could not be created."};
    }
    glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
    if (gl3wInit()) {
        throw std::runtime_error{"GL3W could not be initialized."};
    }
#else
    glewExperimental = 1;
    if (glewInit()) {
        throw std::runtime_error{"GLEW could not be initialized."};
    }
#endif
    glfwSwapInterval(0); // Disable vsync

    GLint gl_version_minor, gl_version_major;
    glGetIntegerv(GL_MINOR_VERSION, &gl_version_minor);
    glGetIntegerv(GL_MAJOR_VERSION, &gl_version_major);

    if (gl_version_major < 3 || (gl_version_major == 3 && gl_version_minor < 1)) {
        throw std::runtime_error{fmt::format("Unsupported OpenGL version {}.{}. instant-ngp requires at least OpenGL 3.1", gl_version_major, gl_version_minor)};
    }

    tlog::success() << "Initialized OpenGL version " << glGetString(GL_VERSION);

	// TODO: Fix window size crashing issues
	// glfwSetWindowSizeCallback(m_glfw_window, );

	init_opengl_shaders();

	return m_glfw_window;
}

void Ui::init_imgui(GLFWwindow* m_glfw_window) {
	this->m_glfw_window = m_glfw_window;

	float xscale, yscale;
	glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;

	// By default, imgui places its configuration (state of the GUI -- size of windows,
	// which regions are expanded, etc.) in ./imgui.ini relative to the working directory.
	// Instead, we would like to place imgui.ini in the directory that instant-ngp project
	// resides in.
	static std::string ini_filename;
	ini_filename = (Utils::get_root_dir()/"imgui.ini").string();
	io.IniFilename = ini_filename.c_str();

	// New ImGui event handling seems to make camera controls laggy if input trickling is true.
	// So disable input trickling.
	io.ConfigInputTrickleEventQueue = false;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 140");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);
}

void Ui::imgui(SyntheticWorld& syn_world, NerfWorld& nerf_world, float frame_time) {
	syn_world.imgui(frame_time);
	nerf_world.imgui(frame_time);
}

void Renderer::init_opengl_shaders() {
	static const char* shader_vert = R"glsl(#version 140
		out vec2 UVs;
		void main() {
			UVs = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);
			gl_Position = vec4(UVs * 2.0 - 1.0, 0.0, 1.0);
		})glsl";

	static const char* shader_frag = R"glsl(#version 140
		in vec2 UVs;
		out vec4 frag_color;
		uniform sampler2D syn_rgba;
		uniform sampler2D syn_depth;
		uniform sampler2D nerf_rgba;
		uniform sampler2D nerf_depth;

		struct FoveationWarp {
			float al, bl, cl;
			float am, bm;
			float ar, br, cr;
			float switch_left, switch_right;
			float inv_switch_left, inv_switch_right;
		};

		uniform FoveationWarp warp_x;
		uniform FoveationWarp warp_y;

		float unwarp(in FoveationWarp warp, float y) {
			y = clamp(y, 0.0, 1.0);
			if (y < warp.inv_switch_left) {
				return (sqrt(-4.0 * warp.al * warp.cl + 4.0 * warp.al * y + warp.bl * warp.bl) - warp.bl) / (2.0 * warp.al);
			} else if (y > warp.inv_switch_right) {
				return (sqrt(-4.0 * warp.ar * warp.cr + 4.0 * warp.ar * y + warp.br * warp.br) - warp.br) / (2.0 * warp.ar);
			} else {
				return (y - warp.bm) / warp.am;
			}
		}

		vec2 unwarp(in vec2 pos) {
			return vec2(unwarp(warp_x, pos.x), unwarp(warp_y, pos.y));
		}

		const float max_nd = 16384.0;

		void main() {
			vec2 tex_coords = UVs;
			tex_coords.y = 1.0 - tex_coords.y;
			tex_coords = unwarp(tex_coords);
			vec4 syn = texture(syn_rgba, tex_coords.xy);
			float sd = texture(syn_depth, tex_coords.xy).r;
			vec4 nerf = texture(nerf_rgba, tex_coords.xy);
			float nd = texture(nerf_depth, tex_coords.xy).r;

			// DEBUG
			// frag_color = vec4(0.0, 0.0, 0.0, 1.0);
			// if (nd < max_nd) {
			// 	frag_color += vec4(0.0, 0.0, nd, 1.0);
			// 	gl_FragDepth = nd;
			// }
			// if (sd < nd) {
			// 	frag_color += vec4(sd, 0.0, 0.0, 1.0);
			// 	gl_FragDepth = nd;
			// }
			// frag_color.rgb /= 5.0;

			if (sd < nd) {
				frag_color = vec4(syn.rgb, 1.0);
				gl_FragDepth = sd;
			} else if (nd < max_nd) {
				frag_color = vec4(nerf.rgb, 1.0);
				gl_FragDepth = nd;
			}
		})glsl";

	GLuint vert = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vert, 1, &shader_vert, NULL);
	glCompileShader(vert);
	ngp::check_shader(vert, "Blit vertex shader", false);

	GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(frag, 1, &shader_frag, NULL);
	glCompileShader(frag);
	ngp::check_shader(frag, "Blit fragment shader", false);

	m_blit_program = glCreateProgram();
	glAttachShader(m_blit_program, vert);
	glAttachShader(m_blit_program, frag);
	glLinkProgram(m_blit_program);
	ngp::check_shader(m_blit_program, "Blit shader program", true);

	glDeleteShader(vert);
	glDeleteShader(frag);

	glGenVertexArrays(1, &m_blit_vao);
}

void Display::init_buffers() {
	// Make sure there's at least one usable render texture
	m_rgba_render_textures = std::make_shared<GLTexture>();
	m_depth_render_textures = std::make_shared<GLTexture>();

	m_render_buffer = std::make_shared<CudaRenderBuffer>(m_rgba_render_textures, m_depth_render_textures);
	m_render_buffer->resize(m_view_res);
	m_render_buffer->disable_dlss();
}

bool Display::begin_frame(CudaDevice& device, bool& is_dirty) {
	if (glfwWindowShouldClose(m_glfw_window) || ImGui::IsKeyPressed(GLFW_KEY_ESCAPE) || ImGui::IsKeyPressed(GLFW_KEY_Q)) {
		destroy();
		return false;
	}

	glfwPollEvents();
	glfwGetFramebufferSize(m_glfw_window, &m_window_res.x, &m_window_res.y);
	if (is_dirty) {
		device.device_guard();
		m_render_buffer->resize(m_window_res);
		is_dirty = false;
	}
	return ui.begin_frame(m_window_res) && renderer.begin_frame(m_window_res);
}

bool Ui::begin_frame(const [[maybe_unused]] ivec2& window_res) {
	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
	ImGuizmo::BeginFrame();

	return true;
}

bool Renderer::begin_frame(const ivec2& window_res) {
	m_window_res = window_res;
	return true;
}

void Display::end_frame() {
	renderer.end_frame();
	ui.end_frame();
	auto time_now = std::chrono::system_clock::now();
	m_last_frame_time = (float)std::chrono::duration_cast<std::chrono::milliseconds>(time_now - m_last_timestamp).count();
	m_last_timestamp = time_now;
 }

void Ui::end_frame() {
	ImGui::EndFrame();
}

void Renderer::end_frame() {
}

bool Display::present(CudaDevice& device, SyntheticWorld& syn_world, NerfWorld& nerf_world) {
	ui.imgui(syn_world, nerf_world, m_last_frame_time);
	m_render_buffer->set_hidden_area_mask(nullptr);
	return renderer.present({1,1}, syn_world.m_rgba_render_textures, syn_world.m_depth_render_textures, syn_world.m_render_buffer_view, 
		nerf_world.m_rgba_render_textures, nerf_world.m_depth_render_textures, nerf_world.m_render_buffer_view, device);
}

bool Renderer::present(const ivec2& m_n_views, std::shared_ptr<ngp::GLTexture> syn_rgba, std::shared_ptr<ngp::GLTexture> syn_depth, const CudaRenderBufferView& syn_view,
		std::shared_ptr<ngp::GLTexture> nerf_rgba, std::shared_ptr<ngp::GLTexture> nerf_depth, const CudaRenderBufferView& nerf_view, CudaDevice& device) { 
	// https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#opengl-interoperability
	if (!m_glfw_window) {
		throw std::runtime_error{"Window must be initialized to be presented."};
	}

	// Make sure all the cuda code finished its business here
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	glfwMakeContextCurrent(m_glfw_window);
	int display_w, display_h;
	glfwGetFramebufferSize(m_glfw_window, &display_w, &display_h);

	// IMAGE RENDER
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_BLEND);
	glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
	glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);

	ivec2 extent = {(int)((float)display_w / m_n_views.x), (int)((float)display_h / m_n_views.y)};

    {
		resize_transfer_bind(device.stream(), syn_view.frame_buffer, m_cpu_frame_buffer_syn, syn_rgba, GL_RGBA32F, GL_RGBA);
		resize_transfer_bind(device.stream(), syn_view.depth_buffer, m_cpu_depth_buffer_syn, syn_depth, GL_R32F, GL_RED);
		resize_transfer_bind(device.stream(), nerf_view.frame_buffer, m_cpu_frame_buffer_nerf, nerf_rgba, GL_RGBA32F, GL_RGBA);
		resize_transfer_bind(device.stream(), nerf_view.depth_buffer, m_cpu_depth_buffer_nerf, nerf_depth, GL_R32F, GL_RED);
    }

	ivec2 top_left{0, display_h - extent.y};
	blit_texture(ngp::Foveation{}, syn_rgba->texture(), nerf_rgba->texture(), GL_LINEAR, syn_depth->texture(), 
		nerf_depth->texture(), 0, top_left, extent);
	glFinish();

	// UI DRAWING
	glViewport(0, 0, display_w, display_h);

	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	// Visualizations are only meaningful when rendering a single view
	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();

	return true;
}

bool Renderer::present(const ivec2& m_n_views, std::shared_ptr<ngp::GLTexture> rgba, std::shared_ptr<ngp::GLTexture> depth, CudaDevice& device) { 
	if (!m_glfw_window) {
		throw std::runtime_error{"Window must be initialized to be presented."};
	}

	// Make sure all the cuda code finished its business here
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	glfwMakeContextCurrent(m_glfw_window);
	int display_w, display_h;
	glfwGetFramebufferSize(m_glfw_window, &display_w, &display_h);

	// IMAGE RENDER
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.0f, 0.0f, 0.5f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_BLEND);
	glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
	glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);

	ivec2 extent = {(int)((float)display_w / m_n_views.x), (int)((float)display_h / m_n_views.y)};

    {
		auto n_elements = m_window_res.x * m_window_res.y;
		if (n_elements != m_cpu_frame_buffer_syn.size()) m_cpu_frame_buffer_syn.resize(n_elements);
		if (n_elements != m_cpu_depth_buffer_syn.size()) m_cpu_depth_buffer_syn.resize(n_elements);
        CUDA_CHECK_THROW(hipMemcpyAsync(m_cpu_frame_buffer_syn.data(), device.render_buffer_view().frame_buffer, n_elements * sizeof(vec4), hipMemcpyDeviceToHost, device.stream()));
        CUDA_CHECK_THROW(hipMemcpyAsync(m_cpu_depth_buffer_syn.data(), device.render_buffer_view().depth_buffer, n_elements * sizeof(float), hipMemcpyDeviceToHost, device.stream()));
		auto rgba_size = rgba->resolution();
		auto depth_size = depth->resolution();
		if (rgba_size != m_window_res) {
			rgba->resize(m_window_res, 4);
		}
		if (depth_size != m_window_res) {
			depth->resize(m_window_res, 1);
		}
		CUDA_CHECK_THROW(hipStreamSynchronize(device.stream()));

		glBindTexture(GL_TEXTURE_2D, rgba->texture());
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, rgba_size.x, rgba_size.y, 0, GL_RGBA, GL_FLOAT, m_cpu_frame_buffer_syn.data());
		glBindTexture(GL_TEXTURE_2D, depth->texture());
		glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, depth_size.x, depth_size.y, 0, GL_RED, GL_FLOAT, m_cpu_depth_buffer_syn.data());
    }

	ivec2 top_left{0, display_h - extent.y};
	// blit_texture(m_foveated_rendering_visualize ? Foveation{} : view.foveation, m_rgba_render_textures.at(i)->texture(), m_foveated_rendering ? GL_LINEAR : GL_NEAREST, m_depth_render_textures.at(i)->texture(), 0, top_left, extent);
	// rgba->blit_from_cuda_mapping()
	blit_texture(ngp::Foveation{}, rgba->texture(), rgba->texture(), GL_LINEAR, depth->texture(), 
		depth->texture(), 0, top_left, extent);
	glFinish();

	// UI DRAWING
	glViewport(0, 0, display_w, display_h);

	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	// Visualizations are only meaningful when rendering a single view
	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();

	return true;
}

void Renderer::blit_texture(const ngp::Foveation& foveation, GLint syn_rgba, GLint nerf_rgba, GLint rgba_filter_mode, 
	GLint syn_depth, GLint nerf_depth, GLint framebuffer, const ivec2& offset, const ivec2& resolution) {
	if (m_blit_program == 0) {
		return;
	}

	// Blit image to OpenXR swapchain.
	// Note that the OpenXR swapchain is 8bit while the rendering is in a float texture.
	// As some XR runtimes do not support float swapchains, we can't render into it directly.

	bool tex = glIsEnabled(GL_TEXTURE_2D);
	bool depth = glIsEnabled(GL_DEPTH_TEST);
	bool cull = glIsEnabled(GL_CULL_FACE);

	if (!tex) 
		glEnable(GL_TEXTURE_2D);
	if (!depth) 
		glEnable(GL_DEPTH_TEST);
	if (cull) 
		glDisable(GL_CULL_FACE);

	glDepthFunc(GL_ALWAYS);
	glDepthMask(GL_TRUE);

	glBindVertexArray(m_blit_vao);
	glUseProgram(m_blit_program);
	auto syn_rgba_uniform = glGetUniformLocation(m_blit_program, "syn_rgba");
	auto syn_depth_uniform = glGetUniformLocation(m_blit_program, "syn_depth");
	glUniform1i(syn_rgba_uniform, 0);
	glUniform1i(syn_depth_uniform, 1);
	auto nerf_rgba_uniform = glGetUniformLocation(m_blit_program, "nerf_rgba");
	auto nerf_depth_uniform = glGetUniformLocation(m_blit_program, "nerf_depth");
	glUniform1i(nerf_rgba_uniform, 2);
	glUniform1i(nerf_depth_uniform, 3);

	auto bind_warp = [&](const ngp::FoveationPiecewiseQuadratic& warp, const std::string& uniform_name) {
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".al").c_str()), warp.al);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bl").c_str()), warp.bl);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cl").c_str()), warp.cl);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".am").c_str()), warp.am);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bm").c_str()), warp.bm);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".ar").c_str()), warp.ar);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".br").c_str()), warp.br);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cr").c_str()), warp.cr);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_left").c_str()), warp.switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_right").c_str()), warp.switch_right);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_left").c_str()), warp.inv_switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_right").c_str()), warp.inv_switch_right);
	};

	bind_warp(foveation.warp_x, "warp_x");
	bind_warp(foveation.warp_y, "warp_y");

	glActiveTexture(GL_TEXTURE3);
	glBindTexture(GL_TEXTURE_2D, nerf_depth);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, nerf_rgba);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, syn_depth);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, syn_rgba);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, rgba_filter_mode);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, rgba_filter_mode);

	glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
	glViewport(offset.x, offset.y, resolution.x, resolution.y);

	glDrawArrays(GL_TRIANGLES, 0, 3);

	glBindVertexArray(0);
	glUseProgram(0);

	glDepthFunc(GL_LESS);

	// restore old state
	if (!tex) glDisable(GL_TEXTURE_2D);
	if (!depth) glDisable(GL_DEPTH_TEST);
	if (cull) glEnable(GL_CULL_FACE);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

void Display::destroy() {
#ifndef NGP_GUI
	throw std::runtime_error{"destroy_window failed: NGP was built without GUI support"};
#else
	if (!Display::m_is_init) {
		return;
	}

	m_render_buffer = nullptr;

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(m_glfw_window);
	glfwTerminate();

	m_glfw_window = nullptr;
	m_is_init = false;
#endif //NGP_GUI
}

}