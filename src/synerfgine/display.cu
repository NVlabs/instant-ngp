#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/marching_cubes.h>

#include <synerfgine/display.h>

namespace sng {

void glfw_error_callback(int error, const char* description) {
	tlog::error() << "GLFW error #" << error << ": " << description;
}

void Display::init_window(int resw, int resh, bool hidden) {
    m_window_res = {resw, resh};

#ifdef NGP_GUI
	renderer.init_glfw(m_glfw_window);
    ui.init(m_glfw_window);
#endif
	renderer.init_buffers();
}

void Renderer::init_glfw(GLFWwindow* m_glfw_window, const ivec2& m_window_res) {
	this->m_window_res = m_window_res;
    glfwSetErrorCallback(glfw_error_callback);
    if (!glfwInit()) {
        throw std::runtime_error{"GLFW could not be initialized."};
    }
    
    glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
    std::string title = "Synthetic Object NeRF Engine";
    m_glfw_window = glfwCreateWindow(m_window_res.x, m_window_res.y, title.c_str(), NULL, NULL);
    if (m_glfw_window == NULL) {
        throw std::runtime_error{"GLFW window could not be created."};
    }
    glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
    if (gl3wInit()) {
        throw std::runtime_error{"GL3W could not be initialized."};
    }
#else
    glewExperimental = 1;
    if (glewInit()) {
        throw std::runtime_error{"GLEW could not be initialized."};
    }
#endif
    glfwSwapInterval(0); // Disable vsync

    GLint gl_version_minor, gl_version_major;
    glGetIntegerv(GL_MINOR_VERSION, &gl_version_minor);
    glGetIntegerv(GL_MAJOR_VERSION, &gl_version_major);

    if (gl_version_major < 3 || (gl_version_major == 3 && gl_version_minor < 1)) {
        throw std::runtime_error{fmt::format("Unsupported OpenGL version {}.{}. instant-ngp requires at least OpenGL 3.1", gl_version_major, gl_version_minor)};
    }

    tlog::success() << "Initialized OpenGL version " << glGetString(GL_VERSION);
}

void Ui::init_imgui(const GLFWwindow* m_glfw_window) {
	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;

	// By default, imgui places its configuration (state of the GUI -- size of windows,
	// which regions are expanded, etc.) in ./imgui.ini relative to the working directory.
	// Instead, we would like to place imgui.ini in the directory that instant-ngp project
	// resides in.
	static std::string ini_filename;
	ini_filename = (root_dir()/"imgui.ini").str();
	io.IniFilename = ini_filename.c_str();

	// New ImGui event handling seems to make camera controls laggy if input trickling is true.
	// So disable input trickling.
	io.ConfigInputTrickleEventQueue = false;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 140");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);

	this->m_glfw_window = m_glfw_window;
}

void Renderer::init_opengl_shaders() {
	static const char* shader_vert = R"glsl(#version 140
		out vec2 UVs;
		void main() {
			UVs = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);
			gl_Position = vec4(UVs * 2.0 - 1.0, 0.0, 1.0);
		})glsl";

	static const char* shader_frag = R"glsl(#version 140
		in vec2 UVs;
		out vec4 frag_color;
		uniform sampler2D rgba_texture;
		uniform sampler2D depth_texture;

		struct FoveationWarp {
			float al, bl, cl;
			float am, bm;
			float ar, br, cr;
			float switch_left, switch_right;
			float inv_switch_left, inv_switch_right;
		};

		uniform FoveationWarp warp_x;
		uniform FoveationWarp warp_y;

		float unwarp(in FoveationWarp warp, float y) {
			y = clamp(y, 0.0, 1.0);
			if (y < warp.inv_switch_left) {
				return (sqrt(-4.0 * warp.al * warp.cl + 4.0 * warp.al * y + warp.bl * warp.bl) - warp.bl) / (2.0 * warp.al);
			} else if (y > warp.inv_switch_right) {
				return (sqrt(-4.0 * warp.ar * warp.cr + 4.0 * warp.ar * y + warp.br * warp.br) - warp.br) / (2.0 * warp.ar);
			} else {
				return (y - warp.bm) / warp.am;
			}
		}

		vec2 unwarp(in vec2 pos) {
			return vec2(unwarp(warp_x, pos.x), unwarp(warp_y, pos.y));
		}

		void main() {
			vec2 tex_coords = UVs;
			tex_coords.y = 1.0 - tex_coords.y;
			tex_coords = unwarp(tex_coords);
			frag_color = texture(rgba_texture, tex_coords.xy);
			//Uncomment the following line of code to visualize debug the depth buffer for debugging.
			// frag_color = vec4(vec3(texture(depth_texture, tex_coords.xy).r), 1.0);
			gl_FragDepth = texture(depth_texture, tex_coords.xy).r;
		})glsl";

	GLuint vert = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vert, 1, &shader_vert, NULL);
	glCompileShader(vert);
	ngp::check_shader(vert, "Blit vertex shader", false);

	GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(frag, 1, &shader_frag, NULL);
	glCompileShader(frag);
	ngp::check_shader(frag, "Blit fragment shader", false);

	m_blit_program = glCreateProgram();
	glAttachShader(m_blit_program, vert);
	glAttachShader(m_blit_program, frag);
	glLinkProgram(m_blit_program);
	ngp::check_shader(m_blit_program, "Blit shader program", true);

	glDeleteShader(vert);
	glDeleteShader(frag);

	glGenVertexArrays(1, &m_blit_vao);
}

void Renderer::init_buffers() {

}

}