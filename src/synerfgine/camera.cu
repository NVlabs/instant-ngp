#include "hip/hip_runtime.h"
#include <synerfgine/camera.h>

namespace sng {

__global__ void g_set_rays(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
 const float focal_length, const mat4x3* cam_mat, vec3* __restrict__ positions, vec3* __restrict__ directions);
Camera::Camera(const vec3& eye, const vec3& at, const vec3& up) : 
    m_eye(eye), 
    m_dir(at - eye), 
    m_up(up), m_scale(length(m_dir)) {
    m_dir = normalize(m_dir);
    g_camera.allocate_memory(sizeof(mat4x3));
    g_camera.copy_from_host({m_camera});
    translate_camera(m_eye, {}, true);
}

Camera::~Camera() {
    g_camera.check_guards();
    g_camera.free_memory();
    g_positions.check_guards();
    g_positions.free_memory();
    g_directions.check_guards();
    g_directions.free_memory();
}

void Camera::translate_camera(const vec3& rel, const mat3& rot, bool allow_up_down) {
    vec3 movement = rot * rel;
    if (!allow_up_down) {
        movement -= dot(movement, m_up) * m_up;
    }

    m_eye += movement;
    m_camera[3] += movement;
}

vec3 Camera::look_at() const {
    return m_eye + m_dir * m_scale;
}

void Camera::set_look_at(const vec3& pos) {
    m_dir = normalize(pos - m_eye);
    m_camera[3] += pos - look_at();
}

void Camera::set_scale(float scale) {
    auto prev_look_at = look_at();
    m_camera[3] = (m_eye - prev_look_at) * (scale / m_scale) + prev_look_at;
    m_scale = scale;
}

void Camera::set_view_dir(const vec3& dir) {
    m_camera[0] = normalize(cross(dir, m_up));
    m_camera[1] = normalize(cross(dir, m_camera[0]));
    m_camera[2] = normalize(dir);
    m_dir = dir;
}

void Camera::set_resolution(const ivec2& res) {
    if (res == m_resolution) return;
    size_t n_elements = res.x * res.y;
    g_positions.check_guards();
    g_positions.allocate_memory(n_elements * sizeof(vec3));
    g_directions.check_guards();
    g_directions.allocate_memory(n_elements * sizeof(vec3));
    m_resolution = res;
}

void Camera::generate_rays(CudaDevice& device) {
    g_camera.copy_from_host({m_camera});
    size_t n_elements = m_resolution.x * m_resolution.y;
    linear_kernel(g_set_rays, 0, device.stream(), n_elements, 
        m_resolution.x, 
        m_resolution.y,
        m_scale,
        g_camera.data(),
        g_positions.data(),
        g_directions.data()
    );
}

__global__ void g_set_rays(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
    const float focal_length, const mat4x3* p_camera, vec3* __restrict__ positions, vec3* __restrict__ directions) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
    float x = (float)(i % width) / (float)width - 0.5;
    float y = (float)(i / height) / (float)height - 0.5;
    positions[i] = (*p_camera)[3];
    vec4 dir = vec4(x, y, focal_length, 1.0);
    directions[i] = (*p_camera) * dir;
}

}