#include "hip/hip_runtime.h"
#include <synerfgine/syn_world.h>

#include <tiny-cuda-nn/common.h>
#include <filesystem>

namespace sng {

namespace fs = std::filesystem;
using namespace tcnn;
using ngp::GLTexture;

static bool is_first = true;

__global__ void debug_paint(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
    vec4* __restrict__ rgba, float* __restrict__ depth);

__global__ void gpu_draw_object(const uint32_t n_elements, const uint32_t width, const uint32_t height, const uint32_t tri_count,
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions, const Triangle* __restrict__ triangles, const Light sun,
    vec4* __restrict__ rgba, float* __restrict__ depth);

__global__ void debug_draw_rays(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions, 
    vec4* __restrict__ rgba, float* __restrict__ depth);

__global__ void debug_triangle_vertices(const uint32_t n_elements, const Triangle* __restrict__ triangles,
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions);

bool SyntheticWorld::handle(CudaDevice& device, const ivec2& resolution) {
    auto stream = device.stream();
    device.render_buffer_view().clear(stream);
    m_resolution = resolution;

    auto& cam = m_camera;
    cam.set_resolution(m_resolution);
    cam.generate_rays_async(device);
    CUDA_CHECK_THROW(hipStreamSynchronize(stream));
    for (auto& vo_kv : m_objects) {
        auto& vo = vo_kv.second;
        // if (is_first) {
        //     const std::string& name = vo_kv.first;
        //     uint32_t tri_count = static_cast<uint32_t>(m_objects.at(name).cpu_triangles().size());
        //     hipStream_t one_timer;
        //     CUDA_CHECK_THROW(hipStreamCreate(&one_timer));
        //     linear_kernel(debug_triangle_vertices, 0, one_timer, tri_count, 
        //         m_objects.at(name).gpu_triangles(), cam.gpu_positions(), cam.gpu_directions());
        //     CUDA_CHECK_THROW(hipStreamSynchronize(one_timer));
        //     is_first = false;
        // }
        draw_object_async(device, vo);
        CUDA_CHECK_THROW(hipStreamSynchronize(stream));
        {
            const std::string& name = vo_kv.first;
            uint32_t tri_count = static_cast<uint32_t>(m_objects.at(name).cpu_triangles().size());
            auto n_elements = m_resolution.x * m_resolution.y;
            // linear_kernel(debug_rt, 0, stream, n_elements,
            //     m_resolution.x, 
            //     m_resolution.y, 
            //     cam.m_world_to_cam[3],
            //     mat4::identity(),
            //     cam.m_world_to_cam,
            //     tri_count,
            //     m_objects.at(name).gpu_triangles(),
            //     device.render_buffer_view().frame_buffer, 
            //     device.render_buffer_view().depth_buffer);
        }
    }
    // {
    //     auto n_elements = m_resolution.x * m_resolution.y;
    //     linear_kernel(debug_draw_rays, 0, stream, n_elements,
    //         m_resolution.x, 
    //         m_resolution.y, 
    //         cam.gpu_positions(),
    //         cam.gpu_directions(),
    //         device.render_buffer_view().frame_buffer, 
    //         device.render_buffer_view().depth_buffer);
    //     CUDA_CHECK_THROW(hipStreamSynchronize(stream));
    // }
    return true;
}

void SyntheticWorld::create_object(const std::string& filename) {
    size_t k = 0;
    fs::path fp = fs::path(filename.c_str());
    std::string name = fp.filename().string();
    while (m_objects.count(name)) {
        ++k;
        name = fp.filename().string() + " " + std::to_string(k);
    }
    m_objects.insert({name, load_virtual_obj(filename.c_str(), name)});
}

void SyntheticWorld::draw_object_async(CudaDevice& device, VirtualObject& virtual_object) {
    auto& cam = m_camera;
    auto stream = device.stream();
    auto n_elements = m_resolution.x * m_resolution.y;
    uint32_t tri_count = static_cast<uint32_t>(virtual_object.cpu_triangles().size());
    linear_kernel(gpu_draw_object, 0, stream, n_elements,
        m_resolution.x, 
        m_resolution.y, 
        tri_count,
        cam.gpu_positions(),
        cam.gpu_directions(),
        virtual_object.gpu_triangles(),
        cam.sun(),
        device.render_buffer_view().frame_buffer, 
        device.render_buffer_view().depth_buffer);
    CUDA_CHECK_THROW(hipStreamSynchronize(stream));
}

__global__ void gpu_draw_object(const uint32_t n_elements, const uint32_t width, const uint32_t height, const uint32_t tri_count,
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions, const Triangle* __restrict__ triangles, const Light sun,
    vec4* __restrict__ rgba, float* __restrict__ depth) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
    rgba[i] = vec4(vec3(0.0), 1.0);
    depth[i] = ngp::MAX_RT_DIST;
    vec3 rd = ray_directions[i];
    vec3 ro = ray_origins[i];
    float dt = ngp::MAX_RT_DIST;
    vec3 normal;
    for (size_t k = 0; k < tri_count; ++k) {
        float t = triangles[k].ray_intersect(ro, rd);
        if (t < dt && t > ngp::MIN_RT_DIST) {
            dt = t;
            normal = triangles[k].normal();
        }
        // if (i == n_elements / 2) {
        //     printf("ro: [%f, %f, %f]; rd: [%f, %f, %f]; t: [%f]\n", ro.r, ro.b, ro.g, rd.r, rd.b, rd.g, t);
        //     printf("TRI: [%f, %f, %f], [%f, %f, %f], [%f, %f, %f]\n", 
        //         triangles[k].a.r, triangles[k].a.b, triangles[k].a.g,
        //         triangles[k].b.r, triangles[k].b.b, triangles[k].b.g,
        //         triangles[k].c.r, triangles[k].c.b, triangles[k].c.g);
        // }
    }

    // depth[i] = max(10.0 - dt, 0.0);
    if (dt < ngp::MAX_RT_DIST) {
        ro += rd * dt;
        vec3 to_sun = normalize(sun.pos - ro);
        // FOR DIFFUSE ONLY, NO AMBIENT / SPEC
        float ndotv = dot(normal, to_sun);
        rgba[i] = vec4(ndotv * vec3(1.0, 0.2, 0.0), 1.0);
    } else {
        rgba[i] = vec4(vec3(0.0), 1.0);
    }
}


__global__ void debug_draw_rays(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions, 
    vec4* __restrict__ rgba, float* __restrict__ depth) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	// if (i == 0) { printf("DIR: %f %f %f\n", ray_directions[i].x, ray_directions[i].y, ray_directions[i].z); }
    rgba[i] = vec4(abs(ray_directions[i]), 1.0);
	// if (i % 100000 == 0) { printf("COL %i: %f %f %f %f\n", i, rgba[i].x, rgba[i].y, rgba[i].z, rgba[i].w); }
}

__global__ void debug_paint(const uint32_t n_elements, const uint32_t width, const uint32_t height, 
    vec4* __restrict__ rgba, float* __restrict__ depth) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
    float x = (float)(i % width) / (float)width;
    float y = (float)(i / height) / (float)height;
    rgba[i] = vec4(x, y, 0.0, 1.0);
    depth[i] = 0.5f;
}

__global__ void debug_triangle_vertices(const uint32_t n_elements, const Triangle* __restrict__ triangles, 
    vec3* __restrict__ ray_origins, vec3* __restrict__ ray_directions) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
    const Triangle* tri = &triangles[i];
    printf("%i: [%f %f %f], [%f %f %f], [%f %f %f]\n", i, 
        tri->a.r, tri->a.g, tri->a.b,
        tri->b.r, tri->b.g, tri->b.b,
        tri->c.r, tri->c.g, tri->c.b);
    // printf("%i: pos [%f %f %f], dir [%f %f %f]\n", i, 
    //     ray_origins[i].r, ray_origins[i].g, ray_origins[i].b, 
    //     ray_directions[i].r, ray_directions[i].g, ray_directions[i].b
    // );
}

}