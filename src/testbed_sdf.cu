#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed_sdf.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/discrete_distribution.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/random_val.cuh> // helpers to generate random values, directions
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyobj_loader_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/trainer.h>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

static constexpr uint32_t MARCH_ITER = 10000;

Testbed::NetworkDims Testbed::network_dims_sdf() const {
	NetworkDims dims;
	dims.n_input = 3;
	dims.n_output = 1;
	dims.n_pos = 3;
	return dims;
}

__device__ inline float square(float x) { return x * x; }
__device__ inline float mix(float a, float b, float t) { return a + (b - a) * t; }
__device__ inline Vector3f mix(const Vector3f &a, const Vector3f &b, float t) { return a + (b - a) * t; }

__device__ inline float SchlickFresnel(float u) {
	float m = __saturatef(1.0 - u);
	return square(square(m)) * m;
}

__device__ inline float G1(float NdotH, float a) {
	if (a >= 1.0) { return 1.0 / PI(); }
	float a2 = square(a);
	float t = 1.0 + (a2 - 1.0) * NdotH * NdotH;
	return (a2 - 1.0) / (PI() * log(a2) * t);
}

__device__ inline float G2(float NdotH, float a) {
	float a2 = square(a);
	float t = 1.0 + (a2 - 1.0) * NdotH * NdotH;
	return a2 / (PI() * t * t);
}

__device__ inline float SmithG_GGX(float NdotV, float alphaG) {
	float a = alphaG * alphaG;
	float b = NdotV * NdotV;
	return 1.0 / (NdotV + sqrtf(a + b - a * b));
}

// this function largely based on:
// https://github.com/wdas/brdf/blob/master/src/brdfs/disney.brdf
// http://blog.selfshadow.com/publications/s2012-shading-course/burley/s2012_pbs_disney_brdf_notes_v3.pdf
__device__ Vector3f evaluate_shading(
	const Vector3f& base_color,
	const Vector3f& ambient_color, // :)
	const Vector3f& light_color, // :)
	float metallic,
	float subsurface,
	float specular,
	float roughness,
	float specular_tint,
	float sheen,
	float sheen_tint,
	float clearcoat,
	float clearcoat_gloss,
	Vector3f L,
	Vector3f V,
	Vector3f N
) {
	float NdotL = N.dot(L);
	float NdotV = N.dot(V);

	Vector3f H = (L + V).normalized();
	float NdotH = N.dot(H);
	float LdotH = L.dot(H);

	// Diffuse fresnel - go from 1 at normal incidence to .5 at grazing
	// and mix in diffuse retro-reflection based on roughness
	float FL = SchlickFresnel(NdotL), FV = SchlickFresnel(NdotV);
	Vector3f amb = (ambient_color * mix(0.2f, FV, metallic));
	amb = amb.array() * base_color.array();
	if (NdotL < 0.f || NdotV < 0.f) {
		return amb;
	}

	float luminance = base_color.dot(Vector3f(0.3f, 0.6f, 0.1f));

	// normalize luminance to isolate hue and saturation components
	Vector3f Ctint = base_color * (1.f/(luminance+0.00001f));
	Vector3f Cspec0 = mix(mix(Vector3f(1.0f,1.0f,1.0f), Ctint, specular_tint) * specular * 0.08f, base_color, metallic);
	Vector3f Csheen = mix(Vector3f(1.0f,1.0f,1.0f), Ctint, sheen_tint);

	float Fd90 = 0.5f + 2.0f * LdotH * LdotH * roughness;
	float Fd = mix(1, Fd90, FL) * mix(1.f, Fd90, FV);

	// Based on Hanrahan-Krueger BRDF approximation of isotropic BSSRDF
	// 1.25 scale is used to (roughly) preserve albedo
	// Fss90 used to "flatten" retroreflection based on roughness
	float Fss90 = LdotH * LdotH * roughness;
	float Fss = mix(1.0f, Fss90, FL) * mix(1.0f, Fss90, FV);
	float ss = 1.25f * (Fss * (1.f / (NdotL + NdotV) - 0.5f) + 0.5f);

	// Specular
	float a= std::max(0.001f, square(roughness));
	float Ds = G2(NdotH, a);
	float FH = SchlickFresnel(LdotH);
	Vector3f Fs = mix(Cspec0, Vector3f(1.0f,1.0f,1.0f), FH);
	float Gs = SmithG_GGX(NdotL, a) * SmithG_GGX(NdotV, a);

	// sheen
	Vector3f Fsheen = FH * sheen * Csheen;

	// clearcoat (ior = 1.5 -> F0 = 0.04)
	float Dr = G1(NdotH, mix(0.1f, 0.001f, clearcoat_gloss));
	float Fr = mix(0.04f, 1.0f, FH);
	float Gr = SmithG_GGX(NdotL, 0.25f) * SmithG_GGX(NdotV, 0.25f);

	float CCs=0.25f * clearcoat * Gr * Fr * Dr;
	Vector3f brdf = (float(1.0f / PI()) * mix(Fd, ss, subsurface) * base_color + Fsheen) * (1.0f - metallic) +
		Gs * Fs * Ds + Vector3f(CCs,CCs,CCs);
	return Vector3f(brdf.array() * light_color.array()) * NdotL + amb;
}

__global__ void advance_pos_kernel_sdf(
	const uint32_t n_elements,
	const float zero_offset,
	Vector3f* __restrict__ positions,
	float* __restrict__ distances,
	SdfPayload* __restrict__ payloads,
	BoundingBox aabb,
	float floor_y,
	const TriangleOctreeNode* __restrict__ octree_nodes,
	int max_depth,
	float distance_scale,
	float maximum_distance,
	float k,
	float* __restrict__ prev_distances,
	float* __restrict__ total_distances,
	float* __restrict__ min_visibility
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	SdfPayload& payload = payloads[i];
	if (!payload.alive) {
		return;
	}

	float distance = distances[i] - zero_offset;

	distance *= distance_scale;

	// Advance by the predicted distance
	Vector3f pos = positions[i];
	pos += distance * payload.dir;

	// Skip over regions not covered by the octree
	if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_depth, pos)) {
		float octree_distance = (TriangleOctree::ray_intersect(octree_nodes, max_depth, pos, payload.dir) + 1e-6f);
		distance += octree_distance;
		pos += octree_distance * payload.dir;
	}
	if (pos.y() < floor_y && payload.dir.y()<0.f) {
		float floor_dist = -(pos.y()-floor_y)/payload.dir.y();
		distance += floor_dist;
		pos += floor_dist * payload.dir;
		payload.alive=false;
	}

	positions[i] = pos;

	if (total_distances && distance > 0.0f) {
		// From https://www.iquilezles.org/www/articles/rmshadows/rmshadows.htm
		float total_distance = total_distances[i];
		float y = distance*distance / (2.0f * prev_distances[i]);
		float d = sqrtf(distance*distance - y*y);

		min_visibility[i] = fminf(min_visibility[i], k * d / fmaxf(0.0f, total_distance - y));
		prev_distances[i] = distance;
		total_distances[i] = total_distance + distance;
	}

	bool stay_alive = distance > maximum_distance && fabsf(distance / 2) > 3*maximum_distance;
	if (!stay_alive) {
		payload.alive = false;
		return;
	}

	if (!aabb.contains(pos)) {
		payload.alive = false;
		return;
	}

	payload.n_steps++;
}

__global__ void perturb_sdf_samples(uint32_t n_elements, const Vector3f* __restrict__ perturbations, Vector3f* __restrict__ positions, float* __restrict__ distances) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	Vector3f perturbation = perturbations[i];
	positions[i] += perturbation;

	// Small epsilon above 1 to ensure a triangle is always found.
	distances[i] = perturbation.norm()*1.001f;
}

__global__ void prepare_shadow_rays(const uint32_t n_elements,
	Vector3f sun_dir,
	Vector3f* __restrict__ positions,
	Vector3f* __restrict__ normals,
	float* __restrict__ distances,
	float* __restrict__ prev_distances,
	float* __restrict__ total_distances,
	float* __restrict__ min_visibility,
	SdfPayload* __restrict__ payloads,
	BoundingBox aabb,
	const TriangleOctreeNode* __restrict__ octree_nodes,
	int max_depth
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	SdfPayload& payload = payloads[i];

	// Step back a little along the ray to prevent self-intersection
	Vector3f view_pos = positions[i] + faceforward(normals[i], -payload.dir, normals[i]).normalized() * 1e-3f;
	Vector3f dir = sun_dir.normalized();

	float t = fmaxf(aabb.ray_intersect(view_pos, dir).x() + 1e-6f, 0.0f);
	view_pos += t * dir;

	if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_depth, view_pos)) {
		t = fmaxf(0.0f, TriangleOctree::ray_intersect(octree_nodes, max_depth, view_pos, dir) + 1e-6f);
		view_pos += t * dir;
	}

	positions[i] = view_pos;

	if (!aabb.contains(view_pos)) {
		distances[i] = 10000.0f;
		payload.alive = false;
		min_visibility[i] = 1.0f;
		return;
	}

	distances[i] = 10000.0f;
	payload.idx = i;
	payload.dir = dir;
	payload.n_steps = 0;
	payload.alive = true;

	if (prev_distances) {
		prev_distances[i] = 1e20f;
	}

	if (total_distances) {
		total_distances[i] = 0.0f;
	}

	if (min_visibility) {
		min_visibility[i] = 1.0f;
	}
}

__global__ void write_shadow_ray_result(const uint32_t n_elements, BoundingBox aabb, const Vector3f* __restrict__ positions, const SdfPayload* __restrict__ shadow_payloads, const float* __restrict__ min_visibility, float* __restrict__ shadow_factors) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	shadow_factors[shadow_payloads[i].idx] = aabb.contains(positions[i]) ? 0.0f : min_visibility[i];
}

__global__ void shade_kernel_sdf(
	const uint32_t n_elements,
	BoundingBox aabb,
	float floor_y,
	const ERenderMode mode,
	const BRDFParams brdf,
	Vector3f sun_dir,
	Vector3f up_dir,
	Matrix<float, 3, 4> camera_matrix,
	Vector3f* __restrict__ positions,
	Vector3f* __restrict__ normals,
	float* __restrict__ distances,
	SdfPayload* __restrict__ payloads,
	Array4f* __restrict__ frame_buffer,
	float* __restrict__ depth_buffer
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	SdfPayload& payload = payloads[i];
	if (!aabb.contains(positions[i])) {
		return;
	}

	// The normal in memory isn't normalized yet
	Vector3f normal = normals[i].normalized();

	Vector3f pos = positions[i];
	bool floor = false;
	if (pos.y() < floor_y+0.001f && payload.dir.y() < 0.f) {
		normal = Vector3f(0.f, 1.f, 0.f);
		floor = true;
	}
	Vector3f cam_pos = camera_matrix.col(3);
	Vector3f cam_fwd = camera_matrix.col(2);
	float ao = powf(0.92f, payload.n_steps * 0.5f) * (1.f / 0.92f);
	Array3f color;
	switch (mode) {
		case ERenderMode::AO: color = Array3f::Constant(powf(0.92f, payload.n_steps)); break;
		case ERenderMode::Shade: {
			float skyam = normal.dot(up_dir)*-0.5f+0.5f;
			Vector3f suncol = Array3f{255.f/255.0f, 225.f/255.0f, 195.f/255.0f} * 4.f * distances[i]; // Distance encodes shadow occlusion. 0=occluded, 1=no shadow
			const Vector3f skycol = Array3f{195.f/255.0f, 215.f/255.0f, 255.f/255.0f} * 4.f * skyam;
			float check_size = 8.f/aabb.diag().x();
			float check=((int(floorf(check_size*(pos.x()-aabb.min.x())))^int(floorf(check_size*(pos.z()-aabb.min.z())))) &1) ? 0.8f : 0.2f;
			const Vector3f floorcol = Array3f{check*check*check, check*check, check};
			Vector3f col = evaluate_shading(floor ? floorcol : brdf.basecolor.array() * brdf.basecolor.array(), brdf.ambientcolor.array() * skycol.array(), suncol, floor ? 0.f : brdf.metallic, floor ? 0.f : brdf.subsurface, floor ? 1.f : brdf.specular, floor ? 0.5f : brdf.roughness, 0.f, floor ? 0.f : brdf.sheen, 0.f, floor ? 0.f : brdf.clearcoat, brdf.clearcoat_gloss, sun_dir, -payload.dir.normalized(), normal);
			color = col.array();
		} break;
		case ERenderMode::Depth: color = Array3f::Constant(cam_fwd.dot(pos - cam_pos)); break;
		case ERenderMode::Positions: {
			color = (pos.array() - Array3f::Constant(0.5f)) / 2.0f + Array3f::Constant(0.5f);
		} break;
		case ERenderMode::Normals: color = 0.5f * normal.array() + Array3f::Constant(0.5f); break;
		case ERenderMode::Cost: color = Array3f::Constant((float)payload.n_steps / 30); break;
		case ERenderMode::EncodingVis: color = normals[i]; break;
	}

	frame_buffer[payload.idx] = {color.x(), color.y(), color.z(), 1.0f};
	depth_buffer[payload.idx] = cam_fwd.dot(pos - cam_pos);
}

__global__ void compact_kernel_shadow_sdf(
	const uint32_t n_elements,
	const float zero_offset,
	Vector3f* src_positions, float* src_distances, SdfPayload* src_payloads, float* src_prev_distances, float* src_total_distances, float* src_min_visibility,
	Vector3f* dst_positions, float* dst_distances, SdfPayload* dst_payloads, float* dst_prev_distances, float* dst_total_distances, float* dst_min_visibility,
	Vector3f* dst_final_positions, float* dst_final_distances, SdfPayload* dst_final_payloads, float* dst_final_prev_distances, float* dst_final_total_distances, float* dst_final_min_visibility,
	BoundingBox aabb,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	SdfPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_positions[idx] = src_positions[i];
		dst_distances[idx] = src_distances[i];
		dst_prev_distances[idx] = src_prev_distances[i];
		dst_total_distances[idx] = src_total_distances[i];
		dst_min_visibility[idx] = src_min_visibility[i];
	} else { // For shadow rays, collect _all_ final samples to keep track of their partial visibility
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_positions[idx] = src_positions[i];
		dst_final_distances[idx] = src_distances[i];
		dst_final_prev_distances[idx] = src_prev_distances[i];
		dst_final_total_distances[idx] = src_total_distances[i];
		dst_final_min_visibility[idx] = aabb.contains(src_positions[i]) ? 0.0f : src_min_visibility[i];
	}
}

__global__ void compact_kernel_sdf(
	const uint32_t n_elements,
	const float zero_offset,
	Vector3f* src_positions, float* src_distances, SdfPayload* src_payloads,
	Vector3f* dst_positions, float* dst_distances, SdfPayload* dst_payloads,
	Vector3f* dst_final_positions, float* dst_final_distances, SdfPayload* dst_final_payloads,
	BoundingBox aabb,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	SdfPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_positions[idx] = src_positions[i];
		dst_distances[idx] = src_distances[i];
	} else if (aabb.contains(src_positions[i])) {
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_positions[idx] = src_positions[i];
		dst_final_distances[idx] = 1.0f; // HACK: Distances encode shadowing factor when shading
	}
}

__global__ void uniform_octree_sample_kernel(
	const uint32_t num_elements,
	default_rng_t rng,
	const TriangleOctreeNode* __restrict__ octree_nodes,
	uint32_t num_nodes,
	uint32_t depth,
	Vector3f* __restrict__ samples
) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= num_elements) return;

	rng.advance(i * (1<<8));

	// Samples random nodes until a leaf is picked
	uint32_t node;
	uint32_t child;
	do {
		node = umin((uint32_t)(random_val(rng) * num_nodes), num_nodes-1);
		child = umin((uint32_t)(random_val(rng) * 8), 8u-1);
	} while (octree_nodes[node].depth < depth-2 || octree_nodes[node].children[child] == -1);

	// Here it should be guaranteed that any child of the node is -1
	float size = scalbnf(1.0f, -depth+1);

	Vector3i16 pos = octree_nodes[node].pos*2;
	if (child&1) ++pos.x();
	if (child&2) ++pos.y();
	if (child&4) ++pos.z();
	samples[i] = size * (pos.cast<float>() + samples[i]);
}

__global__ void scale_to_aabb_kernel(uint32_t n_elements, BoundingBox aabb, Vector3f* __restrict__ inout) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	inout[i] = aabb.min + inout[i].cwiseProduct(aabb.diag());
}

__global__ void compare_signs_kernel(uint32_t n_elements, const Vector3f *positions, const float *distances_ref, const float *distances_model, uint32_t *counters, const TriangleOctreeNode* octree_nodes, int max_depth) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;
	bool inside1 = distances_ref[i]<=0.f;
	bool inside2 = distances_model[i]<=0.f;
	if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_depth, positions[i])) {
		inside2=inside1; // assume, when using the octree, that the model is always correct outside the octree.
		atomicAdd(&counters[6],1); // outside the octree
	} else {
		atomicAdd(&counters[7],1); // inside the octree
	}
	atomicAdd(&counters[inside1 ? 0 : 1],1);
	atomicAdd(&counters[inside2 ? 2 : 3],1);
	if (inside1&&inside2)
		atomicAdd(&counters[4],1);
	if (inside1||inside2)
		atomicAdd(&counters[5],1);

}

__global__ void scale_iou_counters_kernel(uint32_t n_elements, uint32_t *counters, float scale) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	counters[i] = uint32_t(roundf(counters[i]*scale));
}

__global__ void assign_float(uint32_t n_elements, float value, float* __restrict__ out) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	out[i] = value;
}

__global__ void init_rays_with_payload_kernel_sdf(
	uint32_t sample_index,
	Vector3f* __restrict__ positions,
	float* __restrict__ distances,
	SdfPayload* __restrict__ payloads,
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera_matrix,
	Vector2f screen_center,
	Vector3f parallax_shift,
	bool snap_to_pixel_centers,
	BoundingBox aabb,
	float floor_y,
	float plane_z,
	float dof,
	const float* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	Array4f* __restrict__ framebuffer,
	float* __restrict__ depthbuffer,
	const TriangleOctreeNode* __restrict__ octree_nodes = nullptr,
	int max_depth = 0
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	if (plane_z < 0) {
		dof = 0.0;
	}

	Ray ray = pixel_to_ray(sample_index, {x, y}, resolution, focal_length, camera_matrix, screen_center, parallax_shift, snap_to_pixel_centers, plane_z, dof);

	distances[idx] = 10000.0f;

	if (plane_z < 0) {
		float n = ray.d.norm();
		SdfPayload& payload = payloads[idx];
		payload.dir = (1.0f/n) * ray.d;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		positions[idx] = ray.o - plane_z * ray.d;
		depthbuffer[idx] = -plane_z;
		return;
	}

	depthbuffer[idx] = 1e10f;

	ray.d = ray.d.normalized();
	float t = max(aabb.ray_intersect(ray.o, ray.d).x(), 0.0f);
	ray.o = ray.o + (t + 1e-6f) * ray.d;

	if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_depth, ray.o)) {
		t = max(0.0f, TriangleOctree::ray_intersect(octree_nodes, max_depth, ray.o, ray.d));
		if (ray.o.y() > floor_y && ray.d.y()<0.f) {
			float floor_dist = -(ray.o.y() - floor_y)/ray.d.y();
			if (floor_dist>0.f) {
				t=min(t,floor_dist);
			}
		}
		ray.o = ray.o + (t + 1e-6f) * ray.d;
	}

	positions[idx] = ray.o;

	if (envmap_data) {
		framebuffer[idx] = read_envmap(envmap_data, envmap_resolution, ray.d);
	}

	SdfPayload& payload = payloads[idx];
	if (!aabb.contains(ray.o)) {
		payload.alive = false;
		return;
	}

	payload.dir = ray.d;
	payload.idx = idx;
	payload.n_steps = 0;
	payload.alive = true;
}

__host__ __device__ uint32_t sample_discrete(float uniform_sample, const float* __restrict__ cdf, int length) {
	return binary_search(uniform_sample, cdf, length);
}

__global__ void sample_uniform_on_triangle_kernel(uint32_t n_elements, const float* __restrict__ cdf, uint32_t length, const Triangle* __restrict__ triangles, Vector3f* __restrict__ sampled_positions) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	Vector3f sample = sampled_positions[i];
	uint32_t tri_idx = sample_discrete(sample.x(), cdf, length);

	sampled_positions[i] = triangles[tri_idx].sample_uniform_position(sample.tail<2>());
}

void Testbed::SphereTracer::init_rays_from_camera(uint32_t sample_index,
	const Vector2i& resolution,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix,
	const Vector2f& screen_center,
	const Vector3f& parallax_shift,
	bool snap_to_pixel_centers,
	const BoundingBox& aabb,
	float floor_y,
	float plane_z,
	float dof,
	const float* envmap_data,
	const Vector2i& envmap_resolution,
	Array4f* frame_buffer,
	float* depth_buffer,
	const TriangleOctree* octree,
	uint32_t n_octree_levels,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x() * resolution.y();
	enlarge(n_pixels);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x(), threads.x), div_round_up((uint32_t)resolution.y(), threads.y), 1 };
	init_rays_with_payload_kernel_sdf<<<blocks, threads, 0, stream>>>(
		sample_index,
		m_rays[0].pos.data(),
		m_rays[0].distance.data(),
		m_rays[0].payload.data(),
		resolution,
		focal_length,
		camera_matrix,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		aabb,
		floor_y,
		plane_z,
		dof,
		envmap_data,
		envmap_resolution,
		frame_buffer,
		depth_buffer,
		octree ? octree->nodes_gpu() : nullptr,
		octree ? n_octree_levels : 0
	);
	m_n_rays_initialized = (uint32_t)n_pixels;
}

void Testbed::SphereTracer::init_rays_from_data(uint32_t n_elements, const RaysSdfSoa& data, hipStream_t stream) {
	enlarge(n_elements);
	m_rays[0].copy_from_other_async(n_elements, data, stream);
	m_n_rays_initialized = n_elements;
}

uint32_t Testbed::SphereTracer::trace_bvh(TriangleBvh* bvh, const Triangle* triangles, hipStream_t stream) {
	uint32_t n_alive = m_n_rays_initialized;
	m_n_rays_initialized = 0;

	if (!bvh) {
		return 0;
	}

	// Abuse the normal buffer to temporarily hold ray directions
	parallel_for_gpu(stream, n_alive, [payloads=m_rays[0].payload.data(), normals=m_rays[0].normal.data()] __device__ (size_t i) {
		normals[i] = payloads[i].dir;
	});

	bvh->ray_trace_gpu(n_alive, m_rays[0].pos.data(), m_rays[0].normal.data(), triangles, stream);
	return n_alive;
}

uint32_t Testbed::SphereTracer::trace(
	const distance_fun_t& distance_function,
	float zero_offset,
	float distance_scale,
	float maximum_distance,
	const BoundingBox& aabb,
	const float floor_y,
	const TriangleOctree* octree,
	const uint32_t n_octree_levels,
	hipStream_t stream
) {
	if (m_n_rays_initialized == 0) {
		return 0;
	}

	CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter.data(), 0, sizeof(uint32_t), stream));

	const uint32_t STEPS_INBETWEEN_COMPACTION = 4;

	uint32_t n_alive = m_n_rays_initialized;
	m_n_rays_initialized = 0;

	uint32_t i = 1;
	uint32_t double_buffer_index = 0;
	while (i < MARCH_ITER) {
		// Compact more frequently in the first couple of steps
		uint32_t step_size = std::min(i, STEPS_INBETWEEN_COMPACTION);

		RaysSdfSoa& rays_current = m_rays[(double_buffer_index+1)%2];
		RaysSdfSoa& rays_tmp = m_rays[double_buffer_index%2];
		++double_buffer_index;

		// Compact rays that did not diverge yet
		{
			CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter.data(), 0, sizeof(uint32_t), stream));
			if (m_trace_shadow_rays) {
				linear_kernel(compact_kernel_shadow_sdf, 0, stream,
					n_alive,
					zero_offset,
					rays_tmp.pos.data(), rays_tmp.distance.data(), rays_tmp.payload.data(), rays_tmp.prev_distance.data(), rays_tmp.total_distance.data(), rays_tmp.min_visibility.data(),
					rays_current.pos.data(), rays_current.distance.data(), rays_current.payload.data(), rays_current.prev_distance.data(), rays_current.total_distance.data(), rays_current.min_visibility.data(),
					m_rays_hit.pos.data(), m_rays_hit.distance.data(), m_rays_hit.payload.data(), m_rays_hit.prev_distance.data(), m_rays_hit.total_distance.data(), m_rays_hit.min_visibility.data(),
					aabb,
					m_alive_counter.data(), m_hit_counter.data()
				);
			} else {
				linear_kernel(compact_kernel_sdf, 0, stream,
					n_alive,
					zero_offset,
					rays_tmp.pos.data(), rays_tmp.distance.data(), rays_tmp.payload.data(),
					rays_current.pos.data(), rays_current.distance.data(), rays_current.payload.data(),
					m_rays_hit.pos.data(), m_rays_hit.distance.data(), m_rays_hit.payload.data(),
					aabb,
					m_alive_counter.data(), m_hit_counter.data()
				);
			}
			CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		}

		if (n_alive == 0) {
			break;
		}

		for (uint32_t j = 0; j < step_size; ++j) {
			distance_function(n_alive, rays_current.pos, rays_current.distance, stream);
			linear_kernel(advance_pos_kernel_sdf, 0, stream,
				n_alive,
				zero_offset,
				rays_current.pos.data(),
				rays_current.distance.data(),
				rays_current.payload.data(),
				aabb,
				floor_y,
				octree ? octree->nodes_gpu() : nullptr,
				octree ? n_octree_levels : 0,
				distance_scale,
				maximum_distance,
				m_shadow_sharpness,
				m_trace_shadow_rays ? rays_current.prev_distance.data() : nullptr,
				m_trace_shadow_rays ? rays_current.total_distance.data() : nullptr,
				m_trace_shadow_rays ? rays_current.min_visibility.data() : nullptr
			);
		}

		i += step_size;
	}

	uint32_t n_hit;
	CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	return n_hit;
}

void Testbed::SphereTracer::enlarge(size_t n_elements) {
	n_elements = next_multiple(n_elements, size_t(tcnn::batch_size_granularity));
	m_rays[0].enlarge(n_elements);
	m_rays[1].enlarge(n_elements);
	m_rays_hit.enlarge(n_elements);
}

void Testbed::FiniteDifferenceNormalsApproximator::enlarge(uint32_t n_elements) {
	dx.enlarge(n_elements);
	dy.enlarge(n_elements);
	dz.enlarge(n_elements);

	dist_dx_pos.enlarge(n_elements);
	dist_dy_pos.enlarge(n_elements);
	dist_dz_pos.enlarge(n_elements);

	dist_dx_neg.enlarge(n_elements);
	dist_dy_neg.enlarge(n_elements);
	dist_dz_neg.enlarge(n_elements);
}

void Testbed::FiniteDifferenceNormalsApproximator::normal(uint32_t n_elements, const distance_fun_t& distance_function, GPUMemory<Vector3f>& pos, GPUMemory<Vector3f>& normal, float epsilon, hipStream_t stream) {
	enlarge(n_elements);

	parallel_for_gpu(stream, n_elements, [pos=pos.data(), dx=dx.data(), dy=dy.data(), dz=dz.data(), epsilon] __device__ (size_t i) {
		Vector3f p = pos[i];
		dx[i] = Vector3f{p.x() + epsilon, p.y(), p.z()};
		dy[i] = Vector3f{p.x(), p.y() + epsilon, p.z()};
		dz[i] = Vector3f{p.x(), p.y(), p.z() + epsilon};
	});

	distance_function(n_elements, dx, dist_dx_pos, stream);
	distance_function(n_elements, dy, dist_dy_pos, stream);
	distance_function(n_elements, dz, dist_dz_pos, stream);

	parallel_for_gpu(stream, n_elements, [pos=pos.data(), dx=dx.data(), dy=dy.data(), dz=dz.data(), epsilon] __device__ (size_t i) {
		Vector3f p = pos[i];
		dx[i] = Vector3f{p.x() - epsilon, p.y(), p.z()};
		dy[i] = Vector3f{p.x(), p.y() - epsilon, p.z()};
		dz[i] = Vector3f{p.x(), p.y(), p.z() - epsilon};
	});

	distance_function(n_elements, dx, dist_dx_neg, stream);
	distance_function(n_elements, dy, dist_dy_neg, stream);
	distance_function(n_elements, dz, dist_dz_neg, stream);

	parallel_for_gpu(stream, n_elements, [normal=normal.data(), dist_dx_pos=dist_dx_pos.data(), dist_dx_neg=dist_dx_neg.data(), dist_dy_pos=dist_dy_pos.data(), dist_dy_neg=dist_dy_neg.data(), dist_dz_pos=dist_dz_pos.data(), dist_dz_neg=dist_dz_neg.data()] __device__ (size_t i) {
		normal[i] = {dist_dx_pos[i] - dist_dx_neg[i], dist_dy_pos[i] - dist_dy_neg[i], dist_dz_pos[i] - dist_dz_neg[i]};
	});
}

void Testbed::render_sdf(
	const distance_fun_t& distance_function,
	const normals_fun_t& normals_function,
	CudaRenderBuffer& render_buffer,
	const Vector2i& max_res,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix,
	const Vector2f& screen_center,
	hipStream_t stream
) {
	float plane_z = m_slice_plane_z + m_scale;
	if (m_render_mode == ERenderMode::Slice) {
		plane_z = -plane_z;
	}
	auto* octree_ptr = m_sdf.uses_takikawa_encoding || m_sdf.use_triangle_octree ? m_sdf.triangle_octree.get() : nullptr;

	// Reserve the memory for max-res rendering to prevent stuttering
	m_sdf.tracer.enlarge(max_res.x() * max_res.y());

	uint32_t n_octree_levels = octree_ptr ? octree_ptr->depth() : 0;
	if (m_render_ground_truth && m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
		n_octree_levels = std::min(n_octree_levels, m_sdf.brick_level);
	}

	BoundingBox sdf_bounding_box = m_aabb;
	sdf_bounding_box.inflate(m_sdf.zero_offset);
	m_sdf.tracer.init_rays_from_camera(
		render_buffer.spp(),
		render_buffer.in_resolution(),
		focal_length,
		camera_matrix,
		screen_center,
		get_scaled_parallax_shift(),
		m_snap_to_pixel_centers,
		sdf_bounding_box,
		get_floor_y(),
		plane_z,
		m_dof,
		m_envmap.envmap->params_inference(),
		m_envmap.resolution,
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer(),
		octree_ptr,
		n_octree_levels,
		stream
	);

	bool gt_raytrace = m_render_ground_truth && m_sdf.groundtruth_mode == ESDFGroundTruthMode::RaytracedMesh;

	auto trace = [&](SphereTracer& tracer) {
		if (gt_raytrace) {
			return tracer.trace_bvh(m_sdf.triangle_bvh.get(), m_sdf.triangles_gpu.data(), stream);
		} else {
			return tracer.trace(
				distance_function,
				m_sdf.zero_offset,
				m_sdf.distance_scale,
				m_sdf.maximum_distance,
				sdf_bounding_box,
				get_floor_y(),
				octree_ptr,
				n_octree_levels,
				stream
			);
		}
	};

	uint32_t n_hit;
	if (m_render_mode == ERenderMode::Slice) {
		n_hit = m_sdf.tracer.n_rays_initialized();
	} else {
		n_hit = trace(m_sdf.tracer);
	}
	RaysSdfSoa& rays_hit = m_render_mode == ERenderMode::Slice || gt_raytrace ? m_sdf.tracer.rays_init() : m_sdf.tracer.rays_hit();

	if (m_render_mode == ERenderMode::Slice) {
		if (m_visualized_dimension == -1) {
			distance_function(n_hit, rays_hit.pos, rays_hit.distance, stream);
			extract_dimension_pos_neg_kernel<float><<<n_blocks_linear(n_hit*3), n_threads_linear, 0, stream>>>(n_hit*3, 0, 1, 3, rays_hit.distance.data(), CM, (float*)rays_hit.normal.data());
		} else {
			// Store colors in the normal buffer
			uint32_t n_elements = next_multiple(n_hit, tcnn::batch_size_granularity);

			GPUMatrix<float> positions_matrix((float*)rays_hit.pos.data(), 3, n_elements);
			GPUMatrix<float> colors_matrix((float*)rays_hit.normal.data(), 3, n_elements);
			m_network->visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, colors_matrix);
		}
	}

	ERenderMode render_mode = (m_visualized_dimension > -1 || m_render_mode == ERenderMode::Slice) ? ERenderMode::EncodingVis : m_render_mode;
	if (render_mode == ERenderMode::Shade || render_mode == ERenderMode::Normals) {
		if (m_sdf.analytic_normals || gt_raytrace) {
			normals_function(n_hit, rays_hit.pos, rays_hit.normal, stream);
		} else {
			// Prevent spurious enlargements by reserving enough memory to hold a full-res image in any case.
			m_sdf.fd_normals.enlarge(render_buffer.in_resolution().x() * render_buffer.in_resolution().y());
			float fd_normals_epsilon = m_sdf.fd_normals_epsilon;
			if (m_render_ground_truth && m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks && m_sdf.brick_smooth_normals) {
				fd_normals_epsilon = exp2f(-float(n_octree_levels)) * (1.f/(m_sdf.brick_res-1)); // in sdf brick mode, use one voxel as the normal central difference radius
			}
			m_sdf.fd_normals.normal(n_hit, distance_function, rays_hit.pos, rays_hit.normal, fd_normals_epsilon, stream);
		}

		if (render_mode == ERenderMode::Shade && n_hit > 0) {
			// Shadow rays towards the sun
			m_sdf.shadow_tracer.init_rays_from_data(n_hit, rays_hit, stream);
			m_sdf.shadow_tracer.set_trace_shadow_rays(true);
			m_sdf.shadow_tracer.set_shadow_sharpness(m_sdf.shadow_sharpness);
			RaysSdfSoa& shadow_rays_init = m_sdf.shadow_tracer.rays_init();
			linear_kernel(prepare_shadow_rays, 0, stream,
				n_hit,
				m_sun_dir.normalized(),
				shadow_rays_init.pos.data(),
				shadow_rays_init.normal.data(),
				shadow_rays_init.distance.data(),
				shadow_rays_init.prev_distance.data(),
				shadow_rays_init.total_distance.data(),
				shadow_rays_init.min_visibility.data(),
				shadow_rays_init.payload.data(),
				sdf_bounding_box,
				octree_ptr ? octree_ptr->nodes_gpu() : nullptr,
				n_octree_levels
			);
			uint32_t n_hit_shadow = trace(m_sdf.shadow_tracer);
			auto& shadow_rays_hit = gt_raytrace ? m_sdf.shadow_tracer.rays_init() : m_sdf.shadow_tracer.rays_hit();

			linear_kernel(write_shadow_ray_result, 0, stream,
				n_hit_shadow,
				sdf_bounding_box,
				shadow_rays_hit.pos.data(),
				shadow_rays_hit.payload.data(),
				shadow_rays_hit.min_visibility.data(),
				rays_hit.distance.data()
			);

			// todo: Reflection rays?
		}
	} else if (render_mode == ERenderMode::EncodingVis && m_render_mode != ERenderMode::Slice) {
		// HACK: Store colors temporarily in the normal buffer
		uint32_t n_elements = next_multiple(n_hit, tcnn::batch_size_granularity);

		GPUMatrix<float> positions_matrix((float*)rays_hit.pos.data(), 3, n_elements);
		GPUMatrix<float> colors_matrix((float*)rays_hit.normal.data(), 3, n_elements);
		m_network->visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, colors_matrix);
	}

	linear_kernel(shade_kernel_sdf, 0, stream,
		n_hit,
		m_aabb,
		get_floor_y(),
		render_mode,
		m_sdf.brdf,
		m_sun_dir.normalized(),
		m_up_dir.normalized(),
		camera_matrix,
		rays_hit.pos.data(),
		rays_hit.normal.data(),
		rays_hit.distance.data(),
		rays_hit.payload.data(),
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer()
	);

	if (render_mode == ERenderMode::Cost) {
		std::vector<SdfPayload> payloads_final_cpu(n_hit);
		rays_hit.payload.copy_to_host(payloads_final_cpu, n_hit);
		size_t total_n_steps = 0;
		for (uint32_t i = 0; i < n_hit; ++i) {
			total_n_steps += payloads_final_cpu[i].n_steps;
		}
		tlog::info() << "Total steps per hit= " << total_n_steps << "/" << n_hit << " = " << ((float)total_n_steps/(float)n_hit);
	}
}

void Testbed::load_mesh() {
	std::vector<Vector3f> vertices;
	if (equals_case_insensitive(m_data_path.extension(), "obj")) {
		vertices = load_obj(m_data_path.str());
	} else if (equals_case_insensitive(m_data_path.extension(), "stl")) {
		FILE* f = fopen(m_data_path.str().c_str(), "rb");
		if (!f) {
			throw std::runtime_error{"stl file not found"};
		}
		uint32_t buf[21]={};
		if (fread(buf, 4, 21, f) != 4*21) {
			throw std::runtime_error{"stl file too small for header"};
		}
		uint32_t nfaces = buf[20];
		if (memcmp(buf,"solid",5)==0 || buf[20]==0) {
			fclose(f);
			throw std::runtime_error{"ascii stl files are not supported"};
		}
		vertices.reserve(nfaces * 3);
		for (uint32_t i = 0; i < nfaces; ++i) {
			if (fread(buf, 1, 50, f) < 50) {
				nfaces = i;
				break;
			}
			vertices.push_back(*(Vector3f*)(buf + 3));
			vertices.push_back(*(Vector3f*)(buf + 6));
			vertices.push_back(*(Vector3f*)(buf + 9));
		}
		fclose(f);
	} else {
		throw std::runtime_error{"Sdf data path must be a mesh in ascii .obj or binary .stl format."};
	}
	// The expected format is
	// [v1.x][v1.y][v1.z][v2.x]...
	size_t n_vertices = vertices.size();
	size_t n_triangles = n_vertices/3;

	m_raw_aabb.min = Vector3f::Constant(std::numeric_limits<float>::infinity());
	m_raw_aabb.max = Vector3f::Constant(-std::numeric_limits<float>::infinity());
	for (size_t i = 0; i < n_vertices; ++i) {
		m_raw_aabb.enlarge(vertices[i]);
	}

	// Inflate AABB by 1% to give the network a little wiggle room.
	m_raw_aabb.inflate(m_raw_aabb.diag().norm() * 0.005f);
	m_sdf.mesh_scale = m_raw_aabb.diag().maxCoeff();

	// Normalize vertex coordinates to lie within [0,1]^3.
	// This way, none of the constants need to carry around
	// bounding box factors.
	for (size_t i = 0; i < n_vertices; ++i) {
		vertices[i] = (vertices[i] - m_raw_aabb.min - 0.5f * m_raw_aabb.diag()) / m_sdf.mesh_scale + Vector3f::Constant(0.5f);
	}

	m_aabb = {};
	for (size_t i = 0; i < n_vertices; ++i) {
		m_aabb.enlarge(vertices[i]);
	}
	m_aabb = m_aabb.intersection(BoundingBox{Vector3f::Zero(), Vector3f::Ones()});
	m_render_aabb = m_aabb;
	m_mesh.thresh = 0.f;

	m_sdf.triangles_cpu.resize(n_triangles);
	for (size_t i = 0; i < n_vertices; i += 3) {
		m_sdf.triangles_cpu[i/3] = {vertices[i+0], vertices[i+1], vertices[i+2]};
	}

	if (!m_sdf.triangle_bvh)
		m_sdf.triangle_bvh = TriangleBvh::make();
	m_sdf.triangle_bvh->build(m_sdf.triangles_cpu, 8);
	m_sdf.triangles_gpu.resize_and_copy_from_host(m_sdf.triangles_cpu);
	m_sdf.triangle_bvh->build_optix(m_sdf.triangles_gpu, m_inference_stream);

	m_sdf.triangle_octree.reset(new TriangleOctree{});
	m_sdf.triangle_octree->build(*m_sdf.triangle_bvh, m_sdf.triangles_cpu, 10);
	m_sdf.brick_data.free_memory();

	m_bounding_radius = Vector3f::Constant(0.5f).norm();
	set_scale(m_bounding_radius * 1.5f);

	// Compute discrete probability distribution for later sampling of the mesh's surface
	m_sdf.triangle_weights.resize(n_triangles);
	for (size_t i = 0; i < n_triangles; ++i) {
		m_sdf.triangle_weights[i] = m_sdf.triangles_cpu[i].surface_area();
	}
	m_sdf.triangle_distribution.build(m_sdf.triangle_weights);

	// Move CDF to gpu
	m_sdf.triangle_cdf.resize_and_copy_from_host(m_sdf.triangle_distribution.cdf);

	// Clear training data as it's no longer representative
	// of the previously loaded mesh... but don't clear the network.
	// Perhaps it'll look interesting while morphing from one mesh to another.
	m_sdf.training.idx = 0;
	m_sdf.training.size = 0;

	tlog::success() << "Loaded mesh: triangles=" << n_triangles << " AABB=" << m_raw_aabb << " after scaling=" << m_aabb;
}

void Testbed::generate_training_samples_sdf(Vector3f* positions, float* distances, uint32_t n_to_generate, hipStream_t stream, bool uniform_only) {
	uint32_t n_to_generate_base = n_to_generate / 8;
	const uint32_t n_to_generate_surface_exact = uniform_only ? 0 : n_to_generate_base*4;
	const uint32_t n_to_generate_surface_offset = uniform_only ? 0 : n_to_generate_base*3;
	const uint32_t n_to_generate_uniform = uniform_only ? n_to_generate : n_to_generate_base*1;

	const uint32_t n_to_generate_surface = n_to_generate_surface_exact + n_to_generate_surface_offset;

	// Generate uniform 3D samples. Some of these will be transformed to cover the surfaces uniformly. Others will be left as-is.
	generate_random_uniform<float>(stream, m_rng, n_to_generate*3, (float*)positions);

	linear_kernel(sample_uniform_on_triangle_kernel, 0, stream,
		n_to_generate_surface,
		m_sdf.triangle_cdf.data(),
		(uint32_t)m_sdf.triangle_cdf.size(),
		m_sdf.triangles_gpu.data(),
		positions
	);

	// The distances of points on the mesh are zero. Can immediately set.
	CUDA_CHECK_THROW(hipMemsetAsync(distances, 0, n_to_generate_surface_exact*sizeof(float), stream));

	// If we have an octree, generate uniform samples within that octree.
	// Otherwise, at least confine uniform samples to the AABB.
	// (For the uniform_only case, we always use the AABB, then the IoU kernel checks against the octree later)
	float stddev = m_bounding_radius/1024.0f * m_sdf.training.surface_offset_scale;
	if (!uniform_only && (m_sdf.uses_takikawa_encoding || m_sdf.use_triangle_octree)) {
		linear_kernel(uniform_octree_sample_kernel, 0, stream,
			n_to_generate_uniform,
			m_rng,
			m_sdf.triangle_octree->nodes_gpu(),
			m_sdf.triangle_octree->n_nodes(),
			m_sdf.triangle_octree->depth(),
			positions+n_to_generate_surface
		);
		m_rng.advance();

		// If we know the finest discretization of the octree, we can concentrate
		// points MUCH closer to the mesh surface
		float leaf_size = scalbnf(1.0f, -m_sdf.triangle_octree->depth()+1);
		if (leaf_size < stddev) {
			tlog::warning() << "leaf_size < stddev";
			stddev = leaf_size;
		}

		linear_kernel(assign_float, 0, stream,
			n_to_generate_uniform,
			Vector3f::Constant(leaf_size).norm()*1.001f,
			distances+n_to_generate_surface
		);
	} else {
		BoundingBox sdf_aabb = m_aabb;
		sdf_aabb.inflate(m_sdf.zero_offset);
		linear_kernel(scale_to_aabb_kernel, 0, stream,
			n_to_generate_uniform, sdf_aabb,
			positions+n_to_generate_surface
		);

		linear_kernel(assign_float, 0, stream,
			n_to_generate_uniform,
			sdf_aabb.diag().norm()*1.001f,
			distances+n_to_generate_surface
		);
	}

	m_sdf.training.perturbations.enlarge(n_to_generate_surface_offset);
	generate_random_logistic<float>(stream, m_rng, n_to_generate_surface_offset*3, (float*)m_sdf.training.perturbations.data(), 0.0f, stddev);

	linear_kernel(perturb_sdf_samples, 0, stream,
		n_to_generate_surface_offset,
		m_sdf.training.perturbations.data(),
		positions+n_to_generate_surface_exact,
		distances+n_to_generate_surface_exact
	);

	// The following function expects `distances` to contain an upper bound on the
	// true distance. This accelerates lookups.
	m_sdf.triangle_bvh->signed_distance_gpu(
		n_to_generate_uniform+n_to_generate_surface_offset,
		m_sdf.mesh_sdf_mode,
		positions+n_to_generate_surface_exact,
		distances+n_to_generate_surface_exact,
		m_sdf.triangles_gpu.data(),
		true,
		stream
	);

	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
}

__global__ void generate_grid_samples_sdf_uniform(Eigen::Vector3i res_3d, BoundingBox aabb, Vector3f* __restrict__ out) {
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(aabb.max - aabb.min) + aabb.min;
	out[i] = pos;
}

GPUMemory<float> Testbed::get_sdf_gt_on_grid(Vector3i res3d, const BoundingBox& aabb) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<float> density(n_elements);
	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		Vector3f
	>(m_inference_stream, &alloc, n_elements);
	Vector3f* positions = std::get<0>(scratch);
	float* sdf_out = density.data();
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };
	generate_grid_samples_sdf_uniform<<<blocks, threads, 0, m_inference_stream>>>(res3d, aabb, positions);
	CUDA_CHECK_THROW(hipStreamSynchronize(m_inference_stream));
	m_sdf.triangle_bvh->signed_distance_gpu(
			n_elements,
			m_sdf.mesh_sdf_mode,
			positions,
			sdf_out,
			m_sdf.triangles_gpu.data(),
			false,
			m_inference_stream
		);
	CUDA_CHECK_THROW(hipStreamSynchronize(m_inference_stream));
	/*
	std::vector<float> cpudensity(density.size());
	std::vector<Vector3f> cpupositions(n_elements);
	density.copy_to_host(cpudensity);
	hipMemcpy(cpupositions.data(),positions,n_elements*12,hipMemcpyDeviceToHost);
	for (int i=0;i<64;++i)
		printf("[%0.3f %0.3f %0.3f] -> %0.3f\n", cpupositions[i].x(),cpupositions[i].y(),cpupositions[i].z(),cpudensity[i]);
	*/
	return density;
}


void Testbed::train_sdf(size_t target_batch_size, bool get_loss_scalar, hipStream_t stream) {
	const uint32_t n_output_dims = 1;
	const uint32_t n_input_dims = 3;

	if (m_sdf.training.size >= target_batch_size) {
		// Auxiliary matrices for training
		const uint32_t batch_size = (uint32_t)std::min(m_sdf.training.size, target_batch_size);

		// Permute all training records to de-correlate training data
		linear_kernel(shuffle<Vector3f>, 0, stream, m_sdf.training.size, 1, m_training_step, m_sdf.training.positions.data(), m_sdf.training.positions_shuffled.data());
		linear_kernel(shuffle<float>, 0, stream, m_sdf.training.size, 1, m_training_step, m_sdf.training.distances.data(), m_sdf.training.distances_shuffled.data());

		GPUMatrix<float> training_target_matrix(m_sdf.training.distances_shuffled.data(), n_output_dims, batch_size);
		GPUMatrix<float> training_batch_matrix((float*)(m_sdf.training.positions_shuffled.data()), n_input_dims, batch_size);

		auto ctx = m_trainer->training_step(stream, training_batch_matrix, training_target_matrix);

		m_training_step++;

		if (get_loss_scalar) {
			m_loss_scalar.update(m_trainer->loss(stream, *ctx));
		}
	}
}

void Testbed::training_prep_sdf(uint32_t batch_size, hipStream_t stream) {
	if (m_sdf.training.generate_sdf_data_online) {
		m_sdf.training.size = batch_size;
		m_sdf.training.positions.enlarge(m_sdf.training.size);
		m_sdf.training.positions_shuffled.enlarge(m_sdf.training.size);
		m_sdf.training.distances.enlarge(m_sdf.training.size);
		m_sdf.training.distances_shuffled.enlarge(m_sdf.training.size);

		generate_training_samples_sdf(m_sdf.training.positions.data(), m_sdf.training.distances.data(), batch_size, stream, false);
	}
}

// set scale_existing_results_factor=0. to reset any existing results; set it to 1.0 to accumulate more samples onto existing results
// set it to a fraction near 1 to use a sliding EMA
// if blocking is false, then this returns the iou from the *last* call
double Testbed::calculate_iou(uint32_t n_samples, float scale_existing_results_factor, bool blocking, bool force_use_octree) {
	hipStream_t stream = m_training_stream;
	uint32_t countercpu[8];
	m_sdf.iou_counter.enlarge(8);
	if (!blocking) // when not blocking, returns data from the *last* run, then kicks off work to accumulate some more samples
		hipMemcpy(countercpu, m_sdf.iou_counter.data(), 8*4, hipMemcpyDeviceToHost);

	if (scale_existing_results_factor < 1.f) {
		linear_kernel(scale_iou_counters_kernel, 0, stream, 8, m_sdf.iou_counter.data(), scale_existing_results_factor);
	}
	while (n_samples>0) {
		uint32_t batch_size = std::min(uint32_t(128*128*128), n_samples);
		m_sdf.training.size = batch_size;
		n_samples-=batch_size;
		m_sdf.training.positions.enlarge(m_sdf.training.size);
		m_sdf.training.distances.enlarge(m_sdf.training.size); // we use this buffer for the GT distances
		m_sdf.training.distances_shuffled.enlarge(m_sdf.training.size); // we use the shuffled output for the output of inference

		generate_training_samples_sdf(m_sdf.training.positions.data(), m_sdf.training.distances.data(), (uint32_t)(batch_size), stream, true);
		GPUMatrix<float> positions_matrix((float*)m_sdf.training.positions.data(), 3, batch_size);
		GPUMatrix<float> distances_matrix(m_sdf.training.distances_shuffled.data(), 1, batch_size);
		m_network->inference(stream, positions_matrix, distances_matrix);
		auto* octree_ptr = (m_sdf.uses_takikawa_encoding || m_sdf.use_triangle_octree || force_use_octree) ? m_sdf.triangle_octree.get() : nullptr;
		linear_kernel(compare_signs_kernel,0, stream, batch_size, m_sdf.training.positions.data(),
			m_sdf.training.distances.data(), //  ref
			m_sdf.training.distances_shuffled.data(), // model
			m_sdf.iou_counter.data(),
			octree_ptr ? octree_ptr->nodes_gpu() : nullptr, octree_ptr ? octree_ptr->depth() : 0);
	}
	if (blocking) {
		CUDA_CHECK_THROW(hipMemcpyAsync(countercpu, m_sdf.iou_counter.data(), 8*4, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	}

	return countercpu[4]/double(countercpu[5]);
}

NGP_NAMESPACE_END
