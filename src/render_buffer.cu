#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   render_buffer.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <tiny-cuda-nn/gpu_memory.h>

#ifdef NGP_GUI
#  ifdef _WIN32
#    include <GL/gl3w.h>
#  else
#    include <GL/glew.h>
#  endif
#  include <GLFW/glfw3.h>
#endif

#include <stb_image/stb_image.h>

#include <cuda_gl_interop.h>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

extern std::atomic<size_t> g_total_n_bytes_allocated;

void CudaSurface2D::free() {
	if (m_surface) {
		hipDestroySurfaceObject(m_surface);
	}
	m_surface = 0;
	if (m_array) {
		hipFreeArray(m_array);
		g_total_n_bytes_allocated -= m_size.prod() * sizeof(float4);
	}
	m_array = nullptr;
}

void CudaSurface2D::resize(const Vector2i& size) {
	if (size == m_size) {
		return;
	}

	free();

	m_size = size;

	hipChannelFormatDesc desc = hipCreateChannelDesc<float4>();
	CUDA_CHECK_THROW(hipMallocArray(&m_array, &desc, size.x(), size.y(), hipArraySurfaceLoadStore));

	g_total_n_bytes_allocated += m_size.prod() * sizeof(float4);

	struct hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(resource_desc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = m_array;
	CUDA_CHECK_THROW(hipCreateSurfaceObject(&m_surface, &resource_desc));
}

#ifdef NGP_GUI
GLTexture::~GLTexture() {
	m_cuda_mapping.reset();
	if (m_texture_id)
		glDeleteTextures(1, &m_texture_id);
}

GLuint GLTexture::texture() {
	if (!m_texture_id) {
		glGenTextures(1, &m_texture_id);
	}

	return m_texture_id;
}

hipSurfaceObject_t GLTexture::surface() {
	if (!m_cuda_mapping) {
		m_cuda_mapping = std::make_unique<CUDAMapping>(texture(), m_size);
	}
	return m_cuda_mapping->surface();
}

hipArray_t GLTexture::array() {
	if (!m_cuda_mapping) {
		m_cuda_mapping = std::make_unique<CUDAMapping>(texture(), m_size);
	}
	return m_cuda_mapping->array();
}

void GLTexture::blit_from_cuda_mapping() {
	if (!m_cuda_mapping || m_cuda_mapping->is_interop()) {
		return;
	}

	if (m_internal_format != GL_RGBA32F || m_format != GL_RGBA || m_is_8bit) {
		throw std::runtime_error{"Can only blit from CUDA mapping if the texture is RGBA float."};
	}

	const float* data_cpu = m_cuda_mapping->data_cpu();
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, m_size.x(), m_size.y(), 0, GL_RGBA, GL_FLOAT, data_cpu);
}

void GLTexture::load(const char* fname) {
	uint8_t* out; // width * height * RGBA
	int comp,width,height;
	out = stbi_load(fname, &width, &height, &comp, 4);
	if (!out) {
		throw std::runtime_error{std::string{stbi_failure_reason()}};
	}
	ScopeGuard mem_guard{[&]() { stbi_image_free(out); }};
	load(out, { width, height }, 4);
}

void GLTexture::load(const float* data, Vector2i new_size, int n_channels) {
	resize(new_size, n_channels, false);

	glBindTexture(GL_TEXTURE_2D, m_texture_id);
	glTexImage2D(GL_TEXTURE_2D, 0, m_internal_format, new_size.x(), new_size.y(), 0, m_format, GL_FLOAT, data);
}

void GLTexture::load(const uint8_t* data, Vector2i new_size, int n_channels) {
	resize(new_size, n_channels, true);

	glBindTexture(GL_TEXTURE_2D, m_texture_id);
	glTexImage2D(GL_TEXTURE_2D, 0, m_internal_format, new_size.x(), new_size.y(), 0, m_format, GL_UNSIGNED_BYTE, data);
}

void GLTexture::resize(const Vector2i& new_size, int n_channels, bool is_8bit) {
	if (m_size == new_size && m_n_channels == n_channels && m_is_8bit == is_8bit) {
		return;
	}

	if (m_texture_id) {
		m_cuda_mapping.reset();
		glDeleteTextures(1, &m_texture_id);
		m_texture_id = 0;
	}

	glGenTextures(1, &m_texture_id);
	glBindTexture(GL_TEXTURE_2D, m_texture_id);

	switch (n_channels) {
		case 1: m_internal_format = is_8bit ? GL_R8    : GL_R32F;    m_format = GL_RED;  break;
		case 2: m_internal_format = is_8bit ? GL_RG8   : GL_RG32F;   m_format = GL_RG;   break;
		case 3: m_internal_format = is_8bit ? GL_RGB8  : GL_RGB32F;  m_format = GL_RGB;  break;
		case 4: m_internal_format = is_8bit ? GL_RGBA8 : GL_RGBA32F; m_format = GL_RGBA; break;
		default: tlog::error() << "Unsupported number of channels: " << n_channels;
	}
	m_is_8bit = is_8bit;
	m_size = new_size;
	m_n_channels = n_channels;

	glTexImage2D(GL_TEXTURE_2D, 0, m_internal_format, new_size.x(), new_size.y(), 0, m_format, is_8bit ? GL_UNSIGNED_BYTE : GL_FLOAT, nullptr);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
}

GLTexture::CUDAMapping::CUDAMapping(GLuint texture_id, const Vector2i& size) : m_size{size} {
	static bool s_is_cuda_interop_supported = true;
	if (s_is_cuda_interop_supported) {
		hipError_t err = hipGraphicsGLRegisterImage(&m_graphics_resource, texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
		if (err != hipSuccess) {
			s_is_cuda_interop_supported = false;
			hipGetLastError(); // Reset error
		}
	}

	if (!s_is_cuda_interop_supported) {
		// falling back to a regular cuda surface + CPU copy of data
		m_cuda_surface = std::make_unique<CudaSurface2D>();
		m_cuda_surface->resize(size);
		m_data_cpu.resize(m_size.prod() * 4);
		return;
	}

	CUDA_CHECK_THROW(hipGraphicsMapResources(1, &m_graphics_resource));
	CUDA_CHECK_THROW(hipGraphicsSubResourceGetMappedArray(&m_mapped_array, m_graphics_resource, 0, 0));

	struct hipResourceDesc resource_desc;
	memset(&resource_desc, 0, sizeof(resource_desc));
	resource_desc.resType = hipResourceTypeArray;
	resource_desc.res.array.array = m_mapped_array;

	CUDA_CHECK_THROW(hipCreateSurfaceObject(&m_surface, &resource_desc));
}

GLTexture::CUDAMapping::~CUDAMapping() {
	if (m_surface) {
		hipDestroySurfaceObject(m_surface);
		hipGraphicsUnmapResources(1, &m_graphics_resource);
		hipGraphicsUnregisterResource(m_graphics_resource);
	}
}

const float* GLTexture::CUDAMapping::data_cpu() {
	CUDA_CHECK_THROW(hipMemcpy2DFromArray(m_data_cpu.data(), m_size.x() * sizeof(float) * 4, array(), 0, 0, m_size.x() * sizeof(float) * 4, m_size.y(), hipMemcpyDeviceToHost));
	return m_data_cpu.data();
}
#endif //NGP_GUI

__global__ void accumulate_kernel(Vector2i resolution, Array4f* frame_buffer, Array4f* accumulate_buffer, float sample_count, EColorSpace color_space) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	Array4f color = frame_buffer[idx];
	Array4f tmp = accumulate_buffer[idx];

	switch (color_space) {
		case EColorSpace::VisPosNeg:
			{
				float val = color.x() - color.y();
				float tmp_val = tmp.x() - tmp.y();

				tmp_val = (tmp_val * sample_count + val) / (sample_count+1);

				tmp.x() = fmaxf(tmp_val, 0.0f);
				tmp.y() = fmaxf(-tmp_val, 0.0f);
				break;
			}
		case EColorSpace::SRGB:
			color.head<3>() = linear_to_srgb(color.head<3>());
			// fallthrough is intended!
		case EColorSpace::Linear:
			tmp.head<3>() = (tmp.head<3>() * sample_count + color.head<3>()) / (sample_count+1); break;
	}

	tmp.w() = (tmp.w() * sample_count + color.w()) / (sample_count+1);

	accumulate_buffer[idx] = tmp;
}

__device__ Array3f tonemap(Array3f x, ETonemapCurve curve) {
	if (curve == ETonemapCurve::Identity) {
		return x;
	}

	x = x.cwiseMax(0.f);

	float k0, k1, k2, k3, k4, k5;
	if (curve == ETonemapCurve::ACES) {
		// Source:  ACES approximation : https://knarkowicz.wordpress.com/2016/01/06/aces-filmic-tone-mapping-curve/
		// Include pre - exposure cancelation in constants
		k0 = 0.6f * 0.6f * 2.51f;
		k1 = 0.6f * 0.03f;
		k2 = 0.0f;
		k3 = 0.6f * 0.6f * 2.43f;
		k4 = 0.6f * 0.59f;
		k5 = 0.14f;
	} else if (curve == ETonemapCurve::Hable) {
		// Source: https://64.github.io/tonemapping/
		const float A = 0.15f;
		const float B = 0.50f;
		const float C = 0.10f;
		const float D = 0.20f;
		const float E = 0.02f;
		const float F = 0.30f;
		k0 = A * F - A * E;
		k1 = C * B * F - B * E;
		k2 = 0.0f;
		k3 = A * F;
		k4 = B * F;
		k5 = D * F * F;

		const float W = 11.2f;
		const float nom = k0 * (W*W) + k1 * W + k2;
		const float denom = k3 * (W*W) + k4 * W + k5;
		const float white_scale = denom / nom;

		// Include white scale and exposure bias in rational polynomial coefficients
		k0 = 4.0f * k0 * white_scale;
		k1 = 2.0f * k1 * white_scale;
		k2 = k2 * white_scale;
		k3 = 4.0f * k3;
		k4 = 2.0f * k4;
	} else { //if (curve == ETonemapCurve::Reinhard)
		const Vector3f luminance_coefficients = Vector3f(0.2126f, 0.7152f, 0.0722f);
		float Y = luminance_coefficients.dot(x.matrix());

		return x * (1.f / (Y + 1.0f));
	}

	Array3f color_sq = x * x;
	Array3f nom = color_sq * k0 + k1 * x + k2;
	Array3f denom = k3 * color_sq + k4 * x + k5;

	Array3f tonemapped_color = nom / denom;

	return tonemapped_color;
}

__device__ Array3f tonemap(Array3f col, const Array3f& exposure, ETonemapCurve tonemap_curve, EColorSpace color_space, EColorSpace output_color_space) {
	// Conversion to output by
	// 1. converting to linear. (VisPosNeg is treated as linear red/green)
	if (color_space == EColorSpace::SRGB) {
		col = srgb_to_linear(col);
	}

	// 2. applying exposure in linear space
	col *= Array3f::Constant(2.0f).pow(exposure);

	// 3. tonemapping in linear space according to the specified curve
	col = tonemap(col, tonemap_curve);

	// 4. converting to output color space.
	if (output_color_space == EColorSpace::SRGB) {
		col = linear_to_srgb(col);
	}

	return col;
}

__global__ void overlay_image_kernel(
	Vector2i resolution,
	float alpha,
	Array3f exposure,
	Array4f background_color,
	const void* __restrict__ image,
	EImageDataType image_data_type,
	Vector2i image_resolution,
	ETonemapCurve tonemap_curve,
	EColorSpace color_space,
	EColorSpace output_color_space,
	int fov_axis,
	float zoom,
	Eigen::Vector2f screen_center,
	hipSurfaceObject_t surface
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	float scale = image_resolution[fov_axis] / float(resolution[fov_axis]);

	float fx = x+0.5f;
	float fy = y+0.5f;

	fx-=resolution.x()*0.5f; fx/=zoom; fx+=screen_center.x() * resolution.x();
	fy-=resolution.y()*0.5f; fy/=zoom; fy+=screen_center.y() * resolution.y();

	float u = (fx-resolution.x()*0.5f) * scale  + image_resolution.x()*0.5f;
	float v = (fy-resolution.y()*0.5f) * scale  + image_resolution.y()*0.5f;

	int srcx = floorf(u);
	int srcy = floorf(v);
	uint32_t idx = x + resolution.x() * y;
	uint32_t srcidx = srcx + image_resolution.x() * srcy;

	Array4f val;
	if (srcx >= image_resolution.x() || srcy >= image_resolution.y() || srcx < 0 || srcy < 0) {
		val = Array4f::Zero();
	} else {
		val = read_rgba(Vector2i{srcx, srcy}, image_resolution, image, image_data_type);
	}

	Array4f color = {val[0], val[1], val[2], val[3]};

	// The background color is represented in SRGB, so convert
	// to linear if that's not the space in which we're rendering.
	if (color_space != EColorSpace::SRGB) {
		background_color.head<3>() = srgb_to_linear(background_color.head<3>());
	} else {
		if (color.w() > 0) {
			color.head<3>() = linear_to_srgb(color.head<3>() / color.w()) * color.w();
		} else {
			color.head<3>() = Array3f::Zero();
		}
	}

	float weight = (1 - color.w()) * background_color.w();
	color.head<3>() += background_color.head<3>() * weight;
	color.w() += weight;

	color.head<3>() = tonemap(color.head<3>(), exposure, tonemap_curve, color_space, output_color_space);

	Array4f prev_color;
	surf2Dread((float4*)&prev_color, surface, x * sizeof(float4), y);
	color = color * alpha + prev_color * (1.f-alpha);
	surf2Dwrite(to_float4(color), surface, x * sizeof(float4), y);
}

__device__ Array3f colormap_turbo(float x) {
	const Vector4f kRedVec4 =   Vector4f(0.13572138f, 4.61539260f, -42.66032258f, 132.13108234f);
	const Vector4f kGreenVec4 = Vector4f(0.09140261f, 2.19418839f, 4.84296658f, -14.18503333f);
	const Vector4f kBlueVec4 =  Vector4f(0.10667330f, 12.64194608f, -60.58204836f, 110.36276771f);
	const Vector2f kRedVec2 =   Vector2f(-152.94239396f, 59.28637943f);
	const Vector2f kGreenVec2 = Vector2f(4.27729857f, 2.82956604f);
	const Vector2f kBlueVec2 =  Vector2f(-89.90310912f, 27.34824973f);

	x = __saturatef(x);
	Vector4f v4 = Vector4f{ 1.0f, x, x * x, x * x * x };
	Vector2f v2 = Vector2f{ v4.w() * x, v4.w() * v4.z() };
	return Array3f{
		v4.dot(kRedVec4)   + v2.dot(kRedVec2),
		v4.dot(kGreenVec4) + v2.dot(kGreenVec2),
		v4.dot(kBlueVec4)  + v2.dot(kBlueVec2)
	};
}

__device__ Array3f colormap_viridis(float x) {
	const Array3f c0 = Array3f{0.2777273272234177f, 0.005407344544966578f, 0.3340998053353061f};
	const Array3f c1 = Array3f{0.1050930431085774f, 1.404613529898575f, 1.384590162594685f};
	const Array3f c2 = Array3f{-0.3308618287255563f, 0.214847559468213f, 0.09509516302823659f};
	const Array3f c3 = Array3f{-4.634230498983486f, -5.799100973351585f, -19.33244095627987f};
	const Array3f c4 = Array3f{6.228269936347081f, 14.17993336680509f, 56.69055260068105f};
	const Array3f c5 = Array3f{4.776384997670288f, -13.74514537774601f, -65.35303263337234f};
	const Array3f c6 = Array3f{-5.435455855934631f, 4.645852612178535f, 26.3124352495832f};
	x = __saturatef(x);
	return (c0+x*(c1+x*(c2+x*(c3+x*(c4+x*(c5+x*c6))))));
}

__global__ void overlay_false_color_kernel(Vector2i resolution, Vector2i training_resolution, bool to_srgb, int fov_axis, hipSurfaceObject_t surface, const float *error_map, Vector2i error_map_resolution, const float *average, float brightness, bool viridis) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	float error_map_scale = brightness/(0.0000001f+average[0]); // average maps to 1/16th

	float scale = training_resolution[fov_axis] / float(resolution[fov_axis]);
	float u = (x+0.5f-resolution.x()*0.5f) * scale + training_resolution.x()*0.5f;
	float v = (y+0.5f-resolution.y()*0.5f) * scale + training_resolution.y()*0.5f;
	int srcx = floorf(u * error_map_resolution.x() / float(max(1.f, (float)training_resolution.x())));
	int srcy = floorf(v * error_map_resolution.y() / float(max(1.f, (float)training_resolution.y())));

	uint32_t idx = x + resolution.x() * y;
	uint32_t srcidx = srcx + error_map_resolution.x() * srcy;

	if (srcx >= error_map_resolution.x() || srcy >= error_map_resolution.y() || srcx<0 || srcy<0) {
		return;
	}

	float err = error_map[srcidx] * error_map_scale;
	if (viridis) {
		err *= 1.f / (1.f+err);
	}
	Array4f color;
	surf2Dread((float4*)&color, surface, x * sizeof(float4), y);
	Array3f c = viridis ? colormap_viridis(err) : colormap_turbo(err);
	float grey = color.x() * 0.2126f + color.y() * 0.7152f + color.z() * 0.0722f;
	color.x() = grey*__saturatef(c.x());
	color.y() = grey*__saturatef(c.y());
	color.z() = grey*__saturatef(c.z());

	surf2Dwrite(to_float4(color), surface, x * sizeof(float4), y);
}

__global__ void tonemap_kernel(Vector2i resolution, float exposure, Array4f background_color, Array4f* accumulate_buffer, EColorSpace color_space, EColorSpace output_color_space, ETonemapCurve tonemap_curve, bool clamp_output_color, hipSurfaceObject_t surface) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	// The background color is represented in SRGB, so convert
	// to linear if that's not the space in which we're rendering.
	if (color_space != EColorSpace::SRGB) {
		background_color.head<3>() = srgb_to_linear(background_color.head<3>());
	}

	Array4f color = accumulate_buffer[idx];
	float weight = (1 - color.w()) * background_color.w();
	color.head<3>() += background_color.head<3>() * weight;
	color.w() += weight;

	color.head<3>() = tonemap(color.head<3>(), Array3f::Constant(exposure), tonemap_curve, color_space, output_color_space);
	if (clamp_output_color) {
		color = color.cwiseMax(0.0f).cwiseMin(1.0f);
	}

	surf2Dwrite(to_float4(color), surface, x * sizeof(float4), y);
}

__global__ void dlss_splat_kernel(
	Vector2i resolution,
	hipSurfaceObject_t dlss_surface,
	hipSurfaceObject_t surface
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	float4 color;
	surf2Dread(&color, dlss_surface, x * sizeof(float4), y);
	surf2Dwrite(color, surface, x * sizeof(float4), y);
}

void CudaRenderBuffer::resize(const Vector2i& res) {
	m_in_resolution = res;
	m_frame_buffer.enlarge(res.x() * res.y());
	m_depth_buffer.enlarge(res.x() * res.y());
	m_accumulate_buffer.enlarge(res.x() * res.y());

	Vector2i out_res = m_dlss ? m_dlss->out_resolution() : res;
	auto prev_out_res = out_resolution();
	m_surface_provider->resize(out_res);

	if (out_resolution() != prev_out_res) {
		reset_accumulation();
	}
}

void CudaRenderBuffer::clear_frame(hipStream_t stream) {
	CUDA_CHECK_THROW(hipMemsetAsync(m_frame_buffer.data(), 0, m_frame_buffer.bytes(), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_depth_buffer.data(), 0, m_depth_buffer.bytes(), stream));
}

void CudaRenderBuffer::accumulate(float exposure, hipStream_t stream) {
	Vector2i res = in_resolution();

	uint32_t accum_spp = m_dlss ? 0 : m_spp;

	if (accum_spp == 0) {
		CUDA_CHECK_THROW(hipMemsetAsync(m_accumulate_buffer.data(), 0, m_accumulate_buffer.bytes(), stream));
	}

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
	accumulate_kernel<<<blocks, threads, 0, stream>>>(
		res,
		frame_buffer(),
		accumulate_buffer(),
		(float)accum_spp,
		m_color_space
	);

	++m_spp;
}

void CudaRenderBuffer::tonemap(float exposure, const Array4f& background_color, EColorSpace output_color_space, hipStream_t stream) {
	assert(m_dlss || out_resolution() == in_resolution());

	auto res = m_dlss ? in_resolution() : out_resolution();
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
	tonemap_kernel<<<blocks, threads, 0, stream>>>(
		res,
		exposure,
		background_color,
		accumulate_buffer(),
		m_color_space,
		output_color_space,
		m_tonemap_curve,
		m_dlss && output_color_space == EColorSpace::SRGB,
		m_dlss ? m_dlss->frame() : surface()
	);

	if (m_dlss) {
		assert(out_resolution() == m_dlss->out_resolution());

		assert(m_spp >= 1);
		uint32_t sample_index = m_spp - 1;

		m_dlss->run(
			res,
			output_color_space == EColorSpace::Linear, /* HDR mode */
			m_dlss_sharpening,
			Vector2f::Constant(0.5f) - ld_random_pixel_offset(sample_index), /* jitter offset in [-0.5, 0.5] */
			sample_index == 0 /* reset history */
		);

		auto out_res = out_resolution();
		const dim3 out_blocks = { div_round_up((uint32_t)out_res.x(), threads.x), div_round_up((uint32_t)out_res.y(), threads.y), 1 };
		dlss_splat_kernel<<<out_blocks, threads, 0, stream>>>(out_res, m_dlss->output(), surface());
	}
}

void CudaRenderBuffer::overlay_image(
	float alpha,
	const Eigen::Array3f& exposure,
	const Array4f& background_color,
	EColorSpace output_color_space,
	const void* __restrict__ image,
	EImageDataType image_data_type,
	const Vector2i& image_resolution,
	int fov_axis,
	float zoom,
	const Eigen::Vector2f& screen_center,
	hipStream_t stream
) {
	auto res = out_resolution();
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
	overlay_image_kernel<<<blocks, threads, 0, stream>>>(
		res,
		alpha,
		exposure,
		background_color,
		image,
		image_data_type,
		image_resolution,
		m_tonemap_curve,
		m_color_space,
		output_color_space,
		fov_axis,
		zoom,
		screen_center,
		surface()
	);
}

void CudaRenderBuffer::overlay_false_color(Vector2i training_resolution, bool to_srgb, int fov_axis, hipStream_t stream, const float* error_map, Vector2i error_map_resolution, const float* average, float brightness, bool viridis) {
	auto res = out_resolution();
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
	overlay_false_color_kernel<<<blocks, threads, 0, stream>>>(
		res,
		training_resolution,
		to_srgb,
		fov_axis,
		surface(),
		error_map,
		error_map_resolution,
		average,
		brightness,
		viridis
	);
}

void CudaRenderBuffer::enable_dlss(const Eigen::Vector2i& out_res) {
#ifdef NGP_VULKAN
	if (!m_dlss || m_dlss->out_resolution() != out_res) {
		m_dlss = dlss_init(out_res);
	}
	resize(in_resolution());
#else
	throw std::runtime_error{"NGP was compiled without Vulkan/NGX/DLSS support."};
#endif
}

void CudaRenderBuffer::disable_dlss() {
	m_dlss = nullptr;
}

NGP_NAMESPACE_END
