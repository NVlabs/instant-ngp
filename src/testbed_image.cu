#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed_image.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/random_val.cuh>
#include <neural-graphics-primitives/testbed.h>

#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/trainer.h>

#include <fstream>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

Testbed::NetworkDims Testbed::network_dims_image() const {
	NetworkDims dims;
	dims.n_input = 2;
	dims.n_output = 3;
	dims.n_pos = 2;
	return dims;
}

__global__ void halton23_kernel(uint32_t n_elements, size_t base_idx, Vector2f* __restrict__ output) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	output[i] = {halton<2>(base_idx+i), halton<3>(base_idx+i)};
}

__global__ void sobol2_kernel(uint32_t n_elements, size_t base_idx, uint32_t seed, Vector2f* __restrict__ output) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	output[i] = ld_random_val_2d(base_idx + i, seed);
}

__global__ void zip_kernel(uint32_t n_elements, const float* __restrict__ in, Vector2f* __restrict__ output) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	output[i] = {in[i], in[i+n_elements]};
}

__global__ void stratify2_kernel(uint32_t n_elements, uint32_t log2_batch_size, Vector2f* __restrict__ inout) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t log2Size = log2_batch_size / 2;
	uint32_t size = 1 << log2Size;

	uint32_t in_batch_index = i & ((1 << log2_batch_size)-1);

	uint32_t x = in_batch_index & ((1 << log2Size)-1);
	uint32_t y = in_batch_index >> log2Size;

	Vector2f val = inout[i];
	inout[i] = {val.x() / size + ((float)x/size), val.y() / size + ((float)y/size)};
}

__global__ void init_image_coords(
	Vector2f* __restrict__ positions,
	Vector2i resolution,
	Vector2i image_resolution,
	float view_dist,
	Vector2f image_pos,
	Vector2f screen_center,
	bool snap_to_pixel_centers,
	uint32_t sample_index
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;
	positions[idx] = pixel_to_image_uv(
		sample_index,
		{x, y},
		resolution,
		image_resolution,
		screen_center,
		view_dist,
		image_pos,
		snap_to_pixel_centers
	);
}

// #define COLOR_SPACE_CONVERT convert to ycrcb experiment - causes some color shift tho it does lead to very slightly sharper edges. not a net win if you like colors :)
#define CHROMA_SCALE 0.2f

__global__ void colorspace_convert_image_half(Vector2i resolution, const char* __restrict__ texture) {
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= resolution.x() || y >= resolution.y()) return;
	__half val[4];
	*(int2*)&val[0] = ((int2*)texture)[y * resolution.x() + x];
	float R=val[0],G=val[1],B=val[2];
	val[0]=(0.2126f * R + 0.7152f * G + 0.0722f * B);
	val[1]=((-0.1146f * R - 0.3845f * G + 0.5f * B)+0.f)*CHROMA_SCALE;
	val[2]=((0.5f * R - 0.4542f * G - 0.0458f * B)+0.f)*CHROMA_SCALE;
	((int2*)texture)[y * resolution.x() + x] = *(int2*)&val[0];
}

__global__ void colorspace_convert_image_float(Vector2i resolution, const char* __restrict__ texture) {
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	if (x >= resolution.x() || y >= resolution.y()) return;
	float val[4];
	*(float4*)&val[0] = ((float4*)texture)[y * resolution.x() + x];
	float R=val[0],G=val[1],B=val[2];
	val[0]=(0.2126f * R + 0.7152f * G + 0.0722f * B);
	val[1]=((-0.1146f * R - 0.3845f * G + 0.5f * B)+0.f)*CHROMA_SCALE;
	val[2]=((0.5f * R - 0.4542f * G - 0.0458f * B)+0.f)*CHROMA_SCALE;
	((float4*)texture)[y * resolution.x() + x] = *(float4*)&val[0];
}

__global__ void shade_kernel_image(Vector2i resolution, const Vector2f* __restrict__ positions, const Array3f* __restrict__ colors, Array4f* __restrict__ frame_buffer, float* __restrict__ depth_buffer, bool linear_colors) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	const Vector2f uv = positions[idx];
	if (uv.x() < 0.0f || uv.x() > 1.0f || uv.y() < 0.0f || uv.y() > 1.0f) {
		frame_buffer[idx] = Array4f::Zero();
		depth_buffer[idx] = 1e10f;
		return;
	}

	Array3f color = colors[idx];

	if (!linear_colors) {
		color = srgb_to_linear(color);
	}

#ifdef COLOR_SPACE_CONVERT
	float Y=color.x(), Cb =color.y()*(1.f/CHROMA_SCALE) -0.f, Cr = color.z() * (1.f/CHROMA_SCALE) - 0.f;
	float R = Y                + 1.5748f * Cr;
	float G = Y - 0.1873f * Cb - 0.4681 * Cr;
	float B = Y + 1.8556f * Cb;
	frame_buffer[idx] = {R, G, B, 1.0f};
#else
	frame_buffer[idx] = {color.x(), color.y(), color.z(), 1.0f};
#endif
	depth_buffer[idx] = 1.0f;
}

template <typename T, uint32_t stride>
__global__ void eval_image_kernel_and_snap(uint32_t n_elements, const T* __restrict__ texture, Vector2f* __restrict__ positions, Vector2i resolution, float* __restrict__ result, bool snap_to_pixel_centers, bool linear_colors) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t output_idx = i * stride;

	Vector2f pos = positions[i];

	auto read_val = [&](int x, int y) {
		auto val = ((tcnn::vector_t<T, 4>*)texture)[y * resolution.x() + x];
		Array4f result{val[0], val[1], val[2], val[3]};
		if (!linear_colors) {
			result.head<3>() = linear_to_srgb(result.head<3>());
		}
		return result;
	};

	Array4f val;
	if (snap_to_pixel_centers) {
		Vector2i pos_int = pos.cwiseProduct(resolution.cast<float>()).array().floor().matrix().cast<int>();
		positions[i] = (pos_int.cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
		pos_int = pos_int.cwiseMax(0).cwiseMin(resolution - Vector2i::Ones());
		val = read_val(pos_int.x(), pos_int.y());
	} else {
		pos = (pos.cwiseProduct(resolution.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(resolution.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));

		const Vector2i pos_int = pos.cast<int>();
		const Vector2f weight = pos - pos_int.cast<float>();

		const Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

		val =
			(1 - weight.x()) * (1 - weight.y()) * read_val(idx.x(), idx.y()) +
			(weight.x()) * (1 - weight.y()) * read_val(idx.x()+1, idx.y()) +
			(1 - weight.x()) * (weight.y()) * read_val(idx.x(), idx.y()+1) +
			(weight.x()) * (weight.y()) * read_val(idx.x()+1, idx.y()+1);
	}

	result[output_idx + 0] = val.x();
	result[output_idx + 1] = val.y();
	result[output_idx + 2] = val.z();

	for (uint32_t i = 3; i < stride; ++i) {
		result[output_idx + i] = 1;
	}
}

void Testbed::train_image(size_t target_batch_size, bool get_loss_scalar, hipStream_t stream) {
	const uint32_t n_output_dims = 3;
	const uint32_t n_input_dims = 2;

	// Auxiliary matrices for training
	const uint32_t batch_size = (uint32_t)target_batch_size;

	// Permute all training records to de-correlate training data

	const uint32_t n_elements = batch_size;
	m_image.training.positions.enlarge(n_elements);
	m_image.training.targets.enlarge(n_elements);

	if (m_image.random_mode == ERandomMode::Halton) {
		linear_kernel(halton23_kernel, 0, stream, n_elements, (size_t)batch_size * m_training_step, m_image.training.positions.data());
	} else if (m_image.random_mode == ERandomMode::Sobol) {
		linear_kernel(sobol2_kernel, 0, stream, n_elements, (size_t)batch_size * m_training_step, m_seed, m_image.training.positions.data());
	} else {
		generate_random_uniform<float>(stream, m_rng, n_elements * n_input_dims, (float*)m_image.training.positions.data());
		if (m_image.random_mode == ERandomMode::Stratified) {
			uint32_t log2_batch_size = 0;
			if (!is_pot(batch_size, &log2_batch_size)) {
				tlog::warning() << "Can't stratify a non-pot batch size";
			} else if (log2_batch_size % 2 != 0) {
				tlog::warning() << "Can't stratify a non-square batch size";
			} else {
				linear_kernel(stratify2_kernel, 0, stream, n_elements, log2_batch_size, m_image.training.positions.data());
			}
		}
	}

	if (m_image.type == EDataType::Float) {
		linear_kernel(eval_image_kernel_and_snap<float, 3>, 0, stream,
			n_elements,
			(float*)m_image.data.data(),
			m_image.training.positions.data(),
			m_image.resolution,
			(float*)m_image.training.targets.data(),
			m_image.training.snap_to_pixel_centers,
			m_image.training.linear_colors
		);
	} else {
		linear_kernel(eval_image_kernel_and_snap<__half, 3>, 0, stream,
			n_elements,
			(__half*)m_image.data.data(),
			m_image.training.positions.data(),
			m_image.resolution,
			(float*)m_image.training.targets.data(),
			m_image.training.snap_to_pixel_centers,
			m_image.training.linear_colors
		);
	}

	GPUMatrix<float> training_batch_matrix((float*)(m_image.training.positions.data()), n_input_dims, batch_size);
	GPUMatrix<float> training_target_matrix((float*)(m_image.training.targets.data()), n_output_dims, batch_size);

	auto ctx = m_trainer->training_step(stream, training_batch_matrix, training_target_matrix);
	m_training_step++;

	if (get_loss_scalar) {
		m_loss_scalar.update(m_trainer->loss(stream, *ctx));
	}
}

void Testbed::render_image(CudaRenderBuffer& render_buffer, hipStream_t stream) {
	auto res = render_buffer.in_resolution();

	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)res.x() * res.y();
	uint32_t n_elements = next_multiple((uint32_t)n_pixels, tcnn::batch_size_granularity);
	m_image.render_coords.enlarge(n_elements);
	m_image.render_out.enlarge(n_elements);

	// Generate 2D coords at which to query the network
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
	init_image_coords<<<blocks, threads, 0, stream>>>(
		m_image.render_coords.data(),
		res,
		m_image.resolution,
		m_scale,
		m_image.pos,
		m_screen_center - Vector2f::Constant(0.5f),
		m_snap_to_pixel_centers,
		render_buffer.spp()
	);

	// Obtain colors for each 2D coord
	if (m_image.type == EDataType::Float) {
		linear_kernel(eval_image_kernel_and_snap<float, 3>, 0, stream,
			n_elements,
			(float*)m_image.data.data(),
			m_image.render_coords.data(),
			m_image.resolution,
			(float*)m_image.render_out.data(),
			m_image.training.snap_to_pixel_centers,
			m_image.training.linear_colors
		);
	} else {
		linear_kernel(eval_image_kernel_and_snap<__half, 3>, 0, stream,
			n_elements,
			(__half*)m_image.data.data(),
			m_image.render_coords.data(),
			m_image.resolution,
			(float*)m_image.render_out.data(),
			m_image.training.snap_to_pixel_centers,
			m_image.training.linear_colors
		);
	}

	if (!m_render_ground_truth) {
		if (m_visualized_dimension >= 0) {
			GPUMatrix<float> positions_matrix((float*)m_image.render_coords.data(), 2, n_elements);
			GPUMatrix<float> colors_matrix((float*)m_image.render_out.data(), 3, n_elements);
			m_network->visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, colors_matrix);
		} else {
			GPUMatrix<float> positions_matrix((float*)m_image.render_coords.data(), 2, n_elements);
			GPUMatrix<float> colors_matrix((float*)m_image.render_out.data(), 3, n_elements);
			m_network->inference(stream, positions_matrix, colors_matrix);
		}
	}

	// Splat colors to render texture
	shade_kernel_image<<<blocks, threads, 0, stream>>>(
		res,
		m_image.render_coords.data(),
		m_image.render_out.data(),
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer(),
		m_image.training.linear_colors
	);
}

void Testbed::load_image() {
	if (equals_case_insensitive(m_data_path.extension(), "exr")) {
		load_exr_image();
	} else if (equals_case_insensitive(m_data_path.extension(), "bin")) {
		load_binary_image();
	} else {
		load_stbi_image();
	}

#ifdef COLOR_SPACE_CONVERT
	const dim3 threads = { 32, 32, 1 };
	const dim3 blocks = { div_round_up((uint32_t)m_image.resolution.x(), threads.x), div_round_up((uint32_t)m_image.resolution.x(), threads.y), 1 };
	if (m_image.type == EDataType::Half)
		colorspace_convert_image_half<<<blocks, threads, 0>>>(m_image.resolution, m_image.data.data());
	else
		colorspace_convert_image_float<<<blocks, threads, 0>>>(m_image.resolution, m_image.data.data());
#endif

	tlog::success()
		<< "Loaded a " << (m_image.type == EDataType::Half ? "half" : "full") << "-precision image with "
		<< m_image.resolution.x() << "x" << m_image.resolution.y() << " pixels.";
}

void Testbed::load_exr_image() {
	if (!m_data_path.exists()) {
		throw std::runtime_error{m_data_path.str() + " does not exist."};
	}

	tlog::info() << "Loading EXR image from " << m_data_path;

	// First step: load an image that we'd like to learn
	GPUMemory<float> image = load_exr(m_data_path.str(), m_image.resolution.x(), m_image.resolution.y());
	m_image.data.resize(image.size() * sizeof(float));
	CUDA_CHECK_THROW(hipMemcpy(m_image.data.data(), image.data(), image.size() * sizeof(float), hipMemcpyDeviceToDevice));

	m_image.type = EDataType::Float;
}

void Testbed::load_stbi_image() {
	if (!m_data_path.exists()) {
		throw std::runtime_error{m_data_path.str() + " does not exist."};
	}

	tlog::info() << "Loading STBI image from " << m_data_path;

	// First step: load an image that we'd like to learn
	GPUMemory<float> image = load_stbi(m_data_path.str(), m_image.resolution.x(), m_image.resolution.y());
	m_image.data.resize(image.size() * sizeof(float));
	CUDA_CHECK_THROW(hipMemcpy(m_image.data.data(), image.data(), image.size() * sizeof(float), hipMemcpyDeviceToDevice));

	m_image.type = EDataType::Float;
}


void Testbed::load_binary_image() {
	if (!m_data_path.exists()) {
		throw std::runtime_error{m_data_path.str() + " does not exist."};
	}

	tlog::info() << "Loading binary image from " << m_data_path;

	std::ifstream f(m_data_path.str(), std::ios::in | std::ios::binary);
	f.read(reinterpret_cast<char*>(&m_image.resolution.y()), sizeof(int));
	f.read(reinterpret_cast<char*>(&m_image.resolution.x()), sizeof(int));

	size_t n_pixels = (size_t)m_image.resolution.x() * m_image.resolution.y();
	m_image.data.resize(n_pixels * 4 * sizeof(__half));

	// Can directly copy to GPU memory!
	// TODO: uncomment once GDS works everywhere
	// {
	// 	int fd = open(m_data_path.string().c_str(), O_DIRECT);

	// 	CUfileError_t status;
	// 	CUfileDescr_t cf_descr;
	// 	CUfileHandle_t cf_handle;
	// 	memset((void *)&cf_descr, 0, sizeof(CUfileDescr_t));
	// 	cf_descr.handle.fd = fd;
	// 	cf_descr.type = CU_FILE_HANDLE_TYPE_OPAQUE_FD;
	// 	status = cuFileHandleRegister(&cf_handle, &cf_descr);
	// 	if (status.err != CU_FILE_SUCCESS) {
	// 		close(fd);
	// 		throw std::runtime_error{std::string{"cuFileHandleRegister fd "} + std::to_string(fd) + " status " + status.err};
	// 	}

	// 	status = cuFileBufRegister(m_image.data.data(), m_image.data.get_bytes(), 0);
	// 	if (status.err != CU_FILE_SUCCESS) {
	// 		cuFileHandleDeregister(cf_handle);
	// 		close(fd);
	// 		throw std::runtime_error{std::string{"buffer registration failed "} + status.err};
	// 	}

	// 	cuFileRead(cf_handle, m_image.data.data(), m_image.data.get_bytes(), 2 * sizeof(int), 0);

	// 	status = cuFileBufDeregister(devPtr_base);
	// 	if (status.err != CU_FILE_SUCCESS) {
	// 		cuFileHandleDeregister(cf_handle);
	// 		close(fd);
	// 		throw std::runtime_error{std::string{"buffer deregistration failed "} + status.err};
	// 	}

	// 	cuFileHandleDeregister(cf_handle);
	// 	close(fd);
	// }

	std::vector<__half> image(n_pixels * 4);
	f.read(reinterpret_cast<char*>(image.data()), sizeof(__half) * image.size());
	CUDA_CHECK_THROW(hipMemcpy(m_image.data.data(), image.data(), image.size() * sizeof(__half), hipMemcpyHostToDevice));
	m_image.type = EDataType::Half;
}

__global__ void image_coords_from_idx(const uint32_t n_elements, uint32_t offset, Vector2f* __restrict__ pos, Vector2i resolution) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	const uint32_t idx = i + offset;

	int x = idx % resolution.x();
	int y = idx / resolution.x();

	pos[i] = (Vector2i{x, y}.cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
}

__global__ void image_mse_kernel(const uint32_t n_elements, const Array3f* __restrict__ target, const Array3f* __restrict__ prediction, float* __restrict__ result, bool quantize_to_byte) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	Array3f pred = prediction[i];
	if (quantize_to_byte) {
		pred = (pred * 255.0f + Array3f::Constant(0.5f)).cast<int>().cwiseMax(0).cwiseMin(255).cast<float>() / 255.0f;
	}

	const Array3f diff = target[i] - pred;
	result[i] = (diff * diff).mean();
}

float Testbed::compute_image_mse(bool quantize_to_byte) {
	const uint32_t n_output_dims = 3;
	const uint32_t n_input_dims = 2;

	// Auxiliary matrices for training
	const uint32_t n_elements = m_image.resolution.prod();
	const uint32_t max_batch_size = 1u<<20;

	GPUMemory<float> se(n_elements);
	GPUMemory<Vector2f> pos(max_batch_size);
	GPUMemory<Array3f> targets(max_batch_size);
	GPUMemory<Array3f> predictions(max_batch_size);
	const uint32_t n_batches = div_round_up(n_elements, max_batch_size);
	for (uint32_t i = 0; i < n_batches; ++i) {
		uint32_t offset = i * max_batch_size;
		uint32_t batch_size = (std::min(max_batch_size, n_elements - offset) + 255u ) & (~255u);

		GPUMatrix<float> pos_matrix((float*)(pos.data()), n_input_dims, batch_size);
		GPUMatrix<float> targets_matrix((float*)(targets.data()), n_output_dims, batch_size);
		GPUMatrix<float> predictions_matrix((float*)(predictions.data()), n_output_dims, batch_size);

		linear_kernel(image_coords_from_idx, 0, nullptr,
			batch_size,
			offset,
			pos.data(),
			m_image.resolution
		);

		if (m_image.type == EDataType::Float) {
			linear_kernel(eval_image_kernel_and_snap<float, 3>, 0, nullptr,
				batch_size,
				(float*)m_image.data.data(),
				pos.data(),
				m_image.resolution,
				(float*)targets.data(),
				true,
				m_image.training.linear_colors
			);
		} else {
			linear_kernel(eval_image_kernel_and_snap<__half, 3>, 0, nullptr,
				batch_size,
				(__half*)m_image.data.data(),
				pos.data(),
				m_image.resolution,
				(float*)targets.data(),
				true,
				m_image.training.linear_colors
			);
		}

		m_network->inference(pos_matrix, predictions_matrix);

		linear_kernel(image_mse_kernel, 0, nullptr,
			batch_size,
			targets.data(),
			predictions.data(),
			se.data() + offset,
			quantize_to_byte
		);
	}

	return reduce_sum(se.data(), n_elements, nullptr) / n_elements;
}

NGP_NAMESPACE_END
