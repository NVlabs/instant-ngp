#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   marchingcubes.cu
 *  @author Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/bounding_box.cuh>
#include <neural-graphics-primitives/random_val.cuh> // helpers to generate random values, directions
#include <neural-graphics-primitives/thread_pool.h>

#include <tiny-cuda-nn/gpu_memory.h>
#include <filesystem/path.h>

#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <stb_image/stb_image_write.h>
#include <stdarg.h>

#ifdef NGP_GUI
#  ifdef _WIN32
#    include <GL/gl3w.h>
#  else
#    include <GL/glew.h>
#  endif
#  include <GLFW/glfw3.h>
#  include <cuda_gl_interop.h>
#endif

#include <vector>

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

Vector3i get_marching_cubes_res(uint32_t res_1d, const BoundingBox &aabb) {
	float scale = res_1d / (aabb.max - aabb.min).maxCoeff();
	Vector3i res3d = ((aabb.max - aabb.min) * scale + Vector3f::Constant(0.5f)).cast<int>();
	res3d.x() = next_multiple((unsigned int)res3d.x(), 16u);
	res3d.y() = next_multiple((unsigned int)res3d.y(), 16u);
	res3d.z() = next_multiple((unsigned int)res3d.z(), 16u);
	return res3d;
}

#ifdef NGP_GUI

void glCheckError(const char* file, unsigned int line) {
  GLenum errorCode = glGetError();
  while (errorCode != GL_NO_ERROR) {
    std::string fileString(file);
    std::string error = "unknown error";
    // clang-format off
    switch (errorCode) {
      case GL_INVALID_ENUM:      error = "GL_INVALID_ENUM"; break;
      case GL_INVALID_VALUE:     error = "GL_INVALID_VALUE"; break;
      case GL_INVALID_OPERATION: error = "GL_INVALID_OPERATION"; break;
      case GL_STACK_OVERFLOW:    error = "GL_STACK_OVERFLOW"; break;
      case GL_STACK_UNDERFLOW:   error = "GL_STACK_UNDERFLOW"; break;
      case GL_OUT_OF_MEMORY:     error = "GL_OUT_OF_MEMORY"; break;
    }
    // clang-format on

    tlog::error() << "OpenglError : file=" << file << " line=" << line << " error:" << error;
    errorCode = glGetError();
  }
}


bool check_shader(GLuint handle, const char* desc, bool program) {
	GLint status = 0, log_length = 0;
	if (program) {
		glGetProgramiv(handle, GL_LINK_STATUS, &status);
		glGetProgramiv(handle, GL_INFO_LOG_LENGTH, &log_length);
	} else {
		glGetShaderiv(handle, GL_COMPILE_STATUS, &status);
		glGetShaderiv(handle, GL_INFO_LOG_LENGTH, &log_length);
	}
	if ((GLboolean)status == GL_FALSE) {
		tlog::error() << "Failed to compile shader: " << desc;
	}
	if (log_length > 1) {
		std::vector<char> log; log.resize(log_length+1);
		if (program) {
			glGetProgramInfoLog(handle, log_length, NULL, (GLchar*)log.data());
		} else {
			glGetShaderInfoLog(handle, log_length, NULL, (GLchar*)log.data());
		}
		log.back() = 0;
		tlog::error() << log.data();
	}
	return (GLboolean)status == GL_TRUE;
}

GLuint compile_shader(bool pixel, const char* code) {
	GLuint g_VertHandle = glCreateShader(pixel ? GL_FRAGMENT_SHADER : GL_VERTEX_SHADER );
	const char* glsl_version = "#version 330\n";
	const GLchar* strings[2] = { glsl_version, code};
	glShaderSource(g_VertHandle, 2, strings, NULL);
	glCompileShader(g_VertHandle);
	if (!check_shader(g_VertHandle, pixel?"pixel":"vertex", false)) {
		glDeleteShader(g_VertHandle);
		return 0;
	}
	return g_VertHandle;
}

void draw_mesh_gl(
	const GPUMemory<Vector3f>& verts,
	const GPUMemory<Vector3f>& normals,
	const GPUMemory<Vector3f>& colors,
	const GPUMemory<uint32_t>& indices,
	const Vector2i& resolution,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix,
	const Vector2f& screen_center,
	int mesh_render_mode
) {
	if (verts.size() == 0 || indices.size() == 0 || mesh_render_mode == 0) {
		return;
	}

	static GLuint vs = 0, ps = 0, program = 0, VAO = 0, VBO[3] = {}, els = 0, vbosize = 0, elssize = 0;
	if (!VAO) {
		glGenVertexArrays(1, &VAO);
		glBindVertexArray(VAO);
	}
	if (vbosize != verts.size()) {
		for (int i= 0; i < 3; ++i) {
			if (VBO[i]) {
				cudaGLUnregisterBufferObject(VBO[i]);
				glDeleteBuffers(1, &VBO[i]);
			}
			glGenBuffers(1, &VBO[i]);
			vbosize = verts.size();
			glBindBuffer(GL_ARRAY_BUFFER, VBO[i]);
			glBufferData(GL_ARRAY_BUFFER, vbosize * sizeof(Vector3f), NULL, GL_DYNAMIC_COPY);
			cudaGLRegisterBufferObject(VBO[i]);
		}
	}
	if (elssize != indices.size()) {
		if (els) {
			cudaGLUnregisterBufferObject(els);
			glDeleteBuffers(1, &els);
		}
		glGenBuffers(1, &els);
		elssize = indices.size();
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, els);
		glBufferData(GL_ELEMENT_ARRAY_BUFFER, elssize * sizeof(int), NULL, GL_STREAM_DRAW);
		cudaGLRegisterBufferObject(els);
	}
	void *ptr=nullptr;
	cudaGLMapBufferObject(&ptr, VBO[0]);
	if (ptr) hipMemcpy(ptr, verts.data(), vbosize * sizeof(Vector3f), hipMemcpyDeviceToDevice);
	cudaGLMapBufferObject(&ptr, VBO[1]);
	if (ptr) hipMemcpy(ptr, normals.data(), vbosize * sizeof(Vector3f), hipMemcpyDeviceToDevice);
	cudaGLMapBufferObject(&ptr, VBO[2]);
	if (ptr) hipMemcpy(ptr, colors.data(), vbosize * sizeof(Vector3f), hipMemcpyDeviceToDevice);

	//std::vector<Vector3f> cpucols; cpucols.resize(verts.size());
	//colors.copy_to_host(cpucols);

	cudaGLUnmapBufferObject(VBO[2]);
	cudaGLUnmapBufferObject(VBO[1]);
	cudaGLUnmapBufferObject(VBO[0]);
	cudaGLMapBufferObject(&ptr, els);
	if (ptr) hipMemcpy(ptr, indices.data(), indices.get_bytes(), hipMemcpyDeviceToDevice);
	cudaGLUnmapBufferObject(els);

	if (!program) {
		vs = compile_shader(false, R"foo(
layout (location = 0) in vec3 pos;
layout (location = 1) in vec3 nor;
layout (location = 2) in vec3 col;
out vec3 vtxcol;
uniform mat4 camera;
uniform vec2 f;
uniform ivec2 res;
uniform vec2 cen;
uniform int mode;
void main()
{
	vec4 p = camera * vec4(pos, 1.0);
	p.xy *= vec2(2.0, -2.0) * f.xy / vec2(res.xy);
	p.w = p.z;
	p.z = p.z - 0.1;
	p.xy += cen * p.w;
	if (mode == 2) {
		vtxcol = normalize(nor) * 0.5 + vec3(0.5); // visualize vertex normals
	} else {
		vtxcol = col;
	}
	gl_Position = p;
}
)foo");
		ps = compile_shader(true, R"foo(
layout (location = 0) out vec4 o;
in vec3 vtxcol;
uniform int mode;
void main() {
	if (mode == 3) {
		vec3 tricol = vec3((ivec3(923, 3572, 5423) * gl_PrimitiveID) & 255) * (1.0 / 255.0);
		o = vec4(tricol, 1.0);
	} else {
		o = vec4(vtxcol, 1.0);
	}
}
)foo");
		program = glCreateProgram();
		glAttachShader(program, vs);
		glAttachShader(program, ps);
		glLinkProgram(program);
		if (!check_shader(program, "shader program", true)) {
			glDeleteProgram(program);
			program = 0;
		}
	}
	Matrix4f view2world=Matrix4f::Identity();
	view2world.block<3,4>(0,0) = camera_matrix;
	Matrix4f world2view = view2world.inverse();
	glBindVertexArray(VAO);
	glUseProgram(program);
	glUniformMatrix4fv(glGetUniformLocation(program, "camera"), 1, GL_FALSE, (GLfloat*)&world2view);
	glUniform2f(glGetUniformLocation(program, "f"), focal_length.x(), focal_length.y());
	glUniform2f(glGetUniformLocation(program, "cen"), screen_center.x()*2.f-1.f, screen_center.y()*-2.f+1.f);
	glUniform2i(glGetUniformLocation(program, "res"), resolution.x(), resolution.y());
	glUniform1i(glGetUniformLocation(program, "mode"), mesh_render_mode);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, els);
	GLuint posat = (GLuint)glGetAttribLocation(program, "pos");
	GLuint norat = (GLuint)glGetAttribLocation(program, "nor");
	GLuint colat = (GLuint)glGetAttribLocation(program, "col");
	glEnableVertexAttribArray(posat);
	glEnableVertexAttribArray(norat);
	glEnableVertexAttribArray(colat);
	glBindBuffer(GL_ARRAY_BUFFER, VBO[0]);
	glVertexAttribPointer(posat, 3, GL_FLOAT, GL_FALSE, 3*4, 0);
	glBindBuffer(GL_ARRAY_BUFFER, VBO[1]);
	glVertexAttribPointer(norat, 3, GL_FLOAT, GL_FALSE, 3*4, 0);
	glBindBuffer(GL_ARRAY_BUFFER, VBO[2]);
	glVertexAttribPointer(colat, 3, GL_FLOAT, GL_FALSE, 3*4, 0);
	glCullFace(GL_BACK);
	glDisable(GL_CULL_FACE);
	glEnable(GL_DEPTH_TEST);
	glDrawElements(GL_TRIANGLES, (GLsizei)indices.size(), GL_UNSIGNED_INT , (GLvoid*)0);
	glDisable(GL_CULL_FACE);

	glUseProgram(0);
}
#endif //NGP_GUI

/*
vertex indices with z=0
0 -> 1   <-- first edge is 0->1, then 1->2 etc
^    |
|	 v
3 <- 2

with z=1
4 -> 5   <-- fourth edge is 4->5 etc
^    |
|	 v
7 <- 6

edges 8-11 go in +z direction from vertex 0-3
*/
__global__ void gen_vertices(BoundingBox aabb, Vector3i res_3d, const float* __restrict__ density, int*__restrict__ vertidx_grid, Vector3f* verts_out, float thresh, uint32_t* __restrict__ counters) {
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z()) return;
	Vector3f scale=(aabb.max-aabb.min).cwiseQuotient(res_3d.cast<float>());
	Vector3f offset=aabb.min;
	uint32_t res2=res_3d.x()*res_3d.y();
	uint32_t res3=res_3d.x()*res_3d.y()*res_3d.z();
	uint32_t idx=x+y*res_3d.x()+z*res2;
	float f0 = density[idx];
	bool inside=(f0>thresh);
	if (x<res_3d.x()-1) {
		float f1 = density[idx+1];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx]=Vector3f{float(x)+dt, float(y), float(z)}.cwiseProduct(scale) + offset;
			}
		}
	}
	if (y<res_3d.y()-1) {
		float f1 = density[idx+res_3d.x()];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx+res3]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx]=Vector3f{float(x), float(y)+dt, float(z)}.cwiseProduct(scale) + offset;
			}
		}
	}
	if (z<res_3d.z()-1) {
		float f1 = density[idx+res2];
		if (inside != (f1>thresh)) {
			uint32_t vidx = atomicAdd(counters,1);
			if (verts_out) {
				vertidx_grid[idx+res3*2]=vidx+1;
				float prevf=f0,nextf=f1;
				float dt=((thresh-prevf)/(nextf-prevf));
				verts_out[vidx]=Vector3f{float(x), float(y), float(z)+dt}.cwiseProduct(scale) + offset;
			}
		}
	}
}

__global__ void accumulate_1ring(uint32_t num_tris, const uint32_t* indices, const Vector3f* verts_in, Vector4f* verts_out, Vector3f *normals_out) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=num_tris) return;
	uint32_t ia=indices[i*3+0];
	uint32_t ib=indices[i*3+1];
	uint32_t ic=indices[i*3+2];
	Vector3f pa=verts_in[ia];
	Vector3f pb=verts_in[ib];
	Vector3f pc=verts_in[ic];

	atomicAdd(&verts_out[ia][0], pb.x()+pc.x());
	atomicAdd(&verts_out[ia][1], pb.y()+pc.y());
	atomicAdd(&verts_out[ia][2], pb.z()+pc.z());
	atomicAdd(&verts_out[ia][3], 2.f);
	atomicAdd(&verts_out[ib][0], pa.x()+pc.x());
	atomicAdd(&verts_out[ib][1], pa.y()+pc.y());
	atomicAdd(&verts_out[ib][2], pa.z()+pc.z());
	atomicAdd(&verts_out[ib][3], 2.f);
	atomicAdd(&verts_out[ic][0], pb.x()+pa.x());
	atomicAdd(&verts_out[ic][1], pb.y()+pa.y());
	atomicAdd(&verts_out[ic][2], pb.z()+pa.z());
	atomicAdd(&verts_out[ic][3], 2.f);

	if (normals_out) {
		Vector3f n= (pb-pa).cross(pa-pc); // don't normalise so it's weighted by area
		atomicAdd(&normals_out[ia][0], n.x());
		atomicAdd(&normals_out[ia][1], n.y());
		atomicAdd(&normals_out[ia][2], n.z());
		atomicAdd(&normals_out[ib][0], n.x());
		atomicAdd(&normals_out[ib][1], n.y());
		atomicAdd(&normals_out[ib][2], n.z());
		atomicAdd(&normals_out[ic][0], n.x());
		atomicAdd(&normals_out[ic][1], n.y());
		atomicAdd(&normals_out[ic][2], n.z());
	}
}

__global__ void compute_centroids(uint32_t num_verts, Vector3f* centroids_out, const Vector4f* verts_in) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=num_verts) return;
	Vector4f p = verts_in[i];
	if (p.w()<=0.f) return;
	Vector3f c=verts_in[i].head<3>() * (1.f/p.w());
	centroids_out[i]=c;
}

__global__ void gen_faces(Vector3i res_3d, const float* __restrict__ density, const int*__restrict__ vertidx_grid, uint32_t* indices_out, float thresh, uint32_t *__restrict__ counters) {
	// marching cubes tables from https://github.com/pmneila/PyMCubes/blob/master/mcubes/src/marchingcubes.cpp which in turn seems to be from https://web.archive.org/web/20181127124338/http://paulbourke.net/geometry/polygonise/
	// License is BSD 3-clause, which can be found here: https://github.com/pmneila/PyMCubes/blob/master/LICENSE
	/*
	static constexpr uint16_t edge_table[256] =
	{
		0x000, 0x109, 0x203, 0x30a, 0x406, 0x50f, 0x605, 0x70c, 0x80c, 0x905, 0xa0f, 0xb06, 0xc0a, 0xd03, 0xe09, 0xf00,
		0x190, 0x099, 0x393, 0x29a, 0x596, 0x49f, 0x795, 0x69c, 0x99c, 0x895, 0xb9f, 0xa96, 0xd9a, 0xc93, 0xf99, 0xe90,
		0x230, 0x339, 0x033, 0x13a, 0x636, 0x73f, 0x435, 0x53c, 0xa3c, 0xb35, 0x83f, 0x936, 0xe3a, 0xf33, 0xc39, 0xd30,
		0x3a0, 0x2a9, 0x1a3, 0x0aa, 0x7a6, 0x6af, 0x5a5, 0x4ac, 0xbac, 0xaa5, 0x9af, 0x8a6, 0xfaa, 0xea3, 0xda9, 0xca0,
		0x460, 0x569, 0x663, 0x76a, 0x066, 0x16f, 0x265, 0x36c, 0xc6c, 0xd65, 0xe6f, 0xf66, 0x86a, 0x963, 0xa69, 0xb60,
		0x5f0, 0x4f9, 0x7f3, 0x6fa, 0x1f6, 0x0ff, 0x3f5, 0x2fc, 0xdfc, 0xcf5, 0xfff, 0xef6, 0x9fa, 0x8f3, 0xbf9, 0xaf0,
		0x650, 0x759, 0x453, 0x55a, 0x256, 0x35f, 0x055, 0x15c, 0xe5c, 0xf55, 0xc5f, 0xd56, 0xa5a, 0xb53, 0x859, 0x950,
		0x7c0, 0x6c9, 0x5c3, 0x4ca, 0x3c6, 0x2cf, 0x1c5, 0x0cc, 0xfcc, 0xec5, 0xdcf, 0xcc6, 0xbca, 0xac3, 0x9c9, 0x8c0,
		0x8c0, 0x9c9, 0xac3, 0xbca, 0xcc6, 0xdcf, 0xec5, 0xfcc, 0x0cc, 0x1c5, 0x2cf, 0x3c6, 0x4ca, 0x5c3, 0x6c9, 0x7c0,
		0x950, 0x859, 0xb53, 0xa5a, 0xd56, 0xc5f, 0xf55, 0xe5c, 0x15c, 0x055, 0x35f, 0x256, 0x55a, 0x453, 0x759, 0x650,
		0xaf0, 0xbf9, 0x8f3, 0x9fa, 0xef6, 0xfff, 0xcf5, 0xdfc, 0x2fc, 0x3f5, 0x0ff, 0x1f6, 0x6fa, 0x7f3, 0x4f9, 0x5f0,
		0xb60, 0xa69, 0x963, 0x86a, 0xf66, 0xe6f, 0xd65, 0xc6c, 0x36c, 0x265, 0x16f, 0x066, 0x76a, 0x663, 0x569, 0x460,
		0xca0, 0xda9, 0xea3, 0xfaa, 0x8a6, 0x9af, 0xaa5, 0xbac, 0x4ac, 0x5a5, 0x6af, 0x7a6, 0x0aa, 0x1a3, 0x2a9, 0x3a0,
		0xd30, 0xc39, 0xf33, 0xe3a, 0x936, 0x83f, 0xb35, 0xa3c, 0x53c, 0x435, 0x73f, 0x636, 0x13a, 0x033, 0x339, 0x230,
		0xe90, 0xf99, 0xc93, 0xd9a, 0xa96, 0xb9f, 0x895, 0x99c, 0x69c, 0x795, 0x49f, 0x596, 0x29a, 0x393, 0x099, 0x190,
		0xf00, 0xe09, 0xd03, 0xc0a, 0xb06, 0xa0f, 0x905, 0x80c, 0x70c, 0x605, 0x50f, 0x406, 0x30a, 0x203, 0x109, 0x000
	};
	*/
	static constexpr int8_t triangle_table[256][16] =
	{
		{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 8, 3, 9, 8, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 2, 10, 0, 2, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 8, 3, 2, 10, 8, 10, 9, 8, -1, -1, -1, -1, -1, -1, -1},
		{3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 11, 2, 8, 11, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 11, 2, 1, 9, 11, 9, 8, 11, -1, -1, -1, -1, -1, -1, -1},
		{3, 10, 1, 11, 10, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 10, 1, 0, 8, 10, 8, 11, 10, -1, -1, -1, -1, -1, -1, -1},
		{3, 9, 0, 3, 11, 9, 11, 10, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 3, 0, 7, 3, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 1, 9, 4, 7, 1, 7, 3, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 4, 7, 3, 0, 4, 1, 2, 10, -1, -1, -1, -1, -1, -1, -1},
		{9, 2, 10, 9, 0, 2, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
		{2, 10, 9, 2, 9, 7, 2, 7, 3, 7, 9, 4, -1, -1, -1, -1},
		{8, 4, 7, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 4, 7, 11, 2, 4, 2, 0, 4, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 1, 8, 4, 7, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
		{4, 7, 11, 9, 4, 11, 9, 11, 2, 9, 2, 1, -1, -1, -1, -1},
		{3, 10, 1, 3, 11, 10, 7, 8, 4, -1, -1, -1, -1, -1, -1, -1},
		{1, 11, 10, 1, 4, 11, 1, 0, 4, 7, 11, 4, -1, -1, -1, -1},
		{4, 7, 8, 9, 0, 11, 9, 11, 10, 11, 0, 3, -1, -1, -1, -1},
		{4, 7, 11, 4, 11, 9, 9, 11, 10, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 5, 4, 1, 5, 0, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 5, 4, 8, 3, 5, 3, 1, 5, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 1, 2, 10, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
		{5, 2, 10, 5, 4, 2, 4, 0, 2, -1, -1, -1, -1, -1, -1, -1},
		{2, 10, 5, 3, 2, 5, 3, 5, 4, 3, 4, 8, -1, -1, -1, -1},
		{9, 5, 4, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 11, 2, 0, 8, 11, 4, 9, 5, -1, -1, -1, -1, -1, -1, -1},
		{0, 5, 4, 0, 1, 5, 2, 3, 11, -1, -1, -1, -1, -1, -1, -1},
		{2, 1, 5, 2, 5, 8, 2, 8, 11, 4, 8, 5, -1, -1, -1, -1},
		{10, 3, 11, 10, 1, 3, 9, 5, 4, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 5, 0, 8, 1, 8, 10, 1, 8, 11, 10, -1, -1, -1, -1},
		{5, 4, 0, 5, 0, 11, 5, 11, 10, 11, 0, 3, -1, -1, -1, -1},
		{5, 4, 8, 5, 8, 10, 10, 8, 11, -1, -1, -1, -1, -1, -1, -1},
		{9, 7, 8, 5, 7, 9, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 3, 0, 9, 5, 3, 5, 7, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 7, 8, 0, 1, 7, 1, 5, 7, -1, -1, -1, -1, -1, -1, -1},
		{1, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 7, 8, 9, 5, 7, 10, 1, 2, -1, -1, -1, -1, -1, -1, -1},
		{10, 1, 2, 9, 5, 0, 5, 3, 0, 5, 7, 3, -1, -1, -1, -1},
		{8, 0, 2, 8, 2, 5, 8, 5, 7, 10, 5, 2, -1, -1, -1, -1},
		{2, 10, 5, 2, 5, 3, 3, 5, 7, -1, -1, -1, -1, -1, -1, -1},
		{7, 9, 5, 7, 8, 9, 3, 11, 2, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 7, 9, 7, 2, 9, 2, 0, 2, 7, 11, -1, -1, -1, -1},
		{2, 3, 11, 0, 1, 8, 1, 7, 8, 1, 5, 7, -1, -1, -1, -1},
		{11, 2, 1, 11, 1, 7, 7, 1, 5, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 8, 8, 5, 7, 10, 1, 3, 10, 3, 11, -1, -1, -1, -1},
		{5, 7, 0, 5, 0, 9, 7, 11, 0, 1, 0, 10, 11, 10, 0, -1},
		{11, 10, 0, 11, 0, 3, 10, 5, 0, 8, 0, 7, 5, 7, 0, -1},
		{11, 10, 5, 7, 11, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 1, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 8, 3, 1, 9, 8, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 5, 2, 6, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 5, 1, 2, 6, 3, 0, 8, -1, -1, -1, -1, -1, -1, -1},
		{9, 6, 5, 9, 0, 6, 0, 2, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 9, 8, 5, 8, 2, 5, 2, 6, 3, 2, 8, -1, -1, -1, -1},
		{2, 3, 11, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 0, 8, 11, 2, 0, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 1, 9, 2, 9, 11, 2, 9, 8, 11, -1, -1, -1, -1},
		{6, 3, 11, 6, 5, 3, 5, 1, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 11, 0, 11, 5, 0, 5, 1, 5, 11, 6, -1, -1, -1, -1},
		{3, 11, 6, 0, 3, 6, 0, 6, 5, 0, 5, 9, -1, -1, -1, -1},
		{6, 5, 9, 6, 9, 11, 11, 9, 8, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 3, 0, 4, 7, 3, 6, 5, 10, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 5, 10, 6, 8, 4, 7, -1, -1, -1, -1, -1, -1, -1},
		{10, 6, 5, 1, 9, 7, 1, 7, 3, 7, 9, 4, -1, -1, -1, -1},
		{6, 1, 2, 6, 5, 1, 4, 7, 8, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 5, 5, 2, 6, 3, 0, 4, 3, 4, 7, -1, -1, -1, -1},
		{8, 4, 7, 9, 0, 5, 0, 6, 5, 0, 2, 6, -1, -1, -1, -1},
		{7, 3, 9, 7, 9, 4, 3, 2, 9, 5, 9, 6, 2, 6, 9, -1},
		{3, 11, 2, 7, 8, 4, 10, 6, 5, -1, -1, -1, -1, -1, -1, -1},
		{5, 10, 6, 4, 7, 2, 4, 2, 0, 2, 7, 11, -1, -1, -1, -1},
		{0, 1, 9, 4, 7, 8, 2, 3, 11, 5, 10, 6, -1, -1, -1, -1},
		{9, 2, 1, 9, 11, 2, 9, 4, 11, 7, 11, 4, 5, 10, 6, -1},
		{8, 4, 7, 3, 11, 5, 3, 5, 1, 5, 11, 6, -1, -1, -1, -1},
		{5, 1, 11, 5, 11, 6, 1, 0, 11, 7, 11, 4, 0, 4, 11, -1},
		{0, 5, 9, 0, 6, 5, 0, 3, 6, 11, 6, 3, 8, 4, 7, -1},
		{6, 5, 9, 6, 9, 11, 4, 7, 9, 7, 11, 9, -1, -1, -1, -1},
		{10, 4, 9, 6, 4, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 10, 6, 4, 9, 10, 0, 8, 3, -1, -1, -1, -1, -1, -1, -1},
		{10, 0, 1, 10, 6, 0, 6, 4, 0, -1, -1, -1, -1, -1, -1, -1},
		{8, 3, 1, 8, 1, 6, 8, 6, 4, 6, 1, 10, -1, -1, -1, -1},
		{1, 4, 9, 1, 2, 4, 2, 6, 4, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 1, 2, 9, 2, 4, 9, 2, 6, 4, -1, -1, -1, -1},
		{0, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 3, 2, 8, 2, 4, 4, 2, 6, -1, -1, -1, -1, -1, -1, -1},
		{10, 4, 9, 10, 6, 4, 11, 2, 3, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 2, 2, 8, 11, 4, 9, 10, 4, 10, 6, -1, -1, -1, -1},
		{3, 11, 2, 0, 1, 6, 0, 6, 4, 6, 1, 10, -1, -1, -1, -1},
		{6, 4, 1, 6, 1, 10, 4, 8, 1, 2, 1, 11, 8, 11, 1, -1},
		{9, 6, 4, 9, 3, 6, 9, 1, 3, 11, 6, 3, -1, -1, -1, -1},
		{8, 11, 1, 8, 1, 0, 11, 6, 1, 9, 1, 4, 6, 4, 1, -1},
		{3, 11, 6, 3, 6, 0, 0, 6, 4, -1, -1, -1, -1, -1, -1, -1},
		{6, 4, 8, 11, 6, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 10, 6, 7, 8, 10, 8, 9, 10, -1, -1, -1, -1, -1, -1, -1},
		{0, 7, 3, 0, 10, 7, 0, 9, 10, 6, 7, 10, -1, -1, -1, -1},
		{10, 6, 7, 1, 10, 7, 1, 7, 8, 1, 8, 0, -1, -1, -1, -1},
		{10, 6, 7, 10, 7, 1, 1, 7, 3, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 6, 1, 6, 8, 1, 8, 9, 8, 6, 7, -1, -1, -1, -1},
		{2, 6, 9, 2, 9, 1, 6, 7, 9, 0, 9, 3, 7, 3, 9, -1},
		{7, 8, 0, 7, 0, 6, 6, 0, 2, -1, -1, -1, -1, -1, -1, -1},
		{7, 3, 2, 6, 7, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 11, 10, 6, 8, 10, 8, 9, 8, 6, 7, -1, -1, -1, -1},
		{2, 0, 7, 2, 7, 11, 0, 9, 7, 6, 7, 10, 9, 10, 7, -1},
		{1, 8, 0, 1, 7, 8, 1, 10, 7, 6, 7, 10, 2, 3, 11, -1},
		{11, 2, 1, 11, 1, 7, 10, 6, 1, 6, 7, 1, -1, -1, -1, -1},
		{8, 9, 6, 8, 6, 7, 9, 1, 6, 11, 6, 3, 1, 3, 6, -1},
		{0, 9, 1, 11, 6, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 8, 0, 7, 0, 6, 3, 11, 0, 11, 6, 0, -1, -1, -1, -1},
		{7, 11, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 8, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 9, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 1, 9, 8, 3, 1, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
		{10, 1, 2, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 3, 0, 8, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
		{2, 9, 0, 2, 10, 9, 6, 11, 7, -1, -1, -1, -1, -1, -1, -1},
		{6, 11, 7, 2, 10, 3, 10, 8, 3, 10, 9, 8, -1, -1, -1, -1},
		{7, 2, 3, 6, 2, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{7, 0, 8, 7, 6, 0, 6, 2, 0, -1, -1, -1, -1, -1, -1, -1},
		{2, 7, 6, 2, 3, 7, 0, 1, 9, -1, -1, -1, -1, -1, -1, -1},
		{1, 6, 2, 1, 8, 6, 1, 9, 8, 8, 7, 6, -1, -1, -1, -1},
		{10, 7, 6, 10, 1, 7, 1, 3, 7, -1, -1, -1, -1, -1, -1, -1},
		{10, 7, 6, 1, 7, 10, 1, 8, 7, 1, 0, 8, -1, -1, -1, -1},
		{0, 3, 7, 0, 7, 10, 0, 10, 9, 6, 10, 7, -1, -1, -1, -1},
		{7, 6, 10, 7, 10, 8, 8, 10, 9, -1, -1, -1, -1, -1, -1, -1},
		{6, 8, 4, 11, 8, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 6, 11, 3, 0, 6, 0, 4, 6, -1, -1, -1, -1, -1, -1, -1},
		{8, 6, 11, 8, 4, 6, 9, 0, 1, -1, -1, -1, -1, -1, -1, -1},
		{9, 4, 6, 9, 6, 3, 9, 3, 1, 11, 3, 6, -1, -1, -1, -1},
		{6, 8, 4, 6, 11, 8, 2, 10, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 3, 0, 11, 0, 6, 11, 0, 4, 6, -1, -1, -1, -1},
		{4, 11, 8, 4, 6, 11, 0, 2, 9, 2, 10, 9, -1, -1, -1, -1},
		{10, 9, 3, 10, 3, 2, 9, 4, 3, 11, 3, 6, 4, 6, 3, -1},
		{8, 2, 3, 8, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1},
		{0, 4, 2, 4, 6, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 9, 0, 2, 3, 4, 2, 4, 6, 4, 3, 8, -1, -1, -1, -1},
		{1, 9, 4, 1, 4, 2, 2, 4, 6, -1, -1, -1, -1, -1, -1, -1},
		{8, 1, 3, 8, 6, 1, 8, 4, 6, 6, 10, 1, -1, -1, -1, -1},
		{10, 1, 0, 10, 0, 6, 6, 0, 4, -1, -1, -1, -1, -1, -1, -1},
		{4, 6, 3, 4, 3, 8, 6, 10, 3, 0, 3, 9, 10, 9, 3, -1},
		{10, 9, 4, 6, 10, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 5, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 4, 9, 5, 11, 7, 6, -1, -1, -1, -1, -1, -1, -1},
		{5, 0, 1, 5, 4, 0, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
		{11, 7, 6, 8, 3, 4, 3, 5, 4, 3, 1, 5, -1, -1, -1, -1},
		{9, 5, 4, 10, 1, 2, 7, 6, 11, -1, -1, -1, -1, -1, -1, -1},
		{6, 11, 7, 1, 2, 10, 0, 8, 3, 4, 9, 5, -1, -1, -1, -1},
		{7, 6, 11, 5, 4, 10, 4, 2, 10, 4, 0, 2, -1, -1, -1, -1},
		{3, 4, 8, 3, 5, 4, 3, 2, 5, 10, 5, 2, 11, 7, 6, -1},
		{7, 2, 3, 7, 6, 2, 5, 4, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 5, 4, 0, 8, 6, 0, 6, 2, 6, 8, 7, -1, -1, -1, -1},
		{3, 6, 2, 3, 7, 6, 1, 5, 0, 5, 4, 0, -1, -1, -1, -1},
		{6, 2, 8, 6, 8, 7, 2, 1, 8, 4, 8, 5, 1, 5, 8, -1},
		{9, 5, 4, 10, 1, 6, 1, 7, 6, 1, 3, 7, -1, -1, -1, -1},
		{1, 6, 10, 1, 7, 6, 1, 0, 7, 8, 7, 0, 9, 5, 4, -1},
		{4, 0, 10, 4, 10, 5, 0, 3, 10, 6, 10, 7, 3, 7, 10, -1},
		{7, 6, 10, 7, 10, 8, 5, 4, 10, 4, 8, 10, -1, -1, -1, -1},
		{6, 9, 5, 6, 11, 9, 11, 8, 9, -1, -1, -1, -1, -1, -1, -1},
		{3, 6, 11, 0, 6, 3, 0, 5, 6, 0, 9, 5, -1, -1, -1, -1},
		{0, 11, 8, 0, 5, 11, 0, 1, 5, 5, 6, 11, -1, -1, -1, -1},
		{6, 11, 3, 6, 3, 5, 5, 3, 1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 10, 9, 5, 11, 9, 11, 8, 11, 5, 6, -1, -1, -1, -1},
		{0, 11, 3, 0, 6, 11, 0, 9, 6, 5, 6, 9, 1, 2, 10, -1},
		{11, 8, 5, 11, 5, 6, 8, 0, 5, 10, 5, 2, 0, 2, 5, -1},
		{6, 11, 3, 6, 3, 5, 2, 10, 3, 10, 5, 3, -1, -1, -1, -1},
		{5, 8, 9, 5, 2, 8, 5, 6, 2, 3, 8, 2, -1, -1, -1, -1},
		{9, 5, 6, 9, 6, 0, 0, 6, 2, -1, -1, -1, -1, -1, -1, -1},
		{1, 5, 8, 1, 8, 0, 5, 6, 8, 3, 8, 2, 6, 2, 8, -1},
		{1, 5, 6, 2, 1, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 3, 6, 1, 6, 10, 3, 8, 6, 5, 6, 9, 8, 9, 6, -1},
		{10, 1, 0, 10, 0, 6, 9, 5, 0, 5, 6, 0, -1, -1, -1, -1},
		{0, 3, 8, 5, 6, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{10, 5, 6, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 5, 10, 7, 5, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{11, 5, 10, 11, 7, 5, 8, 3, 0, -1, -1, -1, -1, -1, -1, -1},
		{5, 11, 7, 5, 10, 11, 1, 9, 0, -1, -1, -1, -1, -1, -1, -1},
		{10, 7, 5, 10, 11, 7, 9, 8, 1, 8, 3, 1, -1, -1, -1, -1},
		{11, 1, 2, 11, 7, 1, 7, 5, 1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 1, 2, 7, 1, 7, 5, 7, 2, 11, -1, -1, -1, -1},
		{9, 7, 5, 9, 2, 7, 9, 0, 2, 2, 11, 7, -1, -1, -1, -1},
		{7, 5, 2, 7, 2, 11, 5, 9, 2, 3, 2, 8, 9, 8, 2, -1},
		{2, 5, 10, 2, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1},
		{8, 2, 0, 8, 5, 2, 8, 7, 5, 10, 2, 5, -1, -1, -1, -1},
		{9, 0, 1, 5, 10, 3, 5, 3, 7, 3, 10, 2, -1, -1, -1, -1},
		{9, 8, 2, 9, 2, 1, 8, 7, 2, 10, 2, 5, 7, 5, 2, -1},
		{1, 3, 5, 3, 7, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 7, 0, 7, 1, 1, 7, 5, -1, -1, -1, -1, -1, -1, -1},
		{9, 0, 3, 9, 3, 5, 5, 3, 7, -1, -1, -1, -1, -1, -1, -1},
		{9, 8, 7, 5, 9, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{5, 8, 4, 5, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1},
		{5, 0, 4, 5, 11, 0, 5, 10, 11, 11, 3, 0, -1, -1, -1, -1},
		{0, 1, 9, 8, 4, 10, 8, 10, 11, 10, 4, 5, -1, -1, -1, -1},
		{10, 11, 4, 10, 4, 5, 11, 3, 4, 9, 4, 1, 3, 1, 4, -1},
		{2, 5, 1, 2, 8, 5, 2, 11, 8, 4, 5, 8, -1, -1, -1, -1},
		{0, 4, 11, 0, 11, 3, 4, 5, 11, 2, 11, 1, 5, 1, 11, -1},
		{0, 2, 5, 0, 5, 9, 2, 11, 5, 4, 5, 8, 11, 8, 5, -1},
		{9, 4, 5, 2, 11, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 5, 10, 3, 5, 2, 3, 4, 5, 3, 8, 4, -1, -1, -1, -1},
		{5, 10, 2, 5, 2, 4, 4, 2, 0, -1, -1, -1, -1, -1, -1, -1},
		{3, 10, 2, 3, 5, 10, 3, 8, 5, 4, 5, 8, 0, 1, 9, -1},
		{5, 10, 2, 5, 2, 4, 1, 9, 2, 9, 4, 2, -1, -1, -1, -1},
		{8, 4, 5, 8, 5, 3, 3, 5, 1, -1, -1, -1, -1, -1, -1, -1},
		{0, 4, 5, 1, 0, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{8, 4, 5, 8, 5, 3, 9, 0, 5, 0, 3, 5, -1, -1, -1, -1},
		{9, 4, 5, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 11, 7, 4, 9, 11, 9, 10, 11, -1, -1, -1, -1, -1, -1, -1},
		{0, 8, 3, 4, 9, 7, 9, 11, 7, 9, 10, 11, -1, -1, -1, -1},
		{1, 10, 11, 1, 11, 4, 1, 4, 0, 7, 4, 11, -1, -1, -1, -1},
		{3, 1, 4, 3, 4, 8, 1, 10, 4, 7, 4, 11, 10, 11, 4, -1},
		{4, 11, 7, 9, 11, 4, 9, 2, 11, 9, 1, 2, -1, -1, -1, -1},
		{9, 7, 4, 9, 11, 7, 9, 1, 11, 2, 11, 1, 0, 8, 3, -1},
		{11, 7, 4, 11, 4, 2, 2, 4, 0, -1, -1, -1, -1, -1, -1, -1},
		{11, 7, 4, 11, 4, 2, 8, 3, 4, 3, 2, 4, -1, -1, -1, -1},
		{2, 9, 10, 2, 7, 9, 2, 3, 7, 7, 4, 9, -1, -1, -1, -1},
		{9, 10, 7, 9, 7, 4, 10, 2, 7, 8, 7, 0, 2, 0, 7, -1},
		{3, 7, 10, 3, 10, 2, 7, 4, 10, 1, 10, 0, 4, 0, 10, -1},
		{1, 10, 2, 8, 7, 4, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 1, 4, 1, 7, 7, 1, 3, -1, -1, -1, -1, -1, -1, -1},
		{4, 9, 1, 4, 1, 7, 0, 8, 1, 8, 7, 1, -1, -1, -1, -1},
		{4, 0, 3, 7, 4, 3, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{4, 8, 7, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{9, 10, 8, 10, 11, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 9, 3, 9, 11, 11, 9, 10, -1, -1, -1, -1, -1, -1, -1},
		{0, 1, 10, 0, 10, 8, 8, 10, 11, -1, -1, -1, -1, -1, -1, -1},
		{3, 1, 10, 11, 3, 10, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 2, 11, 1, 11, 9, 9, 11, 8, -1, -1, -1, -1, -1, -1, -1},
		{3, 0, 9, 3, 9, 11, 1, 2, 9, 2, 11, 9, -1, -1, -1, -1},
		{0, 2, 11, 8, 0, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{3, 2, 11, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 8, 2, 8, 10, 10, 8, 9, -1, -1, -1, -1, -1, -1, -1},
		{9, 10, 2, 0, 9, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{2, 3, 8, 2, 8, 10, 0, 1, 8, 1, 10, 8, -1, -1, -1, -1},
		{1, 10, 2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{1, 3, 8, 9, 1, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 9, 1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{0, 3, 8, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1},
		{-1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1, -1}
	};

	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	uint32_t z = blockIdx.z * blockDim.z + threadIdx.z;
	if (x>=res_3d.x()-1 || y>=res_3d.y()-1 || z>=res_3d.z()-1) return;
	uint32_t res1=res_3d.x();
	uint32_t res2=res_3d.x()*res_3d.y();
	uint32_t res3=res_3d.x()*res_3d.y()*res_3d.z();
	uint32_t idx=x+y*res_3d.x()+z*res2;

	uint32_t idx_x=idx;
	uint32_t idx_y=idx+res3;
	uint32_t idx_z=idx+res3*2;

	int mask=0;
	if (density[idx]>thresh) mask|=1;
	if (density[idx+1]>thresh) mask|=2;
	if (density[idx+1+res1]>thresh) mask|=4;
	if (density[idx+res1]>thresh) mask|=8;
	idx+=res2;
	if (density[idx]>thresh) mask|=16;
	if (density[idx+1]>thresh) mask|=32;
	if (density[idx+1+res1]>thresh) mask|=64;
	if (density[idx+res1]>thresh) mask|=128;
	idx-=res2;

	if (!mask || mask==255) return;
	int local_edges[12];
	if (vertidx_grid) {
		local_edges[0]=vertidx_grid[idx_x];
		local_edges[1]=vertidx_grid[idx_y+1];
		local_edges[2]=vertidx_grid[idx_x+res1];
		local_edges[3]=vertidx_grid[idx_y];

		local_edges[4]=vertidx_grid[idx_x+res2];
		local_edges[5]=vertidx_grid[idx_y+1+res2];
		local_edges[6]=vertidx_grid[idx_x+res1+res2];
		local_edges[7]=vertidx_grid[idx_y+res2];

		local_edges[8]=vertidx_grid[idx_z];
		local_edges[9]=vertidx_grid[idx_z+1];
		local_edges[10]=vertidx_grid[idx_z+1+res1];
		local_edges[11]=vertidx_grid[idx_z+res1];
	}
	uint32_t tricount=0;
	const int8_t *triangles=triangle_table[mask];
	for (;tricount<15;tricount+=3) if (triangles[tricount]<0) break;
	uint32_t tidx = atomicAdd(counters+1,tricount);
	if (indices_out) {
		for (int i=0;i<15;++i) {
			int j = triangles[i];
			if (j<0) break;
			if (!local_edges[j]) {
				printf("at %d %d %d, mask is %d, j is %d, local_edges is 0\n", x,y,z,mask,j);
			}
			indices_out[tidx+i]=local_edges[j]-1;
		}
	}
}

void compute_mesh_1ring(const tcnn::GPUMemory<Vector3f> &verts, const tcnn::GPUMemory<uint32_t> &indices, tcnn::GPUMemory<Vector4f> &output_pos, tcnn::GPUMemory<Vector3f> &output_normals) { // computes the average of the 1ring of all verts, as homogenous coordinates
	output_pos.resize(verts.size());
	output_pos.memset(0);
	output_normals.resize(verts.size());
	output_normals.memset(0);
	linear_kernel(accumulate_1ring, 0, nullptr, indices.size()/3, indices.data(), verts.data(), output_pos.data(), output_normals.data());
}

__global__ void compute_mesh_opt_gradients_kernel(
	uint32_t n_verts,
	float thresh,
	const Vector3f* verts,
	const Vector3f* normals,
	const Vector4f* verts_smoothed,
	const network_precision_t* densities,
	uint32_t input_gradient_width,
	const float* input_gradients,
	Vector3f* verts_gradient_out,
	float k_smooth_amount,
	float k_density_amount,
	float k_inflate_amount
) {
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_verts) return;

	Vector3f src = verts[i];
	Vector4f p = verts_smoothed[i];

	if (p.w() <= 0.f) {
		p.w() = 1.f;
	}

	Vector3f target=p.head<3>() * (1.f/p.w());
	Vector3f smoothing_grad = src - target; // negative...

	Vector3f input_gradient = *(const Vector3f *)(input_gradients + i * input_gradient_width);

	Vector3f n = input_gradient.normalized();
	float density = densities[i];
	verts_gradient_out[i] = n * sign(density - thresh) * k_density_amount + smoothing_grad * k_smooth_amount - normals[i].normalized() * k_inflate_amount;
}

void compute_mesh_opt_gradients(
	float thresh,
	const tcnn::GPUMemory<Vector3f>& verts,
	const tcnn::GPUMemory<Vector3f>& normals,
	const tcnn::GPUMemory<Vector4f>& verts_smoothed,
	const network_precision_t* densities,
	uint32_t input_gradients_width,
	const float* input_gradients,
	GPUMemory<Vector3f>& verts_gradient_out,
	float k_smooth_amount,
	float k_density_amount,
	float k_inflate_amount
) {
	linear_kernel(
		compute_mesh_opt_gradients_kernel,
		0,
		nullptr,
		verts.size(),
		thresh,
		verts.data(),
		normals.data(),
		verts_smoothed.data(),
		densities,
		input_gradients_width,
		input_gradients,
		verts_gradient_out.data(),
		k_smooth_amount,
		k_density_amount,
		k_inflate_amount
	);
}

void marching_cubes_gpu(hipStream_t stream, BoundingBox aabb, Vector3i res_3d, float thresh, const tcnn::GPUMemory<float>& density, tcnn::GPUMemory<Vector3f>& verts_out, tcnn::GPUMemory<uint32_t>& indices_out) {
	GPUMemory<uint32_t> counters;

	counters.enlarge(4);
	counters.memset(0);

	size_t n_bytes = res_3d.x() * (size_t)res_3d.y() * res_3d.z() * 3 * sizeof(int);
	auto workspace = allocate_workspace(stream, n_bytes);
	CUDA_CHECK_THROW(hipMemsetAsync(workspace.data(), -1, n_bytes, stream));

	int* vertex_grid = (int*)workspace.data();

	const dim3 threads = { 4, 4, 4 };
	const dim3 blocks = { div_round_up((uint32_t)res_3d.x(), threads.x), div_round_up((uint32_t)res_3d.y(), threads.y), div_round_up((uint32_t)res_3d.z(), threads.z) };
	// count only
	gen_vertices<<<blocks, threads, 0>>>(aabb, res_3d, density.data(), nullptr, nullptr, thresh, counters.data());
	gen_faces<<<blocks, threads, 0>>>(res_3d, density.data(), nullptr, nullptr, thresh, counters.data());
	std::vector<uint32_t> cpucounters; cpucounters.resize(4);
	counters.copy_to_host(cpucounters);
	tlog::info() << "#vertices=" << cpucounters[0] << " #triangles=" << (cpucounters[1]/3);

	uint32_t n_verts=(cpucounters[0]+127)&~127; // round for later nn stuff
	verts_out.resize(n_verts);
	verts_out.memset(0);
	indices_out.resize(cpucounters[1]);
	// actually generate verts
	gen_vertices<<<blocks, threads, 0>>>(aabb, res_3d, density.data(), vertex_grid, verts_out.data(), thresh, counters.data()+2);
	gen_faces<<<blocks, threads, 0>>>(res_3d, density.data(), vertex_grid, indices_out.data(), thresh, counters.data()+2);
}

void save_mesh(
	GPUMemory<Vector3f>& verts,
	GPUMemory<Vector3f>& normals,
	GPUMemory<Vector3f>& colors,
	GPUMemory<uint32_t>& indices,
	const char* outputname,
	bool unwrap_it,
	float nerf_scale,
	Vector3f nerf_offset
) {
	std::vector<Vector3f> cpuverts; cpuverts.resize(verts.size());
	std::vector<Vector3f> cpunormals; cpunormals.resize(normals.size());
	std::vector<Vector3f> cpucolors; cpucolors.resize(colors.size());
	std::vector<uint32_t> cpuindices; cpuindices.resize(indices.size());
	verts.copy_to_host(cpuverts);
	normals.copy_to_host(cpunormals);
	colors.copy_to_host(cpucolors);
	indices.copy_to_host(cpuindices);

	uint32_t numquads = ((cpuindices.size()/3)+1)/2;
	uint32_t numquadsx = uint32_t(sqrtf(numquads)+4) & (~3);
	uint32_t numquadsy = (numquads+numquadsx-1)/numquadsx;
	uint32_t quadresy = 8;
	uint32_t quadresx = quadresy+3;
	uint32_t texw = quadresx*numquadsx;
	uint32_t texh = quadresy*numquadsy;

	if (unwrap_it) {
		uint8_t* tex = (uint8_t*)malloc(texw*texh*3);
		for (uint32_t y = 0; y < texh; ++y) {
			for (uint32_t x = 0; x < texw; ++x) {
				uint32_t q = (x/quadresx)+(y/quadresy)*numquadsx;
				// 0 x x 3 - - 4
				// | .\x x\. . |
				// | . .\x x\. |
				// 2 - - 1 x x 5
				uint32_t xi = x % quadresx, yi = y % quadresy;
				uint32_t t = q*2 + (xi>yi+1);
				int r = (t*923)&255;
				int g = (t*3572)&255;
				int b = (t*5423)&255;
				//if (xi==yi+1 || xi==yi+2)
				//	r=g=b=0;
				tex[x*3+y*3*texw+0]=r;
				tex[x*3+y*3*texw+1]=g;
				tex[x*3+y*3*texw+2]=b;
			}
		}
		stbi_write_tga(fs::path(outputname).with_extension(".tga").str().c_str(), texw, texh, 3, tex);
		free(tex);
	}

	FILE* f = fopen(outputname,"wb");
	if (!f) {
		throw std::runtime_error{"Failed to open " + std::string(outputname) + " for writing."};
	}

	if (fs::path(outputname).extension() == "ply") {
		// ply file
		fprintf(f,
			"ply\n"
			"format ascii 1.0\n"
			"comment output from https://github.com/NVlabs/instant-ngp\n"
			"element vertex %u\n"
			"property float x\n"
			"property float y\n"
			"property float z\n"
			"property float nx\n"
			"property float ny\n"
			"property float nz\n"
			"property uchar red\n"
			"property uchar green\n"
			"property uchar blue\n"
			"element face %u\n"
			"property list uchar int vertex_index\n"
			"end_header\n"
			, (unsigned int)cpuverts.size()
			, (unsigned int)cpuindices.size()/3
		);
		for (size_t i=0;i<cpuverts.size();++i) {
			Vector3f p=(cpuverts[i]-nerf_offset)/nerf_scale;
			Vector3f c=cpucolors[i];
			Vector3f n=cpunormals[i].normalized();
			unsigned char c8[3]={(unsigned char)tcnn::clamp(c.x()*255.f,0.f,255.f),(unsigned char)tcnn::clamp(c.y()*255.f,0.f,255.f),(unsigned char)tcnn::clamp(c.z()*255.f,0.f,255.f)};
			fprintf(f,"%0.5f %0.5f %0.5f %0.3f %0.3f %0.3f %d %d %d\n", p.x(), p.y(), p.z(), n.x(), n.y(), n.z(), c8[0], c8[1], c8[2]);
		}
		for (size_t i=0;i<cpuindices.size();i+=3) {
			fprintf(f,"3 %d %d %d\n", cpuindices[i+2], cpuindices[i+1], cpuindices[i+0]);
		}
	} else {
		// obj file
		if (unwrap_it) {
			fprintf(f, "mtllib nerf.mtl\n");
		}
		for (size_t i = 0; i < cpuverts.size(); ++i) {
			Vector3f p = (cpuverts[i]-nerf_offset)/nerf_scale;
			Vector3f c = cpucolors[i];
			fprintf(f,"v %0.5f %0.5f %0.5f %0.3f %0.3f %0.3f\n", p.x(), p.y(), p.z(), tcnn::clamp(c.x(), 0.f, 1.f), tcnn::clamp(c.y(), 0.f, 1.f), tcnn::clamp(c.z(), 0.f, 1.f));
		}
		for (auto &v: cpunormals) {
			auto n = v.normalized();
			fprintf(f,"vn %0.5f %0.5f %0.5f\n", n.x(), n.y(), n.z());
		}
		if (unwrap_it) {
			for (size_t i = 0; i < cpuindices.size(); i++) {
				uint32_t q = (uint32_t)(i/6);
				uint32_t x = (q%numquadsx)*quadresx;
				uint32_t y = (q/numquadsx)*quadresy;
				uint32_t d = quadresy-1;
				switch (i % 6) {
					case 0: break;
					case 1: x += d; y += d; break;
					case 2: y += d; break;
					case 3: x += 3; break;
					case 4: x += 3+d; break;
					case 5: x += 3+d; y += d; break;
				}
				fprintf(f,"vt %0.5f %0.5f\n", ((float)x+0.5f)/float(texw), 1.f-((float)y+0.5f)/float(texh));
			}
			fprintf(f, "g default\nusemtl nerf\ns 1\n");
			for (size_t i = 0; i < cpuindices.size(); i += 3) {
				fprintf(f,"f %u/%u/%u %u/%u/%u %u/%u/%u\n",
					cpuindices[i+2]+1,(uint32_t)i+3,  cpuindices[i+2]+1,
					cpuindices[i+1]+1,(uint32_t)i+2,cpuindices[i+1]+1,
					cpuindices[i+0]+1,(uint32_t)i+1,cpuindices[i+0]+1
				);
			}
		} else {
			for (size_t i = 0; i < cpuindices.size(); i += 3) {
				fprintf(f,"f %u//%u %u//%u %u//%u\n",
					cpuindices[i+2]+1, cpuindices[i+2]+1, cpuindices[i+1]+1, cpuindices[i+1]+1, cpuindices[i+0]+1, cpuindices[i+0]+1
				);
			}
		}
	}
	fclose(f);
}

void save_density_grid_to_png(const GPUMemory<float>& density, const char* filename, Vector3i res3d, float thresh, bool swap_y_z, float density_range) {
	float density_scale = 128.f / density_range; // map from -density_range to density_range into 0-255
	std::vector<float> density_cpu;
	density_cpu.resize(density.size());
	density.copy_to_host(density_cpu);
	uint32_t num_voxels = 0;
	uint32_t num_lattice_points_near_zero_crossing = 0;
	uint32_t N = res3d.x()*res3d.y()*res3d.z();
	for (int z = 1; z < res3d.z() - 1; ++z) {
		for (int y = 1; y < res3d.y() - 1; ++y) {
			for (int x = 1; x < res3d.x() - 1; ++x) {
				int count = 0;
				count += density_cpu[(x+0)+(y+0)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+1)+(y+0)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+0)+(y+1)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+1)+(y+1)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+0)+(y+0)*res3d.x()+(z+1)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+1)+(y+0)*res3d.x()+(z+1)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+0)+(y+1)*res3d.x()+(z+1)*res3d.x()*res3d.y()] < thresh;
				count += density_cpu[(x+1)+(y+1)*res3d.x()+(z+1)*res3d.x()*res3d.y()] < thresh;
				if (count>0 && count<8) {
					num_voxels++;
				}

				bool mysign = density_cpu[(x+0)+(y+0)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh;
				bool near_zero_crossing=false;
				near_zero_crossing |= (density_cpu[(x+1)+(y+0)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh) != mysign;
				near_zero_crossing |= (density_cpu[(x-1)+(y+0)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh) != mysign;
				near_zero_crossing |= (density_cpu[(x+0)+(y+1)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh) != mysign;
				near_zero_crossing |= (density_cpu[(x+0)+(y-1)*res3d.x()+(z+0)*res3d.x()*res3d.y()] < thresh) != mysign;
				near_zero_crossing |= (density_cpu[(x+0)+(y+0)*res3d.x()+(z+1)*res3d.x()*res3d.y()] < thresh) != mysign;
				near_zero_crossing |= (density_cpu[(x+0)+(y+0)*res3d.x()+(z-1)*res3d.x()*res3d.y()] < thresh) != mysign;
				if (near_zero_crossing) {
					num_lattice_points_near_zero_crossing++;
				}
			}
		}
	}


	if (swap_y_z) {
		res3d = {res3d.x(), res3d.z(), res3d.y()};
	}

	uint32_t ndown = uint32_t(sqrtf(res3d.z()));
	uint32_t nacross = (res3d.z()+ndown-1)/ndown;
	uint32_t w = res3d.x() * nacross;
	uint32_t h = res3d.y() * ndown;
	uint8_t* pngpixels = (uint8_t *)malloc(size_t(w)*size_t(h));
	uint8_t* dst = pngpixels;
	for (int v = 0; v < h; ++v) {
		for (int u = 0; u < w; ++u) {
			int x = u % res3d.x();
			int y = v % res3d.y();
			int z = (u / res3d.x()) + (v / res3d.y()) * nacross;
			if (z < res3d.z()) {
				if (swap_y_z) {
					*dst++ = (uint8_t)tcnn::clamp((density_cpu[x + z*res3d.x() + y*res3d.x()*res3d.z()]-thresh)*density_scale + 128.5f, 0.f, 255.f);
				} else {
					*dst++ = (uint8_t)tcnn::clamp((density_cpu[x + (res3d.y()-1-y)*res3d.x() + z*res3d.x()*res3d.y()]-thresh)*density_scale + 128.5f, 0.f, 255.f);
				}
			} else {
				*dst++ = 0;
			}
		}
	}

	stbi_write_png(filename, w, h, 1, pngpixels, w);

	tlog::success() << "Wrote density PNG to " << filename;
	tlog::info()
		<< "  #lattice points=" << N
		<< " #zero-x voxels=" << num_voxels << " (" << ((num_voxels*100.0)/N) << "%%)"
		<< " #lattice near zero-x=" << num_lattice_points_near_zero_crossing << " (" << ((num_lattice_points_near_zero_crossing*100.0)/N) << "%%)";

	free(pngpixels);
}

// Distinct from `save_density_grid_to_png` not just in that is writes RGBA, but also
// in that it writes a sequence of PNGs rather than a single large PNG.
// TODO: make both methods configurable to do either single PNG or PNG sequence.
void save_rgba_grid_to_png_sequence(const GPUMemory<Array4f>& rgba, const char* path, Vector3i res3d, bool swap_y_z) {
	std::vector<Array4f> rgba_cpu;
	rgba_cpu.resize(rgba.size());
	rgba.copy_to_host(rgba_cpu);

	if (swap_y_z) {
		res3d = {res3d.x(), res3d.z(), res3d.y()};
	}

	uint32_t w = res3d.x();
	uint32_t h = res3d.y();

	auto progress = tlog::progress(res3d.z());

	std::atomic<int> n_saved{0};
	ThreadPool{}.parallelFor<int>(0, res3d.z(), [&](int z) {
		uint8_t* pngpixels = (uint8_t*)malloc(size_t(w) * size_t(h) * 4);
		uint8_t* dst = pngpixels;
		for (int y = 0; y < h; ++y) {
			for (int x = 0; x < w; ++x) {
				size_t i = swap_y_z ? (x + z*res3d.x() + y*res3d.x()*res3d.z()) : (x + (res3d.y()-1-y)*res3d.x() + z*res3d.x()*res3d.y());
				*dst++ = (uint8_t)tcnn::clamp(rgba_cpu[i].x() * 255.f, 0.f, 255.f);
				*dst++ = (uint8_t)tcnn::clamp(rgba_cpu[i].y() * 255.f, 0.f, 255.f);
				*dst++ = (uint8_t)tcnn::clamp(rgba_cpu[i].z() * 255.f, 0.f, 255.f);
				*dst++ = (uint8_t)tcnn::clamp(rgba_cpu[i].w() * 255.f, 0.f, 255.f);
			}
		}
		// write slice
		char filename[256];
		snprintf(filename, sizeof(filename), "%s/%04d_%dx%d.png", path, z, w, h);
		stbi_write_png(filename, w, h, 4, pngpixels, w*4);
		free(pngpixels);

		progress.update(++n_saved);
	});
	tlog::success() << "Wrote RGBA PNG sequence to " << path;
}

NGP_NAMESPACE_END
