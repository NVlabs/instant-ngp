#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed_nerf.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#ifdef copysign
#undef copysign
#endif

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

inline constexpr __device__ float NERF_RENDERING_NEAR_DISTANCE() { return 0.05f; }
inline constexpr __device__ uint32_t NERF_STEPS() { return 1024; } // finest number of steps per unit length
inline constexpr __device__ uint32_t NERF_CASCADES() { return 8; }

inline constexpr __device__ float SQRT3() { return 1.73205080757f; }
inline constexpr __device__ float STEPSIZE() { return (SQRT3() / NERF_STEPS()); } // for nerf raymarch
inline constexpr __device__ float MIN_CONE_STEPSIZE() { return STEPSIZE(); }
// Maximum step size is the width of the coarsest gridsize cell.
inline constexpr __device__ float MAX_CONE_STEPSIZE() { return STEPSIZE() * (1<<(NERF_CASCADES()-1)) * NERF_STEPS() / NERF_GRIDSIZE(); }

// Used to index into the PRNG stream. Must be larger than the number of
// samples consumed by any given training ray.
inline constexpr __device__ uint32_t N_MAX_RANDOM_SAMPLES_PER_RAY() { return 8; }

// Any alpha below this is considered "invisible" and is thus culled away.
inline constexpr __device__ float NERF_MIN_OPTICAL_THICKNESS() { return 0.01f; }

static constexpr uint32_t MARCH_ITER = 10000;

static constexpr uint32_t MIN_STEPS_INBETWEEN_COMPACTION = 1;
static constexpr uint32_t MAX_STEPS_INBETWEEN_COMPACTION = 8;

Testbed::NetworkDims Testbed::network_dims_nerf() const {
	NetworkDims dims;
	dims.n_input = sizeof(NerfCoordinate) / sizeof(float);
	dims.n_output = 4;
	dims.n_pos = sizeof(NerfPosition) / sizeof(float);
	return dims;
}

inline __host__ __device__ uint32_t grid_mip_offset(uint32_t mip) {
	return (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE()) * mip;
}

inline __host__ __device__ float calc_cone_angle(float cosine, const Eigen::Vector2f& focal_length, float cone_angle_constant) {
	// Pixel size. Doesn't always yield a good performance vs. quality
	// trade off. Especially if training pixels have a much different
	// size than rendering pixels.
	// return cosine*cosine / focal_length.mean();

	return cone_angle_constant;
}

inline __host__ __device__ float calc_dt(float t, float cone_angle) {
	return tcnn::clamp(t*cone_angle, MIN_CONE_STEPSIZE(), MAX_CONE_STEPSIZE());
}

struct LossAndGradient {
	Eigen::Array3f loss;
	Eigen::Array3f gradient;

	__host__ __device__ LossAndGradient operator*(float scalar) {
		return {loss * scalar, gradient * scalar};
	}

	__host__ __device__ LossAndGradient operator/(float scalar) {
		return {loss / scalar, gradient / scalar};
	}
};

inline __device__ Array3f copysign(const Array3f& a, const Array3f& b) {
	return {
		copysignf(a.x(), b.x()),
		copysignf(a.y(), b.y()),
		copysignf(a.z(), b.z()),
	};
}

inline __device__ LossAndGradient l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference * difference,
		2.0f * difference
	};
}

inline __device__ LossAndGradient relative_l2_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction * prediction + Array3f::Constant(1e-2f)).inverse();
	return {
		difference * difference * factor,
		2.0f * difference * factor
	};
}

inline __device__ LossAndGradient l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	return {
		difference.abs(),
		copysign(Array3f::Ones(), difference),
	};
}

inline __device__ LossAndGradient huber_loss(const Array3f& target, const Array3f& prediction, float alpha = 1) {
	Array3f difference = prediction - target;
	Array3f abs_diff = difference.abs();
	Array3f square = 0.5f/alpha * difference * difference;
	return {
		{
			abs_diff.x() > alpha ? (abs_diff.x() - 0.5f * alpha) : square.x(),
			abs_diff.y() > alpha ? (abs_diff.y() - 0.5f * alpha) : square.y(),
			abs_diff.z() > alpha ? (abs_diff.z() - 0.5f * alpha) : square.z(),
		},
		{
			abs_diff.x() > alpha ? (difference.x() > 0 ? 1.0f : -1.0f) : (difference.x() / alpha),
			abs_diff.y() > alpha ? (difference.y() > 0 ? 1.0f : -1.0f) : (difference.y() / alpha),
			abs_diff.z() > alpha ? (difference.z() > 0 ? 1.0f : -1.0f) : (difference.z() / alpha),
		},
	};
}

inline __device__ LossAndGradient log_l1_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f divisor = difference.abs() + Array3f::Ones();
	return {
		divisor.log(),
		copysign(divisor.inverse(), difference),
	};
}

inline __device__ LossAndGradient smape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (0.5f * (prediction.abs() + target.abs()) + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}

inline __device__ LossAndGradient mape_loss(const Array3f& target, const Array3f& prediction) {
	Array3f difference = prediction - target;
	Array3f factor = (prediction.abs() + Array3f::Constant(1e-2f)).inverse();
	return {
		difference.abs() * factor,
		copysign(factor, difference),
	};
}

inline __device__ float distance_to_next_voxel(const Vector3f& pos, const Vector3f& dir, const Vector3f& idir, uint32_t res) { // dda like step
	Vector3f p = res * pos;
	float tx = (floorf(p.x() + 0.5f + 0.5f * sign(dir.x())) - p.x()) * idir.x();
	float ty = (floorf(p.y() + 0.5f + 0.5f * sign(dir.y())) - p.y()) * idir.y();
	float tz = (floorf(p.z() + 0.5f + 0.5f * sign(dir.z())) - p.z()) * idir.z();
	float t = min(min(tx, ty), tz);

	return fmaxf(t / res, 0.0f);
}

inline __device__ float advance_to_next_voxel(float t, float cone_angle, const Vector3f& pos, const Vector3f& dir, const Vector3f& idir, uint32_t res) {
	// Analytic stepping by a multiple of dt. Make empty space unequal to non-empty space
	// due to the different stepping.
	// float dt = calc_dt(t, cone_angle);
	// return t + ceilf(fmaxf(distance_to_next_voxel(pos, dir, idir, res) / dt, 0.5f)) * dt;

	// Regular stepping (may be slower but matches non-empty space)
	float t_target = t + distance_to_next_voxel(pos, dir, idir, res);
	do {
		t += calc_dt(t, cone_angle);
	} while (t < t_target);
	return t;
}

__device__ float network_to_rgb(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_rgb_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_density(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(val);
		default: assert(false);
	}
	return 0.0f;
}

__device__ float network_to_density_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -15.0f, 15.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ Array3f network_to_rgb(const tcnn::vector_t<tcnn::network_precision_t, 4>& local_network_output, ENerfActivation activation) {
	return {
		network_to_rgb(float(local_network_output[0]), activation),
		network_to_rgb(float(local_network_output[1]), activation),
		network_to_rgb(float(local_network_output[2]), activation)
	};
}

__device__ Vector3f warp_position(const Vector3f& pos, const BoundingBox& aabb) {
	// return {tcnn::logistic(pos.x() - 0.5f), tcnn::logistic(pos.y() - 0.5f), tcnn::logistic(pos.z() - 0.5f)};
	// return pos;

	return aabb.relative_pos(pos);
}

__device__ Vector3f unwarp_position(const Vector3f& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.min + pos.cwiseProduct(aabb.diag());
}

__device__ Vector3f unwarp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.diag();
}

__device__ Vector3f warp_position_derivative(const Vector3f& pos, const BoundingBox& aabb) {
	return unwarp_position_derivative(pos, aabb).cwiseInverse();
}

__host__ __device__ Vector3f warp_direction(const Vector3f& dir) {
	return (dir + Vector3f::Ones()) * 0.5f;
}

__device__ Vector3f unwarp_direction(const Vector3f& dir) {
	return dir * 2.0f - Vector3f::Ones();
}

__device__ Vector3f warp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(0.5f);
}

__device__ Vector3f unwarp_direction_derivative(const Vector3f& dir) {
	return Vector3f::Constant(2.0f);
}

__device__ float warp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return (dt - MIN_CONE_STEPSIZE()) / (max_stepsize - MIN_CONE_STEPSIZE());
}

__device__ float unwarp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return dt * (max_stepsize - MIN_CONE_STEPSIZE()) + MIN_CONE_STEPSIZE();
}

__device__ uint32_t cascaded_grid_idx_at(Vector3f pos, uint32_t mip) {
	float mip_scale = scalbnf(1.0f, -mip);
	pos -= Vector3f::Constant(0.5f);
	pos *= mip_scale;
	pos += Vector3f::Constant(0.5f);

	Vector3i i = (pos * NERF_GRIDSIZE()).cast<int>();

	if (i.x() < -1 || i.x() > NERF_GRIDSIZE() || i.y() < -1 || i.y() > NERF_GRIDSIZE() || i.z() < -1 || i.z() > NERF_GRIDSIZE()) {
		printf("WTF %d %d %d\n", i.x(), i.y(), i.z());
	}

	uint32_t idx = tcnn::morton3D(
		tcnn::clamp(i.x(), 0, (int)NERF_GRIDSIZE()-1),
		tcnn::clamp(i.y(), 0, (int)NERF_GRIDSIZE()-1),
		tcnn::clamp(i.z(), 0, (int)NERF_GRIDSIZE()-1)
	);

	return idx;
}

__device__ bool density_grid_occupied_at(const Vector3f& pos, const uint8_t* density_grid_bitfield, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return density_grid_bitfield[idx/8+grid_mip_offset(mip)/8] & (1<<(idx%8));
}

__device__ float cascaded_grid_at(Vector3f pos, const float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return cascaded_grid[idx+grid_mip_offset(mip)];
}

__device__ float& cascaded_grid_at(Vector3f pos, float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	return cascaded_grid[idx+grid_mip_offset(mip)];
}

__global__ void extract_srgb_with_activation(const uint32_t n_elements,	const uint32_t rgb_stride, const float* __restrict__ rgbd, float* __restrict__ rgb, ENerfActivation rgb_activation, bool from_linear) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	const uint32_t elem_idx = i / 3;
	const uint32_t dim_idx = i - elem_idx * 3;

	float c = network_to_rgb(rgbd[elem_idx*4 + dim_idx], rgb_activation);
	if (from_linear) {
		c = linear_to_srgb(c);
	}

	rgb[elem_idx*rgb_stride + dim_idx] = c;
}

__global__ void mark_untrained_density_grid(const uint32_t n_elements,  float* __restrict__ grid_out,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	bool clear_visible_voxels
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t level = i / (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());
	uint32_t pos_idx = i % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);



	Vector3f pos = ((Vector3f{(float)x+0.5f, (float)y+0.5f, (float)z+0.5f}) / NERF_GRIDSIZE() - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);
	float voxel_radius = 0.5f*SQRT3()*scalbnf(1.0f, level) / NERF_GRIDSIZE();
	int count=0;
	for (uint32_t j=0; j < n_training_images; ++j) {
		if (metadata[j].camera_distortion.mode == ECameraDistortionMode::FTheta) {
			// not supported for now
			count++;
			break;
		}
		float half_resx = metadata[j].resolution.x() * 0.5f;
		float half_resy = metadata[j].resolution.y() * 0.5f;
		Matrix<float, 3, 4> xform = training_xforms[j].start;
		Vector3f ploc = pos - xform.col(3);
		float x = ploc.dot(xform.col(0));
		float y = ploc.dot(xform.col(1));
		float z = ploc.dot(xform.col(2));
		if (z > 0.f) {
			auto focal = metadata[j].focal_length;
			// TODO - add a box / plane intersection to stop thomas from murdering me
			if (fabsf(x) - voxel_radius < z / focal.x() * half_resx && fabsf(y) - voxel_radius < z / focal.y() * half_resy) {
				count++;
				if (count > 0) break;
			}
		}
	}

	if (clear_visible_voxels || (grid_out[i] < 0) != (count <= 0)) {
		grid_out[i] = (count > 0) ? 0.f : -1.f;
	}
}

__global__ void generate_grid_samples_nerf_uniform(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, NerfPosition* __restrict__ out) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	out[i] = { warp_position(pos, train_aabb), warp_dt(MIN_CONE_STEPSIZE()) };
}

// generate samples for uniform grid including constant ray direction
__global__ void generate_grid_samples_nerf_uniform_dir(Eigen::Vector3i res_3d, const uint32_t step, BoundingBox render_aabb, BoundingBox train_aabb, Eigen::Vector3f ray_dir, NerfCoordinate* __restrict__ network_input) {
	// check grid_in for negative values -> must be negative on output
	uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	uint32_t z = threadIdx.z + blockIdx.z * blockDim.z;
	if (x>=res_3d.x() || y>=res_3d.y() || z>=res_3d.z())
		return;
	uint32_t i = x+ y*res_3d.x() + z*res_3d.x()*res_3d.y();
	Vector3f pos = Array3f{(float)x, (float)y, (float)z} * Array3f{1.f/res_3d.x(),1.f/res_3d.y(),1.f/res_3d.z()};
	pos = pos.cwiseProduct(render_aabb.max - render_aabb.min) + render_aabb.min;
	network_input[i] = { warp_position(pos, train_aabb), warp_direction(ray_dir), warp_dt(MIN_CONE_STEPSIZE()) };
}

inline __device__ int mip_from_pos(const Vector3f& pos) {
	int exponent;
	float maxval = (pos - Vector3f::Constant(0.5f)).cwiseAbs().maxCoeff();
	frexpf(maxval, &exponent);
	return min(NERF_CASCADES()-1, max(0, exponent+1));
}

inline __device__ int mip_from_dt(float dt, const Vector3f& pos) {
	int mip = mip_from_pos(pos);
	dt *= 2*NERF_GRIDSIZE();
	if (dt<1.f) return mip;
	int exponent;
	frexpf(dt, &exponent);
	return min(NERF_CASCADES()-1, max(exponent, mip));
}

__global__ void generate_grid_samples_nerf_nonuniform(const uint32_t n_elements, default_rng_t rng, const uint32_t step, BoundingBox aabb, const float* __restrict__ grid_in, NerfPosition* __restrict__ out, uint32_t* __restrict__ indices, uint32_t n_cascades, float thresh) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// 1 random number to select the level, 3 to select the position.
	rng.advance(i*4);
	uint32_t level = (uint32_t)(random_val(rng) * n_cascades) % n_cascades;

	// Select grid cell that has density
	uint32_t idx;
	for (uint32_t j = 0; j < 10; ++j) {
		idx = ((i+step*n_elements) * 56924617 + j * 19349663 + 96925573) % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());
		idx += level * NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE();
		if (grid_in[idx] > thresh) {
			break;
		}
	}

	// Random position within that cellq
	uint32_t pos_idx = idx % (NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE());

	uint32_t x = tcnn::morton3D_invert(pos_idx>>0);
	uint32_t y = tcnn::morton3D_invert(pos_idx>>1);
	uint32_t z = tcnn::morton3D_invert(pos_idx>>2);

	Vector3f pos = ((Vector3f{(float)x, (float)y, (float)z} + random_val_3d(rng)) / NERF_GRIDSIZE() - Vector3f::Constant(0.5f)) * scalbnf(1.0f, level) + Vector3f::Constant(0.5f);

	out[i] = { warp_position(pos, aabb), warp_dt(MIN_CONE_STEPSIZE()) };
	indices[i] = idx;
}

__global__ void splat_grid_samples_nerf_max_nearest_neighbor(const uint32_t n_elements, const uint32_t* __restrict__ indices, const tcnn::network_precision_t* network_output, float* __restrict__ grid_out, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint32_t local_idx = indices[i];

	// Current setting: optical thickness of the smallest possible stepsize.
	// Uncomment for:   optical thickness of the ~expected step size when the observer is in the middle of the scene
	uint32_t level = 0;//local_idx / (NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE());

	float mlp = network_to_density(float(network_output[i]), density_activation);
	float optical_thickness = mlp * scalbnf(MIN_CONE_STEPSIZE(), level);

	// Positive floats are monotonically ordered when their bit pattern is interpretes as uint.
	// uint atomicMax is thus perfectly acceptable.
	atomicMax((uint32_t*)&grid_out[local_idx], __float_as_uint(optical_thickness));
}

__global__ void grid_samples_half_to_float(const uint32_t n_elements, BoundingBox aabb, float* dst, const tcnn::network_precision_t* network_output, ENerfActivation density_activation, const NerfPosition* __restrict__ coords_in, const float* __restrict__ grid_in) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	// let's interpolate for marching cubes based on the raw MLP output, not the density (exponentiated) version
	//float mlp = network_to_density(float(network_output[i * padded_output_width]), density_activation);
	float mlp = float(network_output[i]);

	if (grid_in) {
		Vector3f pos = unwarp_position(coords_in[i].p, aabb);
		float grid_density = cascaded_grid_at(pos, grid_in, mip_from_pos(pos));
		if (grid_density < NERF_MIN_OPTICAL_THICKNESS()) {
			mlp = -10000.f;
		}
	}
	dst[i] = mlp;
}

__global__ void ema_grid_samples_nerf(const uint32_t n_elements,
	float decay,
	const uint32_t count,
	float* __restrict__ grid_out,
	const float* __restrict__ grid_in
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	float importance = grid_in[i];

	// float ema_debias_old = 1 - (float)powf(decay, count);
	// float ema_debias_new = 1 - (float)powf(decay, count+1);

	// float filtered_val = ((grid_out[i] * decay * ema_debias_old + importance * (1 - decay)) / ema_debias_new);
	// grid_out[i] = filtered_val;

	// Maximum instead of EMA allows capture of very thin features.
	// Basically, we want the grid cell turned on as soon as _ANYTHING_ visible is in there.

	float prev_val = grid_out[i];
	float val = (prev_val<0.f) ? prev_val : fmaxf(prev_val * decay, importance);
	grid_out[i] = val;
}

__global__ void decay_sharpness_grid_nerf(const uint32_t n_elements, float decay, float* __restrict__ grid) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	grid[i] *= decay;
}

__global__ void grid_to_bitfield(const uint32_t n_elements,
	const float* __restrict__ grid,
	uint8_t* __restrict__ grid_bitfield,
	const float* __restrict__ mean_density_ptr
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	float thresh = std::min(NERF_MIN_OPTICAL_THICKNESS(), *mean_density_ptr);

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		bits |= grid[i*8+j] > thresh ? ((uint8_t)1 << j) : 0;
	}

	grid_bitfield[i] = bits;
}

__global__ void bitfield_max_pool(const uint32_t n_elements,
	const uint8_t* __restrict__ prev_level,
	uint8_t* __restrict__ next_level
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	uint8_t bits = 0;

	#pragma unroll
	for (uint8_t j = 0; j < 8; ++j) {
		// If any bit is set in the previous level, set this
		// level's bit. (Max pooling.)
		bits |= prev_level[i*8+j] > 0 ? ((uint8_t)1 << j) : 0;
	}

	uint32_t x = tcnn::morton3D_invert(i>>0) + NERF_GRIDSIZE()/8;
	uint32_t y = tcnn::morton3D_invert(i>>1) + NERF_GRIDSIZE()/8;
	uint32_t z = tcnn::morton3D_invert(i>>2) + NERF_GRIDSIZE()/8;

	next_level[tcnn::morton3D(x, y, z)] |= bits;
}

__global__ void advance_pos_nerf(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	Vector3f camera_fwd,
	Vector2f focal_length,
	uint32_t sample_index,
	NerfPayload* __restrict__ payloads,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;
	float dt = calc_dt(t, cone_angle);
	t += ld_random_val(sample_index, i * 786433) * dt;
	Vector3f pos;

	while (1) {
		if (!render_aabb.contains(pos = origin + dir * t)) {
			payload.alive = false;
			break;
		}

		dt = calc_dt(t, cone_angle);
		uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

		if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
			break;
		}

		uint32_t res = NERF_GRIDSIZE()>>mip;
		t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
	}

	payload.t = t;
}

__global__ void generate_nerf_network_inputs_from_positions(const uint32_t n_elements, BoundingBox aabb, const Vector3f* __restrict__ pos, PitchedPtr<NerfCoordinate> network_input, const float* extra_dims) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir=(pos[i]-Vector3f::Constant(0.5f)).normalized(); // choose outward pointing directions, for want of a better choice
	network_input(i)->set_with_optional_extra_dims(warp_position(pos[i], aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()), extra_dims, network_input.stride_in_bytes);
}

__global__ void generate_nerf_network_inputs_at_current_position(const uint32_t n_elements, BoundingBox aabb, const NerfPayload* __restrict__ payloads, PitchedPtr<NerfCoordinate> network_input, const float* extra_dims) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Vector3f dir = payloads[i].dir;
	network_input(i)->set_with_optional_extra_dims(warp_position(payloads[i].origin + dir * payloads[i].t, aabb), warp_direction(dir), warp_dt(MIN_CONE_STEPSIZE()), extra_dims, network_input.stride_in_bytes);
}

__global__ void compute_nerf_density(const uint32_t n_elements, Array4f* network_output, ENerfActivation rgb_activation, ENerfActivation density_activation) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	Array4f rgba = network_output[i];
	rgba.w() = tcnn::clamp(1.f - __expf(-network_to_density(rgba.w(), density_activation) / 100.0f), 0.0f, 1.0f);
	rgba.x() = network_to_rgb(rgba.x(), rgb_activation) * rgba.w();
	rgba.y() = network_to_rgb(rgba.y(), rgb_activation) * rgba.w();
	rgba.z() = network_to_rgb(rgba.z(), rgb_activation) * rgba.w();

	network_output[i] = rgba;
}

__global__ void generate_next_nerf_network_inputs(
	const uint32_t n_elements,
	BoundingBox render_aabb,
	BoundingBox train_aabb,
	Vector2f focal_length,
	Vector3f camera_fwd,
	NerfPayload* __restrict__ payloads,
	PitchedPtr<NerfCoordinate> network_input,
	uint32_t n_steps,
	const uint8_t* __restrict__ density_grid,
	uint32_t min_mip,
	float cone_angle_constant,
	const float* extra_dims
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Vector3f origin = payload.origin;
	Vector3f dir = payload.dir;
	Vector3f idir = dir.cwiseInverse();

	float cone_angle = calc_cone_angle(dir.dot(camera_fwd), focal_length, cone_angle_constant);

	float t = payload.t;

	for (uint32_t j = 0; j < n_steps; ++j) {
		Vector3f pos;
		float dt = 0.0f;
		while (1) {
			if (!render_aabb.contains(pos = origin + dir * t)) {
				payload.n_steps = j;
				return;
			}

			dt = calc_dt(t, cone_angle);
			uint32_t mip = max(min_mip, mip_from_dt(dt, pos));

			if (!density_grid || density_grid_occupied_at(pos, density_grid, mip)) {
				break;
			}

			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, dir, idir, res);
		}

		network_input(i + j * n_elements)->set_with_optional_extra_dims(warp_position(pos, train_aabb), warp_direction(dir), warp_dt(dt), extra_dims, network_input.stride_in_bytes); // XXXCONE
		t += dt;
	}

	payload.t = t;
	payload.n_steps = n_steps;
}

__global__ void composite_kernel_nerf(
	const uint32_t n_elements,
	const uint32_t stride,
	const uint32_t current_step,
	BoundingBox aabb,
	float glow_y_cutoff,
	int glow_mode,
	const uint32_t n_training_images,
	const TrainingXForm* __restrict__ training_xforms,
	Matrix<float, 3, 4> camera_matrix,
	Vector2f focal_length,
	float depth_scale,
	Array4f* __restrict__ rgba,
	float* __restrict__ depth,
	NerfPayload* payloads,
	PitchedPtr<NerfCoordinate> network_input,
	const tcnn::network_precision_t* __restrict__ network_output,
	uint32_t padded_output_width,
	uint32_t n_steps,
	ERenderMode render_mode,
	const uint8_t* __restrict__ density_grid,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_transmittance
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& payload = payloads[i];

	if (!payload.alive) {
		return;
	}

	Array4f local_rgba = rgba[i];
	float local_depth = depth[i];
	Vector3f origin = payload.origin;
	Vector3f cam_fwd = camera_matrix.col(2);
	// Composite in the last n steps
	uint32_t actual_n_steps = payload.n_steps;
	uint32_t j = 0;

	for (; j < actual_n_steps; ++j) {
		tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output;
		local_network_output[0] = network_output[i + j * n_elements + 0 * stride];
		local_network_output[1] = network_output[i + j * n_elements + 1 * stride];
		local_network_output[2] = network_output[i + j * n_elements + 2 * stride];
		local_network_output[3] = network_output[i + j * n_elements + 3 * stride];
		const NerfCoordinate* input = network_input(i + j * n_elements);
		Vector3f warped_pos = input->pos.p;
		Vector3f pos = unwarp_position(warped_pos, aabb);

		float T = 1.f - local_rgba.w();
		float dt = unwarp_dt(input->dt);
		float alpha = 1.f - __expf(-network_to_density(float(local_network_output[3]), density_activation) * dt);
		if (show_accel >= 0) {
			alpha = 1.f;
		}
		float weight = alpha * T;

		Array3f rgb = network_to_rgb(local_network_output, rgb_activation);

		if (glow_mode) { // random grid visualizations ftw!
#if 0
			if (0) {  // extremely startrek edition
				float glow_y = (pos.y() - (glow_y_cutoff - 0.5f)) * 2.f;
				if (glow_y>1.f) glow_y=max(0.f,21.f-glow_y*20.f);
				if (glow_y>0.f) {
					float line;
					line =max(0.f,cosf(pos.y()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.x()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.z()*2.f*3.141592653589793f * 16.f)-0.95f);
					line+=max(0.f,cosf(pos.y()*4.f*3.141592653589793f * 16.f)-0.975f);
					line+=max(0.f,cosf(pos.x()*4.f*3.141592653589793f * 16.f)-0.975f);
					line+=max(0.f,cosf(pos.z()*4.f*3.141592653589793f * 16.f)-0.975f);
					glow_y=glow_y*glow_y*0.5f + glow_y*line*25.f;
					rgb.y()+=glow_y;
					rgb.z()+=glow_y*0.5f;
					rgb.x()+=glow_y*0.25f;
				}
			}
#endif
			float glow = 0.f;

			bool green_grid = glow_mode & 1;
			bool green_cutline = glow_mode & 2;
			bool mask_to_alpha = glow_mode & 4;

			// less used?
			bool radial_mode = glow_mode & 8;
			bool grid_mode = glow_mode & 16; // makes object rgb go black!

			{
				float dist;
				if (radial_mode) {
					dist = (pos - camera_matrix.col(3)).norm();
					dist = min(dist, (4.5f - pos.y()) * 0.333f);
				} else {
					dist = pos.y();
				}

				if (grid_mode) {
					glow = 1.f / max(1.f, dist);
				} else {
					float y = glow_y_cutoff - dist; // - (ii*0.005f);
					float mask = 0.f;
					if (y > 0.f) {
						y *= 80.f;
						mask = min(1.f, y);
						//if (mask_mode) {
						//	rgb.x()=rgb.y()=rgb.z()=mask; // mask mode
						//} else
						{
							if (green_cutline) {
								glow += max(0.f, 1.f - abs(1.f -y)) * 4.f;
							}

							if (y>1.f) {
								y = 1.f - (y - 1.f) * 0.05f;
							}

							if (green_grid) {
								glow += max(0.f, y / max(1.f, dist));
							}
						}
					}
					if (mask_to_alpha) {
						weight *= mask;
					}
				}
			}

			if (glow > 0.f) {
				float line;
				line  = max(0.f, cosf(pos.y() * 2.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.x() * 2.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.z() * 2.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.y() * 4.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.x() * 4.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.z() * 4.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.y() * 8.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.x() * 8.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.z() * 8.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.y() * 16.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.x() * 16.f * 3.141592653589793f * 16.f) - 0.975f);
				line += max(0.f, cosf(pos.z() * 16.f * 3.141592653589793f * 16.f) - 0.975f);
				if (grid_mode) {
					glow = /*glow*glow*0.75f + */ glow * line * 15.f;
					rgb.y() = glow;
					rgb.z() = glow * 0.5f;
					rgb.x() = glow * 0.25f;
				} else {
					glow = glow * glow * 0.25f + glow * line * 15.f;
					rgb.y() += glow;
					rgb.z() += glow * 0.5f;
					rgb.x() += glow * 0.25f;
				}
			}
		} // glow

		if (render_mode == ERenderMode::Normals) {
			// Network input contains the gradient of the network output w.r.t. input.
			// So to compute density gradients, we need to apply the chain rule.
			// The normal is then in the opposite direction of the density gradient (i.e. the direction of decreasing density)
			Vector3f normal = -network_to_density_derivative(float(local_network_output[3]), density_activation) * warped_pos;
			rgb = normal.normalized().array();
		} else if (render_mode == ERenderMode::Positions) {
			if (show_accel >= 0) {
				uint32_t mip = max(show_accel, mip_from_pos(pos));
				uint32_t res = NERF_GRIDSIZE() >> mip;
				int ix = pos.x()*(res);
				int iy = pos.y()*(res);
				int iz = pos.z()*(res);
				default_rng_t rng(ix+iy*232323+iz*727272);
				rgb.x() = 1.f-mip*(1.f/(NERF_CASCADES()-1));
				rgb.y() = rng.next_float();
				rgb.z() = rng.next_float();
			} else {
				rgb = (pos.array() - Array3f::Constant(0.5f)) / 2.0f + Array3f::Constant(0.5f);
			}
		} else if (render_mode == ERenderMode::EncodingVis) {
			rgb = warped_pos.array();
		} else if (render_mode == ERenderMode::Depth) {
			rgb = Array3f::Constant(cam_fwd.dot(pos - origin) * depth_scale);
		} else if (render_mode == ERenderMode::AO) {
			rgb = Array3f::Constant(alpha);
		}

		local_rgba.head<3>() += rgb * weight;
		local_rgba.w() += weight;
		if (weight > payload.max_weight) {
			payload.max_weight = weight;
			local_depth = cam_fwd.dot(pos - camera_matrix.col(3));
		}

		if (local_rgba.w() > (1.0f - min_transmittance)) {
			local_rgba /= local_rgba.w();
			break;
		}
	}

	if (j < n_steps) {
		payload.alive = false;
		payload.n_steps = j + current_step;
	}

	rgba[i] = local_rgba;
	depth[i] = local_depth;
}

static constexpr float UNIFORM_SAMPLING_FRACTION = 0.5f;

inline __device__ Vector2f sample_cdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, float* __restrict__ pdf) {
	if (sample.x() < UNIFORM_SAMPLING_FRACTION) {
		sample.x() /= UNIFORM_SAMPLING_FRACTION;
		return sample;
	}

	sample.x() = (sample.x() - UNIFORM_SAMPLING_FRACTION) / (1.0f - UNIFORM_SAMPLING_FRACTION);

	cdf_y += img * res.y();

	// First select row according to cdf_y
	uint32_t y = binary_search(sample.y(), cdf_y, res.y());
	float prev = y > 0 ? cdf_y[y-1] : 0.0f;
	float pmf_y = cdf_y[y] - prev;
	sample.y() = (sample.y() - prev) / pmf_y;

	cdf_x_cond_y += img * res.y() * res.x() + y * res.x();

	// Then, select col according to x
	uint32_t x = binary_search(sample.x(), cdf_x_cond_y, res.x());
	prev = x > 0 ? cdf_x_cond_y[x-1] : 0.0f;
	float pmf_x = cdf_x_cond_y[x] - prev;
	sample.x() = (sample.x() - prev) / pmf_x;

	if (pdf) {
		*pdf = pmf_x * pmf_y * res.prod();
	}

	return {((float)x + sample.x()) / (float)res.x(), ((float)y + sample.y()) / (float)res.y()};
}

inline __device__ float pdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y) {
	Vector2i p = (sample.cwiseProduct(res.cast<float>())).cast<int>().cwiseMax(0).cwiseMin(res - Vector2i::Ones());

	cdf_y += img * res.y();
	cdf_x_cond_y += img * res.y() * res.x() + p.y() * res.x();

	float pmf_y = cdf_y[p.y()];
	if (p.y() > 0) {
		pmf_y -= cdf_y[p.y()-1];
	}

	float pmf_x = cdf_x_cond_y[p.x()];
	if (p.x() > 0) {
		pmf_x -= cdf_x_cond_y[p.x()-1];
	}

	// Probability mass of picking the pixel
	float pmf = pmf_x * pmf_y;

	// To convert to probability density, divide by area of pixel
	return UNIFORM_SAMPLING_FRACTION + pmf * res.prod() * (1.0f - UNIFORM_SAMPLING_FRACTION);
}

inline __device__ Vector2f nerf_random_image_pos_training(default_rng_t& rng, const Vector2i& resolution, bool snap_to_pixel_centers, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, const Vector2i& cdf_res, uint32_t img, float* __restrict__ pdf = nullptr) {
	Vector2f xy = random_val_2d(rng);

	if (cdf_x_cond_y) {
		xy = sample_cdf_2d(xy, img, cdf_res, cdf_x_cond_y, cdf_y, pdf);
	} else if (pdf) {
		*pdf = 1.0f;
	}

	if (snap_to_pixel_centers) {
		xy = (xy.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	}
	return xy;
}

inline __device__ uint32_t image_idx(uint32_t base_idx, uint32_t n_rays, uint32_t n_rays_total, uint32_t n_training_images, const float* __restrict__ cdf = nullptr, float* __restrict__ pdf = nullptr) {
	if (cdf) {
		float sample = ld_random_val(base_idx + n_rays_total, 0xdeadbeef);
		// float sample = random_val(base_idx + n_rays_total);
		uint32_t img = binary_search(sample, cdf, n_training_images);

		if (pdf) {
			float prev = img > 0 ? cdf[img-1] : 0.0f;
			*pdf = (cdf[img] - prev) * n_training_images;
		}

		return img;
	}

	// return ((base_idx + n_rays_total) * 56924617 + 96925573) % n_training_images;

	// Neighboring threads in the warp process the same image. Increases locality.
	if (pdf) {
		*pdf = 1.0f;
	}
	return (((base_idx + n_rays_total) * n_training_images) / n_rays) % n_training_images;
}

__global__ void generate_training_samples_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	const uint32_t n_rays_total,
	default_rng_t rng,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out_unnormalized,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<NerfCoordinate> coords_out,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	const uint8_t* __restrict__ density_grid,
	bool max_level_rand_training,
	float* __restrict__ max_level_ptr,
	bool snap_to_pixel_centers,
	bool train_envmap,
	float cone_angle_constant,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i cdf_res,
	const float* __restrict__ extra_dims_gpu,
	uint32_t n_extra_dims
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

	uint32_t img = image_idx(i, n_rays, n_rays_total, n_training_images, cdf_img);
	Eigen::Vector2i resolution = metadata[img].resolution;

	rng.advance(i * N_MAX_RANDOM_SAMPLES_PER_RAY());
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, cdf_res, img);

	// Negative values indicate masked-away regions
	size_t pix_idx = pixel_idx(xy, resolution, 0);
	if (read_rgba(xy, resolution, metadata[img].pixels, metadata[img].image_data_type).x() < 0.0f) {
		return;
	}

	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	float motionblur_time = random_val(rng);

	const Vector2f focal_length = metadata[img].focal_length;
	const Vector2f principal_point = metadata[img].principal_point;
	const float* extra_dims = extra_dims_gpu + img * n_extra_dims;
	const CameraDistortion camera_distortion = metadata[img].camera_distortion;

	const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, motionblur_time);

	Ray ray_unnormalized;
	const Ray* rays_in_unnormalized = metadata[img].rays;
	if (rays_in_unnormalized) {
		// Rays have been explicitly supplied. Read them.
		ray_unnormalized = rays_in_unnormalized[pix_idx];

		/* DEBUG - compare the stored rays to the computed ones
		const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, 0.f);
		Ray ray2;
		ray2.o = xform.col(3);
		ray2.d = f_theta_distortion(xy, principal_point, camera_distortion);
		ray2.d = (xform.block<3, 3>(0, 0) * ray2.d).normalized();
		if (i==1000) {
			printf("\n%d uv %0.3f,%0.3f pixel %0.2f,%0.2f transform from [%0.5f %0.5f %0.5f] to [%0.5f %0.5f %0.5f]\n"
				" origin    [%0.5f %0.5f %0.5f] vs [%0.5f %0.5f %0.5f]\n"
				" direction [%0.5f %0.5f %0.5f] vs [%0.5f %0.5f %0.5f]\n"
			, img,xy.x(), xy.y(), xy.x()*resolution.x(), xy.y()*resolution.y(),
				training_xforms[img].start.col(3).x(),training_xforms[img].start.col(3).y(),training_xforms[img].start.col(3).z(),
				training_xforms[img].end.col(3).x(),training_xforms[img].end.col(3).y(),training_xforms[img].end.col(3).z(),
				ray_unnormalized.o.x(),ray_unnormalized.o.y(),ray_unnormalized.o.z(),
				ray2.o.x(),ray2.o.y(),ray2.o.z(),
				ray_unnormalized.d.x(),ray_unnormalized.d.y(),ray_unnormalized.d.z(),
				ray2.d.x(),ray2.d.y(),ray2.d.z());
		}
		*/
	} else {
		// Rays need to be inferred from the camera matrix
		ray_unnormalized.o = xform.col(3);
		if (camera_distortion.mode == ECameraDistortionMode::FTheta) {
			ray_unnormalized.d = f_theta_undistortion(xy - principal_point, camera_distortion.params, {0.f, 0.f, 1.f});
		} else {
			ray_unnormalized.d = {
				(xy.x()-principal_point.x())*resolution.x() / focal_length.x(),
				(xy.y()-principal_point.y())*resolution.y() / focal_length.y(),
				1.0f,
			};

			if (camera_distortion.mode == ECameraDistortionMode::Iterative) {
				iterative_camera_undistortion(camera_distortion.params, &ray_unnormalized.d.x(), &ray_unnormalized.d.y());
			}
		}

		if (distortion_data) {
			ray_unnormalized.d.head<2>() += read_image<2>(distortion_data, distortion_resolution, xy);
		}

		ray_unnormalized.d = (xform.block<3, 3>(0, 0) * ray_unnormalized.d); // NOT normalized
	}

	Eigen::Vector3f ray_d_normalized = ray_unnormalized.d.normalized();

	Vector2f tminmax = aabb.ray_intersect(ray_unnormalized.o, ray_d_normalized);
	float cone_angle = calc_cone_angle(ray_d_normalized.dot(xform.col(2)), focal_length, cone_angle_constant);

	// The near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), 0.0f);

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle) * random_val(rng);
	Vector3f idir = ray_d_normalized.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t=startt;
	Vector3f pos;

	while (aabb.contains(pos = ray_unnormalized.o + t * ray_d_normalized) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray_d_normalized, idir, res);
		}
	}
	if (j == 0 && !train_envmap) {
		return;
	}
	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = i;
	rays_out_unnormalized[ray_idx] = ray_unnormalized;
	numsteps_out[ray_idx*2+0] = numsteps;
	numsteps_out[ray_idx*2+1] = base;

	Vector3f warped_dir = warp_direction(ray_d_normalized);
	t=startt;
	j=0;
	while (aabb.contains(pos = ray_unnormalized.o + t * ray_d_normalized) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			coords_out(j)->set_with_optional_extra_dims(warp_position(pos, aabb), warped_dir, warp_dt(dt), extra_dims, coords_out.stride_in_bytes);
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray_d_normalized, idir, res);
		}
	}
	if (max_level_rand_training) {
		max_level_ptr += base;
		for (j = 0; j < numsteps; ++j) {
			max_level_ptr[j] = max_level;
		}
	}
}


__device__ LossAndGradient loss_and_gradient(const Vector3f& target, const Vector3f& prediction, ELossType loss_type) {
	switch (loss_type) {
		case ELossType::RelativeL2:  return relative_l2_loss(target, prediction); break;
		case ELossType::L1:          return l1_loss(target, prediction); break;
		case ELossType::Mape:        return mape_loss(target, prediction); break;
		case ELossType::Smape:       return smape_loss(target, prediction); break;
		// Note: we divide the huber loss by a factor of 5 such that its L2 region near zero
		// matches with the L2 loss and error numbers become more comparable. This allows reading
		// off dB numbers of ~converged models and treating them as approximate PSNR to compare
		// with other NeRF methods. Self-normalizing optimizers such as Adam are agnostic to such
		// constant factors; optimization is therefore unaffected.
		case ELossType::Huber:       return huber_loss(target, prediction, 0.1f) / 5.0f; break;
		case ELossType::LogL1:       return log_l1_loss(target, prediction); break;
		default: case ELossType::L2: return l2_loss(target, prediction); break;
	}
}

__global__ void compute_loss_kernel_train_nerf(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const uint32_t max_samples_compacted,
	const uint32_t* __restrict__ rays_counter,
	float loss_scale,
	int padded_output_width,
	const float* __restrict__ envmap_data,
	float* __restrict__ envmap_gradient,
	const Vector2i envmap_resolution,
	ELossType envmap_loss_type,
	Array3f background_color,
	EColorSpace color_space,
	bool train_with_random_bg_color,
	bool train_in_linear_colors,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const tcnn::network_precision_t* network_output,
	uint32_t* __restrict__ numsteps_counter,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in_unnormalized,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<const NerfCoordinate> coords_in,
	PitchedPtr<NerfCoordinate> coords_out,
	tcnn::network_precision_t* dloss_doutput,
	ELossType loss_type,
	float* __restrict__ loss_output,
	bool max_level_rand_training,
	float* __restrict__ max_level_compacted_ptr,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	bool snap_to_pixel_centers,
	float* __restrict__ error_map,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res,
	const Vector2i error_map_cdf_res,
	const float* __restrict__ sharpness_data,
	Eigen::Vector2i sharpness_resolution,
	float* __restrict__ sharpness_grid,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	const Eigen::Array3f* __restrict__ exposure,
	Eigen::Array3f* __restrict__ exposure_gradient,
	float depth_supervision_lambda,
	float near_distance
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	uint32_t base = numsteps_in[i*2+1];

	coords_in += base;
	network_output += base * padded_output_width;

	float T = 1.f;

	float EPSILON = 1e-4f;

	Array3f rgb_ray = Array3f::Zero();
	Vector3f hitpoint = Vector3f::Zero();

	float depth_ray = 0.f;
	uint32_t compacted_numsteps = 0;
	Eigen::Vector3f ray_o = rays_in_unnormalized[i].o;
	for (; compacted_numsteps < numsteps; ++compacted_numsteps) {
		if (T < EPSILON) {
			break;
		}

		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const Vector3f pos = unwarp_position(coords_in.ptr->pos.p, aabb);
		const float dt = unwarp_dt(coords_in.ptr->dt);
		float cur_depth = (pos - ray_o).norm();
		float density = network_to_density(float(local_network_output[3]), density_activation);


		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray += weight * rgb;
		hitpoint += weight * pos;
		depth_ray += weight * cur_depth;
		T *= (1.f - alpha);

		network_output += padded_output_width;
		coords_in += 1;
	}
	hitpoint /= (1.0f - T);

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());

	float img_pdf = 1.0f;
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img, &img_pdf);
	Eigen::Vector2i resolution = metadata[img].resolution;

	float xy_pdf = 1.0f;
	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_cdf_res, img, &xy_pdf);
	float max_level = max_level_rand_training ? (random_val(rng) * 2.0f) : 1.0f; // Multiply by 2 to ensure 50% of training is at max level

	if (train_with_random_bg_color) {
		background_color = random_val_3d(rng);
	}
	Array3f pre_envmap_background_color = background_color = srgb_to_linear(background_color);

	// Composit background behind envmap
	Array4f envmap_value;
	Vector3f dir;
	if (envmap_data) {
		dir = rays_in_unnormalized[i].d.normalized();
		envmap_value = read_envmap(envmap_data, envmap_resolution, dir);
		background_color = envmap_value.head<3>() + background_color * (1.0f - envmap_value.w());
	}

	Array3f exposure_scale = (0.6931471805599453f * exposure[img]).exp();
	// Array3f rgbtarget = composit_and_lerp(xy, resolution, img, training_images, background_color, exposure_scale);
	// Array3f rgbtarget = composit(xy, resolution, img, training_images, background_color, exposure_scale);
	Array4f texsamp = read_rgba(xy, resolution, metadata[img].pixels, metadata[img].image_data_type);

	Array3f rgbtarget;
	if (train_in_linear_colors || color_space == EColorSpace::Linear) {
		rgbtarget = exposure_scale * texsamp.head<3>() + (1.0f - texsamp.w()) * background_color;

		if (!train_in_linear_colors) {
			rgbtarget = linear_to_srgb(rgbtarget);
			background_color = linear_to_srgb(background_color);
		}
	} else if (color_space == EColorSpace::SRGB) {
		background_color = linear_to_srgb(background_color);
		if (texsamp.w() > 0) {
			rgbtarget = linear_to_srgb(exposure_scale * texsamp.head<3>() / texsamp.w()) * texsamp.w() + (1.0f - texsamp.w()) * background_color;
		} else {
			rgbtarget = background_color;
		}
	}

	if (compacted_numsteps == numsteps) {
		// support arbitrary background colors
		rgb_ray += T * background_color;
	}

	// Step again, this time computing loss
	network_output -= padded_output_width * compacted_numsteps; // rewind the pointer
	coords_in -= compacted_numsteps;

	uint32_t compacted_base = atomicAdd(numsteps_counter, compacted_numsteps); // first entry in the array is a counter
	compacted_numsteps = min(max_samples_compacted - min(max_samples_compacted, compacted_base), compacted_numsteps);
	numsteps_in[i*2+0] = compacted_numsteps;
	numsteps_in[i*2+1] = compacted_base;
	if (compacted_numsteps == 0) {
		return;
	}

	max_level_compacted_ptr += compacted_base;
	coords_out += compacted_base;

	dloss_doutput += compacted_base * padded_output_width;

	LossAndGradient lg = loss_and_gradient(rgbtarget, rgb_ray, loss_type);
	lg.loss /= img_pdf * xy_pdf;

	float target_depth = rays_in_unnormalized[i].d.norm() * ((depth_supervision_lambda > 0.0f && metadata[img].depth) ? read_depth(xy, resolution, metadata[img].depth) : -1.0f);
	float depth_loss_gradient = target_depth > 0.0f ? (depth_ray - target_depth) * 2.f * depth_supervision_lambda : 0.0f;

	// Note: dividing the gradient by the PDF would cause unbiased loss estimates.
	// Essentially: variance reduction, but otherwise the same optimization.
	// We _dont_ want that. If importance sampling is enabled, we _do_ actually want
	// to change the weighting of the loss function. So don't divide.
	// lg.gradient /= img_pdf * xy_pdf;

	float mean_loss = lg.loss.mean();
	if (loss_output) {
		loss_output[i] = mean_loss / (float)n_rays;
	}

	if (error_map) {
		const Vector2f pos = (xy.cwiseProduct(error_map_res.cast<float>()) - Vector2f::Constant(0.5f)).cwiseMax(0.0f).cwiseMin(error_map_res.cast<float>() - Vector2f::Constant(1.0f + 1e-4f));
		const Vector2i pos_int = pos.cast<int>();
		const Vector2f weight = pos - pos_int.cast<float>();

		Vector2i idx = pos_int.cwiseMin(resolution - Vector2i::Constant(2)).cwiseMax(0);

		auto deposit_val = [&](int x, int y, float val) {
			atomicAdd(&error_map[img * error_map_res.prod() + y * error_map_res.x() + x], val);
		};

		if (sharpness_data && aabb.contains(hitpoint)) {
			Vector2i sharpness_pos = xy.cwiseProduct(sharpness_resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(sharpness_resolution - Vector2i::Constant(1));
			float sharp = sharpness_data[img * sharpness_resolution.prod() + sharpness_pos.y() * sharpness_resolution.x() + sharpness_pos.x()] + 1e-6f;

			// The maximum value of positive floats interpreted in uint format is the same as the maximum value of the floats.
			float grid_sharp = __uint_as_float(atomicMax((uint32_t*)&cascaded_grid_at(hitpoint, sharpness_grid, mip_from_pos(hitpoint)), __float_as_uint(sharp)));
			grid_sharp = fmaxf(sharp, grid_sharp); // atomicMax returns the old value, so compute the new one locally.

			mean_loss *= fmaxf(sharp / grid_sharp, 0.01f);
		}

		deposit_val(idx.x(),   idx.y(),   (1 - weight.x()) * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x()+1, idx.y(),        weight.x()  * (1 - weight.y()) * mean_loss);
		deposit_val(idx.x(),   idx.y()+1, (1 - weight.x()) *      weight.y()  * mean_loss);
		deposit_val(idx.x()+1, idx.y()+1,      weight.x()  *      weight.y()  * mean_loss);
	}

	loss_scale /= n_rays;

	const float output_l2_reg = rgb_activation == ENerfActivation::Exponential ? 1e-4f : 0.0f;
	const float output_l1_reg_density = *mean_density_ptr < NERF_MIN_OPTICAL_THICKNESS() ? 1e-4f : 0.0f;

	// now do it again computing gradients
	Array3f rgb_ray2 = { 0.f,0.f,0.f };
	float depth_ray2 = 0.f;
	T = 1.f;
	for (uint32_t j = 0; j < compacted_numsteps; ++j) {
		if (max_level_rand_training) {
			max_level_compacted_ptr[j] = max_level;
		}
		// Compact network inputs
		NerfCoordinate* coord_out = coords_out(j);
		const NerfCoordinate* coord_in = coords_in(j);
		coord_out->copy(*coord_in, coords_out.stride_in_bytes);

		const Vector3f pos = unwarp_position(coord_in->pos.p, aabb);
		float depth = (pos - ray_o).norm();

		float dt = unwarp_dt(coord_in->dt);
		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const float density = network_to_density(float(local_network_output[3]), density_activation);
		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray2 += weight * rgb;
		depth_ray2 += weight * depth;
		T *= (1.f - alpha);

		// we know the suffix of this ray compared to where we are up to. note the suffix depends on this step's alpha as suffix = (1-alpha)*(somecolor), so dsuffix/dalpha = -somecolor = -suffix/(1-alpha)
		const Array3f suffix = rgb_ray - rgb_ray2;
		const Array3f dloss_by_drgb = weight * lg.gradient;

		tcnn::vector_t<tcnn::network_precision_t, 4> local_dL_doutput;

		// chain rule to go from dloss/drgb to dloss/dmlp_output
		local_dL_doutput[0] = loss_scale * (dloss_by_drgb.x() * network_to_rgb_derivative(local_network_output[0], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[0])); // Penalize way too large color values
		local_dL_doutput[1] = loss_scale * (dloss_by_drgb.y() * network_to_rgb_derivative(local_network_output[1], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[1]));
		local_dL_doutput[2] = loss_scale * (dloss_by_drgb.z() * network_to_rgb_derivative(local_network_output[2], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[2]));

		float density_derivative = network_to_density_derivative(float(local_network_output[3]), density_activation);
		const float depth_suffix = depth_ray - depth_ray2;
		const float depth_supervision = depth_loss_gradient * (T * depth - depth_suffix);

		float dloss_by_dmlp = density_derivative * (
			dt * (lg.gradient.matrix().dot((T * rgb - suffix).matrix()) + depth_supervision)
		);

		//static constexpr float mask_supervision_strength = 1.f; // we are already 'leaking' mask information into the nerf via the random bg colors; setting this to eg between 1 and  100 encourages density towards 0 in such regions.
		//dloss_by_dmlp += (texsamp.w()<0.001f) ? mask_supervision_strength * weight : 0.f;

		local_dL_doutput[3] =
			loss_scale * dloss_by_dmlp +
			(float(local_network_output[3]) < 0.0f ? -output_l1_reg_density : 0.0f) +
			(float(local_network_output[3]) > -10.0f && depth < near_distance ? 1e-4f : 0.0f);
			;

		*(tcnn::vector_t<tcnn::network_precision_t, 4>*)dloss_doutput = local_dL_doutput;

		dloss_doutput += padded_output_width;
		network_output += padded_output_width;
	}

	if (exposure_gradient) {
		// Assume symmetric loss
		Array3f dloss_by_dgt = -lg.gradient / xy_pdf;

		if (!train_in_linear_colors) {
			dloss_by_dgt /= srgb_to_linear_derivative(rgbtarget);
		}

		// 2^exposure * log(2)
		Array3f dloss_by_dexposure = loss_scale * dloss_by_dgt * exposure_scale * 0.6931471805599453f;
		atomicAdd(&exposure_gradient[img].x(), dloss_by_dexposure.x());
		atomicAdd(&exposure_gradient[img].y(), dloss_by_dexposure.y());
		atomicAdd(&exposure_gradient[img].z(), dloss_by_dexposure.z());
	}

	if (compacted_numsteps == numsteps && envmap_gradient) {
		Array3f loss_gradient = lg.gradient;
		if (envmap_loss_type != loss_type) {
			loss_gradient = loss_and_gradient(rgbtarget, rgb_ray, envmap_loss_type).gradient;
		}

		Array3f dloss_by_dbackground = T * loss_gradient;
		if (!train_in_linear_colors) {
			dloss_by_dbackground /= srgb_to_linear_derivative(background_color);
		}

		tcnn::vector_t<tcnn::network_precision_t, 4> dL_denvmap;
		dL_denvmap[0] = loss_scale * dloss_by_dbackground.x();
		dL_denvmap[1] = loss_scale * dloss_by_dbackground.y();
		dL_denvmap[2] = loss_scale * dloss_by_dbackground.z();


		float dloss_by_denvmap_alpha = dloss_by_dbackground.matrix().dot(-pre_envmap_background_color.matrix());

		// dL_denvmap[3] = loss_scale * dloss_by_denvmap_alpha;
		dL_denvmap[3] = (tcnn::network_precision_t)0;

		deposit_envmap_gradient(dL_denvmap, envmap_gradient, envmap_resolution, dir);
	}
}


__global__ void compute_cam_gradient_train_nerf(
	const uint32_t n_rays,
	const uint32_t n_rays_total,
	default_rng_t rng,
	const BoundingBox aabb,
	const uint32_t* __restrict__ rays_counter,
	const TrainingXForm* training_xforms,
	bool snap_to_pixel_centers,
	Vector3f* cam_pos_gradient,
	Vector3f* cam_rot_gradient,
	const uint32_t n_training_images,
	const TrainingImageMetadata* __restrict__ metadata,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in_unnormalized,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<NerfCoordinate> coords,
	PitchedPtr<NerfCoordinate> coords_gradient,
	float* __restrict__ distortion_gradient,
	float* __restrict__ distortion_gradient_weight,
	const Vector2i distortion_resolution,
	Vector2f* cam_focal_length_gradient,
	const float* __restrict__ cdf_x_cond_y,
	const float* __restrict__ cdf_y,
	const float* __restrict__ cdf_img,
	const Vector2i error_map_res
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	if (numsteps == 0) {
		// The ray doesn't matter. So no gradient onto the camera
		return;
	}

	uint32_t base = numsteps_in[i*2+1];
	coords += base;
	coords_gradient += base;

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img);
	Eigen::Vector2i resolution = metadata[img].resolution;

	const Matrix<float, 3, 4>& xform = training_xforms[img].start;

	Ray ray = rays_in_unnormalized[i];
	ray.d = ray.d.normalized();
	Ray ray_gradient = { Vector3f::Zero(), Vector3f::Zero() };

	// Compute ray gradient
	for (uint32_t j = 0; j < numsteps; ++j) {
		// pos = ray.o + t * ray.d;

		const Vector3f warped_pos = coords(j)->pos.p;
		const Vector3f pos_gradient = coords_gradient(j)->pos.p.cwiseProduct(warp_position_derivative(warped_pos, aabb));
		ray_gradient.o += pos_gradient;
		const Vector3f pos = unwarp_position(warped_pos, aabb);

		// Scaled by t to account for the fact that further-away objects' position
		// changes more rapidly as the direction changes.
		float t = (pos - ray.o).norm();
		const Vector3f dir_gradient = coords_gradient(j)->dir.d.cwiseProduct(warp_direction_derivative(coords(j)->dir.d));
		ray_gradient.d += pos_gradient * t + dir_gradient;
	}

	// Projection of the raydir gradient onto the plane normal to raydir,
	// because that's the only degree of motion that the raydir has.
	ray_gradient.d -= ray.d * ray_gradient.d.dot(ray.d);

	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());
	float xy_pdf = 1.0f;

	Vector2f xy = nerf_random_image_pos_training(rng, resolution, snap_to_pixel_centers, cdf_x_cond_y, cdf_y, error_map_res, img, &xy_pdf);

	if (distortion_gradient) {
		// Rotate ray gradient to obtain image plane gradient.
		// This has the effect of projecting the (already projected) ray gradient from the
		// tangent plane of the sphere onto the image plane (which is correct!).
		Vector3f image_plane_gradient = xform.block<3,3>(0,0).inverse() * ray_gradient.d;

		// Splat the resulting 2D image plane gradient into the distortion params
		deposit_image_gradient<2>(image_plane_gradient.head<2>() / xy_pdf, distortion_gradient, distortion_gradient_weight, distortion_resolution, xy);
	}

	if (cam_pos_gradient) {
		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_pos_gradient[img][j], ray_gradient.o[j] / xy_pdf);
		}
	}

	if (cam_rot_gradient) {
		// Rotation is averaged in log-space (i.e. by averaging angle-axes).
		// Due to our construction of ray_gradient.d, ray_gradient.d and ray.d are
		// orthogonal, leading to the angle_axis magnitude to equal the magnitude
		// of ray_gradient.d.
		Vector3f angle_axis = ray.d.cross(ray_gradient.d);

		// Atomically reduce the ray gradient into the xform gradient
		#pragma unroll
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_rot_gradient[img][j], angle_axis[j] / xy_pdf);
		}
	}
}

__global__ void compute_extra_dims_gradient_train_nerf(
	const uint32_t n_rays,
	const uint32_t n_rays_total,
	const uint32_t* __restrict__ rays_counter,
	float* extra_dims_gradient,
	uint32_t n_extra_dims,
	const uint32_t n_training_images,
	const uint32_t* __restrict__ ray_indices_in,
	uint32_t* __restrict__ numsteps_in,
	PitchedPtr<NerfCoordinate> coords_gradient,
	const float* __restrict__ cdf_img
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	if (numsteps == 0) {
		// The ray doesn't matter. So no gradient onto the camera
		return;
	}
	uint32_t base = numsteps_in[i*2+1];
	coords_gradient += base;
	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices_in[i];
	uint32_t img = image_idx(ray_idx, n_rays, n_rays_total, n_training_images, cdf_img);

	extra_dims_gradient += n_extra_dims * img;

	for (uint32_t j = 0; j < numsteps; ++j) {
		const float *src = coords_gradient(j)->get_extra_dims();
		for (uint32_t k = 0; k < n_extra_dims; ++k) {
			atomicAdd(&extra_dims_gradient[k], src[k]);
		}
	}
}

__global__ void shade_kernel_nerf(
	const uint32_t n_elements,
	Array4f* __restrict__ rgba,
	float* __restrict__ depth,
	NerfPayload* __restrict__ payloads,
	ERenderMode render_mode,
	bool train_in_linear_colors,
	Array4f* __restrict__ frame_buffer,
	float* __restrict__ depth_buffer
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;
	NerfPayload& payload = payloads[i];

	Array4f tmp = rgba[i];

	if (render_mode == ERenderMode::Normals) {
		Array3f n = tmp.head<3>().matrix().normalized().array();
		tmp.head<3>() = (0.5f * n + Array3f::Constant(0.5f)) * tmp.w();
	} else if (render_mode == ERenderMode::Cost) {
		float col = (float)payload.n_steps / 128;
		tmp = {col, col, col, 1.0f};
	}

	if (!train_in_linear_colors && (render_mode == ERenderMode::Shade || render_mode == ERenderMode::Slice)) {
		// Accumulate in linear colors
		tmp.head<3>() = srgb_to_linear(tmp.head<3>());
	}

	frame_buffer[payload.idx] = tmp + frame_buffer[payload.idx] * (1.0f - tmp.w());
	if (render_mode != ERenderMode::Slice && tmp.w() > 0.2f) {
		depth_buffer[payload.idx] = depth[i];
	}
}

__global__ void compact_kernel_nerf(
	const uint32_t n_elements,
	Array4f* src_rgba, float* src_depth, NerfPayload* src_payloads,
	Array4f* dst_rgba, float* dst_depth, NerfPayload* dst_payloads,
	Array4f* dst_final_rgba, float* dst_final_depth, NerfPayload* dst_final_payloads,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	NerfPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_rgba[idx] = src_rgba[i];
		dst_depth[idx] = src_depth[i];
	} else if (src_rgba[i].w() > 0.001f) {
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_rgba[idx] = src_rgba[i];
		dst_final_depth[idx] = src_depth[i];
	}
}

__global__ void init_rays_with_payload_kernel_nerf(
	uint32_t sample_index,
	NerfPayload* __restrict__ payloads,
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera_matrix0,
	Matrix<float, 3, 4> camera_matrix1,
	Vector4f rolling_shutter,
	Vector2f screen_center,
	Vector3f parallax_shift,
	bool snap_to_pixel_centers,
	BoundingBox aabb,
	float plane_z,
	float dof,
	CameraDistortion camera_distortion,
	const float* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	Array4f* __restrict__ framebuffer,
	float* __restrict__ depthbuffer,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	ERenderMode render_mode
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	if (plane_z < 0) {
		dof = 0.0;
	}


	// TODO: pixel_to_ray also immediately computes u,v for the pixel, so this is somewhat redundant
	float u = (x + 0.5f) * (1.f / resolution.x());
	float v = (y + 0.5f) * (1.f / resolution.y());
	float ray_time = rolling_shutter.x() + rolling_shutter.y() * u + rolling_shutter.z() * v + rolling_shutter.w() * ld_random_val(sample_index, idx * 72239731);
	Ray ray = pixel_to_ray(
		sample_index,
		{x, y},
		resolution,
		focal_length,
		camera_matrix0 * ray_time + camera_matrix1 * (1.f - ray_time),
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		plane_z,
		dof,
		camera_distortion,
		distortion_data,
		distortion_resolution
	);

	NerfPayload& payload = payloads[idx];
	payload.max_weight = 0.0f;

	if (plane_z < 0) {
		float n = ray.d.norm();
		payload.origin = ray.o;
		payload.dir = (1.0f/n) * ray.d;
		payload.t = -plane_z*n;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		depthbuffer[idx] = -plane_z;
		return;
	}

	depthbuffer[idx] = 1e10f;

	ray.d = ray.d.normalized();

	if (envmap_data) {
		framebuffer[idx] = read_envmap(envmap_data, envmap_resolution, ray.d);
	}

	float t = fmaxf(aabb.ray_intersect(ray.o, ray.d).x(), NERF_RENDERING_NEAR_DISTANCE()) + 1e-6f;

	if (!aabb.contains(ray.o + ray.d * t)) {
		payload.origin = ray.o;
		payload.alive = false;
		return;
	}

	if (render_mode == ERenderMode::Distortion) {
		Vector2f offset = Vector2f::Zero();
		if (distortion_data) {
			offset += read_image<2>(distortion_data, distortion_resolution, Vector2f((float)x + 0.5f, (float)y + 0.5f).cwiseQuotient(resolution.cast<float>()));
		}
		framebuffer[idx].head<3>() = to_rgb(offset * 50.0f);
		framebuffer[idx].w() = 1.0f;
		depthbuffer[idx] = 1.0f;
		payload.origin = ray.o + ray.d * 10000.0f;
		payload.alive = false;
		return;
	}

	payload.origin = ray.o;
	payload.dir = ray.d;
	payload.t = t;
	payload.idx = idx;
	payload.n_steps = 0;
	payload.alive = true;
}

static constexpr float MIN_PDF = 0.01f;

__global__ void construct_cdf_2d(
	uint32_t n_images,
	uint32_t height,
	uint32_t width,
	const float* __restrict__ data,
	float* __restrict__ cdf_x_cond_y,
	float* __restrict__ cdf_y
) {
	const uint32_t y = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t img = threadIdx.y + blockIdx.y * blockDim.y;
	if (y >= height || img >= n_images) return;

	const uint32_t offset_xy = img * height * width + y * width;
	data += offset_xy;
	cdf_x_cond_y += offset_xy;

	float cum = 0;
	for (uint32_t x = 0; x < width; ++x) {
		cum += data[x] + 1e-10f;
		cdf_x_cond_y[x] = cum;
	}

	cdf_y[img * height + y] = cum;
	float norm = __frcp_rn(cum);

	for (uint32_t x = 0; x < width; ++x) {
		cdf_x_cond_y[x] = (1.0f - MIN_PDF) * cdf_x_cond_y[x] * norm + MIN_PDF * (float)(x+1) / (float)width;
	}
}

__global__ void construct_cdf_1d(
	uint32_t n_images,
	uint32_t height,
	float* __restrict__ cdf_y,
	float* __restrict__ cdf_img
) {
	const uint32_t img = threadIdx.x + blockIdx.x * blockDim.x;
	if (img >= n_images) return;

	cdf_y += img * height;

	float cum = 0;
	for (uint32_t y = 0; y < height; ++y) {
		cum += cdf_y[y];
		cdf_y[y] = cum;
	}

	cdf_img[img] = cum;

	float norm = __frcp_rn(cum);
	for (uint32_t y = 0; y < height; ++y) {
		cdf_y[y] = (1.0f - MIN_PDF) * cdf_y[y] * norm + MIN_PDF * (float)(y+1) / (float)height;
	}
}

__global__ void safe_divide(const uint32_t num_elements, float* __restrict__ inout, const float* __restrict__ divisor) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	float local_divisor = divisor[i];
	inout[i] = local_divisor > 0.0f ? (inout[i] / local_divisor) : 0.0f;
}

void Testbed::NerfTracer::init_rays_from_camera(
	uint32_t sample_index,
	uint32_t padded_output_width,
	uint32_t n_extra_dims,
	const Vector2i& resolution,
	const Vector2f& focal_length,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& camera_matrix1,
	const Vector4f& rolling_shutter,
	Vector2f screen_center,
	Vector3f parallax_shift,
	bool snap_to_pixel_centers,
	const BoundingBox& render_aabb,
	float plane_z,
	float dof,
	const CameraDistortion& camera_distortion,
	const float* envmap_data,
	const Vector2i& envmap_resolution,
	const float* distortion_data,
	const Vector2i& distortion_resolution,
	Eigen::Array4f* frame_buffer,
	float* depth_buffer,
	uint8_t *grid,
	int show_accel,
	float cone_angle_constant,
	ERenderMode render_mode,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x() * resolution.y();
	enlarge(n_pixels, padded_output_width, n_extra_dims, stream);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x(), threads.x), div_round_up((uint32_t)resolution.y(), threads.y), 1 };
	init_rays_with_payload_kernel_nerf<<<blocks, threads, 0, stream>>>(
		sample_index,
		m_rays[0].payload,
		resolution,
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		render_aabb,
		plane_z,
		dof,
		camera_distortion,
		envmap_data,
		envmap_resolution,
		frame_buffer,
		depth_buffer,
		distortion_data,
		distortion_resolution,
		render_mode
	);

	m_n_rays_initialized = resolution.x() * resolution.y();

	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].rgba, 0, m_n_rays_initialized * sizeof(Array4f), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].depth, 0, m_n_rays_initialized * sizeof(float), stream));

	linear_kernel(advance_pos_nerf, 0, stream,
		m_n_rays_initialized,
		render_aabb,
		camera_matrix1.col(2),
		focal_length,
		sample_index,
		m_rays[0].payload,
		grid,
		(show_accel >= 0) ? show_accel : 0,
		cone_angle_constant
	);
}

uint32_t Testbed::NerfTracer::trace(
	NerfNetwork<network_precision_t>& network,
	const BoundingBox& render_aabb,
	const BoundingBox& train_aabb,
	const uint32_t n_training_images,
	const TrainingXForm* training_xforms,
	const Vector2f& focal_length,
	float cone_angle_constant,
	const uint8_t* grid,
	ERenderMode render_mode,
	const Eigen::Matrix<float, 3, 4> &camera_matrix,
	float depth_scale,
	int visualized_layer,
	int visualized_dim,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	float min_transmittance,
	float glow_y_cutoff,
	int glow_mode,
	const float* extra_dims_gpu,
	hipStream_t stream
) {
	if (m_n_rays_initialized == 0) {
		return 0;
	}

	CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter.data(), 0, sizeof(uint32_t), stream));

	uint32_t n_alive = m_n_rays_initialized;
	// m_n_rays_initialized = 0;

	uint32_t i = 1;
	uint32_t double_buffer_index = 0;
	while (i < MARCH_ITER) {
		RaysNerfSoa& rays_current = m_rays[(double_buffer_index + 1) % 2];
		RaysNerfSoa& rays_tmp = m_rays[double_buffer_index % 2];
		++double_buffer_index;

		// Compact rays that did not diverge yet
		{
			CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter.data(), 0, sizeof(uint32_t), stream));
			linear_kernel(compact_kernel_nerf, 0, stream,
				n_alive,
				rays_tmp.rgba, rays_tmp.depth, rays_tmp.payload,
				rays_current.rgba, rays_current.depth, rays_current.payload,
				m_rays_hit.rgba, m_rays_hit.depth, m_rays_hit.payload,
				m_alive_counter.data(), m_hit_counter.data()
			);
			CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		}

		if (n_alive == 0) {
			break;
		}

		uint32_t n_steps_between_compaction = tcnn::clamp(m_n_rays_initialized / n_alive, (uint32_t)MIN_STEPS_INBETWEEN_COMPACTION, (uint32_t)MAX_STEPS_INBETWEEN_COMPACTION);

		uint32_t extra_stride = network.n_extra_dims() * sizeof(float);
		PitchedPtr<NerfCoordinate> input_data((NerfCoordinate*)m_network_input, 1, 0, extra_stride);
		linear_kernel(generate_next_nerf_network_inputs, 0, stream,
			n_alive,
			render_aabb,
			train_aabb,
			focal_length,
			camera_matrix.col(2),
			rays_current.payload,
			input_data,
			n_steps_between_compaction,
			grid,
			(show_accel>=0) ? show_accel : 0,
			cone_angle_constant,
			extra_dims_gpu
		);
		uint32_t n_elements = next_multiple(n_alive * n_steps_between_compaction, tcnn::batch_size_granularity);
		GPUMatrix<float> positions_matrix((float*)m_network_input, (sizeof(NerfCoordinate) + extra_stride) / sizeof(float), n_elements);
		GPUMatrix<network_precision_t, RM> rgbsigma_matrix((network_precision_t*)m_network_output, network.padded_output_width(), n_elements);
		network.inference_mixed_precision(stream, positions_matrix, rgbsigma_matrix);

		if (render_mode == ERenderMode::Normals) {
			network.input_gradient(stream, 3, positions_matrix, positions_matrix);
		} else if (render_mode == ERenderMode::EncodingVis) {
			network.visualize_activation(stream, visualized_layer, visualized_dim, positions_matrix, positions_matrix);
		}

		linear_kernel(composite_kernel_nerf, 0, stream,
			n_alive,
			n_elements,
			i,
			train_aabb,
			glow_y_cutoff,
			glow_mode,
			n_training_images,
			training_xforms,
			camera_matrix,
			focal_length,
			depth_scale,
			rays_current.rgba,
			rays_current.depth,
			rays_current.payload,
			input_data,
			m_network_output,
			network.padded_output_width(),
			n_steps_between_compaction,
			render_mode,
			grid,
			rgb_activation,
			density_activation,
			show_accel,
			min_transmittance
		);

		i += n_steps_between_compaction;
	}

	uint32_t n_hit;
	CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter.data(), sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	return n_hit;
}

void Testbed::NerfTracer::enlarge(size_t n_elements, uint32_t padded_output_width, uint32_t n_extra_dims, hipStream_t stream) {
	n_elements = next_multiple(n_elements, size_t(tcnn::batch_size_granularity));
	size_t num_floats = sizeof(NerfCoordinate) / 4 + n_extra_dims;
	auto scratch = allocate_workspace_and_distribute<
		Array4f, float, NerfPayload, // m_rays[0]
		Array4f, float, NerfPayload, // m_rays[1]
		Array4f, float, NerfPayload, // m_rays_hit

		network_precision_t,
		float
	>(
		stream, &m_scratch_alloc,
		n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * padded_output_width,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * num_floats
	);

	m_rays[0].set(std::get<0>(scratch), std::get<1>(scratch), std::get<2>(scratch), n_elements);
	m_rays[1].set(std::get<3>(scratch), std::get<4>(scratch), std::get<5>(scratch), n_elements);
	m_rays_hit.set(std::get<6>(scratch), std::get<7>(scratch), std::get<8>(scratch), n_elements);

	m_network_output = std::get<9>(scratch);
	m_network_input = std::get<10>(scratch);
}

void Testbed::Nerf::Training::reset_extra_dims(default_rng_t &rng) {
	uint32_t n_extra_dims = dataset.n_extra_dims();
	std::vector<float> extra_dims_cpu(n_extra_dims * (dataset.n_images + 1)); // n_images + 1 since we use an extra 'slot' for the inference latent code
	float *dst = extra_dims_cpu.data();
	ArrayXf zero(n_extra_dims);
	extra_dims_opt.resize(dataset.n_images, AdamOptimizer<ArrayXf>(1e-4f, zero));
	for (uint32_t i = 0; i < dataset.n_images; ++i) {
		Eigen::Vector3f light_dir = warp_direction(dataset.metadata[i].light_dir.normalized());
		extra_dims_opt[i].reset_state(Eigen::VectorXf(n_extra_dims));
		Eigen::ArrayXf &optimzer_value = extra_dims_opt[i].variable();
		for (uint32_t j = 0; j < n_extra_dims; ++j) {
			if (dataset.has_light_dirs && j < 3)
				dst[j] = light_dir[j];
			else
				dst[j] = random_val(rng) * 2.f - 1.f;
			optimzer_value[j] = dst[j];
		}
		dst += n_extra_dims;
	}
	extra_dims_gpu.resize_and_copy_from_host(extra_dims_cpu);
}

const float* Testbed::get_inference_extra_dims(hipStream_t stream) const {
	if (m_nerf_network->n_extra_dims() == 0) {
		return nullptr;
	}
	const float* extra_dims_src = m_nerf.training.extra_dims_gpu.data() + m_nerf.extra_dim_idx_for_inference * m_nerf.training.dataset.n_extra_dims();
	if (!m_nerf.training.dataset.has_light_dirs) {
		return extra_dims_src;
	}

	// the dataset has light directions, so we must construct a temporary buffer and fill it as requested.
	// we use an extra 'slot' that was pre-allocated for us at the end of the extra_dims array.
	size_t size = m_nerf_network->n_extra_dims() * sizeof(float);
	float* dims_gpu = m_nerf.training.extra_dims_gpu.data() + m_nerf.training.dataset.n_images * m_nerf.training.dataset.n_extra_dims();
	CUDA_CHECK_THROW(hipMemcpyAsync(dims_gpu, extra_dims_src, size, hipMemcpyDeviceToDevice, stream));
	Eigen::Vector3f light_dir = warp_direction(m_nerf.light_dir.normalized());
	CUDA_CHECK_THROW(hipMemcpyAsync(dims_gpu, &light_dir, min(size, sizeof(Eigen::Vector3f)), hipMemcpyHostToDevice, stream));
	return dims_gpu;
}

void Testbed::render_nerf(CudaRenderBuffer& render_buffer, const Vector2i& max_res, const Vector2f& focal_length, const Matrix<float, 3, 4>& camera_matrix0, const Matrix<float, 3, 4>& camera_matrix1, const Vector4f& rolling_shutter, const Vector2f& screen_center, hipStream_t stream) {
	float plane_z = m_slice_plane_z + m_scale;
	if (m_render_mode == ERenderMode::Slice) {
		plane_z = -plane_z;
	}

	ERenderMode render_mode = m_visualized_dimension > -1 ? ERenderMode::EncodingVis : m_render_mode;

	const float* extra_dims_gpu = get_inference_extra_dims(stream);

	ScopeGuard tmp_memory_guard{[&]() {
		m_nerf.tracer.clear();
	}};

	// Our motion vector code can't undo f-theta and grid distortions -- so don't render these if DLSS is enabled.
	bool render_opencv_camera_distortion = m_nerf.render_with_camera_distortion && (!render_buffer.dlss() || m_nerf.render_distortion.mode == ECameraDistortionMode::Iterative);
	bool render_grid_camera_distortion = m_nerf.render_with_camera_distortion && !render_buffer.dlss();

	CameraDistortion camera_distortion = render_opencv_camera_distortion ? m_nerf.render_distortion : CameraDistortion{};


	m_nerf.tracer.init_rays_from_camera(
		render_buffer.spp(),
		m_network->padded_output_width(),
		m_nerf_network->n_extra_dims(),
		render_buffer.in_resolution(),
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		get_scaled_parallax_shift(),
		m_snap_to_pixel_centers,
		m_render_aabb,
		plane_z,
		m_dof,
		camera_distortion,
		m_envmap.envmap->params_inference(),
		m_envmap.resolution,
		render_grid_camera_distortion ? m_distortion.map->params_inference() : nullptr,
		m_distortion.resolution,
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer(),
		m_nerf.density_grid_bitfield.data(),
		m_nerf.show_accel,
		m_nerf.cone_angle_constant,
		render_mode,
		stream
	);

	uint32_t n_hit;
	if (m_render_mode == ERenderMode::Slice) {
		n_hit = m_nerf.tracer.n_rays_initialized();
	} else {
		float depth_scale = 1.0f / m_nerf.training.dataset.scale;
		n_hit = m_nerf.tracer.trace(
			*m_nerf_network,
			m_render_aabb,
			m_aabb,
			m_nerf.training.n_images_for_training,
			m_nerf.training.transforms.data(),
			focal_length,
			m_nerf.cone_angle_constant,
			m_nerf.density_grid_bitfield.data(),
			render_mode,
			camera_matrix1,
			depth_scale,
			m_visualized_layer,
			m_visualized_dimension,
			m_nerf.rgb_activation,
			m_nerf.density_activation,
			m_nerf.show_accel,
			m_nerf.rendering_min_transmittance,
			m_nerf.m_glow_y_cutoff,
			m_nerf.m_glow_mode,
			extra_dims_gpu,
			stream
		);
	}
	RaysNerfSoa& rays_hit = m_render_mode == ERenderMode::Slice ? m_nerf.tracer.rays_init() : m_nerf.tracer.rays_hit();

	if (m_render_mode == ERenderMode::Slice) {
		// Store colors in the normal buffer
		uint32_t n_elements = next_multiple(n_hit, tcnn::batch_size_granularity);
		const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
		const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

		m_nerf.vis_input.enlarge(n_elements * floats_per_coord);
		m_nerf.vis_rgba.enlarge(n_elements);
		linear_kernel(generate_nerf_network_inputs_at_current_position, 0, stream, n_hit, m_aabb, rays_hit.payload, PitchedPtr<NerfCoordinate>((NerfCoordinate*)m_nerf.vis_input.data(), 1, 0, extra_stride), extra_dims_gpu );

		GPUMatrix<float> positions_matrix((float*)m_nerf.vis_input.data(), floats_per_coord, n_elements);
		GPUMatrix<float> rgbsigma_matrix((float*)m_nerf.vis_rgba.data(), 4, n_elements);

		if (m_visualized_dimension == -1) {
			m_network->inference(stream, positions_matrix, rgbsigma_matrix);
			linear_kernel(compute_nerf_density, 0, stream, n_hit, m_nerf.vis_rgba.data(), m_nerf.rgb_activation, m_nerf.density_activation);
		} else {
			m_network->visualize_activation(stream, m_visualized_layer, m_visualized_dimension, positions_matrix, rgbsigma_matrix);
		}

		linear_kernel(shade_kernel_nerf, 0, stream,
			n_hit,
			m_nerf.vis_rgba.data(),
			nullptr,
			rays_hit.payload,
			m_render_mode,
			m_nerf.training.linear_colors,
			render_buffer.frame_buffer(),
			render_buffer.depth_buffer()
		);
		return;
	}

	linear_kernel(shade_kernel_nerf, 0, stream,
		n_hit,
		rays_hit.rgba,
		rays_hit.depth,
		rays_hit.payload,
		m_render_mode,
		m_nerf.training.linear_colors,
		render_buffer.frame_buffer(),
		render_buffer.depth_buffer()
	);

	if (render_mode == ERenderMode::Cost) {
		std::vector<NerfPayload> payloads_final_cpu(n_hit);
		CUDA_CHECK_THROW(hipMemcpyAsync(payloads_final_cpu.data(), rays_hit.payload, n_hit * sizeof(NerfPayload), hipMemcpyDeviceToHost, stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));

		size_t total_n_steps = 0;
		for (uint32_t i = 0; i < n_hit; ++i) {
			total_n_steps += payloads_final_cpu[i].n_steps;
		}
		tlog::info() << "Total steps per hit= " << total_n_steps << "/" << n_hit << " = " << ((float)total_n_steps/(float)n_hit);
	}
}

void Testbed::Nerf::Training::set_camera_intrinsics(int frame_idx, float fx, float fy, float cx, float cy, float k1, float k2, float p1, float p2) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return;
	}
	if (fx <= 0.f) fx = fy;
	if (fy <= 0.f) fy = fx;
	auto &m = dataset.metadata[frame_idx];
	if (cx < 0.f) cx = -cx; else cx = cx / m.resolution.x();
	if (cy < 0.f) cy = -cy; else cy = cy / m.resolution.y();
	ECameraDistortionMode mode = (k1 || k2 || p1 || p2) ? ECameraDistortionMode::Iterative : ECameraDistortionMode::None;
	m.camera_distortion = { mode, k1, k2, p1, p2 };
	m.principal_point = { cx, cy };
	m.focal_length = { fx, fy };
	update_metadata(frame_idx, frame_idx + 1);
}

void Testbed::Nerf::Training::set_camera_extrinsics(int frame_idx, const Eigen::Matrix<float, 3, 4> &camera_to_world) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return;
	}

	dataset.xforms[frame_idx].start = dataset.xforms[frame_idx].end = dataset.nerf_matrix_to_ngp(camera_to_world);
	cam_rot_offset[frame_idx].reset_state();
	cam_pos_offset[frame_idx].reset_state();
	cam_exposure[frame_idx].reset_state();
	update_transforms(frame_idx, frame_idx + 1);
}

void Testbed::Nerf::Training::reset_camera_extrinsics() {
	for (auto&& opt : cam_rot_offset) {
		opt.reset_state();
	}

	for (auto&& opt : cam_pos_offset) {
		opt.reset_state();
	}

	for (auto&& opt : cam_exposure) {
		opt.reset_state();
	}
}

void Testbed::Nerf::Training::export_camera_extrinsics(const std::string& filename, bool export_extrinsics_in_quat_format) {
	tlog::info() << "Saving a total of " << n_images_for_training << " poses to " << filename;
	nlohmann::json trajectory;
	for(int i = 0; i < n_images_for_training; ++i) {
		nlohmann::json frame {{"id", i}};

		const Eigen::Matrix<float, 3, 4> p_nerf = get_camera_extrinsics(i);
		if (export_extrinsics_in_quat_format) {
			// Assume 30 fps
			frame["time"] =  i*0.033f;
			// Convert the pose from NeRF to Quaternion format.
			const Eigen::Matrix<float, 3, 3> conv_coords_l {{ 0.f,  1.f,  0.f},
															{ 0.f,  0.f, -1.f},
															{-1.f,  0.f,  0.f}};
			const Eigen::Matrix<float, 4, 4> conv_coords_r {{ 1.f,  0.f,  0.f,  0.f},
															{ 0.f, -1.f,  0.f,  0.f},
															{ 0.f,  0.f, -1.f,  0.f},
															{ 0.f,  0.f,  0.f,  1.f}};
			const Eigen::Matrix<float, 3, 4> p_quat = conv_coords_l * p_nerf * conv_coords_r;

			const Eigen::Quaternionf rot_q {p_quat.block<3, 3>(0, 0)};
			frame["q"] = {rot_q.w(), rot_q.x(), rot_q.y(), rot_q.z()};
			frame["t"] = {p_quat(0, 3), p_quat(1, 3), p_quat(2, 3)};
		} else {
			frame["transform_matrix"] = {p_nerf.row(0), p_nerf.row(1), p_nerf.row(2)};
		}

		trajectory.emplace_back(frame);
	}
	std::ofstream file(filename);
    file << std::setw(2) << trajectory << std::endl;
}

Eigen::Matrix<float, 3, 4> Testbed::Nerf::Training::get_camera_extrinsics(int frame_idx) {
	if (frame_idx < 0 || frame_idx >= dataset.n_images) {
		return Eigen::Matrix<float, 3, 4>::Identity();
	}
	return dataset.ngp_matrix_to_nerf(transforms[frame_idx].start);
}

void Testbed::Nerf::Training::update_metadata(int first, int last) {
	if (last < 0) {
		last = dataset.n_images;
	}

	if (last > dataset.n_images) {
		last = dataset.n_images;
	}

	int n = last - first;
	if (n <= 0) {
		return;
	}
	metadata_gpu.enlarge(last);
	CUDA_CHECK_THROW(hipMemcpy(metadata_gpu.data() + first, dataset.metadata.data() + first, n * sizeof(TrainingImageMetadata), hipMemcpyHostToDevice));
}

void Testbed::Nerf::Training::update_transforms(int first, int last) {
	if (last < 0) {
		last=dataset.n_images;
	}

	if (last > dataset.n_images) {
		last = dataset.n_images;
	}

	int n = last - first;
	if (n <= 0) {
		return;
	}

	if (transforms.size() < last) {
		transforms.resize(last);
	}

	for (uint32_t i = 0; i < n; ++i) {
		auto xform = dataset.xforms[i + first];
		Vector3f rot = cam_rot_offset[i + first].variable();
		float angle = rot.norm();
		rot /= angle;

		if (angle > 0) {
			xform.start.block<3, 3>(0, 0) = AngleAxisf(angle, rot) * xform.start.block<3, 3>(0, 0);
			xform.end.block<3, 3>(0, 0) = AngleAxisf(angle, rot) * xform.end.block<3, 3>(0, 0);
		}

		xform.start.col(3) += cam_pos_offset[i + first].variable();
		xform.end.col(3) += cam_pos_offset[i + first].variable();
		transforms[i + first] = xform;
	}

	transforms_gpu.enlarge(last);
	CUDA_CHECK_THROW(hipMemcpy(transforms_gpu.data() + first, transforms.data() + first, n * sizeof(TrainingXForm), hipMemcpyHostToDevice));
}

void Testbed::create_empty_nerf_dataset(size_t n_images, int aabb_scale, bool is_hdr) {
	m_nerf.training.dataset = ngp::create_empty_nerf_dataset(n_images, aabb_scale, is_hdr);
	load_nerf();
	m_nerf.training.n_images_for_training = 0;
	m_training_data_available = true;
}

void Testbed::load_nerf() {
	if (!m_data_path.empty()) {
		std::vector<fs::path> json_paths;
		if (m_data_path.is_directory()) {
			for (const auto& path : fs::directory{m_data_path}) {
				if (path.is_file() && equals_case_insensitive(path.extension(), "json")) {
					json_paths.emplace_back(path);
				}
			}
		} else if (equals_case_insensitive(m_data_path.extension(), "msgpack")) {
			load_snapshot(m_data_path.str());
			set_train(false);
			return;
		} else if (equals_case_insensitive(m_data_path.extension(), "json")) {
			json_paths.emplace_back(m_data_path);
		} else {
			throw std::runtime_error{"NeRF data path must either be a json file or a directory containing json files."};
		}

		m_nerf.training.dataset = ngp::load_nerf(json_paths, m_nerf.sharpen);
	}

	m_nerf.rgb_activation = m_nerf.training.dataset.is_hdr ? ENerfActivation::Exponential : ENerfActivation::Logistic;

	m_nerf.training.n_images_for_training = (int)m_nerf.training.dataset.n_images;

	m_nerf.training.update_metadata();

	m_nerf.training.cam_pos_gradient.resize(m_nerf.training.dataset.n_images, Vector3f::Zero());
	m_nerf.training.cam_pos_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_pos_gradient);

	m_nerf.training.cam_exposure.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Array3f>(1e-3f));
	m_nerf.training.cam_pos_offset.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Vector3f>(1e-4f));
	m_nerf.training.cam_rot_offset.resize(m_nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
	m_nerf.training.cam_focal_length_offset = AdamOptimizer<Vector2f>(1e-5f);

	m_nerf.training.cam_rot_gradient.resize(m_nerf.training.dataset.n_images, Vector3f::Zero());
	m_nerf.training.cam_rot_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_rot_gradient);

	m_nerf.training.cam_exposure_gradient.resize(m_nerf.training.dataset.n_images, Array3f::Zero());
	m_nerf.training.cam_exposure_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);
	m_nerf.training.cam_exposure_gradient_gpu.resize_and_copy_from_host(m_nerf.training.cam_exposure_gradient);

	m_nerf.training.cam_focal_length_gradient = Vector2f::Zero();
	m_nerf.training.cam_focal_length_gradient_gpu.resize_and_copy_from_host(&m_nerf.training.cam_focal_length_gradient, 1);

	m_nerf.training.reset_extra_dims(m_rng);

	if (m_nerf.training.dataset.has_rays) {
		m_nerf.training.near_distance = 0.0f;
		// m_nerf.training.optimize_exposure = true;
	}

	// Uncomment the following line to see how the network learns distortion from scratch rather than
	// starting from the distortion that's described by the training data.
	// m_nerf.training.dataset.camera_distortion = {};

	// Perturbation of the training cameras -- for debugging the online extrinsics learning code
	float perturb_amount = 0.0f;
	if (perturb_amount > 0.f) {
		for (uint32_t i = 0; i < m_nerf.training.dataset.n_images; ++i) {
			Vector3f rot = random_val_3d(m_rng) * perturb_amount;
			float angle = rot.norm();
			rot /= angle;
			auto trans = random_val_3d(m_rng);
			m_nerf.training.dataset.xforms[i].start.block<3,3>(0,0) = AngleAxisf(angle, rot).matrix() * m_nerf.training.dataset.xforms[i].start.block<3,3>(0,0);
			m_nerf.training.dataset.xforms[i].start.col(3) += trans * perturb_amount;
			m_nerf.training.dataset.xforms[i].end.block<3,3>(0,0) = AngleAxisf(angle, rot).matrix() * m_nerf.training.dataset.xforms[i].end.block<3,3>(0,0);
			m_nerf.training.dataset.xforms[i].end.col(3) += trans * perturb_amount;
		}
	}

	m_nerf.training.update_transforms();

	if (!m_nerf.training.dataset.metadata.empty()) {
		m_nerf.render_distortion = m_nerf.training.dataset.metadata[0].camera_distortion;
		m_screen_center = Eigen::Vector2f::Constant(1.f) - m_nerf.training.dataset.metadata[0].principal_point;
	}

	if (!is_pot(m_nerf.training.dataset.aabb_scale)) {
		throw std::runtime_error{std::string{"NeRF dataset's `aabb_scale` must be a power of two, but is "} + std::to_string(m_nerf.training.dataset.aabb_scale)};
	}

	int max_aabb_scale = 1 << (NERF_CASCADES()-1);
	if (m_nerf.training.dataset.aabb_scale > max_aabb_scale) {
		throw std::runtime_error{
			std::string{"NeRF dataset must have `aabb_scale <= "} + std::to_string(max_aabb_scale) +
			"`, but is " + std::to_string(m_nerf.training.dataset.aabb_scale) +
			". You can increase this limit by factors of 2 by incrementing `NERF_CASCADES()` and re-compiling."
		};
	}

	m_aabb = BoundingBox{Vector3f::Constant(0.5f), Vector3f::Constant(0.5f)};
	m_aabb.inflate(0.5f * std::min(1 << (NERF_CASCADES()-1), m_nerf.training.dataset.aabb_scale));
	m_raw_aabb = m_aabb;
	m_render_aabb = m_aabb;
	if (!m_nerf.training.dataset.render_aabb.is_empty()) {
		m_render_aabb = m_nerf.training.dataset.render_aabb.intersection(m_aabb);
	}

	m_nerf.max_cascade = 0;
	while ((1 << m_nerf.max_cascade) < m_nerf.training.dataset.aabb_scale) {
		++m_nerf.max_cascade;
	}

	// Perform fixed-size stepping in unit-cube scenes (like original NeRF) and exponential
	// stepping in larger scenes.
	m_nerf.cone_angle_constant = m_nerf.training.dataset.aabb_scale <= 1 ? 0.0f : (1.0f / 256.0f);

	m_up_dir = m_nerf.training.dataset.up;
}

void Testbed::update_density_grid_nerf(float decay, uint32_t n_uniform_density_grid_samples, uint32_t n_nonuniform_density_grid_samples, hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();

	m_nerf.density_grid.enlarge(n_elements);

	const uint32_t n_density_grid_samples = n_uniform_density_grid_samples + n_nonuniform_density_grid_samples;

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,       // positions at which the NN will be queried for density evaluation
		uint32_t,           // indices of corresponding density grid cells
		float,              // the resulting densities `density_grid_tmp` to be merged with the running estimate of the grid
		network_precision_t // output of the MLP before being converted to densities.
	>(stream, &alloc, n_density_grid_samples, n_elements, n_elements, n_density_grid_samples * padded_output_width);

	NerfPosition* density_grid_positions = std::get<0>(scratch);
	uint32_t* density_grid_indices = std::get<1>(scratch);
	float* density_grid_tmp = std::get<2>(scratch);
	network_precision_t* mlp_out = std::get<3>(scratch);

	if (m_training_step == 0 || m_nerf.training.n_images_for_training != m_nerf.training.n_images_for_training_prev) {
		m_nerf.training.n_images_for_training_prev = m_nerf.training.n_images_for_training;
		if (m_training_step == 0) {
			m_nerf.density_grid_ema_step = 0;
		}
		// Only cull away empty regions where no camera is looking when the cameras are actually meaningful.
		if (!m_nerf.training.dataset.has_rays) {
			linear_kernel(mark_untrained_density_grid, 0, stream, n_elements, m_nerf.density_grid.data(),
				m_nerf.training.n_images_for_training,
				m_nerf.training.metadata_gpu.data(),
				m_nerf.training.transforms_gpu.data(),
				m_training_step == 0
			);
		} else {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid.data(), 0, sizeof(float)*n_elements, stream));
		}
	}

	uint32_t n_steps = 1;
	for (uint32_t i = 0; i < n_steps; ++i) {
		CUDA_CHECK_THROW(hipMemsetAsync(density_grid_tmp, 0, sizeof(float)*n_elements, stream));

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_uniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions,
			density_grid_indices,
			m_nerf.max_cascade+1,
			-0.01f
		);
		m_rng.advance();

		linear_kernel(generate_grid_samples_nerf_nonuniform, 0, stream,
			n_nonuniform_density_grid_samples,
			m_rng,
			m_nerf.density_grid_ema_step,
			m_aabb,
			m_nerf.density_grid.data(),
			density_grid_positions+n_uniform_density_grid_samples,
			density_grid_indices+n_uniform_density_grid_samples,
			m_nerf.max_cascade+1,
			NERF_MIN_OPTICAL_THICKNESS()
		);
		m_rng.advance();

		GPUMatrix<network_precision_t, RM> density_matrix(mlp_out, padded_output_width, n_density_grid_samples);
		GPUMatrix<float> density_grid_position_matrix((float*)density_grid_positions, sizeof(NerfPosition)/sizeof(float), n_density_grid_samples);
		m_nerf_network->density(stream, density_grid_position_matrix, density_matrix, false);

		linear_kernel(splat_grid_samples_nerf_max_nearest_neighbor, 0, stream, n_density_grid_samples, density_grid_indices, mlp_out, density_grid_tmp, m_nerf.rgb_activation, m_nerf.density_activation);
		linear_kernel(ema_grid_samples_nerf, 0, stream, n_elements, decay, m_nerf.density_grid_ema_step, m_nerf.density_grid.data(), density_grid_tmp);

		++m_nerf.density_grid_ema_step;
	}

	update_density_grid_mean_and_bitfield(stream);
}

void Testbed::update_density_grid_mean_and_bitfield(hipStream_t stream) {
	const uint32_t n_elements = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE();

	size_t size_including_mips = grid_mip_offset(NERF_CASCADES())/8;
	m_nerf.density_grid_bitfield.enlarge(size_including_mips);
	m_nerf.density_grid_mean.enlarge(reduce_sum_workspace_size(n_elements));

	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.density_grid_mean.data(), 0, sizeof(float), stream));
	reduce_sum(m_nerf.density_grid.data(), [n_elements] __device__ (float val) { return fmaxf(val, 0.f) / (n_elements); }, m_nerf.density_grid_mean.data(), n_elements, stream);

	linear_kernel(grid_to_bitfield, 0, stream, n_elements/8 * NERF_CASCADES(), m_nerf.density_grid.data(), m_nerf.density_grid_bitfield.data(), m_nerf.density_grid_mean.data());

	for (uint32_t level = 1; level < NERF_CASCADES(); ++level) {
		linear_kernel(bitfield_max_pool, 0, stream, n_elements/64, m_nerf.get_density_grid_bitfield_mip(level-1), m_nerf.get_density_grid_bitfield_mip(level));
	}
}

void Testbed::Nerf::Training::Counters::prepare_for_training_steps(hipStream_t stream) {
	numsteps_counter.enlarge(1);
	numsteps_counter_compacted.enlarge(1);
	loss.enlarge(rays_per_batch);
	CUDA_CHECK_THROW(hipMemsetAsync(numsteps_counter.data(), 0, sizeof(uint32_t), stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(numsteps_counter_compacted.data(), 0, sizeof(uint32_t), stream)); // clear the counter in the first slot
	CUDA_CHECK_THROW(hipMemsetAsync(loss.data(), 0, sizeof(float)*rays_per_batch, stream));
}

float Testbed::Nerf::Training::Counters::update_after_training(uint32_t target_batch_size, bool get_loss_scalar, hipStream_t stream) {
	std::vector<uint32_t> counter_cpu(1);
	std::vector<uint32_t> compacted_counter_cpu(1);
	numsteps_counter.copy_to_host(counter_cpu);
	numsteps_counter_compacted.copy_to_host(compacted_counter_cpu);
	measured_batch_size = 0;
	measured_batch_size_before_compaction = 0;

	if (counter_cpu[0] == 0 || compacted_counter_cpu[0] == 0) {
		return 0.f;
	}

	measured_batch_size_before_compaction = counter_cpu[0];
	measured_batch_size = compacted_counter_cpu[0];

	float loss_scalar = 0.0;
	if (get_loss_scalar) {
		loss_scalar = reduce_sum(loss.data(), rays_per_batch, stream) * (float)measured_batch_size / (float)target_batch_size;
	}

	rays_per_batch = (uint32_t)((float)rays_per_batch * (float)target_batch_size / (float)measured_batch_size);
	rays_per_batch = std::min(next_multiple(rays_per_batch, tcnn::batch_size_granularity), 1u << 18);

	return loss_scalar;
}

void Testbed::train_nerf(uint32_t target_batch_size, bool get_loss_scalar, hipStream_t stream) {
	if (m_nerf.training.n_images_for_training == 0) {
		return;
	}

	if (m_nerf.training.include_sharpness_in_error) {
		size_t n_cells = NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES();
		if (m_nerf.training.sharpness_grid.size() < n_cells) {
			m_nerf.training.sharpness_grid.enlarge(NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_CASCADES());
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		}

		if (m_training_step == 0) {
			CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.sharpness_grid.data(), 0, m_nerf.training.sharpness_grid.get_bytes(), stream));
		} else {
			linear_kernel(decay_sharpness_grid_nerf, 0, stream, m_nerf.training.sharpness_grid.size(), 0.95f, m_nerf.training.sharpness_grid.data());
		}
	}
	m_nerf.training.counters_rgb.prepare_for_training_steps(stream);

	if (m_nerf.training.n_steps_since_cam_update == 0) {
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_pos_gradient_gpu.data(), 0, m_nerf.training.cam_pos_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_rot_gradient_gpu.data(), 0, m_nerf.training.cam_rot_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_exposure_gradient_gpu.data(), 0, m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradients(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_distortion.map->gradient_weights(), 0, sizeof(float)*m_distortion.map->n_params(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_focal_length_gradient_gpu.data(), 0, m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(), stream));
	}

	bool train_extra_dims = m_nerf.training.dataset.n_extra_learnable_dims > 0 && m_nerf.training.optimize_extra_dims;
	uint32_t n_extra_dims = m_nerf.training.dataset.n_extra_dims();
	if (train_extra_dims) {
		uint32_t n = n_extra_dims * m_nerf.training.n_images_for_training;
		m_nerf.training.extra_dims_gradient_gpu.enlarge(n);
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.extra_dims_gradient_gpu.data(), 0, m_nerf.training.extra_dims_gradient_gpu.get_bytes(), stream));
	}

	if (m_nerf.training.n_steps_since_error_map_update == 0 && !m_nerf.training.dataset.metadata.empty()) {
		uint32_t n_samples_per_image = (m_nerf.training.n_steps_between_error_map_updates * m_nerf.training.counters_rgb.rays_per_batch) / m_nerf.training.dataset.n_images;
		Eigen::Vector2i res = m_nerf.training.dataset.metadata[0].resolution;
		m_nerf.training.error_map.resolution = Vector2i::Constant((int)(std::sqrt(std::sqrt((float)n_samples_per_image)) * 3.5f)).cwiseMin(res);
		m_nerf.training.error_map.data.resize(m_nerf.training.error_map.resolution.prod() * m_nerf.training.dataset.n_images);
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.data.data(), 0, m_nerf.training.error_map.data.get_bytes(), stream));
	}

	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;
	if (envmap_gradient) {
		CUDA_CHECK_THROW(hipMemsetAsync(envmap_gradient, 0, sizeof(float)*m_envmap.envmap->n_params(), stream));
	}

	train_nerf_step(
		target_batch_size,
		m_nerf.training.counters_rgb.rays_per_batch,
		m_nerf.training.counters_rgb.numsteps_counter.data(),
		m_nerf.training.counters_rgb.numsteps_counter_compacted.data(),
		m_nerf.training.counters_rgb.loss.data(),
		m_training_stream
	);

	m_trainer->optimizer_step(stream, LOSS_SCALE);

	++m_training_step;

	if (envmap_gradient) {
		m_envmap.trainer->optimizer_step(stream, LOSS_SCALE);
	}

	float loss_scalar = m_nerf.training.counters_rgb.update_after_training(target_batch_size, get_loss_scalar, stream);
	bool zero_records = m_nerf.training.counters_rgb.measured_batch_size == 0;
	if (get_loss_scalar) {
		m_loss_scalar.update(loss_scalar);
	}

	if (zero_records) {
		m_loss_scalar.set(0.f);
		tlog::warning() << "Nerf training generated 0 samples. Aborting training.";
		m_train = false;
	}

	// Compute CDFs from the error map
	m_nerf.training.n_steps_since_error_map_update += 1;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;
	if (accumulate_error && m_nerf.training.n_steps_since_error_map_update >= m_nerf.training.n_steps_between_error_map_updates) {
		m_nerf.training.error_map.cdf_resolution = m_nerf.training.error_map.resolution;
		m_nerf.training.error_map.cdf_x_cond_y.resize(m_nerf.training.error_map.cdf_resolution.prod() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_y.resize(m_nerf.training.error_map.cdf_resolution.y() * m_nerf.training.dataset.n_images);
		m_nerf.training.error_map.cdf_img.resize(m_nerf.training.dataset.n_images);

		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_x_cond_y.data(), 0, m_nerf.training.error_map.cdf_x_cond_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_y.data(), 0, m_nerf.training.error_map.cdf_y.get_bytes(), stream));
		CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.error_map.cdf_img.data(), 0, m_nerf.training.error_map.cdf_img.get_bytes(), stream));

		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)m_nerf.training.error_map.cdf_resolution.y(), threads.x), div_round_up((uint32_t)m_nerf.training.dataset.n_images, threads.y), 1 };
		construct_cdf_2d<<<blocks, threads, 0, stream>>>(
			m_nerf.training.dataset.n_images, m_nerf.training.error_map.cdf_resolution.y(), m_nerf.training.error_map.cdf_resolution.x(),
			m_nerf.training.error_map.data.data(),
			m_nerf.training.error_map.cdf_x_cond_y.data(),
			m_nerf.training.error_map.cdf_y.data()
		);
		linear_kernel(construct_cdf_1d, 0, stream,
			m_nerf.training.dataset.n_images,
			m_nerf.training.error_map.cdf_resolution.y(),
			m_nerf.training.error_map.cdf_y.data(),
			m_nerf.training.error_map.cdf_img.data()
		);

		// Compute image CDF on the CPU. It's single-threaded anyway. No use parallelizing.
		m_nerf.training.error_map.pmf_img_cpu.resize(m_nerf.training.error_map.cdf_img.size());
		m_nerf.training.error_map.cdf_img.copy_to_host(m_nerf.training.error_map.pmf_img_cpu);
		std::vector<float> cdf_img_cpu = m_nerf.training.error_map.pmf_img_cpu; // Copy unnormalized PDF into CDF buffer
		float cum = 0;
		for (float& f : cdf_img_cpu) {
			cum += f;
			f = cum;
		}
		float norm = 1.0f / cum;
		for (size_t i = 0; i < cdf_img_cpu.size(); ++i) {
			constexpr float MIN_PMF = 0.1f;
			m_nerf.training.error_map.pmf_img_cpu[i] = (1.0f - MIN_PMF) * m_nerf.training.error_map.pmf_img_cpu[i] * norm + MIN_PMF / (float)m_nerf.training.dataset.n_images;
			cdf_img_cpu[i] = (1.0f - MIN_PMF) * cdf_img_cpu[i] * norm + MIN_PMF * (float)(i+1) / (float)m_nerf.training.dataset.n_images;
		}
		m_nerf.training.error_map.cdf_img.copy_from_host(cdf_img_cpu);

		// Reset counters and decrease update rate.
		m_nerf.training.n_steps_since_error_map_update = 0;
		m_nerf.training.n_rays_since_error_map_update = 0;
		m_nerf.training.error_map.is_cdf_valid = true;

		m_nerf.training.n_steps_between_error_map_updates = (uint32_t)(m_nerf.training.n_steps_between_error_map_updates * 1.5f);
	}

	// Get extrinsics gradients
	m_nerf.training.n_steps_since_cam_update += 1;


	if (train_extra_dims) {
		std::vector<float> extra_dims_gradient(m_nerf.training.extra_dims_gradient_gpu.size());
		std::vector<float> &extra_dims_new_values = extra_dims_gradient; // just create an alias to make the code clearer.
		m_nerf.training.extra_dims_gradient_gpu.copy_to_host(extra_dims_gradient);
		// Optimization step
		for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
			ArrayXf gradient(n_extra_dims);
			gradient.setZero();
			for (uint32_t j = 0; j<n_extra_dims; ++j) {
				gradient[j] = extra_dims_gradient[i * n_extra_dims + j] / LOSS_SCALE;
				if (isnan(gradient[j])) {
					printf("OH NO %d %d %0.3f\n", i,j, gradient[j]);
				}
			}

			float l2_reg = 1e-4f;
			gradient = m_nerf.training.extra_dims_opt[i].variable() * l2_reg;

			//m_nerf.training.extra_dims_opt[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.extra_dims_opt[i].step() / 128)), m_optimizer->learning_rate()/1000.0f));

			m_nerf.training.extra_dims_opt[i].step(gradient);

			const ArrayXf &value = m_nerf.training.extra_dims_opt[i].variable();
			for (uint32_t j = 0; j < n_extra_dims; ++j) {
				extra_dims_new_values[i * n_extra_dims + j] = value[j];
			}
		}

		//m_nerf.training.extra_dims_gpu.copy_from_host(extra_dims_new_values);
		CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.extra_dims_gpu.data(), extra_dims_new_values.data(), m_nerf.training.n_images_for_training * n_extra_dims * sizeof(float) , hipMemcpyHostToDevice, stream));
	}

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length || m_nerf.training.optimize_exposure;
	if (train_camera && m_nerf.training.n_steps_since_cam_update >= m_nerf.training.n_steps_between_cam_updates) {
		float per_camera_loss_scale = (float)m_nerf.training.n_images_for_training / LOSS_SCALE / (float)m_nerf.training.n_steps_between_cam_updates;

		if (m_nerf.training.optimize_extrinsics) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_pos_gradient.data(), m_nerf.training.cam_pos_gradient_gpu.data(), m_nerf.training.cam_pos_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_rot_gradient.data(), m_nerf.training.cam_rot_gradient_gpu.data(), m_nerf.training.cam_rot_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			CUDA_CHECK_THROW(hipStreamSynchronize(stream));

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				Vector3f pos_gradient = m_nerf.training.cam_pos_gradient[i] * per_camera_loss_scale;
				Vector3f rot_gradient = m_nerf.training.cam_rot_gradient[i] * per_camera_loss_scale;

				float l2_reg = m_nerf.training.extrinsic_l2_reg;
				pos_gradient += m_nerf.training.cam_pos_offset[i].variable() * l2_reg;
				rot_gradient += m_nerf.training.cam_rot_offset[i].variable() * l2_reg;

				m_nerf.training.cam_pos_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_pos_offset[i].step() / 128)), m_optimizer->learning_rate()/1000.0f));
				m_nerf.training.cam_rot_offset[i].set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_rot_offset[i].step() / 128)), m_optimizer->learning_rate()/1000.0f));

				m_nerf.training.cam_pos_offset[i].step(pos_gradient);
				m_nerf.training.cam_rot_offset[i].step(rot_gradient);
			}

			m_nerf.training.update_transforms();
		}

		if (m_nerf.training.optimize_distortion) {
			linear_kernel(safe_divide, 0, stream,
				m_distortion.map->n_params(),
				m_distortion.map->gradients(),
				m_distortion.map->gradient_weights()
			);
			m_distortion.trainer->optimizer_step(stream, LOSS_SCALE*(float)m_nerf.training.n_steps_between_cam_updates);
		}

		if (m_nerf.training.optimize_focal_length) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_focal_length_gradient.data(),m_nerf.training.cam_focal_length_gradient_gpu.data(),m_nerf.training.cam_focal_length_gradient_gpu.get_bytes(),hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
			Vector2f focal_length_gradient = m_nerf.training.cam_focal_length_gradient * per_camera_loss_scale;
			float l2_reg = m_nerf.training.intrinsic_l2_reg;
			focal_length_gradient += m_nerf.training.cam_focal_length_offset.variable() * l2_reg;
			m_nerf.training.cam_focal_length_offset.set_learning_rate(std::max(1e-3f * std::pow(0.33f, (float)(m_nerf.training.cam_focal_length_offset.step() / 128)),m_optimizer->learning_rate() / 1000.0f));
			m_nerf.training.cam_focal_length_offset.step(focal_length_gradient);
			m_nerf.training.update_metadata();
		}

		if (m_nerf.training.optimize_exposure) {
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gradient.data(), m_nerf.training.cam_exposure_gradient_gpu.data(), m_nerf.training.cam_exposure_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));

			Array3f mean_exposure = Array3f::Constant(0.0f);

			// Optimization step
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				Array3f gradient = m_nerf.training.cam_exposure_gradient[i] * per_camera_loss_scale;

				float l2_reg = m_nerf.training.exposure_l2_reg;
				gradient += m_nerf.training.cam_exposure[i].variable() * l2_reg;

				m_nerf.training.cam_exposure[i].set_learning_rate(m_optimizer->learning_rate());
				m_nerf.training.cam_exposure[i].step(gradient);

				mean_exposure += m_nerf.training.cam_exposure[i].variable();
			}

			mean_exposure /= m_nerf.training.n_images_for_training;

			// Renormalize
			std::vector<Array3f> cam_exposures(m_nerf.training.n_images_for_training);
			for (uint32_t i = 0; i < m_nerf.training.n_images_for_training; ++i) {
				cam_exposures[i] = m_nerf.training.cam_exposure[i].variable() -= mean_exposure;
			}

			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_exposure_gpu.data(), cam_exposures.data(), m_nerf.training.cam_exposure_gpu.get_bytes(), hipMemcpyHostToDevice, stream));
		}

		m_nerf.training.n_steps_since_cam_update = 0;
	}
}

void Testbed::train_nerf_step(uint32_t target_batch_size, uint32_t n_rays_per_batch, uint32_t* counter, uint32_t* compacted_counter, float* loss, hipStream_t stream) {
	const uint32_t padded_output_width = m_network->padded_output_width();
	const uint32_t max_samples = target_batch_size * 16; // Somewhat of a worst case
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		uint32_t, // ray_indices
		Ray, // rays
		uint32_t, // numsteps
		float, // coords
		float, // max_level
		network_precision_t, // mlp_out
		network_precision_t, // dloss_dmlp_out
		float, // coords_compacted
		float, // coords_gradient
		float, // max_level_compacted
		uint32_t // ray_counter
	>(
		stream, &alloc,
		n_rays_per_batch,
		n_rays_per_batch,
		n_rays_per_batch * 2,
		max_samples * floats_per_coord,
		max_samples,
		std::max(target_batch_size, max_samples) * padded_output_width,
		target_batch_size * padded_output_width,
		target_batch_size * floats_per_coord,
		target_batch_size * floats_per_coord,
		target_batch_size,
		1
	);

	// TODO: C++17 structured binding
	uint32_t* ray_indices = std::get<0>(scratch);
	Ray* rays_unnormalized = std::get<1>(scratch);
	uint32_t* numsteps = std::get<2>(scratch);
	float* coords = std::get<3>(scratch);
	float* max_level = std::get<4>(scratch);
	network_precision_t* mlp_out = std::get<5>(scratch);
	network_precision_t* dloss_dmlp_out = std::get<6>(scratch);
	float* coords_compacted = std::get<7>(scratch);
	float* coords_gradient = std::get<8>(scratch);
	float* max_level_compacted = std::get<9>(scratch);
	uint32_t* ray_counter = std::get<10>(scratch);

	uint32_t max_inference;
	if (m_nerf.training.counters_rgb.measured_batch_size_before_compaction == 0) {
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = max_inference = max_samples;
	} else {
		max_inference = next_multiple(std::min(m_nerf.training.counters_rgb.measured_batch_size_before_compaction, max_samples), tcnn::batch_size_granularity);
	}

	GPUMatrix<float> coords_matrix((float*)coords, floats_per_coord, max_inference);
	GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, max_inference);

	GPUMatrix<float> compacted_coords_matrix((float*)coords_compacted, floats_per_coord, target_batch_size);
	GPUMatrix<network_precision_t> compacted_rgbsigma_matrix(mlp_out, padded_output_width, target_batch_size);

	GPUMatrix<network_precision_t> gradient_matrix(dloss_dmlp_out, padded_output_width, target_batch_size);

	if (m_training_step == 0) {
		m_nerf.training.counters_rgb.n_rays_total = 0;
	}

	uint32_t n_rays_total = m_nerf.training.counters_rgb.n_rays_total;
	m_nerf.training.counters_rgb.n_rays_total += n_rays_per_batch;
	m_nerf.training.n_rays_since_error_map_update += n_rays_per_batch;

	// If we have an envmap, prepare its gradient buffer
	float* envmap_gradient = m_nerf.training.train_envmap ? m_envmap.envmap->gradients() : nullptr;

	bool sample_focal_plane_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_focal_plane_proportional_to_error;
	bool sample_image_proportional_to_error = m_nerf.training.error_map.is_cdf_valid && m_nerf.training.sample_image_proportional_to_error;
	bool include_sharpness_in_error = m_nerf.training.include_sharpness_in_error;
	// This is low-overhead enough to warrant always being on.
	// It makes for useful visualizations of the training error.
	bool accumulate_error = true;

	CUDA_CHECK_THROW(hipMemsetAsync(ray_counter, 0, sizeof(uint32_t), stream));

	linear_kernel(generate_training_samples_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		max_inference,
		n_rays_total,
		m_rng,
		ray_counter,
		counter,
		ray_indices,
		rays_unnormalized,
		numsteps,
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		m_nerf.training.n_images_for_training,
		m_nerf.training.metadata_gpu.data(),
		m_nerf.training.transforms_gpu.data(),
		m_nerf.density_grid_bitfield.data(),
		m_max_level_rand_training,
		max_level,
		m_nerf.training.snap_to_pixel_centers,
		m_nerf.training.train_envmap,
		m_nerf.cone_angle_constant,
		m_distortion.map->params(),
		m_distortion.resolution,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.cdf_resolution,
		m_nerf.training.extra_dims_gpu.data(),
		m_nerf_network->n_extra_dims()
	);

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level : nullptr);
	}

	m_network->inference_mixed_precision(stream, coords_matrix, rgbsigma_matrix, false);

	if (hg_enc) {
		hg_enc->set_max_level_gpu(m_max_level_rand_training ? max_level_compacted : nullptr);
	}

	linear_kernel(compute_loss_kernel_train_nerf, 0, stream,
		n_rays_per_batch,
		m_aabb,
		n_rays_total,
		m_rng,
		target_batch_size,
		ray_counter,
		LOSS_SCALE,
		padded_output_width,
		m_envmap.envmap->params(),
		envmap_gradient,
		m_envmap.resolution,
		m_envmap.loss_type,
		m_background_color.head<3>(),
		m_color_space,
		m_nerf.training.random_bg_color,
		m_nerf.training.linear_colors,
		m_nerf.training.n_images_for_training,
		m_nerf.training.metadata_gpu.data(),
		mlp_out,
		compacted_counter,
		ray_indices,
		rays_unnormalized,
		numsteps,
		PitchedPtr<const NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1 ,0, extra_stride),
		dloss_dmlp_out,
		m_nerf.training.loss_type,
		loss,
		m_max_level_rand_training,
		max_level_compacted,
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.training.snap_to_pixel_centers,
		accumulate_error ? m_nerf.training.error_map.data.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
		sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
		sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
		m_nerf.training.error_map.resolution,
		m_nerf.training.error_map.cdf_resolution,
		include_sharpness_in_error ? m_nerf.training.dataset.sharpness_data.data() : nullptr,
		m_nerf.training.dataset.sharpness_resolution,
		m_nerf.training.sharpness_grid.data(),
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.cam_exposure_gpu.data(),
		m_nerf.training.optimize_exposure ? m_nerf.training.cam_exposure_gradient_gpu.data() : nullptr,
		m_nerf.training.depth_supervision_lambda,
		m_nerf.training.near_distance
	);

	fill_rollover_and_rescale<network_precision_t><<<n_blocks_linear(target_batch_size*padded_output_width), n_threads_linear, 0, stream>>>(
		target_batch_size, padded_output_width, compacted_counter, dloss_dmlp_out
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size * floats_per_coord), n_threads_linear, 0, stream>>>(
		target_batch_size, floats_per_coord, compacted_counter, (float*)coords_compacted
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size), n_threads_linear, 0, stream>>>(
		target_batch_size, 1, compacted_counter, max_level_compacted
	);

	bool train_camera = m_nerf.training.optimize_extrinsics || m_nerf.training.optimize_distortion || m_nerf.training.optimize_focal_length;
	bool train_extra_dims = m_nerf.training.dataset.n_extra_learnable_dims > 0 && m_nerf.training.optimize_extra_dims;
	bool prepare_input_gradients = train_camera || train_extra_dims;
	GPUMatrix<float> coords_gradient_matrix((float*)coords_gradient, floats_per_coord, target_batch_size);

	{
		auto ctx = m_network->forward(stream, compacted_coords_matrix, &compacted_rgbsigma_matrix, false, prepare_input_gradients);
		m_network->backward(stream, *ctx, compacted_coords_matrix, compacted_rgbsigma_matrix, gradient_matrix, prepare_input_gradients ? &coords_gradient_matrix : nullptr, false, EGradientMode::Overwrite);
	}

	if (train_extra_dims) {
		// Compute extra-dim gradients
		linear_kernel(compute_extra_dims_gradient_train_nerf, 0, stream,
		n_rays_per_batch,
			n_rays_total,
			ray_counter,
			m_nerf.training.extra_dims_gradient_gpu.data(),
			m_nerf.training.dataset.n_extra_dims(),
			m_nerf.training.n_images_for_training,
			ray_indices,
			numsteps,
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_gradient, 1, 0, extra_stride),
			sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr
		);
	}

	if (train_camera) {
		// Compute camera gradients
		linear_kernel(compute_cam_gradient_train_nerf, 0, stream,
			n_rays_per_batch,
			n_rays_total,
			m_rng,
			m_aabb,
			ray_counter,
			m_nerf.training.transforms_gpu.data(),
			m_nerf.training.snap_to_pixel_centers,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_pos_gradient_gpu.data() : nullptr,
			m_nerf.training.optimize_extrinsics ? m_nerf.training.cam_rot_gradient_gpu.data() : nullptr,
			m_nerf.training.n_images_for_training,
			m_nerf.training.metadata_gpu.data(),
			ray_indices,
			rays_unnormalized,
			numsteps,
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1, 0, extra_stride),
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_gradient, 1, 0, extra_stride),
			m_nerf.training.optimize_distortion ? m_distortion.map->gradients() : nullptr,
			m_nerf.training.optimize_distortion ? m_distortion.map->gradient_weights() : nullptr,
			m_distortion.resolution,
			m_nerf.training.optimize_focal_length ? m_nerf.training.cam_focal_length_gradient_gpu.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_x_cond_y.data() : nullptr,
			sample_focal_plane_proportional_to_error ? m_nerf.training.error_map.cdf_y.data() : nullptr,
			sample_image_proportional_to_error ? m_nerf.training.error_map.cdf_img.data() : nullptr,
			m_nerf.training.error_map.cdf_resolution
		);
	}

	m_rng.advance();

	if (hg_enc) {
		hg_enc->set_max_level_gpu(nullptr);
	}
}


void Testbed::training_prep_nerf(uint32_t batch_size, hipStream_t stream) {
	if (m_nerf.training.n_images_for_training == 0) {
		return;
	}

	float alpha = m_nerf.training.density_grid_decay;
	uint32_t n_cascades = m_nerf.max_cascade+1;

	if (m_training_step < 256) {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()*n_cascades, 0, stream);
	} else {
		update_density_grid_nerf(alpha, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, NERF_GRIDSIZE()*NERF_GRIDSIZE()*NERF_GRIDSIZE()/4*n_cascades, stream);
	}
}

void Testbed::optimise_mesh_step(uint32_t n_steps) {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	const uint32_t padded_output_width = m_nerf_network->padded_density_output_width();
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float);
	GPUMemory<float> coords(n_verts * floats_per_coord);
	GPUMemory<network_precision_t> mlp_out(n_verts * padded_output_width);

	GPUMatrix<float> positions_matrix((float*)coords.data(), floats_per_coord, n_verts);
	GPUMatrix<network_precision_t, RM> density_matrix(mlp_out.data(), padded_output_width, n_verts);

	const float* extra_dims_gpu = get_inference_extra_dims(m_inference_stream);

	for (uint32_t i = 0; i < n_steps; ++i) {
		linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream,
			n_verts,
			m_aabb,
			m_mesh.verts.data(),
			PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords.data(), 1, 0, extra_stride),
			extra_dims_gpu
		);

		// For each optimizer step, we need the density at the given pos...
		m_nerf_network->density(m_inference_stream, positions_matrix, density_matrix);
		// ...as well as the input gradient w.r.t. density, which we will store in the nerf coords.
		m_nerf_network->input_gradient(m_inference_stream, 3, positions_matrix, positions_matrix);
		// and the 1ring centroid for laplacian smoothing
		compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);

		// With these, we can compute a gradient that points towards the threshold-crossing of density...
		compute_mesh_opt_gradients(
			m_mesh.thresh,
			m_mesh.verts,
			m_mesh.vert_normals,
			m_mesh.verts_smoothed,
			mlp_out.data(),
			floats_per_coord,
			(const float*)coords.data(),
			m_mesh.verts_gradient,
			m_mesh.smooth_amount,
			m_mesh.density_amount,
			m_mesh.inflate_amount
		);

		// ...that we can pass to the optimizer.
		m_mesh.verts_optimizer->step(m_inference_stream, 1.0f, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	}
}

void Testbed::compute_mesh_vertex_colors() {
	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	if (!n_verts) {
		return;
	}

	m_mesh.vert_colors.resize(n_verts);
	m_mesh.vert_colors.memset(0);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		const float* extra_dims_gpu = get_inference_extra_dims(m_inference_stream);

		const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
		const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float);
		GPUMemory<float> coords(n_verts * floats_per_coord);
		GPUMemory<float> mlp_out(n_verts * 4);

		GPUMatrix<float> positions_matrix((float*)coords.data(), floats_per_coord, n_verts);
		GPUMatrix<float> color_matrix(mlp_out.data(), 4, n_verts);
		linear_kernel(generate_nerf_network_inputs_from_positions, 0, m_inference_stream, n_verts, m_aabb, m_mesh.verts.data(), PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords.data(), 1, 0, extra_stride), extra_dims_gpu);
		m_network->inference(m_inference_stream, positions_matrix, color_matrix);
		linear_kernel(extract_srgb_with_activation, 0, m_inference_stream, n_verts * 3, 3, mlp_out.data(), (float*)m_mesh.vert_colors.data(), m_nerf.rgb_activation, m_nerf.training.linear_colors);
	}
}

GPUMemory<float> Testbed::get_density_on_grid(Vector3i res3d, const BoundingBox& aabb) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<float> density(n_elements);

	const uint32_t batch_size = std::min(n_elements, 1u<<20);
	bool nerf_mode = m_testbed_mode == ETestbedMode::Nerf;

	const uint32_t padded_output_width = nerf_mode ? m_nerf_network->padded_density_output_width() : m_network->padded_output_width();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		NerfPosition,
		network_precision_t
	>(m_inference_stream, &alloc, n_elements, batch_size * padded_output_width);

	NerfPosition* positions = std::get<0>(scratch);
	network_precision_t* mlp_out = std::get<1>(scratch);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };

	BoundingBox unit_cube = BoundingBox{Vector3f::Zero(), Vector3f::Ones()};
	generate_grid_samples_nerf_uniform<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, aabb, nerf_mode ? m_aabb : unit_cube , positions);

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		GPUMatrix<network_precision_t, RM> density_matrix(mlp_out, padded_output_width, local_batch_size);

		GPUMatrix<float> positions_matrix((float*)(positions + offset), sizeof(NerfPosition)/sizeof(float), local_batch_size);
		if (nerf_mode) {
			m_nerf_network->density(m_inference_stream, positions_matrix, density_matrix);
		} else {
			m_network->inference_mixed_precision(m_inference_stream, positions_matrix, density_matrix);
		}
		linear_kernel(grid_samples_half_to_float, 0, m_inference_stream,
			local_batch_size,
			m_aabb,
			density.data() + offset , //+ axis_step * n_elements,
			mlp_out,
			m_nerf.density_activation,
			positions + offset,
			nerf_mode ? m_nerf.density_grid.data() : nullptr
		);
	}

	return density;
}

GPUMemory<Eigen::Array4f> Testbed::get_rgba_on_grid(Vector3i res3d, Eigen::Vector3f ray_dir) {
	const uint32_t n_elements = (res3d.x()*res3d.y()*res3d.z());
	GPUMemory<Eigen::Array4f> rgba(n_elements);
	GPUMemory<NerfCoordinate> positions(n_elements);
	const uint32_t batch_size = std::min(n_elements, 1u<<20);

	// generate inputs
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)res3d.x(), threads.x), div_round_up((uint32_t)res3d.y(), threads.y), div_round_up((uint32_t)res3d.z(), threads.z) };
	generate_grid_samples_nerf_uniform_dir<<<blocks, threads, 0, m_inference_stream>>>(res3d, m_nerf.density_grid_ema_step, m_render_aabb, m_aabb, ray_dir, positions.data());

	// Only process 1m elements at a time
	for (uint32_t offset = 0; offset < n_elements; offset += batch_size) {
		uint32_t local_batch_size = std::min(n_elements - offset, batch_size);

		// run network
		GPUMatrix<float> positions_matrix((float*) (positions.data() + offset), sizeof(NerfCoordinate)/sizeof(float), local_batch_size);
		GPUMatrix<float> rgbsigma_matrix((float*) (rgba.data() + offset), 4, local_batch_size);
		m_network->inference(m_inference_stream, positions_matrix, rgbsigma_matrix);

		// convert network output to RGBA (in place)
		linear_kernel(compute_nerf_density, 0, m_inference_stream, local_batch_size, rgba.data() + offset, m_nerf.rgb_activation, m_nerf.density_activation);
	}
	return rgba;
}

int Testbed::marching_cubes(Vector3i res3d, const BoundingBox& aabb, float thresh) {
	res3d.x() = next_multiple((unsigned int)res3d.x(), 16u);
	res3d.y() = next_multiple((unsigned int)res3d.y(), 16u);
	res3d.z() = next_multiple((unsigned int)res3d.z(), 16u);

	if (thresh == std::numeric_limits<float>::max()) {
		thresh = m_mesh.thresh;
	}

	GPUMemory<float> density = get_density_on_grid(res3d, aabb);
	marching_cubes_gpu(m_inference_stream, m_render_aabb, res3d, thresh, density, m_mesh.verts, m_mesh.indices);

	uint32_t n_verts = (uint32_t)m_mesh.verts.size();
	m_mesh.verts_gradient.resize(n_verts);

	m_mesh.trainable_verts = std::make_shared<TrainableBuffer<3, 1, float>>(Matrix<int, 1, 1>{(int)n_verts});
	m_mesh.verts_gradient.copy_from_device(m_mesh.verts); // Make sure the vertices don't get destroyed in the initialization

	pcg32 rnd{m_seed};
	m_mesh.trainable_verts->initialize_params(rnd, (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts.data(), (float*)m_mesh.verts_gradient.data());
	m_mesh.verts.copy_from_device(m_mesh.verts_gradient);

	m_mesh.verts_optimizer.reset(create_optimizer<float>({
		{"otype", "Adam"},
		{"learning_rate", 1e-4},
		{"beta1", 0.9f},
		{"beta2", 0.99f},
	}));

	m_mesh.verts_optimizer->allocate(m_mesh.trainable_verts);

	compute_mesh_1ring(m_mesh.verts, m_mesh.indices, m_mesh.verts_smoothed, m_mesh.vert_normals);
	compute_mesh_vertex_colors();

	return (int)(m_mesh.indices.size()/3);
}

uint8_t* Testbed::Nerf::get_density_grid_bitfield_mip(uint32_t mip) {
	return density_grid_bitfield.data() + grid_mip_offset(mip)/8;
}

int Testbed::find_best_training_view(int default_view) {
	int bestimage = default_view;
	float bestscore = 1000.f;
	for (int i = 0; i < m_nerf.training.n_images_for_training; ++i) {
		float score = (m_nerf.training.transforms[i].start.col(3) - m_camera.col(3)).norm();
		score += 0.25f * (m_nerf.training.transforms[i].start.col(2) - m_camera.col(2)).norm();
		if (score < bestscore) {
			bestscore = score;
			bestimage = i;
		}
	}
	return bestimage;
}

NGP_NAMESPACE_END
