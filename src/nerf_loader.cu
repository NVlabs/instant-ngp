#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   nerfloader.cu
 *  @author Alex Evans & Thomas Müller, NVIDIA
 *  @brief  Loads a NeRF data set from NeRF's original format
 */

#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/thread_pool.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <json/json.hpp>

#include <filesystem/path.h>

#define _USE_MATH_DEFINES
#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>

#define STB_IMAGE_IMPLEMENTATION

#if defined(__NVCC__)
#if defined __NVCC_DIAG_PRAGMA_SUPPORT__
#  pragma nv_diag_suppress 550
#else
#  pragma diag_suppress 550
#endif
#endif
#include <stb_image/stb_image.h>
#if defined(__NVCC__)
#if defined __NVCC_DIAG_PRAGMA_SUPPORT__
#  pragma nv_diag_default 550
#else
#  pragma diag_default 550
#endif
#endif

using namespace tcnn;
using namespace std::literals;
using namespace Eigen;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

__global__ void convert_rgba32(const uint64_t num_pixels, const uint8_t* __restrict__ pixels, uint8_t* __restrict__ out, bool white_2_transparent = false, bool black_2_transparent = false, uint32_t mask_color = 0) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_pixels) return;

	uint8_t rgba[4];
	*((uint32_t*)&rgba[0]) = *((uint32_t*)&pixels[i*4]);

	// NSVF dataset has 'white = transparent' madness
	if (white_2_transparent && rgba[0] == 255 && rgba[1] == 255 && rgba[2] == 255) {
		rgba[3] = 0;
	}

	if (black_2_transparent && rgba[0] == 0 && rgba[1] == 0 && rgba[2] == 0) {
		rgba[3] = 0;
	}

	if (mask_color != 0 && mask_color == *((uint32_t*)&rgba[0])) {
		// turn the mask into hot pink
		rgba[0] = 0xFF; rgba[1] = 0x00; rgba[2] = 0xFF; rgba[3] = 0x00;
	}

	*((uint32_t*)&out[i*4]) = *((uint32_t*)&rgba[0]);
}

__global__ void from_fullp(const uint64_t num_elements, const float* __restrict__ pixels, __half* __restrict__ out) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	out[i] = (__half)pixels[i];
}

template <typename T>
__global__ void copy_depth(const uint64_t num_elements, float* __restrict__ depth_dst, const T* __restrict__ depth_pixels, float depth_scale) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_elements) return;

	if (depth_pixels == nullptr || depth_scale <= 0.f) {
		depth_dst[i] = 0.f; // no depth data for this entire image. zero it out
	} else {
		depth_dst[i] = depth_pixels[i] * depth_scale;
	}
}

template <typename T>
__global__ void sharpen(const uint64_t num_pixels, const uint32_t w, const T* __restrict__ pix, T* __restrict__ destpix, float center_w, float inv_totalw) {
	const uint64_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= num_pixels) return;

	float rgba[4] = {
		(float)pix[i*4+0]*center_w,
		(float)pix[i*4+1]*center_w,
		(float)pix[i*4+2]*center_w,
		(float)pix[i*4+3]*center_w
	};

	int64_t i2=i-1; if (i2<0) i2=0; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i-w; if (i2<0) i2=0; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i+1; if (i2>=num_pixels) i2-=num_pixels; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	i2=i+w; if (i2>=num_pixels) i2-=num_pixels; i2*=4;
	for (int j=0;j<4;++j) rgba[j]-=(float)pix[i2++];
	for (int j=0;j<4;++j) destpix[i*4+j]=(T)max(0.f, rgba[j] * inv_totalw);
}

__device__ inline float luma(const Array4f& c) {
	return c[0] * 0.2126f + c[1] * 0.7152f + c[2] * 0.0722f;
}

__global__ void compute_sharpness(Eigen::Vector2i sharpness_resolution, Eigen::Vector2i image_resolution, uint32_t n_images, const void* __restrict__ images_data, EImageDataType image_data_type, float* __restrict__ sharpness_data) {
	const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
	const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;
	const uint32_t i = threadIdx.z + blockIdx.z * blockDim.z;
	if (x >= sharpness_resolution.x() || y >= sharpness_resolution.y() || i>=n_images) return;
	const size_t sharp_size = sharpness_resolution.x() * sharpness_resolution.y();
	sharpness_data += sharp_size * i + x + y * sharpness_resolution.x();

	// overlap patches a bit
	int x_border = 0; // (image_resolution.x()/sharpness_resolution.x())/4;
	int y_border = 0; // (image_resolution.y()/sharpness_resolution.y())/4;

	int x1 = (x*image_resolution.x())/sharpness_resolution.x()-x_border, x2 = ((x+1)*image_resolution.x())/sharpness_resolution.x()+x_border;
	int y1 = (y*image_resolution.y())/sharpness_resolution.y()-y_border, y2 = ((y+1)*image_resolution.y())/sharpness_resolution.y()+y_border;
	// clamp to 1 pixel in from edge
	x1=max(x1,1); y1=max(y1,1);
	x2=min(x2,image_resolution.x()-2); y2=min(y2,image_resolution.y()-2);
	// yes, yes I know I should do a parallel reduction and shared memory and stuff. but we have so many tiles in flight, and this is load-time, meh.
	float tot_lap=0.f,tot_lap2=0.f,tot_lum=0.f;
	float scal=1.f/((x2-x1)*(y2-y1));
	for (int yy=y1;yy<y2;++yy) {
		for (int xx=x1; xx<x2; ++xx) {
			Array4f n, e, s, w, c;
			c = read_rgba(Vector2i{xx, yy}, image_resolution, images_data, image_data_type, i);
			n = read_rgba(Vector2i{xx, yy-1}, image_resolution, images_data, image_data_type, i);
			w = read_rgba(Vector2i{xx-1, yy}, image_resolution, images_data, image_data_type, i);
			s = read_rgba(Vector2i{xx, yy+1}, image_resolution, images_data, image_data_type, i);
			e = read_rgba(Vector2i{xx+1, yy}, image_resolution, images_data, image_data_type, i);
			float lum = luma(c);
			float lap = lum * 4.f - luma(n) - luma(e) - luma(s) - luma(w);
			tot_lap += lap;
			tot_lap2 += lap*lap;
			tot_lum += lum;
		}
	}
	tot_lap*=scal;
	tot_lap2*=scal;
	tot_lum*=scal;
	float variance_of_laplacian = tot_lap2 - tot_lap * tot_lap;
	*sharpness_data = (variance_of_laplacian) ; // / max(0.00001f,tot_lum*tot_lum); // var / (tot+0.001f);
}

bool ends_with(const std::string& str, const std::string& suffix) {
	return str.size() >= suffix.size() && 0 == str.compare(str.size()-suffix.size(), suffix.size(), suffix);
}

NerfDataset create_empty_nerf_dataset(size_t n_images, int aabb_scale, bool is_hdr) {
	NerfDataset result{};
	result.n_images = n_images;
	result.sharpness_resolution = { 128, 72 };
	result.sharpness_data.enlarge( result.sharpness_resolution.x() * result.sharpness_resolution.y() *  result.n_images );
	result.xforms.resize(n_images);
	result.metadata.resize(n_images);
	result.pixelmemory.resize(n_images);
	result.depthmemory.resize(n_images);
	result.raymemory.resize(n_images);
	result.scale = NERF_SCALE;
	result.offset = {0.5f, 0.5f, 0.5f};
	result.aabb_scale = aabb_scale;
	result.is_hdr = is_hdr;
	for (size_t i = 0; i < n_images; ++i) {
		result.xforms[i].start = Eigen::Matrix<float, 3, 4>::Identity();
		result.xforms[i].end = Eigen::Matrix<float, 3, 4>::Identity();
	}
	return result;
}

NerfDataset load_nerf(const std::vector<filesystem::path>& jsonpaths, float sharpen_amount) {
	if (jsonpaths.empty()) {
		throw std::runtime_error{"Cannot load NeRF data from an empty set of paths."};
	}

	tlog::info() << "Loading NeRF dataset from";

	NerfDataset result{};

	std::ifstream f{jsonpaths.front().str()};
	nlohmann::json transforms = nlohmann::json::parse(f, nullptr, true, true);

	ThreadPool pool;

	struct LoadedImageInfo {
		Eigen::Vector2i res = Eigen::Vector2i::Zero();
		bool image_data_on_gpu = false;
		EImageDataType image_type = EImageDataType::None;
		bool white_transparent = false;
		bool black_transparent = false;
		uint32_t mask_color = 0;
		void *pixels = nullptr;
		uint16_t *depth_pixels = nullptr;
		Ray *rays = nullptr;
		float depth_scale = -1.f;
	};
	std::vector<LoadedImageInfo> images;
	LoadedImageInfo info = {};

	if (transforms["camera"].is_array()) {
		throw std::runtime_error{"hdf5 is no longer supported. please use the hdf52nerf.py conversion script"};
	}

	// nerf original format
	std::vector<nlohmann::json> jsons;
	std::transform(
		jsonpaths.begin(), jsonpaths.end(),
		std::back_inserter(jsons), [](const auto& path) {
			return nlohmann::json::parse(std::ifstream{path.str()}, nullptr, true, true);
		}
	);

	result.n_images = 0;
	for (size_t i = 0; i < jsons.size(); ++i) {
		auto& json = jsons[i];
		fs::path basepath = jsonpaths[i].parent_path();
		if (!json.contains("frames") || !json["frames"].is_array()) {
			tlog::warning() << "  " << jsonpaths[i] << " does not contain any frames. Skipping.";
			continue;
		}
		tlog::info() << "  " << jsonpaths[i];
		auto& frames = json["frames"];

		float sharpness_discard_threshold = json.value("sharpness_discard_threshold", 0.0f); // Keep all by default

		std::sort(frames.begin(), frames.end(), [](const auto& frame1, const auto& frame2) {
			return frame1["file_path"] < frame2["file_path"];
		});

		if (json.contains("n_frames")) {
			size_t cull_idx = std::min(frames.size(), (size_t)json["n_frames"]);
			frames.get_ptr<nlohmann::json::array_t*>()->resize(cull_idx);
		}

		if (frames[0].contains("sharpness")) {
			auto frames_copy = frames;
			frames.clear();

			// Kill blurrier frames than their neighbors
			const int neighborhood_size = 3;
			for (int i = 0; i < (int)frames_copy.size(); ++i) {
				float mean_sharpness = 0.0f;
				int mean_start = std::max(0, i-neighborhood_size);
				int mean_end = std::min(i+neighborhood_size, (int)frames_copy.size()-1);
				for (int j = mean_start; j < mean_end; ++j) {
					mean_sharpness += float(frames_copy[j]["sharpness"]);
				}
				mean_sharpness /= (mean_end - mean_start);

				// Compatibility with Windows paths on Linux. (Breaks linux filenames with "\\" in them, which is acceptable for us.)
				frames_copy[i]["file_path"] = replace_all(frames_copy[i]["file_path"], "\\", "/");

				if ((basepath / fs::path(std::string(frames_copy[i]["file_path"]))).exists() && frames_copy[i]["sharpness"] > sharpness_discard_threshold * mean_sharpness) {
					frames.emplace_back(frames_copy[i]);
				} else {
					// tlog::info() << "discarding frame " << frames_copy[i]["file_path"];
					// fs::remove(basepath / fs::path(std::string(frames_copy[i]["file_path"])));
				}
			}
		}

		result.n_images += frames.size();
	}

	images.resize(result.n_images);
	result.xforms.resize(result.n_images);
	result.metadata.resize(result.n_images);
	result.pixelmemory.resize(result.n_images);
	result.depthmemory.resize(result.n_images);
	result.raymemory.resize(result.n_images);

	result.scale = NERF_SCALE;
	result.offset = {0.5f, 0.5f, 0.5f};

	std::vector<std::future<void>> futures;

	size_t image_idx = 0;
	if (result.n_images==0) {
		throw std::invalid_argument{"No training images were found for NeRF training!"};
	}

	auto progress = tlog::progress(result.n_images);

	result.from_mitsuba = false;
	bool fix_premult = false;
	bool enable_ray_loading = true;
	bool enable_depth_loading = true;
	std::atomic<int> n_loaded{0};
	BoundingBox cam_aabb;
	for (size_t i = 0; i < jsons.size(); ++i) {
		auto& json = jsons[i];

		fs::path basepath = jsonpaths[i].parent_path();
		std::string jp = jsonpaths[i].str();
		auto lastdot=jp.find_last_of('.'); if (lastdot==std::string::npos) lastdot=jp.length();
		auto lastunderscore=jp.find_last_of('_'); if (lastunderscore==std::string::npos) lastunderscore=lastdot; else lastunderscore++;
		std::string part_after_underscore(jp.begin()+lastunderscore,jp.begin()+lastdot);

		if (json.contains("enable_ray_loading")) {
			enable_ray_loading = bool(json["enable_ray_loading"]);
			tlog::info() << "enable_ray_loading=" << enable_ray_loading;
		}
		if (json.contains("enable_depth_loading")) {
			enable_depth_loading = bool(json["enable_depth_loading"]);
			tlog::info() << "enable_depth_loading is " << enable_depth_loading;
		}

		if (json.contains("normal_mts_args")) {
			result.from_mitsuba = true;
		}

		if (json.contains("fix_premult")) {
			fix_premult = (bool)json["fix_premult"];
		}

		if (result.from_mitsuba) {
			result.scale = 0.66f;
			result.offset = {0.25f * result.scale, 0.25f * result.scale, 0.25f * result.scale};
		}

		if (json.contains("render_aabb")) {
			result.render_aabb.min={float(json["render_aabb"][0][0]),float(json["render_aabb"][0][1]),float(json["render_aabb"][0][2])};
			result.render_aabb.max={float(json["render_aabb"][1][0]),float(json["render_aabb"][1][1]),float(json["render_aabb"][1][2])};
		}

		if (json.contains("sharpen")) {
			sharpen_amount = json["sharpen"];
		}

		if (json.contains("white_transparent")) {
			info.white_transparent = bool(json["white_transparent"]);
		}

		if (json.contains("black_transparent")) {
			info.black_transparent = bool(json["black_transparent"]);
		}

		if (json.contains("scale")) {
			result.scale = json["scale"];
		}

		if (json.contains("importance_sampling")) {
			result.wants_importance_sampling = json["importance_sampling"];
		}

		if (json.contains("n_extra_learnable_dims")) {
			result.n_extra_learnable_dims = json["n_extra_learnable_dims"];
		}

		CameraDistortion camera_distortion = {};
		Vector2f principal_point = Vector2f::Constant(0.5f);
		Vector4f rolling_shutter = Vector4f::Zero();

		if (json.contains("integer_depth_scale")) {
			info.depth_scale = json["integer_depth_scale"];
		}

		// Camera distortion
		{
			if (json.contains("k1")) {
				camera_distortion.params[0] = json["k1"];
				if (camera_distortion.params[0] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("k2")) {
				camera_distortion.params[1] = json["k2"];
				if (camera_distortion.params[1] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("p1")) {
				camera_distortion.params[2] = json["p1"];
				if (camera_distortion.params[2] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("p2")) {
				camera_distortion.params[3] = json["p2"];
				if (camera_distortion.params[3] != 0.f) {
					camera_distortion.mode = ECameraDistortionMode::Iterative;
				}
			}

			if (json.contains("cx")) {
				principal_point.x() = (float)json["cx"] / (float)json["w"];
			}

			if (json.contains("cy")) {
				principal_point.y() = (float)json["cy"] / (float)json["h"];
			}

			if (json.contains("rolling_shutter")) {
				// the rolling shutter is a float3 of [A,B,C] where the time
				// for each pixel is t= A + B * u + C * v
				// where u and v are the pixel coordinates (0-1),
				// and the resulting t is used to interpolate between the start
				// and end transforms for each training xform
				float motionblur_amount = 0.f;
				if (json["rolling_shutter"].size() >= 4) {
					motionblur_amount = float(json["rolling_shutter"][3]);
				}

				rolling_shutter = {float(json["rolling_shutter"][0]), float(json["rolling_shutter"][1]), float(json["rolling_shutter"][2]), motionblur_amount};
			}

			if (json.contains("ftheta_p0")) {
				camera_distortion.params[0] = json["ftheta_p0"];
				camera_distortion.params[1] = json["ftheta_p1"];
				camera_distortion.params[2] = json["ftheta_p2"];
				camera_distortion.params[3] = json["ftheta_p3"];
				camera_distortion.params[4] = json["ftheta_p4"];
				camera_distortion.params[5] = json["w"];
				camera_distortion.params[6] = json["h"];
				camera_distortion.mode = ECameraDistortionMode::FTheta;
			}
		}

		if (json.contains("aabb_scale")) {
			result.aabb_scale = json["aabb_scale"];
		}

		if (json.contains("offset")) {
			result.offset =
				json["offset"].is_array() ?
				Vector3f{float(json["offset"][0]), float(json["offset"][1]), float(json["offset"][2])} :
				Vector3f{float(json["offset"]), float(json["offset"]), float(json["offset"])};
		}

		if (json.contains("aabb")) {
			// map the given aabb of the form [[minx,miny,minz],[maxx,maxy,maxz]] via an isotropic scale and translate to fit in the (0,0,0)-(1,1,1) cube, with the given center at 0.5,0.5,0.5
			const auto& aabb=json["aabb"];
			float length = std::max(0.000001f,std::max(std::max(std::abs(float(aabb[1][0])-float(aabb[0][0])),std::abs(float(aabb[1][1])-float(aabb[0][1]))),std::abs(float(aabb[1][2])-float(aabb[0][2]))));
			result.scale = 1.f/length;
			result.offset = { ((float(aabb[1][0])+float(aabb[0][0]))*0.5f)*-result.scale + 0.5f , ((float(aabb[1][1])+float(aabb[0][1]))*0.5f)*-result.scale + 0.5f,((float(aabb[1][2])+float(aabb[0][2]))*0.5f)*-result.scale + 0.5f};
		}

		if (json.contains("frames") && json["frames"].is_array()) {
			for (int j = 0; j < json["frames"].size(); ++j) {
				auto& frame = json["frames"][j];
				nlohmann::json& jsonmatrix_start = frame.contains("transform_matrix_start") ? frame["transform_matrix_start"] : frame["transform_matrix"];
				nlohmann::json& jsonmatrix_end = frame.contains("transform_matrix_end") ? frame["transform_matrix_end"] : jsonmatrix_start;
				const Vector3f p = Vector3f{float(jsonmatrix_start[0][3]), float(jsonmatrix_start[1][3]), float(jsonmatrix_start[2][3])} * result.scale + result.offset;
				const Vector3f q = Vector3f{float(jsonmatrix_end[0][3]), float(jsonmatrix_end[1][3]), float(jsonmatrix_end[2][3])} * result.scale + result.offset;
				cam_aabb.enlarge(p);
				cam_aabb.enlarge(q);
			}
		}

		if (json.contains("up")) {
			// axes are permuted as for the xforms below
			result.up[0] = float(json["up"][1]);
			result.up[1] = float(json["up"][2]);
			result.up[2] = float(json["up"][0]);
		}

		if (json.contains("envmap") && result.envmap_resolution.isZero()) {
			std::string json_provided_path = json["envmap"];
			fs::path envmap_path = basepath / json_provided_path;
			if (!envmap_path.exists()) {
				throw std::runtime_error{std::string{"Environment map path "} + envmap_path.str() + " does not exist."};
			}

			if (equals_case_insensitive(envmap_path.extension(), "exr")) {
				result.envmap_data = load_exr(envmap_path.str(), result.envmap_resolution.x(), result.envmap_resolution.y());
				result.is_hdr = true;
			} else {
				result.envmap_data = load_stbi(envmap_path.str(), result.envmap_resolution.x(), result.envmap_resolution.y());
			}
		}

		if (json.contains("frames") && json["frames"].is_array()) pool.parallelForAsync<size_t>(0, json["frames"].size(), [&, basepath, image_idx, info](size_t i) {
			size_t i_img = i + image_idx;
			auto& frame = json["frames"][i];
			LoadedImageInfo& dst = images[i_img];
			dst = info; // copy defaults

			std::string json_provided_path(frame["file_path"]);
			if (json_provided_path == "") {
				char buf[256];
				snprintf(buf, 256, "%s_%03d/rgba.png", part_after_underscore.c_str(), (int)i);
				json_provided_path = buf;
			}
			fs::path path = basepath / json_provided_path;

			if (path.extension() == "") {
				path = path.with_extension("png");
				if (!path.exists()) {
					path = path.with_extension("exr");
				}
				if (!path.exists()) {
					throw std::runtime_error{ "Could not find image file: " + path.str()};
				}
			}

			int comp = 0;
			if (equals_case_insensitive(path.extension(), "exr")) {
				dst.pixels = load_exr_to_gpu(&dst.res.x(), &dst.res.y(), path.str().c_str(), fix_premult);
				dst.image_type = EImageDataType::Half;
				dst.image_data_on_gpu = true;
				result.is_hdr = true;
			} else {
				dst.image_data_on_gpu = false;
				uint8_t* img = stbi_load(path.str().c_str(), &dst.res.x(), &dst.res.y(), &comp, 4);


				fs::path alphapath = basepath / (std::string{frame["file_path"]} + ".alpha."s + path.extension());
				if (alphapath.exists()) {
					int wa=0,ha=0;
					uint8_t* alpha_img = stbi_load(alphapath.str().c_str(), &wa, &ha, &comp, 4);
					if (!alpha_img) {
						throw std::runtime_error{"Could not load alpha image "s + alphapath.str()};
					}
					ScopeGuard mem_guard{[&]() { stbi_image_free(alpha_img); }};
					if (wa != dst.res.x() || ha != dst.res.y()) {
						throw std::runtime_error{std::string{"Alpha image has wrong resolution: "} + alphapath.str()};
					}
					tlog::success() << "Alpha loaded from " << alphapath;
					for (int i=0;i<dst.res.prod();++i) {
						img[i*4+3] = uint8_t(255.0f*srgb_to_linear(alpha_img[i*4]*(1.f/255.f))); // copy red channel of alpha to alpha.png to our alpha channel
					}
				}

				fs::path maskpath = path.parent_path()/(std::string{"dynamic_mask_"} + path.basename() + ".png");
				if (maskpath.exists()) {
					int wa=0,ha=0;
					uint8_t* mask_img = stbi_load(maskpath.str().c_str(), &wa, &ha, &comp, 4);
					if (!mask_img) {
						throw std::runtime_error{std::string{"Could not load mask image "} + maskpath.str()};
					}
					ScopeGuard mem_guard{[&]() { stbi_image_free(mask_img); }};
					if (wa != dst.res.x() || ha != dst.res.y()) {
						throw std::runtime_error{std::string{"Mask image has wrong resolution: "} + maskpath.str()};
					}
					dst.mask_color = 0x00FF00FF; // HOT PINK
					for (int i = 0; i < dst.res.prod(); ++i) {
						if (mask_img[i*4] != 0) {
							*(uint32_t*)&img[i*4] = dst.mask_color;
						}
					}
				}

				dst.pixels = img;
				dst.image_type = EImageDataType::Byte;
			}

			if (!dst.pixels) {
				throw std::runtime_error{ "image not found: " + path.str() };
			}

			if (enable_depth_loading && info.depth_scale > 0.f && frame.contains("depth_path")) {
				fs::path depthpath = basepath / std::string{frame["depth_path"]};
				if (depthpath.exists()) {
					int wa=0,ha=0;
					dst.depth_pixels = stbi_load_16(depthpath.str().c_str(), &wa, &ha, &comp, 1);
					if (!dst.depth_pixels) {
						throw std::runtime_error{"Could not load depth image "s + depthpath.str()};
					}
					if (wa != dst.res.x() || ha != dst.res.y()) {
						throw std::runtime_error{std::string{"Depth image has wrong resolution: "} + depthpath.str()};
					}
					//tlog::success() << "Depth loaded from " << depthpath;
				}
			}

			fs::path rayspath = path.parent_path()/(std::string{"rays_"} + path.basename() + ".dat");
			if (enable_ray_loading && rayspath.exists()) {
				uint32_t n_pixels = dst.res.prod();
				dst.rays = (Ray*)malloc(n_pixels * sizeof(Ray));

				std::ifstream rays_file{rayspath.str(), std::ios::binary};
				rays_file.read((char*)dst.rays, n_pixels * sizeof(Ray));

				std::streampos fsize = 0;
				fsize = rays_file.tellg();
				rays_file.seekg(0, std::ios::end);
				fsize = rays_file.tellg() - fsize;

				if (fsize > 0) {
					tlog::warning() << fsize << " bytes remaining in rays file " << rayspath;
				}

				for (uint32_t px = 0; px < n_pixels; ++px) {
					result.nerf_ray_to_ngp(dst.rays[px]);
				}
				result.has_rays = true;
			}

			nlohmann::json& jsonmatrix_start = frame.contains("transform_matrix_start") ? frame["transform_matrix_start"] : frame["transform_matrix"];
			nlohmann::json& jsonmatrix_end =   frame.contains("transform_matrix_end") ? frame["transform_matrix_end"] : jsonmatrix_start;

			if (frame.contains("driver_parameters")) {
				Eigen::Vector3f light_dir(
					frame["driver_parameters"].value("LightX", 0.f),
					frame["driver_parameters"].value("LightY", 0.f),
					frame["driver_parameters"].value("LightZ", 0.f)
				);
				result.metadata[i_img].light_dir = result.nerf_direction_to_ngp(light_dir.normalized());
				result.has_light_dirs = true;
				result.n_extra_learnable_dims = 0;
			}

			auto read_focal_length = [&](int resolution, const std::string& axis) {
				if (frame.contains(axis + "_fov")) {
					return fov_to_focal_length(resolution, (float)frame[axis + "_fov"]);
				} else if (json.contains("fl_"s + axis)) {
					return (float)json["fl_"s + axis];
				} else if (json.contains("camera_angle_"s + axis)) {
					return fov_to_focal_length(resolution, (float)json["camera_angle_"s + axis] * 180 / PI());
				} else {
					return 0.0f;
				}
			};

			// x_fov is in degrees, camera_angle_x in radians. Yes, it's silly.
			float x_fl = read_focal_length(dst.res.x(), "x");
			float y_fl = read_focal_length(dst.res.y(), "y");

			if (x_fl != 0) {
				result.metadata[i_img].focal_length = Vector2f::Constant(x_fl);
				if (y_fl != 0) {
					result.metadata[i_img].focal_length.y() = y_fl;
				}
			} else if (y_fl != 0) {
				result.metadata[i_img].focal_length = Vector2f::Constant(y_fl);
			} else {
				throw std::runtime_error{"Couldn't read fov."};
			}

			for (int m = 0; m < 3; ++m) {
				for (int n = 0; n < 4; ++n) {
					result.xforms[i_img].start(m, n) = float(jsonmatrix_start[m][n]);
					result.xforms[i_img].end(m, n) = float(jsonmatrix_end[m][n]);
				}
			}

			result.metadata[i_img].rolling_shutter = rolling_shutter;
			result.metadata[i_img].principal_point = principal_point;
			result.metadata[i_img].camera_distortion = camera_distortion;

			result.xforms[i_img].start = result.nerf_matrix_to_ngp(result.xforms[i_img].start);
			result.xforms[i_img].end = result.nerf_matrix_to_ngp(result.xforms[i_img].end);

			progress.update(++n_loaded);
		}, futures);

		if (json.contains("frames")) {
			image_idx += json["frames"].size();
		}

	}

	waitAll(futures);

	tlog::success() << "Loaded " << images.size() << " images after " << tlog::durationToString(progress.duration());
	tlog::info() << "  cam_aabb=" << cam_aabb;

	if (result.has_rays) {
		tlog::success() << "Loaded per-pixel rays.";
	}
	if (!images.empty() && images[0].mask_color) {
		tlog::success() << "Loaded dynamic masks.";
	}

	result.sharpness_resolution = { 128, 72 };
	result.sharpness_data.enlarge( result.sharpness_resolution.x() * result.sharpness_resolution.y() *  result.n_images );

	// copy / convert images to the GPU
	for (uint32_t i = 0; i < result.n_images; ++i) {
		const LoadedImageInfo& m = images[i];
		result.set_training_image(i, m.res, m.pixels, m.depth_pixels, m.depth_scale * result.scale, m.image_data_on_gpu, m.image_type, EDepthDataType::UShort, sharpen_amount, m.white_transparent, m.black_transparent, m.mask_color, m.rays);
		CUDA_CHECK_THROW(hipDeviceSynchronize());
	}
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	// free memory
	for (uint32_t i = 0; i < result.n_images; ++i) {
		if (images[i].image_data_on_gpu) {
			CUDA_CHECK_THROW(hipFree(images[i].pixels));
		} else {
			free(images[i].pixels);
		}
		free(images[i].rays);
		free(images[i].depth_pixels);
	}
	return result;
}

void NerfDataset::set_training_image(int frame_idx, const Eigen::Vector2i& image_resolution, const void* pixels, const void* depth_pixels, float depth_scale, bool image_data_on_gpu, EImageDataType image_type, EDepthDataType depth_type, float sharpen_amount, bool white_transparent, bool black_transparent, uint32_t mask_color, const Ray *rays) {
	if (frame_idx < 0 || frame_idx >= n_images) {
		throw std::runtime_error{"NerfDataset::set_training_image: invalid frame index"};
	}
	size_t n_pixels = image_resolution.prod();
	size_t img_size = n_pixels * 4; // 4 channels
	size_t image_type_stride = image_type_size(image_type);
	// copy to gpu if we need to do a conversion
	GPUMemory<uint8_t> images_data_gpu_tmp;
	GPUMemory<uint8_t> depth_tmp;
	if (!image_data_on_gpu && image_type == EImageDataType::Byte) {
		images_data_gpu_tmp.resize(img_size * image_type_stride);
		images_data_gpu_tmp.copy_from_host((uint8_t*)pixels);
		pixels = images_data_gpu_tmp.data();

		if (depth_pixels) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		image_data_on_gpu = true;
	}

	// copy or convert the pixels
	pixelmemory[frame_idx].resize(img_size * image_type_size(image_type));
	void* dst = pixelmemory[frame_idx].data();

	switch (image_type) {
		default: throw std::runtime_error{"unknown image type in set_training_image"};
		case EImageDataType::Byte: linear_kernel(convert_rgba32, 0, nullptr, n_pixels, (uint8_t*)pixels, (uint8_t*)dst, white_transparent, black_transparent, mask_color); break;
		case EImageDataType::Half: // fallthrough is intended
		case EImageDataType::Float: CUDA_CHECK_THROW(hipMemcpy(dst, pixels, img_size * image_type_size(image_type), image_data_on_gpu ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice)); break;
	}

	// copy over depths if provided
	if (depth_scale >= 0.f) {
		depthmemory[frame_idx].resize(img_size);
		float* depth_dst = depthmemory[frame_idx].data();

		if (depth_pixels && !image_data_on_gpu) {
			depth_tmp.resize(n_pixels * depth_type_size(depth_type));
			depth_tmp.copy_from_host((uint8_t*)depth_pixels);
			depth_pixels = depth_tmp.data();
		}

		switch (depth_type) {
			default: throw std::runtime_error{"unknown depth type in set_training_image"};
			case EDepthDataType::UShort: linear_kernel(copy_depth<uint16_t>, 0, nullptr, n_pixels, depth_dst, (const uint16_t*)depth_pixels, depth_scale); break;
			case EDepthDataType::Float: linear_kernel(copy_depth<float>, 0, nullptr, n_pixels, depth_dst, (const float*)depth_pixels, depth_scale); break;
		}
	} else {
		depthmemory[frame_idx].free_memory();
	}

	// apply requested sharpening
	if (sharpen_amount > 0.f) {
		if (image_type == EImageDataType::Byte) {
			tcnn::GPUMemory<uint8_t> images_data_half(img_size * sizeof(__half));
			linear_kernel(from_rgba32<__half>, 0, nullptr, n_pixels, (uint8_t*)pixels, (__half*)images_data_half.data(), white_transparent, black_transparent, mask_color);
			pixelmemory[frame_idx] = std::move(images_data_half);
			dst = pixelmemory[frame_idx].data();
			image_type = EImageDataType::Half;
		}

		assert(image_type == EImageDataType::Half || image_type == EImageDataType::Float);

		tcnn::GPUMemory<uint8_t> images_data_sharpened(img_size * image_type_size(image_type));

		float center_w = 4.f + 1.f / sharpen_amount; // center_w ranges from 5 (strong sharpening) to infinite (no sharpening)
		if (image_type == EImageDataType::Half) {
			linear_kernel(sharpen<__half>, 0, nullptr, n_pixels, image_resolution.x(), (__half*)dst, (__half*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		} else {
			linear_kernel(sharpen<float>, 0, nullptr, n_pixels, image_resolution.x(), (float*)dst, (float*)images_data_sharpened.data(), center_w, 1.f / (center_w - 4.f));
		}

		pixelmemory[frame_idx] = std::move(images_data_sharpened);
		dst = pixelmemory[frame_idx].data();
	}

	if (sharpness_data.size()>0) {
		// compute overall sharpness
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)sharpness_resolution.x(), threads.x), div_round_up((uint32_t)sharpness_resolution.y(), threads.y), 1 };
		sharpness_data.enlarge(sharpness_resolution.x() * sharpness_resolution.y());
		compute_sharpness<<<blocks, threads, 0, nullptr>>>(sharpness_resolution, image_resolution, 1, dst, image_type, sharpness_data.data() + sharpness_resolution.x() * sharpness_resolution.y() * (size_t)frame_idx);
	}

	metadata[frame_idx].pixels = pixelmemory[frame_idx].data();
	metadata[frame_idx].depth = depthmemory[frame_idx].data();
	metadata[frame_idx].resolution = image_resolution;
	metadata[frame_idx].image_data_type = image_type;
	if (rays) {
		raymemory[frame_idx].resize(n_pixels);
		CUDA_CHECK_THROW(hipMemcpy(raymemory[frame_idx].data(), rays, n_pixels * sizeof(Ray), hipMemcpyHostToDevice));
	} else {
		raymemory[frame_idx].free_memory();
	}
	metadata[frame_idx].rays = raymemory[frame_idx].data();
}


NGP_NAMESPACE_END
