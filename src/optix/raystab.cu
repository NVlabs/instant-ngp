#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   raystab.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  Minimal optix program.
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/random_val.cuh>

#include <optix.h>

#include "raystab.h"

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

extern "C" {
	__constant__ Raystab::Params params;
}

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	Vector3f ray_origin = params.ray_origins[idx.x];

	default_rng_t rng;
	rng.advance(idx.x * 2);
	Vector2f offset = random_val_2d(rng);

	static constexpr uint32_t N_STAB_RAYS = 32;
	for (uint32_t i = 0; i < N_STAB_RAYS; ++i) {
		// Use a Fibonacci lattice (with random offset) to regularly
		// distribute the stab rays over the sphere.
		Vector3f ray_direction = fibonacci_dir<N_STAB_RAYS>(i, offset);

		// Trace the stab ray against our scene hierarchy
		unsigned int p0;
		optixTrace(
			params.handle,
			to_float3(ray_origin),
			to_float3(ray_direction),
			0.0f,                // Min intersection distance
			1e16f,               // Max intersection distance
			0.0f,                // rayTime
			OptixVisibilityMask(255), // Specify always visible
			OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT | OPTIX_RAY_FLAG_DISABLE_ANYHIT,
			0,                   // SBT offset
			1,                   // SBT stride
			0,                   // missSBTIndex
			p0
		);

		if (p0 == 0) {
			// One ray escaped. We are outside. Distance doesn't need to be signed.
			return;
		}
	}

	params.distances[idx.x] = -params.distances[idx.x];
}

extern "C" __global__ void __miss__ms() {
	optixSetPayload_0(0);
}

extern "C" __global__ void __closesthit__ch() {
	optixSetPayload_0(1);
}

NGP_NAMESPACE_END
