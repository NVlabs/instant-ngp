#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   pathescape.cu
 *  @author Thomas Müller, NVIDIA
 *  @brief  Minimal optix program.
 */

#include <neural-graphics-primitives/random_val.cuh>
#include <neural-graphics-primitives/common_device.cuh>
#include <optix.h>

#include "pathescape.h"

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

extern "C" {
	__constant__ PathEscape::Params params;
}

struct Onb {
	inline __device__ Onb(const Vector3f& normal) {
		m_normal = normal;

		if (fabs(m_normal.x()) > fabs(m_normal.z())) {
			m_binormal.x() = -m_normal.y();
			m_binormal.y() =  m_normal.x();
			m_binormal.z() =  0;
		} else {
			m_binormal.x() =  0;
			m_binormal.y() = -m_normal.z();
			m_binormal.z() =  m_normal.y();
		}

		m_binormal = m_binormal.normalized();
		m_tangent = m_binormal.cross(m_normal);
	}

	inline __device__ void inverse_transform(Vector3f& p) const {
		p = p.x()*m_tangent + p.y()*m_binormal + p.z()*m_normal;
	}

	Vector3f m_tangent;
	Vector3f m_binormal;
	Vector3f m_normal;
};

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();

	Vector3f query_point = params.ray_origins[idx.x];

	static constexpr uint32_t N_PATHS = 32;
	static constexpr uint32_t N_BOUNCES = 4;

	default_rng_t rng;
	rng.advance(idx.x * 4 * N_PATHS * N_BOUNCES);
	uint32_t n_escaped = 0;
	for (uint32_t i = 0; i < N_PATHS; ++i) {
		Vector3f ray_origin = query_point;
		Vector3f ray_direction = random_dir(rng);

		for (uint32_t j = 0; j < N_BOUNCES; ++j) {
			// Trace the stab ray against our scene hierarchy
			unsigned int p0;
			optixTrace(
				params.handle,
				to_float3(ray_origin),
				to_float3(ray_direction),
				0.0f,                // Min intersection distance
				1e16f,               // Max intersection distance
				0.0f,                // rayTime
				OptixVisibilityMask(255), // Specify always visible
				OPTIX_RAY_FLAG_DISABLE_ANYHIT,
				0,                   // SBT offset
				1,                   // SBT stride
				0,                   // missSBTIndex
				p0
			);

			// If the ray didn't escape, p0 contains the index of the triangle that was hit.
			if ((int)p0 == -1) {
				// 2 rays escaped. We are definitely outside and the escape was likely not a numerical fluke.
				// Distance doesn't need to be signed.
				if (++n_escaped > 2) {
					return;
				}
				break;
			}

			Vector3f N_0;
			float t = params.triangles[p0].ray_intersect(ray_origin, ray_direction, N_0);
			const Vector3f N = faceforward(N_0, -ray_direction, N_0).normalized();

			// Prevent self-intersections by subtracting 1e-3f from the target distance.
			ray_origin += ray_direction * fmaxf(0.0f, t - 1e-3f);

			ray_direction = random_dir_cosine(rng);
			Onb onb(N);
			onb.inverse_transform(ray_direction);
		}
	}
	params.distances[idx.x] = -params.distances[idx.x];
}

extern "C" __global__ void __miss__ms() {
	optixSetPayload_0((uint32_t)-1);
}

extern "C" __global__ void __closesthit__ch() {
	optixSetPayload_0(optixGetPrimitiveIndex());
}

NGP_NAMESPACE_END
