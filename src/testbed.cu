#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <fstream>
#include <set>

#ifdef NGP_GUI
#  include <imgui/imgui.h>
#  include <imgui/backends/imgui_impl_glfw.h>
#  include <imgui/backends/imgui_impl_opengl3.h>
#  include <imguizmo/ImGuizmo.h>
#  include <stb_image/stb_image.h>
#  ifdef _WIN32
#    include <GL/gl3w.h>
#  else
#    include <GL/glew.h>
#  endif
#  include <GLFW/glfw3.h>


#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace Eigen;
using namespace std::literals::chrono_literals;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN

std::atomic<size_t> g_total_n_bytes_allocated{0};

json merge_parent_network_config(const json &child, const fs::path &child_filename) {
	if (!child.contains("parent")) {
		return child;
	}
	fs::path parent_filename = child_filename.parent_path() / std::string(child["parent"]);
	tlog::info() << "Loading parent network config from: " << parent_filename.str();
	std::ifstream f{parent_filename.str()};
	json parent = json::parse(f, nullptr, true, true);
	parent = merge_parent_network_config(parent, parent_filename);
	parent.merge_patch(child);
	return parent;
}

static bool ends_with(const std::string& str, const std::string& ending) {
	if (ending.length() > str.length()) {
		return false;
	}
	return std::equal(std::rbegin(ending), std::rend(ending), std::rbegin(str));
}

void Testbed::load_training_data(const std::string& data_path) {
	m_data_path = data_path;

	if (!m_data_path.exists()) {
		throw std::runtime_error{fmt::format("Data path {} does not exist.", m_data_path.str())};
	}

	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:  load_nerf(); break;
		case ETestbedMode::Sdf:   load_mesh(); break;
		case ETestbedMode::Image: load_image(); break;
		case ETestbedMode::Volume:load_volume(); break;
		default: throw std::runtime_error{"Invalid testbed mode."};
	}

	m_training_data_available = true;
}

void Testbed::clear_training_data() {
	m_training_data_available = false;
	m_nerf.training.dataset.metadata.clear();
}

json Testbed::load_network_config(const fs::path& network_config_path) {
	if (!network_config_path.empty()) {
		m_network_config_path = network_config_path;
	}

	tlog::info() << "Loading network config from: " << network_config_path;

	if (network_config_path.empty() || !network_config_path.exists()) {
		throw std::runtime_error{fmt::format("Network config {} does not exist.", network_config_path.str())};
	}

	json result;
	if (equals_case_insensitive(network_config_path.extension(), "json")) {
		std::ifstream f{network_config_path.str()};
		result = json::parse(f, nullptr, true, true);
		result = merge_parent_network_config(result, network_config_path);
	} else if (equals_case_insensitive(network_config_path.extension(), "msgpack")) {
		std::ifstream f{network_config_path.str(), std::ios::in | std::ios::binary};
		result = json::from_msgpack(f);
		// we assume parent pointers are already resolved in snapshots.
	}

	return result;
}

void Testbed::reload_network_from_file(const std::string& network_config_path) {
	if (!network_config_path.empty()) {
		m_network_config_path = network_config_path;
	}

	m_network_config = load_network_config(m_network_config_path);
	reset_network();
}

void Testbed::reload_network_from_json(const json& json, const std::string& config_base_path) {
	// config_base_path is needed so that if the passed in json uses the 'parent' feature, we know where to look...
	// be sure to use a filename, or if a directory, end with a trailing slash
	m_network_config = merge_parent_network_config(json, config_base_path);
	reset_network();
}

void Testbed::handle_file(const std::string& file) {
	if (ends_with(file, ".msgpack")) {
		load_snapshot(file);
	}
	else if (ends_with(file, ".json")) {
		reload_network_from_file(file);
	} else if (ends_with(file, ".obj") || ends_with(file, ".stl")) {
		m_data_path = file;
		m_testbed_mode = ETestbedMode::Sdf;
		load_mesh();
	} else if (ends_with(file, ".exr") || ends_with(file, ".bin")) {
		m_data_path = file;
		m_testbed_mode = ETestbedMode::Image;
		try {
			load_image();
		} catch (std::runtime_error& e) {
			tlog::error() << "Failed to open image: " << e.what();
			return;
		}
	} else if (ends_with(file, ".nvdb")) {
		m_data_path = file;
		m_testbed_mode = ETestbedMode::Volume;
		try {
			load_volume();
		} catch (std::runtime_error& e) {
			tlog::error() << "Failed to open volume: " << e.what();
			return;
		}
	} else {
		tlog::error() << "Tried to open unknown file type: " << file;
	}
}

void Testbed::reset_accumulation(bool due_to_camera_movement, bool immediate_redraw) {
	if (immediate_redraw) {
		redraw_next_frame();
	}

	if (!due_to_camera_movement || !reprojection_available()) {
		m_windowless_render_surface.reset_accumulation();
		for (auto& tex : m_render_surfaces) {
			tex.reset_accumulation();
		}
	}
}

void Testbed::set_visualized_dim(int dim) {
	m_visualized_dimension = dim;
	reset_accumulation();
}

void Testbed::translate_camera(const Vector3f& rel) {
	m_camera.col(3) += m_camera.block<3,3>(0,0) * rel * m_bounding_radius;
	reset_accumulation(true);
}

void Testbed::set_nerf_camera_matrix(const Matrix<float, 3, 4>& cam) {
	m_camera = m_nerf.training.dataset.nerf_matrix_to_ngp(cam);
}

Vector3f Testbed::look_at() const {
	return view_pos() + view_dir() * m_scale;
}

void Testbed::set_look_at(const Vector3f& pos) {
	m_camera.col(3) += pos - look_at();
}

void Testbed::set_scale(float scale) {
	auto prev_look_at = look_at();
	m_camera.col(3) = (view_pos() - prev_look_at) * (scale / m_scale) + prev_look_at;
	m_scale = scale;
}

void Testbed::set_view_dir(const Vector3f& dir) {
	auto old_look_at = look_at();
	m_camera.col(0) = dir.cross(m_up_dir).normalized();
	m_camera.col(1) = dir.cross(m_camera.col(0)).normalized();
	m_camera.col(2) = dir.normalized();
	set_look_at(old_look_at);
}

void Testbed::first_training_view() {
	m_nerf.training.view = 0;
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::last_training_view() {
	m_nerf.training.view = m_nerf.training.dataset.n_images-1;
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::previous_training_view() {
	if (m_nerf.training.view != 0) {
		m_nerf.training.view -= 1;
	}
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::next_training_view() {
	if (m_nerf.training.view != m_nerf.training.dataset.n_images-1) {
		m_nerf.training.view += 1;
	}
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::set_camera_to_training_view(int trainview) {
	auto old_look_at = look_at();
	m_camera = m_smoothed_camera = get_xform_given_rolling_shutter(m_nerf.training.transforms[trainview], m_nerf.training.dataset.metadata[trainview].rolling_shutter, Vector2f{0.5f, 0.5f}, 0.0f);
	m_relative_focal_length = m_nerf.training.dataset.metadata[trainview].focal_length / (float)m_nerf.training.dataset.metadata[trainview].resolution[m_fov_axis];
	m_scale = std::max((old_look_at - view_pos()).dot(view_dir()), 0.1f);
	m_nerf.render_with_lens_distortion = true;
	m_nerf.render_lens = m_nerf.training.dataset.metadata[trainview].lens;
	m_screen_center = Vector2f::Constant(1.0f) - m_nerf.training.dataset.metadata[0].principal_point;
}

void Testbed::reset_camera() {
	m_fov_axis = 1;
	set_fov(50.625f);
	m_zoom = 1.f;
	m_screen_center = Vector2f::Constant(0.5f);
	m_scale = m_testbed_mode == ETestbedMode::Image ? 1.0f : 1.5f;
	m_camera <<
		1.0f, 0.0f, 0.0f, 0.5f,
		0.0f, -1.0f, 0.0f, 0.5f,
		0.0f, 0.0f, -1.0f, 0.5f;
	m_camera.col(3) -= m_scale * view_dir();
	m_smoothed_camera = m_camera;
	m_up_dir = {0.0f, 1.0f, 0.0f};
	m_sun_dir = Vector3f::Ones().normalized();
	reset_accumulation();
}

void Testbed::set_train(bool mtrain) {
	if (m_train && !mtrain && m_max_level_rand_training) {
		set_max_level(1.f);
	}
	m_train = mtrain;
}

std::string get_filename_in_data_path_with_suffix(fs::path data_path, fs::path network_config_path, const char* suffix) {
	// use the network config name along with the data path to build a filename with the requested suffix & extension
	std::string default_name = network_config_path.basename();
	if (default_name == "") default_name = "base";
	if (data_path.empty())
		return default_name + std::string(suffix);
	if (data_path.is_directory())
		return (data_path / (default_name + std::string{suffix})).str();
	else
		return data_path.stem().str() + "_" + default_name + std::string(suffix);
}

void Testbed::compute_and_save_marching_cubes_mesh(const char* filename, Vector3i res3d , BoundingBox aabb, float thresh, bool unwrap_it) {
	Matrix3f render_aabb_to_local = Matrix3f::Identity();
	if (aabb.is_empty()) {
		aabb = m_testbed_mode == ETestbedMode::Nerf ? m_render_aabb : m_aabb;
		render_aabb_to_local = m_render_aabb_to_local;
	}
	marching_cubes(res3d, aabb, render_aabb_to_local, thresh);
	save_mesh(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, filename, unwrap_it, m_nerf.training.dataset.scale, m_nerf.training.dataset.offset);
}

Eigen::Vector3i Testbed::compute_and_save_png_slices(const char* filename, int res, BoundingBox aabb, float thresh, float density_range, bool flip_y_and_z_axes) {
	Matrix3f render_aabb_to_local = Matrix3f::Identity();
	if (aabb.is_empty()) {
		aabb = m_testbed_mode == ETestbedMode::Nerf ? m_render_aabb : m_aabb;
		render_aabb_to_local = m_render_aabb_to_local;
	}
	if (thresh == std::numeric_limits<float>::max()) {
		thresh = m_mesh.thresh;
	}
	float range = density_range;
	if (m_testbed_mode == ETestbedMode::Sdf) {
		auto res3d = get_marching_cubes_res(res, aabb);
		aabb.inflate(range * aabb.diag().x()/res3d.x());
	}
	auto res3d = get_marching_cubes_res(res, aabb);
	if (m_testbed_mode == ETestbedMode::Sdf)
		range *= -aabb.diag().x()/res3d.x(); // rescale the range to be in output voxels. ie this scale factor is mapped back to the original world space distances.
			// negated so that black = outside, white = inside
	char fname[128];
	snprintf(fname, sizeof(fname), ".density_slices_%dx%dx%d.png", res3d.x(), res3d.y(), res3d.z());
	GPUMemory<float> density = (m_render_ground_truth && m_testbed_mode == ETestbedMode::Sdf) ? get_sdf_gt_on_grid(res3d, aabb, render_aabb_to_local) : get_density_on_grid(res3d, aabb, render_aabb_to_local);
	save_density_grid_to_png(density, (std::string(filename) + fname).c_str(), res3d, thresh, flip_y_and_z_axes, range);
	return res3d;
}

inline float linear_to_db(float x) {
	return -10.f*logf(x)/logf(10.f);
}

template <typename T>
void Testbed::dump_parameters_as_images(const T* params, const std::string& filename_base) {
	size_t non_layer_params_width = 2048;

	size_t layer_params = 0;
	for (auto size : m_network->layer_sizes()) {
		layer_params += size.first * size.second;
	}

	size_t n_params = m_network->n_params();
	size_t n_non_layer_params = n_params - layer_params;

	std::vector<T> params_cpu_network_precision(layer_params + next_multiple(n_non_layer_params, non_layer_params_width));
	std::vector<float> params_cpu(params_cpu_network_precision.size(), 0.0f);
	CUDA_CHECK_THROW(hipMemcpy(params_cpu_network_precision.data(), params, n_params * sizeof(T), hipMemcpyDeviceToHost));

	for (size_t i = 0; i < n_params; ++i) {
		params_cpu[i] = (float)params_cpu_network_precision[i];
	}

	size_t offset = 0;
	size_t layer_id = 0;
	for (auto size : m_network->layer_sizes()) {
		save_exr(params_cpu.data() + offset, size.second, size.first, 1, 1, fmt::format("{}-layer-{}.exr", filename_base, layer_id).c_str());
		offset += size.first * size.second;
		++layer_id;
	}

	if (n_non_layer_params > 0) {
		std::string filename = fmt::format("{}-non-layer.exr", filename_base);
		save_exr(params_cpu.data() + offset, non_layer_params_width, n_non_layer_params / non_layer_params_width, 1, 1, filename.c_str());
	}
}

template void Testbed::dump_parameters_as_images<__half>(const __half*, const std::string&);
template void Testbed::dump_parameters_as_images<float>(const float*, const std::string&);

Eigen::Matrix<float, 3, 4> Testbed::crop_box(bool nerf_space) const {
	Eigen::Vector3f cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
	Eigen::Vector3f radius = m_render_aabb.diag() * 0.5f;
	Eigen::Vector3f x = m_render_aabb_to_local.row(0) * radius.x();
	Eigen::Vector3f y = m_render_aabb_to_local.row(1) * radius.y();
	Eigen::Vector3f z = m_render_aabb_to_local.row(2) * radius.z();
	Eigen::Matrix<float, 3, 4> rv;
	rv.col(0) = x;
	rv.col(1) = y;
	rv.col(2) = z;
	rv.col(3) = cen;
	if (nerf_space) {
		rv = m_nerf.training.dataset.ngp_matrix_to_nerf(rv, true);
	}
	return rv;
}

void Testbed::set_crop_box(Eigen::Matrix<float, 3, 4> m, bool nerf_space) {
	if (nerf_space) {
		m = m_nerf.training.dataset.nerf_matrix_to_ngp(m, true);
	}
	Eigen::Vector3f radius(m.col(0).norm(), m.col(1).norm(), m.col(2).norm());
	Eigen::Vector3f cen(m.col(3));
	m_render_aabb_to_local.row(0) = m.col(0) / radius.x();
	m_render_aabb_to_local.row(1) = m.col(1) / radius.y();
	m_render_aabb_to_local.row(2) = m.col(2) / radius.z();
	cen = m_render_aabb_to_local * cen;
	m_render_aabb.min = cen - radius;
	m_render_aabb.max = cen + radius;
}

std::vector<Eigen::Vector3f> Testbed::crop_box_corners(bool nerf_space) const {
	Eigen::Matrix<float, 3, 4> m = crop_box(nerf_space);
	std::vector<Eigen::Vector3f> rv(8);
	for (int i = 0; i < 8; ++i) {
		rv[i] = m * Eigen::Vector4f((i & 1) ? 1.f : -1.f, (i & 2) ? 1.f : -1.f, (i & 4) ? 1.f : -1.f, 1.f);
		/* debug print out corners to check math is all lined up */
		if (0) {
			tlog::info() << rv[i].x() << "," << rv[i].y() << "," << rv[i].z() << " [" << i << "]";
			Eigen::Vector3f mn = m_render_aabb.min;
			Eigen::Vector3f mx = m_render_aabb.max;
			Eigen::Matrix3f m = m_render_aabb_to_local.transpose();
			Eigen::Vector3f a;

			a.x() = (i&1) ? mx.x() : mn.x();
			a.y() = (i&2) ? mx.y() : mn.y();
			a.z() = (i&4) ? mx.z() : mn.z();
			a = m * a;
			if (nerf_space) {
				a = m_nerf.training.dataset.ngp_position_to_nerf(a);
			}
			tlog::info() << a.x() << "," << a.y() << "," << a.z() << " [" << i << "]";
		}
	}
	return rv;
}

#ifdef NGP_GUI
bool imgui_colored_button(const char *name, float hue) {
	ImGui::PushStyleColor(ImGuiCol_Button, (ImVec4)ImColor::HSV(hue, 0.6f, 0.6f));
	ImGui::PushStyleColor(ImGuiCol_ButtonHovered, (ImVec4)ImColor::HSV(hue, 0.7f, 0.7f));
	ImGui::PushStyleColor(ImGuiCol_ButtonActive, (ImVec4)ImColor::HSV(hue, 0.8f, 0.8f));
	bool rv = ImGui::Button(name);
	ImGui::PopStyleColor(3);
	return rv;
}

void Testbed::imgui() {
	m_picture_in_picture_res = 0;
	if (int read = ImGui::Begin("Camera path", 0, ImGuiWindowFlags_NoScrollbar)) {
		static char path_filename_buf[128] = "";
		if (path_filename_buf[0] == '\0') {
			snprintf(path_filename_buf, sizeof(path_filename_buf), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_cam.json").c_str());
		}

		if (m_camera_path.imgui(path_filename_buf, m_render_ms.val(), m_camera, m_slice_plane_z, m_scale, fov(), m_aperture_size, m_bounding_radius, !m_nerf.training.dataset.xforms.empty() ? m_nerf.training.dataset.xforms[0].start : Matrix<float, 3, 4>::Identity(), m_nerf.glow_mode, m_nerf.glow_y_cutoff)) {
			if (m_camera_path.m_update_cam_from_path) {
				set_camera_from_time(m_camera_path.m_playtime);
				if (read > 1) {
					m_smoothed_camera = m_camera;
				}
			}

			m_pip_render_surface->reset_accumulation();
			reset_accumulation(true);
		}
		if (!m_camera_path.m_keyframes.empty()) {
			float w = ImGui::GetContentRegionAvail().x;
			m_picture_in_picture_res = (float)std::min((int(w)+31)&(~31),1920/4);
			if (m_camera_path.m_update_cam_from_path) {
				ImGui::Image((ImTextureID)(size_t)m_render_textures.front()->texture(), ImVec2(w,w*9.f/16.f));
			} else {
				ImGui::Image((ImTextureID)(size_t)m_pip_render_texture->texture(), ImVec2(w,w*9.f/16.f));
			}
		}
	}
	ImGui::End();


	ImGui::Begin("instant-ngp v" NGP_VERSION);

	size_t n_bytes = tcnn::total_n_bytes_allocated() + g_total_n_bytes_allocated + dlss_allocated_bytes();
	ImGui::Text("Frame: %.2f ms (%.1f FPS); Mem: %s", m_frame_ms.ema_val(), 1000.0f / m_frame_ms.ema_val(), bytes_to_string(n_bytes).c_str());
	bool accum_reset = false;

	if (!m_training_data_available) { ImGui::BeginDisabled(); }

	if (ImGui::CollapsingHeader("Training", m_training_data_available ? ImGuiTreeNodeFlags_DefaultOpen : 0)) {
		if (imgui_colored_button(m_train ? "Stop training" : "Start training", 0.4)) {
			set_train(!m_train);
		}
		ImGui::SameLine();
		ImGui::Checkbox("Train encoding", &m_train_encoding);
		ImGui::SameLine();
		ImGui::Checkbox("Train network", &m_train_network);
		ImGui::SameLine();
		ImGui::Checkbox("Random levels", &m_max_level_rand_training);
		if (m_testbed_mode == ETestbedMode::Nerf) {
			ImGui::Checkbox("Train envmap", &m_nerf.training.train_envmap);
			ImGui::SameLine();
			ImGui::Checkbox("Train extrinsics", &m_nerf.training.optimize_extrinsics);
			ImGui::SameLine();
			ImGui::Checkbox("Train exposure", &m_nerf.training.optimize_exposure);
			ImGui::SameLine();
			ImGui::Checkbox("Train distortion", &m_nerf.training.optimize_distortion);
			if (m_nerf.training.dataset.n_extra_learnable_dims) {
				ImGui::Checkbox("Train latent codes", &m_nerf.training.optimize_extra_dims);
			}
			static char opt_extr_filename_buf[1024] = "./trajectory.json";
			static bool export_extrinsics_in_quat_format = true;
			if (imgui_colored_button("Export extrinsics", 0.4f)) {
				m_nerf.training.export_camera_extrinsics(opt_extr_filename_buf, export_extrinsics_in_quat_format);
			}
			ImGui::SameLine();
			ImGui::PushItemWidth(400.f);
			ImGui::InputText("File", opt_extr_filename_buf, sizeof(opt_extr_filename_buf));
			ImGui::PopItemWidth();
			ImGui::SameLine();
			ImGui::Checkbox("Quaternion format", &export_extrinsics_in_quat_format);
		}
		if (imgui_colored_button("Reset training", 0.f)) {
			reload_network_from_file("");
		}
		ImGui::SameLine();
		ImGui::DragInt("Seed", (int*)&m_seed, 1.0f, 0, std::numeric_limits<int>::max());
		ImGui::SliderInt("Batch size", (int*)&m_training_batch_size, 1 << 12, 1 << 22, "%d", ImGuiSliderFlags_Logarithmic);
		m_training_batch_size = next_multiple(m_training_batch_size, batch_size_granularity);

		if (m_train) {
			std::vector<std::string> timings;
			if (m_testbed_mode == ETestbedMode::Nerf) {
				timings.emplace_back(fmt::format("Grid: {:.01f}ms", m_training_prep_ms.ema_val()));
			} else {
				timings.emplace_back(fmt::format("Datagen: {:.01f}ms", m_training_prep_ms.ema_val()));
			}

			timings.emplace_back(fmt::format("Training: {:.01f}ms", m_training_ms.ema_val()));
			ImGui::Text("%s", join(timings, ", ").c_str());
		} else {
			ImGui::Text("Training paused");
		}
		if (m_testbed_mode == ETestbedMode::Nerf) {
			ImGui::Text("Rays/batch: %d, Samples/ray: %.2f, Batch size: %d/%d", m_nerf.training.counters_rgb.rays_per_batch, (float)m_nerf.training.counters_rgb.measured_batch_size / (float)m_nerf.training.counters_rgb.rays_per_batch, m_nerf.training.counters_rgb.measured_batch_size, m_nerf.training.counters_rgb.measured_batch_size_before_compaction);
		}
		float elapsed_training = std::chrono::duration<float>(std::chrono::steady_clock::now() - m_training_start_time_point).count();
		ImGui::Text("Steps: %d, Loss: %0.6f (%0.2f dB), Elapsed: %.1fs", m_training_step, m_loss_scalar.ema_val(), linear_to_db(m_loss_scalar.ema_val()), elapsed_training);
		ImGui::PlotLines("loss graph", m_loss_graph.data(), std::min(m_loss_graph_samples, m_loss_graph.size()), (m_loss_graph_samples < m_loss_graph.size()) ? 0 : (m_loss_graph_samples % m_loss_graph.size()), 0, FLT_MAX, FLT_MAX, ImVec2(0, 50.f));

		if (m_testbed_mode == ETestbedMode::Nerf && ImGui::TreeNode("NeRF training options")) {
			ImGui::Checkbox("Random bg color", &m_nerf.training.random_bg_color);
			ImGui::SameLine();
			ImGui::Checkbox("Snap to pixel centers", &m_nerf.training.snap_to_pixel_centers);
			ImGui::SliderFloat("Near distance", &m_nerf.training.near_distance, 0.0f, 1.0f);
			accum_reset |= ImGui::Checkbox("Linear colors", &m_nerf.training.linear_colors);
			ImGui::Combo("Loss", (int*)&m_nerf.training.loss_type, LossTypeStr);
			ImGui::Combo("Depth Loss", (int*)&m_nerf.training.depth_loss_type, LossTypeStr);
			ImGui::Combo("RGB activation", (int*)&m_nerf.rgb_activation, NerfActivationStr);
			ImGui::Combo("Density activation", (int*)&m_nerf.density_activation, NerfActivationStr);
			ImGui::SliderFloat("Cone angle", &m_nerf.cone_angle_constant, 0.0f, 1.0f/128.0f);
			ImGui::SliderFloat("Depth supervision strength", &m_nerf.training.depth_supervision_lambda, 0.f, 1.f);

			// Importance sampling options, but still related to training
			ImGui::Checkbox("Sample focal plane ~error", &m_nerf.training.sample_focal_plane_proportional_to_error);
			ImGui::SameLine();
			ImGui::Checkbox("Sample focal plane ~sharpness", &m_nerf.training.include_sharpness_in_error);
			ImGui::Checkbox("Sample image ~error", &m_nerf.training.sample_image_proportional_to_error);
			ImGui::Text("%dx%d error res w/ %d steps between updates", m_nerf.training.error_map.resolution.x(), m_nerf.training.error_map.resolution.y(), m_nerf.training.n_steps_between_error_map_updates);
			ImGui::Checkbox("Display error overlay", &m_nerf.training.render_error_overlay);
			if (m_nerf.training.render_error_overlay) {
				ImGui::SliderFloat("Error overlay brightness", &m_nerf.training.error_overlay_brightness, 0.f, 1.f);
			}
			ImGui::SliderFloat("Density grid decay", &m_nerf.training.density_grid_decay, 0.f, 1.f,"%.4f");
			ImGui::SliderFloat("Extrinsic L2 reg.", &m_nerf.training.extrinsic_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::SliderFloat("Intrinsic L2 reg.", &m_nerf.training.intrinsic_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::SliderFloat("Exposure L2 reg.", &m_nerf.training.exposure_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Sdf && ImGui::TreeNode("SDF training options")) {
			accum_reset |= ImGui::Checkbox("Use octree for acceleration", &m_sdf.use_triangle_octree);
			accum_reset |= ImGui::Combo("Mesh SDF mode", (int*)&m_sdf.mesh_sdf_mode, MeshSdfModeStr);

			accum_reset |= ImGui::SliderFloat("Surface offset scale", &m_sdf.training.surface_offset_scale, 0.125f, 1024.0f, "%.4f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);

			if (ImGui::Checkbox("Calculate IoU", &m_sdf.calculate_iou_online)) {
				m_sdf.iou_decay = 0;
			}

			ImGui::SameLine();
			ImGui::Text("%0.6f", m_sdf.iou);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Image && ImGui::TreeNode("Image training options")) {
			ImGui::Combo("Training coords", (int*)&m_image.random_mode, RandomModeStr);
			ImGui::Checkbox("Snap to pixel centers", &m_image.training.snap_to_pixel_centers);
			accum_reset |= ImGui::Checkbox("Linear colors", &m_image.training.linear_colors);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Volume && ImGui::CollapsingHeader("Volume training options")) {
			accum_reset |= ImGui::SliderFloat("Albedo", &m_volume.albedo, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Scattering", &m_volume.scattering, -2.f, 2.f);
			accum_reset |= ImGui::SliderFloat("Distance scale", &m_volume.inv_distance_scale, 1.f, 100.f, "%.3g", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}
	}

	if (!m_training_data_available) { ImGui::EndDisabled(); }

	if (ImGui::CollapsingHeader("Rendering", ImGuiTreeNodeFlags_DefaultOpen)) {
		ImGui::Checkbox("Render", &m_render);
		ImGui::SameLine();

		const auto& render_tex = m_render_surfaces.front();
		std::string spp_string = m_dlss ? std::string{""} : fmt::format("({} spp)", std::max(render_tex.spp(), 1u));
		ImGui::Text(": %.01fms for %dx%d %s", m_render_ms.ema_val(), render_tex.in_resolution().x(), render_tex.in_resolution().y(), spp_string.c_str());

		if (m_dlss_supported) {
			if (!m_single_view) {
				ImGui::BeginDisabled();
				m_dlss = false;
			}

			if (ImGui::Checkbox("DLSS", &m_dlss)) {
				accum_reset = true;
			}

			if (render_tex.dlss()) {
				ImGui::SameLine();
				ImGui::Text("(automatic quality setting: %s)", DlssQualityStrArray[(int)render_tex.dlss()->quality()]);
				ImGui::SliderFloat("DLSS sharpening", &m_dlss_sharpening, 0.0f, 1.0f, "%.02f");
			}

			if (!m_single_view) {
				ImGui::EndDisabled();
			}
		}

		ImGui::Checkbox("Dynamic resolution", &m_dynamic_res);
		if (ImGui::Checkbox("VSync", &m_vsync)) {
			glfwSwapInterval(m_vsync ? 1 : 0);
		}
		ImGui::SliderFloat("Target FPS", &m_dynamic_res_target_fps, 2.0f, 144.0f, "%.01f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
		ImGui::SliderInt("Max spp", &m_max_spp, 0, 1024, "%d", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);

		if (!m_dynamic_res) {
			ImGui::SliderInt("Fixed resolution factor", &m_fixed_res_factor, 8, 64);
		}

		if (m_testbed_mode == ETestbedMode::Nerf && m_nerf.training.dataset.has_light_dirs) {
			Vector3f light_dir = m_nerf.light_dir.normalized();
			if (ImGui::TreeNodeEx("Light Dir (Polar)", ImGuiTreeNodeFlags_DefaultOpen)) {
				float phi = atan2f(m_nerf.light_dir.x(), m_nerf.light_dir.z());
				float theta = asinf(m_nerf.light_dir.y());
				bool spin = ImGui::SliderFloat("Light Dir Theta", &theta, -PI() / 2.0f, PI() / 2.0f);
				spin |= ImGui::SliderFloat("Light Dir Phi", &phi, -PI(), PI());
				if (spin) {
					float sin_phi, cos_phi;
					sincosf(phi, &sin_phi, &cos_phi);
					float cos_theta=cosf(theta);
					m_nerf.light_dir = {sin_phi * cos_theta,sinf(theta),cos_phi * cos_theta};
					accum_reset = true;
				}
				ImGui::TreePop();
			}
			if (ImGui::TreeNode("Light Dir (Cartesian)")) {
				accum_reset |= ImGui::SliderFloat("Light Dir X", ((float*)(&m_nerf.light_dir)) + 0, -1.0f, 1.0f);
				accum_reset |= ImGui::SliderFloat("Light Dir Y", ((float*)(&m_nerf.light_dir)) + 1, -1.0f, 1.0f);
				accum_reset |= ImGui::SliderFloat("Light Dir Z", ((float*)(&m_nerf.light_dir)) + 2, -1.0f, 1.0f);
				ImGui::TreePop();
			}
		}
		if (m_testbed_mode == ETestbedMode::Nerf && m_nerf.training.dataset.n_extra_learnable_dims) {
			accum_reset |= ImGui::SliderInt("training image latent code for inference", (int*)&m_nerf.extra_dim_idx_for_inference, 0, m_nerf.training.dataset.n_images-1);
		}
		accum_reset |= ImGui::Combo("Render mode", (int*)&m_render_mode, RenderModeStr);
		if (m_testbed_mode == ETestbedMode::Nerf)  {
			accum_reset |= ImGui::Combo("Groundtruth Render mode", (int*)&m_ground_truth_render_mode, GroundTruthRenderModeStr);
			accum_reset |= ImGui::SliderFloat("Groundtruth Alpha", &m_ground_truth_alpha, 0.0f, 1.0f, "%.02f", ImGuiSliderFlags_AlwaysClamp);
		}
		accum_reset |= ImGui::Combo("Color space", (int*)&m_color_space, ColorSpaceStr);
		accum_reset |= ImGui::Combo("Tonemap curve", (int*)&m_tonemap_curve, TonemapCurveStr);
		accum_reset |= ImGui::ColorEdit4("Background", &m_background_color[0]);
		if (ImGui::SliderFloat("Exposure", &m_exposure, -5.f, 5.f)) {
			set_exposure(m_exposure);
		}

		accum_reset |= ImGui::Checkbox("Snap to pixel centers", &m_snap_to_pixel_centers);

		float max_diam = (m_aabb.max-m_aabb.min).maxCoeff();
		float render_diam = (m_render_aabb.max-m_render_aabb.min).maxCoeff();
		float old_render_diam = render_diam;
		if (ImGui::SliderFloat("Crop size", &render_diam, 0.1f, max_diam, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat)) {
			accum_reset = true;
			if (old_render_diam > 0.f && render_diam > 0.f) {
				const Vector3f center = (m_render_aabb.max + m_render_aabb.min) * 0.5f;
				float scale = render_diam / old_render_diam;
				m_render_aabb.max = ((m_render_aabb.max-center) * scale + center).cwiseMin(m_aabb.max);
				m_render_aabb.min = ((m_render_aabb.min-center) * scale + center).cwiseMax(m_aabb.min);
			}
		}

		if (ImGui::TreeNode("Crop aabb")) {
			m_edit_render_aabb = true;
			accum_reset |= ImGui::SliderFloat("Min x", ((float*)&m_render_aabb.min)+0, m_aabb.min.x(), m_render_aabb.max.x(), "%.3f");
			accum_reset |= ImGui::SliderFloat("Min y", ((float*)&m_render_aabb.min)+1, m_aabb.min.y(), m_render_aabb.max.y(), "%.3f");
			accum_reset |= ImGui::SliderFloat("Min z", ((float*)&m_render_aabb.min)+2, m_aabb.min.z(), m_render_aabb.max.z(), "%.3f");
			ImGui::Separator();
			accum_reset |= ImGui::SliderFloat("Max x", ((float*)&m_render_aabb.max)+0, m_render_aabb.min.x(), m_aabb.max.x(), "%.3f");
			accum_reset |= ImGui::SliderFloat("Max y", ((float*)&m_render_aabb.max)+1, m_render_aabb.min.y(), m_aabb.max.y(), "%.3f");
			accum_reset |= ImGui::SliderFloat("Max z", ((float*)&m_render_aabb.max)+2, m_render_aabb.min.z(), m_aabb.max.z(), "%.3f");
			ImGui::Separator();
			Vector3f diag = m_render_aabb.diag();
			bool edit_diag = false;
			float max_diag = m_aabb.diag().maxCoeff();
			edit_diag |= ImGui::SliderFloat("Size x", ((float*)&diag)+0, 0.001f, max_diag, "%.3f");
			edit_diag |= ImGui::SliderFloat("Size y", ((float*)&diag)+1, 0.001f, max_diag, "%.3f");
			edit_diag |= ImGui::SliderFloat("Size z", ((float*)&diag)+2, 0.001f, max_diag, "%.3f");
			if (edit_diag) {
				accum_reset = true;
				Vector3f cen = m_render_aabb.center();
				m_render_aabb = BoundingBox(cen - diag * 0.5f, cen + diag * 0.5f);
			}
			if (ImGui::Button("Reset")) {
				accum_reset = true;
				m_render_aabb = m_aabb;
				m_render_aabb_to_local = Matrix3f::Identity();
			}
			ImGui::SameLine();
			if (ImGui::Button("Reset Rotation Only")) {
				accum_reset = true;
				Eigen::Vector3f world_cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
				m_render_aabb_to_local = Matrix3f::Identity();
				Eigen::Vector3f new_cen = m_render_aabb_to_local * world_cen;
				Eigen::Vector3f old_cen = m_render_aabb.center();
				m_render_aabb.min += new_cen - old_cen;
				m_render_aabb.max += new_cen - old_cen;
			}
			if (/*m_visualize_unit_cube*/ 1) {
				if (ImGui::RadioButton("Translate", m_camera_path.m_gizmo_op == ImGuizmo::TRANSLATE))
					m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
				ImGui::SameLine();
				if (ImGui::RadioButton("Rotate", m_camera_path.m_gizmo_op == ImGuizmo::ROTATE))
					m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
			}
			ImGui::TreePop();
		} else {
			m_edit_render_aabb = false;
		}

		if (m_testbed_mode == ETestbedMode::Nerf && ImGui::TreeNode("NeRF rendering options")) {
			accum_reset |= ImGui::Checkbox("Apply lens distortion", &m_nerf.render_with_lens_distortion);

			if (m_nerf.render_with_lens_distortion) {
				accum_reset |= ImGui::Combo("Lens mode", (int*)&m_nerf.render_lens.mode, "Perspective\0OpenCV\0F-Theta\0LatLong\0");
				if (m_nerf.render_lens.mode == ELensMode::OpenCV) {
					accum_reset |= ImGui::InputFloat("k1", &m_nerf.render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k2", &m_nerf.render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p1", &m_nerf.render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p2", &m_nerf.render_lens.params[3], 0.f, 0.f, "%.5f");
				} else if (m_nerf.render_lens.mode == ELensMode::FTheta) {
					accum_reset |= ImGui::InputFloat("width", &m_nerf.render_lens.params[5], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("height", &m_nerf.render_lens.params[6], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("f_theta p0", &m_nerf.render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p1", &m_nerf.render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p2", &m_nerf.render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p3", &m_nerf.render_lens.params[3], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p4", &m_nerf.render_lens.params[4], 0.f, 0.f, "%.5f");
				}
			}

			accum_reset |= ImGui::SliderFloat("Min transmittance", &m_nerf.render_min_transmittance, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Sdf && ImGui::TreeNode("SDF rendering options")) {
			accum_reset |= ImGui::Combo("Ground Truth Rendering Mode", (int*)&m_sdf.groundtruth_mode,
				"Raytraced Mesh\0"
				"Sphere Traced Mesh\0"
				"SDF Bricks\0"
			);
			if (m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
				accum_reset |= ImGui::SliderInt("Brick Octree Level", (int*)&m_sdf.brick_level, 1, 10);
				accum_reset |= ImGui::Checkbox("Brick Normals track Octree Level", &m_sdf.brick_smooth_normals);
				accum_reset |= ImGui::SliderInt("Brick Quantize Bits", (int*)&m_sdf.brick_quantise_bits, 0, 16);
			}
			accum_reset |= ImGui::Checkbox("Analytic normals", &m_sdf.analytic_normals);

			accum_reset |= ImGui::SliderFloat("Normals epsilon", &m_sdf.fd_normals_epsilon, 0.00001f, 0.1f, "%.6g", ImGuiSliderFlags_Logarithmic);
			accum_reset |= ImGui::SliderFloat("Maximum distance", &m_sdf.maximum_distance, 0.00001f, 0.1f, "%.6g", ImGuiSliderFlags_Logarithmic);
			accum_reset |= ImGui::SliderFloat("Shadow sharpness", &m_sdf.shadow_sharpness, 0.1f, 2048.0f, "%.6g", ImGuiSliderFlags_Logarithmic);

			accum_reset |= ImGui::SliderFloat("Inflate (offset the zero set)", &m_sdf.zero_offset, -0.25f, 0.25f);
			accum_reset |= ImGui::SliderFloat("Distance scale", &m_sdf.distance_scale, 0.25f, 1.f);

			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Image && ImGui::TreeNode("Image rendering options")) {
			static bool quantize_to_byte = false;
			static float mse = 0.0f;

			if (imgui_colored_button("Compute PSNR", 0.4)) {
				mse = compute_image_mse(quantize_to_byte);
			}

			float psnr = -10.0f * std::log(mse) / std::log(10.0f);

			ImGui::SameLine();
			ImGui::Text("%0.6f", psnr);
			ImGui::SameLine();
			ImGui::Checkbox("Quantize", &quantize_to_byte);

			ImGui::TreePop();
		}

		if (ImGui::TreeNode("Debug visualization")) {
			ImGui::Checkbox("Visualize unit cube", &m_visualize_unit_cube);
			if (m_testbed_mode == ETestbedMode::Nerf) {
				ImGui::SameLine();
				ImGui::Checkbox("Visualize cameras", &m_nerf.visualize_cameras);
				accum_reset |= ImGui::SliderInt("Show acceleration", &m_nerf.show_accel, -1, 7);
			}

			if (!m_single_view) { ImGui::BeginDisabled(); }
			if (ImGui::SliderInt("Visualized dimension", &m_visualized_dimension, -1, (int)network_width(m_visualized_layer)-1)) {
				set_visualized_dim(m_visualized_dimension);
			}
			if (!m_single_view) { ImGui::EndDisabled(); }

			if (ImGui::SliderInt("Visualized layer", &m_visualized_layer, 0, (int)network_num_forward_activations()-1)) {
				set_visualized_layer(m_visualized_layer);
			}
			if (ImGui::Checkbox("Single view", &m_single_view)) {
				set_visualized_dim(-1);
				accum_reset = true;
			}

			if (m_testbed_mode == ETestbedMode::Nerf) {
				if (ImGui::Button("First")) {
					first_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Previous")) {
					previous_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Next")) {
					next_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Last")) {
					last_training_view();
				}
				ImGui::SameLine();
				ImGui::Text("%s", m_nerf.training.dataset.paths.at(m_nerf.training.view).c_str());

				if (ImGui::SliderInt("Training view", &m_nerf.training.view, 0, (int)m_nerf.training.dataset.n_images-1)) {
					set_camera_to_training_view(m_nerf.training.view);
					accum_reset = true;
				}
				ImGui::PlotLines("Training view error", m_nerf.training.error_map.pmf_img_cpu.data(), m_nerf.training.error_map.pmf_img_cpu.size(), 0, nullptr, 0.0f, FLT_MAX, ImVec2(0, 60.f));

				if (m_nerf.training.optimize_exposure) {
					std::vector<float> exposures(m_nerf.training.dataset.n_images);
					for (uint32_t i = 0; i < m_nerf.training.dataset.n_images; ++i) {
						exposures[i] = m_nerf.training.cam_exposure[i].variable().x();
					}

					ImGui::PlotLines("Training view exposures", exposures.data(), exposures.size(), 0, nullptr, FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
				}

				if (ImGui::SliderInt("glow mode", &m_nerf.glow_mode, 0, 16)) {
					accum_reset = true;
				}

				if (m_nerf.glow_mode && ImGui::SliderFloat("glow pos", &m_nerf.glow_y_cutoff, -2.f, 3.f)) {
					accum_reset = true;
				}
			}

			ImGui::TreePop();
		}
	}

	if (ImGui::CollapsingHeader("Camera", ImGuiTreeNodeFlags_DefaultOpen)) {
		if (ImGui::SliderFloat("Aperture size", &m_aperture_size, 0.0f, 0.1f)) {
			m_dlss = false;
			accum_reset = true;
		}
		float local_fov = fov();
		if (ImGui::SliderFloat("Field of view", &local_fov, 0.0f, 120.0f)) {
			set_fov(local_fov);
			accum_reset = true;
		}
		accum_reset |= ImGui::SliderFloat("Zoom", &m_zoom, 1.f, 10.f);
		if (m_testbed_mode == ETestbedMode::Sdf) {
			accum_reset |= ImGui::Checkbox("Floor", &m_floor_enable);
			ImGui::SameLine();
		}

		ImGui::Checkbox("First person controls", &m_fps_camera);
		ImGui::SameLine();
		ImGui::Checkbox("Smooth camera motion", &m_camera_smoothing);
		ImGui::SameLine();
		ImGui::Checkbox("Autofocus", &m_autofocus);

		if (ImGui::TreeNode("Advanced camera settings")) {
			accum_reset |= ImGui::SliderFloat2("Screen center", &m_screen_center.x(), 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat2("Parallax shift", &m_parallax_shift.x(), -1.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Slice / focus depth", &m_slice_plane_z, -m_bounding_radius, m_bounding_radius);
			accum_reset |= ImGui::SliderFloat("Render near distance", &m_render_near_distance, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			char buf[2048];
			Vector3f v = view_dir();
			Vector3f p = look_at();
			Vector3f s = m_sun_dir;
			Vector3f u = m_up_dir;
			Array4f b = m_background_color;
			snprintf(buf, sizeof(buf),
				"testbed.background_color = [%0.3f, %0.3f, %0.3f, %0.3f]\n"
				"testbed.exposure = %0.3f\n"
				"testbed.sun_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.up_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.view_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.look_at = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.scale = %0.3f\n"
				"testbed.fov,testbed.aperture_size,testbed.slice_plane_z = %0.3f,%0.3f,%0.3f\n"
				"testbed.autofocus_target = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.autofocus = %s\n\n"
				, b.x(), b.y(), b.z(), b.w()
				, m_exposure
				, s.x(), s.y(), s.z()
				, u.x(), u.y(), u.z()
				, v.x(), v.y(), v.z()
				, p.x(), p.y(), p.z()
				, scale()
				, fov(), m_aperture_size, m_slice_plane_z
				, m_autofocus_target.x(), m_autofocus_target.y(), m_autofocus_target.z()
				, m_autofocus ? "True" : "False"
			);

			if (m_testbed_mode == ETestbedMode::Sdf) {
				size_t n = strlen(buf);
				snprintf(buf+n, sizeof(buf)-n,
					"testbed.sdf.shadow_sharpness = %0.3f\n"
					"testbed.sdf.analytic_normals = %s\n"
					"testbed.sdf.use_triangle_octree = %s\n\n"
					"testbed.sdf.brdf.metallic = %0.3f\n"
					"testbed.sdf.brdf.subsurface = %0.3f\n"
					"testbed.sdf.brdf.specular = %0.3f\n"
					"testbed.sdf.brdf.roughness = %0.3f\n"
					"testbed.sdf.brdf.sheen = %0.3f\n"
					"testbed.sdf.brdf.clearcoat = %0.3f\n"
					"testbed.sdf.brdf.clearcoat_gloss = %0.3f\n"
					"testbed.sdf.brdf.basecolor = [%0.3f,%0.3f,%0.3f]\n\n"
					, m_sdf.shadow_sharpness
					, m_sdf.analytic_normals ? "True" : "False"
					, m_sdf.use_triangle_octree ? "True" : "False"
					, m_sdf.brdf.metallic
					, m_sdf.brdf.subsurface
					, m_sdf.brdf.specular
					, m_sdf.brdf.roughness
					, m_sdf.brdf.sheen
					, m_sdf.brdf.clearcoat
					, m_sdf.brdf.clearcoat_gloss
					, m_sdf.brdf.basecolor.x()
					, m_sdf.brdf.basecolor.y()
					, m_sdf.brdf.basecolor.z()
				);
			}
			ImGui::InputTextMultiline("Params", buf, sizeof(buf));
			ImGui::TreePop();
		}

	}
	if (ImGui::CollapsingHeader("Snapshot")) {
		static char snapshot_filename_buf[128] = "";
		if (snapshot_filename_buf[0] == '\0') {
			snprintf(snapshot_filename_buf, sizeof(snapshot_filename_buf), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".msgpack").c_str());
		}

		ImGui::Text("Snapshot");
		ImGui::SameLine();
		if (ImGui::Button("Save")) {
			save_snapshot(snapshot_filename_buf, m_include_optimizer_state_in_snapshot);
		}
		ImGui::SameLine();
		static std::string snapshot_load_error_string = "";
		if (ImGui::Button("Load")) {
			try {
				load_snapshot(snapshot_filename_buf);
			} catch (std::exception& e) {
				ImGui::OpenPopup("Snapshot load error");
				snapshot_load_error_string = std::string{"Failed to load snapshot: "} + e.what();
			}
		}
		ImGui::SameLine();
		if (ImGui::Button("Dump parameters as images")) {
			dump_parameters_as_images(m_trainer->params(), "params");
		}
		if (ImGui::BeginPopupModal("Snapshot load error", NULL, ImGuiWindowFlags_AlwaysAutoResize)) {
			ImGui::Text("%s", snapshot_load_error_string.c_str());
			if (ImGui::Button("OK", ImVec2(120, 0))) {
				ImGui::CloseCurrentPopup();
			}
			ImGui::EndPopup();
		}
		ImGui::SameLine();
		ImGui::Checkbox("w/ Optimizer State", &m_include_optimizer_state_in_snapshot);
		ImGui::InputText("File", snapshot_filename_buf, sizeof(snapshot_filename_buf));
	}

	if (m_testbed_mode == ETestbedMode::Nerf || m_testbed_mode == ETestbedMode::Sdf) {
		if (ImGui::CollapsingHeader("Marching Cubes Mesh Output")) {
			static bool flip_y_and_z_axes = false;
			static float density_range = 4.f;
			BoundingBox aabb = (m_testbed_mode == ETestbedMode::Nerf) ? m_render_aabb : m_aabb;

			auto res3d = get_marching_cubes_res(m_mesh.res, aabb);

			// If we use an octree to fit the SDF only close to the surface, then marching cubes will not work (SDF not defined everywhere)
			bool disable_marching_cubes = m_testbed_mode == ETestbedMode::Sdf && (m_sdf.uses_takikawa_encoding || m_sdf.use_triangle_octree);
			if (disable_marching_cubes) { ImGui::BeginDisabled(); }

			if (imgui_colored_button("Mesh it!", 0.4f)) {
				marching_cubes(res3d, aabb, m_render_aabb_to_local, m_mesh.thresh);
				m_nerf.render_with_lens_distortion = false;
			}
			if (m_mesh.indices.size()>0) {
				ImGui::SameLine();
				if (imgui_colored_button("Clear Mesh", 0.f)) {
					m_mesh.clear();
				}
			}

			if (disable_marching_cubes) { ImGui::EndDisabled(); }

			ImGui::SameLine();

			if (imgui_colored_button("Save density PNG",-0.4f)) {
				Testbed::compute_and_save_png_slices(m_data_path.str().c_str(), m_mesh.res, {}, m_mesh.thresh, density_range, flip_y_and_z_axes);
			}

			if (m_testbed_mode == ETestbedMode::Nerf) {
				ImGui::SameLine();
				if (imgui_colored_button("Save RGBA PNG sequence", 0.2f)) {
					auto effective_view_dir = flip_y_and_z_axes ? Vector3f{0.0f, 1.0f, 0.0f} : Vector3f{0.0f, 0.0f, 1.0f};
					// Depth of 0.01f is arbitrarily chosen to produce a visually interpretable range of alpha values.
					// Alternatively, if the true transparency of a given voxel is desired, one could use the voxel size,
					// the voxel diagonal, or some form of expected ray length through the voxel, given random directions.
					GPUMemory<Array4f> rgba = get_rgba_on_grid(res3d, effective_view_dir, true, 0.01f);
					auto dir = m_data_path / "rgba_slices";
					if (!dir.exists()) {
						fs::create_directory(dir);
					}
					save_rgba_grid_to_png_sequence(rgba, dir.str().c_str(), res3d, flip_y_and_z_axes);
				}
				if (imgui_colored_button("Save raw volumes", 0.4f)) {
					auto effective_view_dir = flip_y_and_z_axes ? Vector3f{0.0f, 1.0f, 0.0f} : Vector3f{0.0f, 0.0f, 1.0f};
					auto old_local = m_render_aabb_to_local;
					auto old_aabb = m_render_aabb;
					m_render_aabb_to_local = Eigen::Matrix3f::Identity();
					auto dir = m_data_path / "volume_raw";
					if (!dir.exists()) {
						fs::create_directory(dir);
					}
					for (int cascade = 0; (1<<cascade)<= m_aabb.diag().x()+0.5f; ++cascade) {
						float radius = (1<<cascade) * 0.5f;
						m_render_aabb = BoundingBox(Eigen::Vector3f::Constant(0.5f-radius), Eigen::Vector3f::Constant(0.5f+radius));
						// Dump raw density values that the user can then convert to alpha as they please.
						GPUMemory<Array4f> rgba = get_rgba_on_grid(res3d, effective_view_dir, true, 0.0f, true);
						save_rgba_grid_to_raw_file(rgba, dir.str().c_str(), res3d, flip_y_and_z_axes, cascade);
					}
					m_render_aabb_to_local = old_local;
					m_render_aabb = old_aabb;
				}
			}

			ImGui::SameLine();
			ImGui::Checkbox("Swap Y&Z", &flip_y_and_z_axes);
			ImGui::SliderFloat("PNG Density Range", &density_range, 0.001f, 8.f);

			static char obj_filename_buf[128] = "";
			ImGui::SliderInt("Res:", &m_mesh.res, 16, 2048, "%d", ImGuiSliderFlags_Logarithmic);
			ImGui::SameLine();

			ImGui::Text("%dx%dx%d", res3d.x(), res3d.y(), res3d.z());
			if (obj_filename_buf[0] == '\0') {
				snprintf(obj_filename_buf, sizeof(obj_filename_buf), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".obj").c_str());
			}
			float thresh_range = (m_testbed_mode == ETestbedMode::Sdf) ? 0.5f : 10.f;
			ImGui::SliderFloat("MC density threshold",&m_mesh.thresh, -thresh_range, thresh_range);
			ImGui::Combo("Mesh render mode", (int*)&m_mesh_render_mode, "Off\0Vertex Colors\0Vertex Normals\0Face IDs\0");
			ImGui::Checkbox("Unwrap mesh", &m_mesh.unwrap);
			if (uint32_t tricount = m_mesh.indices.size()/3) {
				ImGui::InputText("##OBJFile", obj_filename_buf, sizeof(obj_filename_buf));
				if (ImGui::Button("Save it!")) {
					save_mesh(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, obj_filename_buf, m_mesh.unwrap, m_nerf.training.dataset.scale, m_nerf.training.dataset.offset);
				}
				ImGui::SameLine();
				ImGui::Text("Mesh has %d triangles\n", tricount);
				ImGui::Checkbox("Optimize mesh", &m_mesh.optimize_mesh);
				ImGui::SliderFloat("Laplacian smoothing", &m_mesh.smooth_amount, 0.f, 2048.f);
				ImGui::SliderFloat("Density push", &m_mesh.density_amount, 0.f, 128.f);
				ImGui::SliderFloat("Inflate", &m_mesh.inflate_amount, 0.f, 128.f);
			}
		}
	}

	if (m_testbed_mode == ETestbedMode::Sdf) {
		if (ImGui::CollapsingHeader("BRDF parameters")) {
			accum_reset |= ImGui::ColorEdit3("Base color", (float*)&m_sdf.brdf.basecolor );
			accum_reset |= ImGui::SliderFloat("Roughness", &m_sdf.brdf.roughness, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Specular", &m_sdf.brdf.specular, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Metallic", &m_sdf.brdf.metallic, 0.f, 1.f);
			ImGui::Separator();
			accum_reset |= ImGui::SliderFloat("Subsurface", &m_sdf.brdf.subsurface, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Sheen", &m_sdf.brdf.sheen, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Clearcoat", &m_sdf.brdf.clearcoat, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Clearcoat gloss", &m_sdf.brdf.clearcoat_gloss, 0.f, 1.f);
		}
		m_sdf.brdf.ambientcolor = (m_background_color * m_background_color).head<3>();
	}

	if (ImGui::CollapsingHeader("Histograms of trainable encoding parameters")) {
		ImGui::Checkbox("Gather histograms", &m_gather_histograms);

		static float minlevel = 0.f;
		static float maxlevel = 1.f;
		if (ImGui::SliderFloat("Max level", &maxlevel, 0.f, 1.f))
			set_max_level(maxlevel);
		if (ImGui::SliderFloat("##Min level", &minlevel, 0.f, 1.f))
			set_min_level(minlevel);
		ImGui::SameLine();
		ImGui::Text("%0.1f%% values snapped to 0", m_quant_percent);

		std::vector<float> f(m_num_levels);


		// Hashgrid statistics
		for (int i = 0; i < m_num_levels; ++i) {
			f[i] = m_level_stats[i].mean();
		}
		ImGui::PlotHistogram("Grid means", f.data(), m_num_levels, 0, "means", FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
		for (int i = 0; i < m_num_levels; ++i) {
			f[i] = m_level_stats[i].sigma();
		}
		ImGui::PlotHistogram("Grid sigmas", f.data(), m_num_levels, 0, "sigma", FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
		ImGui::Separator();


		// Histogram of trained hashgrid params
		ImGui::SliderInt("Show details for level", &m_histo_level, 0, m_num_levels - 1);
		if (m_histo_level < m_num_levels) {
			LevelStats& s = m_level_stats[m_histo_level];
			static bool excludezero = false;
			if (excludezero)
				m_histo[128] = 0.f;
			ImGui::PlotHistogram("Values histogram", m_histo, 257, 0, "", FLT_MAX, FLT_MAX, ImVec2(0, 120.f));
			ImGui::SliderFloat("Histogram horizontal scale", &m_histo_scale, 0.01f, 2.f);
			ImGui::Checkbox("Exclude 'zero' from histogram", &excludezero);
			ImGui::Text("Range: %0.5f - %0.5f", s.min, s.max);
			ImGui::Text("Mean: %0.5f Sigma: %0.5f", s.mean(), s.sigma());
			ImGui::Text("Num Zero: %d (%0.1f%%)", s.numzero, s.fraczero() * 100.f);
		}
	}

	if (accum_reset) {
		reset_accumulation();
	}

	if (ImGui::Button("Go to python REPL")) {
		m_want_repl = true;
	}

	ImGui::End();
}

void Testbed::visualize_nerf_cameras(ImDrawList* list, const Matrix<float, 4, 4>& world2proj) {
	for (int i = 0; i < m_nerf.training.n_images_for_training; ++i) {
		auto res = m_nerf.training.dataset.metadata[i].resolution;
		float aspect = float(res.x())/float(res.y());
		auto current_xform = get_xform_given_rolling_shutter(m_nerf.training.transforms[i], m_nerf.training.dataset.metadata[i].rolling_shutter, Vector2f{0.5f, 0.5f}, 0.0f);
		visualize_nerf_camera(list, world2proj, m_nerf.training.dataset.xforms[i].start, aspect, 0x40ffff40);
		visualize_nerf_camera(list, world2proj, m_nerf.training.dataset.xforms[i].end, aspect, 0x40ffff40);
		visualize_nerf_camera(list, world2proj, current_xform, aspect, 0x80ffffff);

		// Visualize near distance
		add_debug_line(list, world2proj, current_xform.col(3), current_xform.col(3) + current_xform.col(2) * m_nerf.training.near_distance, 0x20ffffff);
	}

}

void Testbed::draw_visualizations(ImDrawList* list, const Matrix<float, 3, 4>& camera_matrix) {
	// Visualize 3D cameras for SDF or NeRF use cases
	if (m_testbed_mode != ETestbedMode::Image) {
		Matrix<float, 4, 4> world2view, view2world, view2proj, world2proj;
		view2world.setIdentity();
		view2world.block<3,4>(0,0) = camera_matrix;

		auto focal = calc_focal_length(Vector2i::Ones(), m_fov_axis, m_zoom);
		float zscale = 1.0f / focal[m_fov_axis];

		float xyscale = (float)m_window_res[m_fov_axis];
		Vector2f screen_center = render_screen_center();
		view2proj <<
			xyscale, 0,       (float)m_window_res.x()*screen_center.x()*zscale, 0,
			0,       xyscale, (float)m_window_res.y()*screen_center.y()*zscale, 0,
			0,       0,       1,                                                0,
			0,       0,       zscale,                                           0;

		world2view = view2world.inverse();
		world2proj = view2proj * world2view;
		float aspect = (float)m_window_res.x() / (float)m_window_res.y();

		// Visualize NeRF training poses
		if (m_testbed_mode == ETestbedMode::Nerf) {
			if (m_nerf.visualize_cameras) {
				visualize_nerf_cameras(list, world2proj);
			}
		}

		if (m_visualize_unit_cube) {
			visualize_unit_cube(list, world2proj, Eigen::Vector3f::Constant(0.f), Eigen::Vector3f::Constant(1.f), Eigen::Matrix3f::Identity());
		}
		if (m_edit_render_aabb) {
			if (m_testbed_mode == ETestbedMode::Nerf) {
				visualize_unit_cube(list, world2proj, m_render_aabb.min, m_render_aabb.max, m_render_aabb_to_local);
				ImGuiIO& io = ImGui::GetIO();
				float flx = focal.x();
				float fly = focal.y();
				Matrix<float, 4, 4> view2proj_guizmo;
				float zfar = 100.f;
				float znear = 0.1f;
				view2proj_guizmo <<
					fly*2.f/aspect, 0, 0, 0,
					0, -fly*2.f, 0, 0,
					0, 0, (zfar+znear)/(zfar-znear), -(2.f*zfar*znear) / (zfar-znear),
					0, 0, 1, 0;
				ImGuizmo::SetRect(0, 0, io.DisplaySize.x, io.DisplaySize.y);

				Eigen::Matrix4f matrix=Eigen::Matrix4f::Identity();
				matrix.block<3,3>(0,0) = m_render_aabb_to_local.transpose();
				Eigen::Vector3f cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
				matrix.block<3,4>(0,0).col(3) = cen;
				if (ImGuizmo::Manipulate((const float*)&world2view, (const float*)&view2proj_guizmo, m_camera_path.m_gizmo_op, ImGuizmo::LOCAL, (float*)&matrix, NULL, NULL)) {
					m_render_aabb_to_local = matrix.block<3,3>(0,0).transpose();
					Eigen::Vector3f new_cen = m_render_aabb_to_local * matrix.block<3,4>(0,0).col(3);
					Eigen::Vector3f old_cen = m_render_aabb.center();
					m_render_aabb.min += new_cen - old_cen;
					m_render_aabb.max += new_cen - old_cen;
					reset_accumulation();
				}
			}
		}

		if (m_camera_path.imgui_viz(list, view2proj, world2proj, world2view, focal, aspect)) {
			m_pip_render_surface->reset_accumulation();
		}
	}
}

void glfw_error_callback(int error, const char* description) {
	tlog::error() << "GLFW error #" << error << ": " << description;
}

bool Testbed::keyboard_event() {
	if (ImGui::GetIO().WantCaptureKeyboard) {
		return false;
	}

	if (m_keyboard_event_callback && m_keyboard_event_callback()) {
		return false;
	}

	for (int idx = 0; idx < std::min((int)ERenderMode::NumRenderModes, 10); ++idx) {
		char c[] = { "1234567890" };
		if (ImGui::IsKeyPressed(c[idx])) {
			m_render_mode = (ERenderMode)idx;
			reset_accumulation();
		}
	}

	bool shift = ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Shift;

	if (ImGui::IsKeyPressed('Z')) {
		m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
	}

	if (ImGui::IsKeyPressed('X')) {
		m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
	}

	if (ImGui::IsKeyPressed('E')) {
		set_exposure(m_exposure + (shift ? -0.5f : 0.5f));
		redraw_next_frame();
	}
	if (ImGui::IsKeyPressed('R')) {
		if (shift) {
			reset_camera();
		} else {
			reload_network_from_file("");
		}
	}
	if (ImGui::IsKeyPressed('O')) {
		m_nerf.training.render_error_overlay = !m_nerf.training.render_error_overlay;
	}
	if (ImGui::IsKeyPressed('G')) {
		m_render_ground_truth = !m_render_ground_truth;
		reset_accumulation();
		if (m_render_ground_truth) {
			m_nerf.training.view = find_best_training_view(m_nerf.training.view);
		}
	}
	if (ImGui::IsKeyPressed('.')) {
		if (m_single_view) {
			if (m_visualized_dimension == m_network->width(m_visualized_layer)-1 && m_visualized_layer < m_network->num_forward_activations()-1) {
				set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer+1)));
				set_visualized_dim(0);
			} else {
				set_visualized_dim(std::max(-1, std::min((int)m_network->width(m_visualized_layer)-1, m_visualized_dimension+1)));
			}
		} else {
			set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer+1)));
		}
	}
	if (ImGui::IsKeyPressed(',')) {
		if (m_single_view) {
			if (m_visualized_dimension == 0 && m_visualized_layer > 0) {
				set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer-1)));
				set_visualized_dim(m_network->width(m_visualized_layer)-1);
			} else {
				set_visualized_dim(std::max(-1, std::min((int)m_network->width(m_visualized_layer)-1, m_visualized_dimension-1)));
			}
		} else {
			set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer-1)));
		}
	}
	if (ImGui::IsKeyPressed('M')) {
		m_single_view = !m_single_view;
		set_visualized_dim(-1);
		reset_accumulation();
	}
	if (ImGui::IsKeyPressed('T')) {
		set_train(!m_train);
	}
	if (ImGui::IsKeyPressed('N')) {
		m_sdf.analytic_normals = !m_sdf.analytic_normals;
		reset_accumulation();
	}

	if (ImGui::IsKeyPressed('[')) {
		if (shift) {
			first_training_view();
		} else {
			previous_training_view();
		}
	}

	if (ImGui::IsKeyPressed(']')) {
		if (shift) {
			last_training_view();
		} else {
			next_training_view();
		}
	}

	if (ImGui::IsKeyPressed('=') || ImGui::IsKeyPressed('+')) {
		m_camera_velocity *= 1.5f;
	}

	if (ImGui::IsKeyPressed('-') || ImGui::IsKeyPressed('_')) {
		m_camera_velocity /= 1.5f;
	}

	// WASD camera movement
	Vector3f translate_vec = Vector3f::Zero();
	if (ImGui::IsKeyDown('W')) {
		translate_vec.z() += 1.0f;
	}
	if (ImGui::IsKeyDown('A')) {
		translate_vec.x() += -1.0f;
	}
	if (ImGui::IsKeyDown('S')) {
		translate_vec.z() += -1.0f;
	}
	if (ImGui::IsKeyDown('D')) {
		translate_vec.x() += 1.0f;
	}
	if (ImGui::IsKeyDown(' ')) {
		translate_vec.y() += -1.0f;
	}
	if (ImGui::IsKeyDown('C')) {
		translate_vec.y() += 1.0f;
	}
	translate_vec *= m_camera_velocity * m_frame_ms.val() / 1000.0f;
	if (shift) {
		translate_vec *= 5;
	}
	if (translate_vec != Vector3f::Zero()) {
		m_fps_camera = true;
		translate_camera(translate_vec);
	}
	return false;
}

void Testbed::mouse_wheel(Vector2f m, float delta) {
	if (delta == 0) {
		return;
	}

	if (!ImGui::GetIO().WantCaptureMouse) {
		float scale_factor = pow(1.1f, -delta);
		m_image.pos = (m_image.pos - m) / scale_factor + m;
		set_scale(m_scale * scale_factor);
	}

	reset_accumulation(true);
}

void Testbed::mouse_drag(const Vector2f& rel, int button) {
	Vector3f up = m_up_dir;
	Vector3f side = m_camera.col(0);

	bool is_left_held = (button & 1) != 0;
	bool is_right_held = (button & 2) != 0;

	bool shift = ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Shift;
	if (is_left_held) {
		if (shift) {
			auto mouse = ImGui::GetMousePos();
			determine_autofocus_target_from_pixel({mouse.x, mouse.y});
			reset_accumulation();
		} else {
			float rot_sensitivity = m_fps_camera ? 0.35f : 1.0f;
			Matrix3f rot =
				(AngleAxisf(static_cast<float>(-rel.x() * 2 * PI() * rot_sensitivity), up) * // Scroll sideways around up vector
				AngleAxisf(static_cast<float>(-rel.y() * 2 * PI() * rot_sensitivity), side)).matrix(); // Scroll around side vector

			m_image.pos += rel;
			if (m_fps_camera) {
				m_camera.block<3,3>(0,0) = rot * m_camera.block<3,3>(0,0);
			} else {
				// Turntable
				auto old_look_at = look_at();
				set_look_at({0.0f, 0.0f, 0.0f});
				m_camera = rot * m_camera;
				set_look_at(old_look_at);
			}

			reset_accumulation(true);
		}
	}

	if (is_right_held) {
		Matrix3f rot =
			(AngleAxisf(static_cast<float>(-rel.x() * 2 * PI()), up) * // Scroll sideways around up vector
			AngleAxisf(static_cast<float>(-rel.y() * 2 * PI()), side)).matrix(); // Scroll around side vector

		if (m_render_mode == ERenderMode::Shade) {
			m_sun_dir = rot.transpose() * m_sun_dir;
		}

		m_slice_plane_z += -rel.y() * m_bounding_radius;
		reset_accumulation();
	}

	bool is_middle_held = (button & 4) != 0;
	if (is_middle_held) {
		translate_camera({-rel.x(), -rel.y(), 0.0f});
	}
}

bool Testbed::begin_frame_and_handle_user_input() {
	if (glfwWindowShouldClose(m_glfw_window) || ImGui::IsKeyDown(GLFW_KEY_ESCAPE) || ImGui::IsKeyDown(GLFW_KEY_Q)) {
		destroy_window();
		return false;
	}

	{
		auto now = std::chrono::steady_clock::now();
		auto elapsed = now - m_last_frame_time_point;
		m_last_frame_time_point = now;
		m_frame_ms.update(std::chrono::duration<float, std::milli>(elapsed).count());
	}

	glfwPollEvents();
	glfwGetFramebufferSize(m_glfw_window, &m_window_res.x(), &m_window_res.y());

	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
	ImGuizmo::BeginFrame();

	if (ImGui::IsKeyPressed(GLFW_KEY_TAB) || ImGui::IsKeyPressed(GLFW_KEY_GRAVE_ACCENT)) {
		m_imgui_enabled = !m_imgui_enabled;
	}

	ImVec2 m = ImGui::GetMousePos();
	int mb = 0;
	float mw = 0.f;
	ImVec2 relm = {};
	if (!ImGui::IsAnyItemActive() && !ImGuizmo::IsUsing() && !ImGuizmo::IsOver()) {
		relm = ImGui::GetIO().MouseDelta;
		if (ImGui::GetIO().MouseDown[0]) mb |= 1;
		if (ImGui::GetIO().MouseDown[1]) mb |= 2;
		if (ImGui::GetIO().MouseDown[2]) mb |= 4;
		mw = ImGui::GetIO().MouseWheel;
		relm = {relm.x / (float)m_window_res.y(), relm.y / (float)m_window_res.y()};
	}

	if (m_testbed_mode == ETestbedMode::Nerf && (m_render_ground_truth || m_nerf.training.render_error_overlay)) {
		// find nearest training view to current camera, and set it
		int bestimage = find_best_training_view(-1);
		if (bestimage >= 0) {
			m_nerf.training.view = bestimage;
			if (mb == 0) {// snap camera to ground truth view on mouse up
				set_camera_to_training_view(m_nerf.training.view);
			}
		}
	}

	keyboard_event();
	mouse_wheel({m.x / (float)m_window_res.y(), m.y / (float)m_window_res.y()}, mw);
	mouse_drag({relm.x, relm.y}, mb);

	if (m_imgui_enabled) {
		imgui();
	}

	return true;
}

void Testbed::SecondWindow::draw(GLuint texture) {
	if (!window)
		return;
	int display_w, display_h;
	GLFWwindow *old_context = glfwGetCurrentContext();
	glfwMakeContextCurrent(window);
	glfwGetFramebufferSize(window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.f,0.f,0.f, 1.f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glBindVertexArray(vao);
	if (program)
		glUseProgram(program);
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindVertexArray(0);
	glUseProgram(0);
	glfwSwapBuffers(window);
	glfwMakeContextCurrent(old_context);
}

void Testbed::draw_gui() {
	// Make sure all the cuda code finished its business here
	CUDA_CHECK_THROW(hipDeviceSynchronize());
	if (!m_render_textures.empty())
		m_second_window.draw((GLuint)m_render_textures.front()->texture());
	glfwMakeContextCurrent(m_glfw_window);
	int display_w, display_h;
	glfwGetFramebufferSize(m_glfw_window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.f, 0.f, 0.f, 0.f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);


	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback([](const ImDrawList*, const ImDrawCmd*) {
		glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
		glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);
	}, nullptr);

	if (m_single_view) {
		list->AddImageQuad((ImTextureID)(size_t)m_render_textures.front()->texture(), ImVec2{0.f, 0.f}, ImVec2{(float)display_w, 0.f}, ImVec2{(float)display_w, (float)display_h}, ImVec2{0.f, (float)display_h}, ImVec2(0, 0), ImVec2(1, 0), ImVec2(1, 1), ImVec2(0, 1));
	} else {
		m_dlss = false;

		int i = 0;
		for (int y = 0; y < m_n_views.y(); ++y) {
			for (int x = 0; x < m_n_views.x(); ++x) {
				if (i >= m_render_surfaces.size()) {
					break;
				}

				Vector2f top_left{x * m_view_size.x(), y * m_view_size.y()};

				list->AddImageQuad(
					(ImTextureID)(size_t)m_render_textures[i]->texture(),
					ImVec2{top_left.x(),                          top_left.y()                         },
					ImVec2{top_left.x() + (float)m_view_size.x(), top_left.y()                         },
					ImVec2{top_left.x() + (float)m_view_size.x(), top_left.y() + (float)m_view_size.y()},
					ImVec2{top_left.x(),                          top_left.y() + (float)m_view_size.y()},
					ImVec2(0, 0),
					ImVec2(1, 0),
					ImVec2(1, 1),
					ImVec2(0, 1)
				);

				++i;
			}
		}
	}

	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	auto draw_mesh = [&]() {
		glClear(GL_DEPTH_BUFFER_BIT);
		Vector2i res(display_w, display_h);
		Vector2f focal_length = calc_focal_length(res, m_fov_axis, m_zoom);
		Vector2f screen_center = render_screen_center();
		draw_mesh_gl(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, res, focal_length, m_smoothed_camera, screen_center, (int)m_mesh_render_mode);
	};

	// Visualizations are only meaningful when rendering a single view
	if (m_single_view) {
		if (m_mesh.verts.size() != 0 && m_mesh.indices.size() != 0 && m_mesh_render_mode != EMeshRenderMode::Off) {
			list->AddCallback([](const ImDrawList*, const ImDrawCmd* cmd) {
				(*(decltype(draw_mesh)*)cmd->UserCallbackData)();
			}, &draw_mesh);
			list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);
		}

		draw_visualizations(list, m_smoothed_camera);
	}

	if (m_render_ground_truth) {
		list->AddText(ImVec2(4.f, 4.f), 0xffffffff, "Ground Truth");
	}

	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();
}
#endif //NGP_GUI

void Testbed::train_and_render(bool skip_rendering) {
	if (m_train) {
		train(m_training_batch_size);
	}

	if (m_mesh.optimize_mesh) {
		optimise_mesh_step(1);
	}

	apply_camera_smoothing(m_frame_ms.val());

	if (!m_render_window || !m_render || skip_rendering) {
		return;
	}

	auto start = std::chrono::steady_clock::now();
	ScopeGuard timing_guard{[&]() {
		m_render_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
	}};

	if ((m_smoothed_camera - m_camera).norm() < 0.001f) {
		m_smoothed_camera = m_camera;
	} else {
		reset_accumulation(true);
	}

	if (m_autofocus) {
		autofocus();
	}

	if (m_single_view) {
		// Should have been created when the window was created.
		assert(!m_render_surfaces.empty());

		auto& render_buffer = m_render_surfaces.front();

		if (m_dlss) {
			render_buffer.enable_dlss(m_window_res);
			m_aperture_size = 0.0f;
		} else {
			render_buffer.disable_dlss();
		}

		auto render_res = render_buffer.in_resolution();
		if (render_res.isZero() || (m_train && m_training_step == 0)) {
			render_res = m_window_res/16;
		} else {
			render_res = render_res.cwiseMin(m_window_res);
		}

		float render_time_per_fullres_frame = m_render_ms.val() / (float)render_res.x() / (float)render_res.y() * (float)m_window_res.x() * (float)m_window_res.y();

		// Make sure we don't starve training with slow rendering
		float factor = std::sqrt(1000.0f / m_dynamic_res_target_fps / render_time_per_fullres_frame);
		if (!m_dynamic_res) {
			factor = 8.f/(float)m_fixed_res_factor;
		}

		factor = tcnn::clamp(factor, 1.0f/16.0f, 1.0f);

		if (factor > m_last_render_res_factor * 1.2f || factor < m_last_render_res_factor * 0.8f || factor == 1.0f || !m_dynamic_res) {
			render_res = (m_window_res.cast<float>() * factor).cast<int>().cwiseMin(m_window_res).cwiseMax(m_window_res/16);
			m_last_render_res_factor = factor;
		}

		if (render_buffer.dlss()) {
			render_res = render_buffer.dlss()->clamp_resolution(render_res);
		}

		render_buffer.resize(render_res);
		render_frame(m_smoothed_camera, m_smoothed_camera, Eigen::Vector4f::Zero(), render_buffer);

#ifdef NGP_GUI
		m_render_textures.front()->blit_from_cuda_mapping();

		if (m_picture_in_picture_res > 0) {
			Vector2i res(m_picture_in_picture_res, m_picture_in_picture_res*9/16);
			m_pip_render_surface->resize(res);
			if (m_pip_render_surface->spp() < 8) {
				// a bit gross, but let's copy the keyframe's state into the global state in order to not have to plumb through the fov etc to render_frame.
				CameraKeyframe backup = copy_camera_to_keyframe();
				CameraKeyframe pip_kf = m_camera_path.eval_camera_path(m_camera_path.m_playtime);
				set_camera_from_keyframe(pip_kf);
				render_frame(pip_kf.m(), pip_kf.m(), Eigen::Vector4f::Zero(), *m_pip_render_surface);
				set_camera_from_keyframe(backup);

				m_pip_render_texture->blit_from_cuda_mapping();
			}
		}
#endif
	} else {
#ifdef NGP_GUI
		// Don't do DLSS when multi-view rendering
		m_dlss = false;
		m_render_surfaces.front().disable_dlss();

		int n_views = n_dimensions_to_visualize()+1;

		float d = std::sqrt((float)m_window_res.x() * (float)m_window_res.y() / (float)n_views);

		int nx = (int)std::ceil((float)m_window_res.x() / d);
		int ny = (int)std::ceil((float)n_views / (float)nx);

		m_n_views = {nx, ny};
		m_view_size = {m_window_res.x() / nx, m_window_res.y() / ny};

		while (m_render_surfaces.size() > n_views) {
			m_render_surfaces.pop_back();
		}

		m_render_textures.resize(n_views);
		while (m_render_surfaces.size() < n_views) {
			size_t idx = m_render_surfaces.size();
			m_render_textures[idx] = std::make_shared<GLTexture>();
			m_render_surfaces.emplace_back(m_render_textures[idx]);
		}

		int i = 0;
		for (int y = 0; y < ny; ++y) {
			for (int x = 0; x < nx; ++x) {
				if (i >= n_views) {
					return;
				}

				m_visualized_dimension = i-1;
				m_render_surfaces[i].resize(m_view_size);
				render_frame(m_smoothed_camera, m_smoothed_camera, Eigen::Vector4f::Zero(), m_render_surfaces[i]);
				m_render_textures[i]->blit_from_cuda_mapping();
				++i;
			}
		}
#else
		throw std::runtime_error{"Multi-view rendering is only supported when compiling with NGP_GUI."};
#endif
	}
}


#ifdef NGP_GUI
void Testbed::create_second_window() {
	if (m_second_window.window) {
		return;
	}
	bool frameless = false;
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
	glfwWindowHint(GLFW_RESIZABLE, !frameless);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_CENTER_CURSOR, false);
	glfwWindowHint(GLFW_DECORATED, !frameless);
	glfwWindowHint(GLFW_SCALE_TO_MONITOR, frameless);
	glfwWindowHint(GLFW_TRANSPARENT_FRAMEBUFFER, true);
	// get the window size / coordinates
	int win_w=0,win_h=0,win_x=0,win_y=0;
	GLuint ps=0,vs=0;
	{
		win_w = 1920;
		win_h = 1080;
		win_x = 0x40000000;
		win_y = 0x40000000;
		static const char* copy_shader_vert = "\
			layout (location = 0)\n\
			in vec2 vertPos_data;\n\
			out vec2 texCoords;\n\
			void main(){\n\
				gl_Position = vec4(vertPos_data.xy, 0.0, 1.0);\n\
				texCoords = (vertPos_data.xy + 1.0) * 0.5; texCoords.y=1.0-texCoords.y;\n\
			}";
		static const char* copy_shader_frag = "\
			in vec2 texCoords;\n\
			out vec4 fragColor;\n\
			uniform sampler2D screenTex;\n\
			void main(){\n\
				fragColor = texture(screenTex, texCoords.xy);\n\
			}";
		vs = compile_shader(false, copy_shader_vert);
		ps = compile_shader(true, copy_shader_frag);
	}
	m_second_window.window = glfwCreateWindow(win_w, win_h, "Fullscreen Output", NULL, m_glfw_window);
	if (win_x!=0x40000000) glfwSetWindowPos(m_second_window.window, win_x, win_y);
	glfwMakeContextCurrent(m_second_window.window);
	m_second_window.program = glCreateProgram();
	glAttachShader(m_second_window.program, vs);
	glAttachShader(m_second_window.program, ps);
	glLinkProgram(m_second_window.program);
	if (!check_shader(m_second_window.program, "shader program", true)) {
		glDeleteProgram(m_second_window.program);
		m_second_window.program = 0;
	}
	// vbo and vao
	glGenVertexArrays(1, &m_second_window.vao);
	glGenBuffers(1, &m_second_window.vbo);
	glBindVertexArray(m_second_window.vao);
	const float fsquadVerts[] = {
		-1.0f, -1.0f,
		-1.0f, 1.0f,
		1.0f, 1.0f,
		1.0f, 1.0f,
		1.0f, -1.0f,
		-1.0f, -1.0f};
	glBindBuffer(GL_ARRAY_BUFFER, m_second_window.vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(fsquadVerts), fsquadVerts, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void *)0);
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);
}
#endif //NGP_GUI

void Testbed::init_window(int resw, int resh, bool hidden, bool second_window) {
#ifndef NGP_GUI
	throw std::runtime_error{"init_window failed: NGP was built without GUI support"};
#else
	m_window_res = {resw, resh};

	glfwSetErrorCallback(glfw_error_callback);
	if (!glfwInit()) {
		throw std::runtime_error{"GLFW could not be initialized."};
	}

#ifdef NGP_VULKAN
	try {
		vulkan_and_ngx_init();
		m_dlss_supported = true;
	} catch (const std::runtime_error& e) {
		tlog::warning() << "Could not initialize Vulkan and NGX. DLSS not supported. (" << e.what() << ")";
	}
#else
	m_dlss_supported = false;
#endif

	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GLFW_TRUE);
	glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
	std::string title = "Neural graphics primitives (";
	switch (m_testbed_mode) {
		case ETestbedMode::Image: title += "Image"; break;
		case ETestbedMode::Sdf: title += "SDF"; break;
		case ETestbedMode::Nerf: title += "NeRF"; break;
		case ETestbedMode::Volume: title += "Volume"; break;
	}
	title += ")";
	m_glfw_window = glfwCreateWindow(m_window_res.x(), m_window_res.y(), title.c_str(), NULL, NULL);
	if (m_glfw_window == NULL) {
		throw std::runtime_error{"GLFW window could not be created."};
	}
	glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
	if (gl3wInit()) {
		throw std::runtime_error{"GL3W could not be initialized."};
	}
#else
	glewExperimental = 1;
	if (glewInit()) {
		throw std::runtime_error{"GLEW could not be initialized."};
	}
#endif
	glfwSwapInterval(0); // Disable vsync

	glfwSetWindowUserPointer(m_glfw_window, this);
	glfwSetDropCallback(m_glfw_window, [](GLFWwindow* window, int count, const char** paths) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (!testbed) {
			return;
		}

		testbed->redraw_gui_next_frame();
		for (int i = 0; i < count; i++) {
			testbed->handle_file(paths[i]);
		}
	});

	glfwSetKeyCallback(m_glfw_window, [](GLFWwindow* window, int key, int scancode, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetCursorPosCallback(m_glfw_window, [](GLFWwindow* window, double xpos, double ypos) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetMouseButtonCallback(m_glfw_window, [](GLFWwindow* window, int button, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetScrollCallback(m_glfw_window, [](GLFWwindow* window, double xoffset, double yoffset) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetWindowSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	glfwSetFramebufferSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	float xscale, yscale;
	glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;
	//io.ConfigFlags |= ImGuiConfigFlags_NavEnableKeyboard;     // Enable Keyboard Controls
	io.ConfigInputTrickleEventQueue = false; // new ImGui event handling seems to make camera controls laggy if this is true.
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 330 core");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);

	// Make sure there's at least one usable render texture
	m_render_textures = { std::make_shared<GLTexture>() };

	m_render_surfaces.clear();
	m_render_surfaces.emplace_back(m_render_textures.front());
	m_render_surfaces.front().resize(m_window_res);

	m_pip_render_texture = std::make_shared<GLTexture>();
	m_pip_render_surface = std::make_unique<CudaRenderBuffer>(m_pip_render_texture);

	m_render_window = true;

	if (m_second_window.window == nullptr && second_window) {
		create_second_window();
	}
#endif // NGP_GUI
}

void Testbed::destroy_window() {
#ifndef NGP_GUI
	throw std::runtime_error{"destroy_window failed: NGP was built without GUI support"};
#else
	if (!m_render_window) {
		throw std::runtime_error{"Window must be initialized to be destroyed."};
	}

	m_render_surfaces.clear();
	m_render_textures.clear();

	m_pip_render_surface.reset();
	m_pip_render_texture.reset();

#ifdef NGP_VULKAN
	m_dlss_supported = m_dlss = false;
	vulkan_and_ngx_destroy();
#endif

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(m_glfw_window);
	glfwTerminate();

	m_glfw_window = nullptr;
	m_render_window = false;
#endif //NGP_GUI
}

bool Testbed::frame() {
#ifdef NGP_GUI
	if (m_render_window) {
		if (!begin_frame_and_handle_user_input()) {
			return false;
		}
	}
#endif

	// Render against the trained neural network. If we're training and already close to convergence,
	// we can skip rendering if the scene camera doesn't change
	uint32_t n_to_skip = m_train ? tcnn::clamp(m_training_step / 16u, 15u, 255u) : 0;
	if (m_render_skip_due_to_lack_of_camera_movement_counter > n_to_skip) {
		m_render_skip_due_to_lack_of_camera_movement_counter = 0;
	}
	bool skip_rendering = m_render_skip_due_to_lack_of_camera_movement_counter++ != 0;

	if (!m_dlss && m_max_spp > 0 && !m_render_surfaces.empty() && m_render_surfaces.front().spp() >= m_max_spp) {
		skip_rendering = true;
		if (!m_train) {
			std::this_thread::sleep_for(1ms);
		}
	}

	if (!skip_rendering || (std::chrono::steady_clock::now() - m_last_gui_draw_time_point) > 25ms) {
		redraw_gui_next_frame();
	}

	try {
		while (true) {
			(*m_task_queue.tryPop())();
		}
	} catch (SharedQueueEmptyException&) {}


	train_and_render(skip_rendering);
	if (m_testbed_mode == ETestbedMode::Sdf && m_sdf.calculate_iou_online) {
		m_sdf.iou = calculate_iou(m_train ? 64*64*64 : 128*128*128, m_sdf.iou_decay, false, true);
		m_sdf.iou_decay = 0.f;
	}

#ifdef NGP_GUI
	if (m_render_window) {
		if (m_gui_redraw) {
			// Gather histogram statistics of the encoding in use
			if (m_gather_histograms) {
				gather_histograms();
			}

			draw_gui();
			m_gui_redraw = false;

			m_last_gui_draw_time_point = std::chrono::steady_clock::now();
		}

		ImGui::EndFrame();
	}
#endif

	return true;
}

fs::path Testbed::training_data_path() const {
	return m_data_path.with_extension("training");
}

bool Testbed::want_repl() {
	bool b=m_want_repl;
	m_want_repl=false;
	return b;
}

void Testbed::apply_camera_smoothing(float elapsed_ms) {
	if (m_camera_smoothing) {
		float decay = std::pow(0.02f, elapsed_ms/1000.0f);
		m_smoothed_camera = log_space_lerp(m_smoothed_camera, m_camera, 1.0f - decay);
	} else {
		m_smoothed_camera = m_camera;
	}
}

CameraKeyframe Testbed::copy_camera_to_keyframe() const {
	return CameraKeyframe(m_camera, m_slice_plane_z, m_scale, fov(), m_aperture_size, m_nerf.glow_mode, m_nerf.glow_y_cutoff);
}

void Testbed::set_camera_from_keyframe(const CameraKeyframe& k) {
	m_camera = k.m();
	m_slice_plane_z = k.slice;
	m_scale = k.scale;
	set_fov(k.fov);
	m_aperture_size = k.aperture_size;
	m_nerf.glow_mode = k.glow_mode;
	m_nerf.glow_y_cutoff = k.glow_y_cutoff;
}

void Testbed::set_camera_from_time(float t) {
	if (m_camera_path.m_keyframes.empty())
		return;
	set_camera_from_keyframe(m_camera_path.eval_camera_path(t));
}

void Testbed::update_loss_graph() {
	m_loss_graph[m_loss_graph_samples++ % m_loss_graph.size()] = std::log(m_loss_scalar.val());
}

uint32_t Testbed::n_dimensions_to_visualize() const {
	return m_network->width(m_visualized_layer);
}

float Testbed::fov() const {
	return focal_length_to_fov(1.0f, m_relative_focal_length[m_fov_axis]);
}

void Testbed::set_fov(float val) {
	m_relative_focal_length = Vector2f::Constant(fov_to_focal_length(1, val));
}

Vector2f Testbed::fov_xy() const {
	return focal_length_to_fov(Vector2i::Ones(), m_relative_focal_length);
}

void Testbed::set_fov_xy(const Vector2f& val) {
	m_relative_focal_length = fov_to_focal_length(Vector2i::Ones(), val);
}

size_t Testbed::n_params() {
	return m_network->n_params();
}

size_t Testbed::n_encoding_params() {
	return m_network->n_params() - first_encoder_param();
}

size_t Testbed::first_encoder_param() {
	auto layer_sizes = m_network->layer_sizes();
	size_t first_encoder = 0;
	for (auto size : layer_sizes) {
		first_encoder += size.first * size.second;
	}
	return first_encoder;
}

uint32_t Testbed::network_width(uint32_t layer) const {
	return m_network->width(layer);
}

uint32_t Testbed::network_num_forward_activations() const {
	return m_network->num_forward_activations();
}

void Testbed::set_max_level(float maxlevel) {
	if (!m_network) return;
	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level(maxlevel);
	}
	reset_accumulation();
}

void Testbed::set_min_level(float minlevel) {
	if (!m_network) return;
	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_quantize_threshold(powf(minlevel, 4.f) * 0.2f);
	}
	reset_accumulation();
}

void Testbed::set_visualized_layer(int layer) {
	m_visualized_layer = layer;
	m_visualized_dimension = std::max(-1, std::min(m_visualized_dimension, (int)m_network->width(layer)-1));
	reset_accumulation();
}

ELossType Testbed::string_to_loss_type(const std::string& str) {
	if (equals_case_insensitive(str, "L2")) {
		return ELossType::L2;
	} else if (equals_case_insensitive(str, "RelativeL2")) {
		return ELossType::RelativeL2;
	} else if (equals_case_insensitive(str, "L1")) {
		return ELossType::L1;
	} else if (equals_case_insensitive(str, "Mape")) {
		return ELossType::Mape;
	} else if (equals_case_insensitive(str, "Smape")) {
		return ELossType::Smape;
	} else if (equals_case_insensitive(str, "Huber") || equals_case_insensitive(str, "SmoothL1")) {
		// Legacy: we used to refer to the Huber loss (L2 near zero, L1 further away) as "SmoothL1".
		return ELossType::Huber;
	} else if (equals_case_insensitive(str, "LogL1")) {
		return ELossType::LogL1;
	} else {
		throw std::runtime_error{"Unknown loss type."};
	}
}

Testbed::NetworkDims Testbed::network_dims() const {
	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:   return network_dims_nerf(); break;
		case ETestbedMode::Sdf:    return network_dims_sdf(); break;
		case ETestbedMode::Image:  return network_dims_image(); break;
		case ETestbedMode::Volume: return network_dims_volume(); break;
		default: throw std::runtime_error{"Invalid mode."};
	}
}

void Testbed::reset_network(bool clear_density_grid) {
	m_sdf.iou_decay = 0;

	m_rng = default_rng_t{m_seed};

	// Start with a low rendering resolution and gradually ramp up
	m_render_ms.set(10000);

	reset_accumulation();
	m_nerf.training.counters_rgb.rays_per_batch = 1 << 12;
	m_nerf.training.counters_rgb.measured_batch_size_before_compaction = 0;
	m_nerf.training.n_steps_since_cam_update = 0;
	m_nerf.training.n_steps_since_error_map_update = 0;
	m_nerf.training.n_rays_since_error_map_update = 0;
	m_nerf.training.n_steps_between_error_map_updates = 128;
	m_nerf.training.error_map.is_cdf_valid = false;
	m_nerf.training.density_grid_rng = default_rng_t{m_rng.next_uint()};

	m_nerf.training.reset_camera_extrinsics();

	m_loss_graph_samples = 0;

	// Default config
	json config = m_network_config;

	json& encoding_config = config["encoding"];
	json& loss_config = config["loss"];
	json& optimizer_config = config["optimizer"];
	json& network_config = config["network"];

	auto dims = network_dims();

	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.loss_type = string_to_loss_type(loss_config.value("otype", "L2"));

		// Some of the Nerf-supported losses are not supported by tcnn::Loss,
		// so just create a dummy L2 loss there. The NeRF code path will bypass
		// the tcnn::Loss in any case.
		loss_config["otype"] = "L2";
	}

	// Automatically determine certain parameters if we're dealing with the (hash)grid encoding
	if (to_lower(encoding_config.value("otype", "OneBlob")).find("grid") != std::string::npos) {
		encoding_config["n_pos_dims"] = dims.n_pos;

		const uint32_t n_features_per_level = encoding_config.value("n_features_per_level", 2u);

		if (encoding_config.contains("n_features") && encoding_config["n_features"] > 0) {
			m_num_levels = (uint32_t)encoding_config["n_features"] / n_features_per_level;
		} else {
			m_num_levels = encoding_config.value("n_levels", 16u);
		}

		m_level_stats.resize(m_num_levels);
		m_first_layer_column_stats.resize(m_num_levels);

		const uint32_t log2_hashmap_size = encoding_config.value("log2_hashmap_size", 15);

		m_base_grid_resolution = encoding_config.value("base_resolution", 0);
		if (!m_base_grid_resolution) {
			m_base_grid_resolution = 1u << ((log2_hashmap_size) / dims.n_pos);
			encoding_config["base_resolution"] = m_base_grid_resolution;
		}

		float desired_resolution = 2048.0f; // Desired resolution of the finest hashgrid level over the unit cube
		if (m_testbed_mode == ETestbedMode::Image) {
			desired_resolution = m_image.resolution.maxCoeff() / 2.0f;
		} else if (m_testbed_mode == ETestbedMode::Volume) {
			desired_resolution = m_volume.world2index_scale;
		}

		// Automatically determine suitable per_level_scale
		m_per_level_scale = encoding_config.value("per_level_scale", 0.0f);
		if (m_per_level_scale <= 0.0f && m_num_levels > 1) {
			m_per_level_scale = std::exp(std::log(desired_resolution * (float)m_nerf.training.dataset.aabb_scale / (float)m_base_grid_resolution) / (m_num_levels-1));
			encoding_config["per_level_scale"] = m_per_level_scale;
		}

		tlog::info()
			<< "GridEncoding: "
			<< " Nmin=" << m_base_grid_resolution
			<< " b=" << m_per_level_scale
			<< " F=" << n_features_per_level
			<< " T=2^" << log2_hashmap_size
			<< " L=" << m_num_levels
			;
	}

	m_loss.reset(create_loss<precision_t>(loss_config));
	m_optimizer.reset(create_optimizer<precision_t>(optimizer_config));

	size_t n_encoding_params = 0;
	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.cam_exposure.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Array3f>(1e-3f, Array3f::Zero()));
		m_nerf.training.cam_pos_offset.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Vector3f>(1e-4f, Vector3f::Zero()));
		m_nerf.training.cam_rot_offset.resize(m_nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
		m_nerf.training.cam_focal_length_offset = AdamOptimizer<Vector2f>(1e-5f);

		m_nerf.training.reset_extra_dims(m_rng);

		json& dir_encoding_config = config["dir_encoding"];
		json& rgb_network_config = config["rgb_network"];

		uint32_t n_dir_dims = 3;
		uint32_t n_extra_dims = m_nerf.training.dataset.n_extra_dims();
		m_network = m_nerf_network = std::make_shared<NerfNetwork<precision_t>>(
			dims.n_pos,
			n_dir_dims,
			n_extra_dims,
			dims.n_pos + 1, // The offset of 1 comes from the dt member variable of NerfCoordinate. HACKY
			encoding_config,
			dir_encoding_config,
			network_config,
			rgb_network_config
		);

		m_encoding = m_nerf_network->encoding();
		n_encoding_params = m_encoding->n_params() + m_nerf_network->dir_encoding()->n_params();

		tlog::info()
			<< "Density model: " << dims.n_pos
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_nerf_network->encoding()->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 1
			;

		tlog::info()
			<< "Color model:   " << n_dir_dims
			<< "--[" << std::string(dir_encoding_config["otype"])
			<< "]-->" << m_nerf_network->dir_encoding()->padded_output_width() << "+" << network_config.value("n_output_dims", 16u)
			<< "--[" << std::string(rgb_network_config["otype"])
			<< "(neurons=" << (int)rgb_network_config["n_neurons"] << ",layers=" << ((int)rgb_network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 3
			;

		// Create distortion map model
		{
			json& distortion_map_optimizer_config =  config.contains("distortion_map") && config["distortion_map"].contains("optimizer") ? config["distortion_map"]["optimizer"] : optimizer_config;

			m_distortion.resolution = Vector2i::Constant(32);
			if (config.contains("distortion_map") && config["distortion_map"].contains("resolution")) {
				from_json(config["distortion_map"]["resolution"], m_distortion.resolution);
			}
			m_distortion.map = std::make_shared<TrainableBuffer<2, 2, float>>(m_distortion.resolution);
			m_distortion.optimizer.reset(create_optimizer<float>(distortion_map_optimizer_config));
			m_distortion.trainer = std::make_shared<Trainer<float, float>>(m_distortion.map, m_distortion.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(loss_config)}, m_seed);
		}
	} else {
		uint32_t alignment = network_config.contains("otype") && (equals_case_insensitive(network_config["otype"], "FullyFusedMLP") || equals_case_insensitive(network_config["otype"], "MegakernelMLP")) ? 16u : 8u;

		if (encoding_config.contains("otype") && equals_case_insensitive(encoding_config["otype"], "Takikawa")) {
			if (m_sdf.octree_depth_target == 0) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}

			if (!m_sdf.triangle_octree || m_sdf.triangle_octree->depth() != m_sdf.octree_depth_target) {
				m_sdf.triangle_octree.reset(new TriangleOctree{});
				m_sdf.triangle_octree->build(*m_sdf.triangle_bvh, m_sdf.triangles_cpu, m_sdf.octree_depth_target);
				m_sdf.octree_depth_target = m_sdf.triangle_octree->depth();
				m_sdf.brick_data.free_memory();
			}

			m_encoding.reset(new TakikawaEncoding<precision_t>(
				encoding_config["starting_level"],
				m_sdf.triangle_octree,
				tcnn::string_to_interpolation_type(encoding_config.value("interpolation", "linear"))
			));

			m_network = std::make_shared<NetworkWithInputEncoding<precision_t>>(m_encoding, dims.n_output, network_config);
			m_sdf.uses_takikawa_encoding = true;
		} else {
			m_encoding.reset(create_encoding<precision_t>(dims.n_input, encoding_config));
			m_network = std::make_shared<NetworkWithInputEncoding<precision_t>>(m_encoding, dims.n_output, network_config);
			m_sdf.uses_takikawa_encoding = false;
			if (m_sdf.octree_depth_target == 0 && encoding_config.contains("n_levels")) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}
		}

		n_encoding_params = m_encoding->n_params();

		tlog::info()
			<< "Model:         " << dims.n_input
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_encoding->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << dims.n_output;
	}

	size_t n_network_params = m_network->n_params() - n_encoding_params;

	tlog::info() << "  total_encoding_params=" << n_encoding_params << " total_network_params=" << n_network_params;

	m_trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(m_network, m_optimizer, m_loss, m_seed);
	m_training_step = 0;
	m_training_start_time_point = std::chrono::steady_clock::now();

	// Create envmap model
	{
		json& envmap_loss_config = config.contains("envmap") && config["envmap"].contains("loss") ? config["envmap"]["loss"] : loss_config;
		json& envmap_optimizer_config =  config.contains("envmap") && config["envmap"].contains("optimizer") ? config["envmap"]["optimizer"] : optimizer_config;

		m_envmap.loss_type = string_to_loss_type(envmap_loss_config.value("otype", "L2"));

		m_envmap.resolution = m_nerf.training.dataset.envmap_resolution;
		m_envmap.envmap = std::make_shared<TrainableBuffer<4, 2, float>>(m_envmap.resolution);
		m_envmap.optimizer.reset(create_optimizer<float>(envmap_optimizer_config));
		m_envmap.trainer = std::make_shared<Trainer<float, float, float>>(m_envmap.envmap, m_envmap.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(envmap_loss_config)}, m_seed);

		if (m_nerf.training.dataset.envmap_data.data()) {
			m_envmap.trainer->set_params_full_precision(m_nerf.training.dataset.envmap_data.data(), m_nerf.training.dataset.envmap_data.size());
		}
	}

	if (clear_density_grid) {
		m_nerf.density_grid.memset(0);
		m_nerf.density_grid_bitfield.memset(0);
	}
}

Testbed::Testbed(ETestbedMode mode)
: m_testbed_mode(mode)
{
	uint32_t compute_capability = cuda_compute_capability();
	if (compute_capability < MIN_GPU_ARCH) {
		tlog::warning() << "Insufficient compute capability " << compute_capability << " detected.";
		tlog::warning() << "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly.";
	}

	m_network_config = {
		{"loss", {
			{"otype", "L2"}
		}},
		{"optimizer", {
			{"otype", "Adam"},
			{"learning_rate", 1e-3},
			{"beta1", 0.9f},
			{"beta2", 0.99f},
			{"epsilon", 1e-15f},
			{"l2_reg", 1e-6f},
		}},
		{"encoding", {
			{"otype", "HashGrid"},
			{"n_levels", 16},
			{"n_features_per_level", 2},
			{"log2_hashmap_size", 19},
			{"base_resolution", 16},
		}},
		{"network", {
			{"otype", "FullyFusedMLP"},
			{"n_neurons", 64},
			{"n_layers", 2},
			{"activation", "ReLU"},
			{"output_activation", "None"},
		}},
	};

	reset_camera();

	if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
		throw std::runtime_error{"Testbed required CUDA 10.2 or later."};
	}

	set_exposure(0);
	set_min_level(0.f);
	set_max_level(1.f);
}

Testbed::~Testbed() {

	if (m_render_window) {
		destroy_window();
	}
}

void Testbed::train(uint32_t batch_size) {
	if (!m_training_data_available) {
		m_train = false;
		return;
	}

	if (!m_dlss) {
		// No immediate redraw necessary
		reset_accumulation(false, false);
	}

	uint32_t n_prep_to_skip = m_testbed_mode == ETestbedMode::Nerf ? tcnn::clamp(m_training_step / 16u, 1u, 16u) : 1u;
	if (m_training_step % n_prep_to_skip == 0) {
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_prep_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count() / n_prep_to_skip);
		}};

		switch (m_testbed_mode) {
			case ETestbedMode::Nerf: training_prep_nerf(batch_size, m_stream.get()); break;
			case ETestbedMode::Sdf: training_prep_sdf(batch_size, m_stream.get()); break;
			case ETestbedMode::Image: training_prep_image(batch_size, m_stream.get()); break;
			case ETestbedMode::Volume: training_prep_volume(batch_size, m_stream.get()); break;
			default: throw std::runtime_error{"Invalid training mode."};
		}

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	// Find leaf optimizer and update its settings
	json* leaf_optimizer_config = &m_network_config["optimizer"];
	while (leaf_optimizer_config->contains("nested")) {
		leaf_optimizer_config = &(*leaf_optimizer_config)["nested"];
	}
	(*leaf_optimizer_config)["optimize_matrix_params"] = m_train_network;
	(*leaf_optimizer_config)["optimize_non_matrix_params"] = m_train_encoding;
	m_optimizer->update_hyperparams(m_network_config["optimizer"]);

	bool get_loss_scalar = m_training_step % 16 == 0;

	{
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
		}};

		switch (m_testbed_mode) {
			case ETestbedMode::Nerf:   train_nerf(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Sdf:    train_sdf(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Image:  train_image(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Volume: train_volume(batch_size, get_loss_scalar, m_stream.get()); break;
			default: throw std::runtime_error{"Invalid training mode."};
		}

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	if (get_loss_scalar) {
		update_loss_graph();
	}
}

Vector2f Testbed::calc_focal_length(const Vector2i& resolution, int fov_axis, float zoom) const {
	return m_relative_focal_length * resolution[fov_axis] * zoom;
}

Vector2f Testbed::render_screen_center() const {
	// see pixel_to_ray for how screen center is used; 0.5,0.5 is 'normal'. we flip so that it becomes the point in the original image we want to center on.
	auto screen_center = m_screen_center;
	return {(0.5f-screen_center.x())*m_zoom + 0.5f, (0.5-screen_center.y())*m_zoom + 0.5f};
}

__global__ void dlss_prep_kernel(
	ETestbedMode mode,
	Vector2i resolution,
	uint32_t sample_index,
	Vector2f focal_length,
	Vector2f screen_center,
	Vector3f parallax_shift,
	bool snap_to_pixel_centers,
	float* depth_buffer,
	Matrix<float, 3, 4> camera,
	Matrix<float, 3, 4> prev_camera,
	hipSurfaceObject_t depth_surface,
	hipSurfaceObject_t mvec_surface,
	hipSurfaceObject_t exposure_surface,
	Lens lens,
	const float view_dist,
	const float prev_view_dist,
	const Vector2f image_pos,
	const Vector2f prev_image_pos,
	const Vector2i image_resolution,
	const Vector2i quilting_dims
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	uint32_t x_orig = x;
	uint32_t y_orig = y;

	if (quilting_dims != Vector2i::Ones()) {
		apply_quilting(&x, &y, resolution, parallax_shift, quilting_dims);
	}

	const float depth = depth_buffer[idx];
	Vector2f mvec = mode == ETestbedMode::Image ? motion_vector_2d(
		sample_index,
		{x, y},
		resolution.cwiseQuotient(quilting_dims),
		image_resolution,
		screen_center,
		view_dist,
		prev_view_dist,
		image_pos,
		prev_image_pos,
		snap_to_pixel_centers
	) : motion_vector_3d(
		sample_index,
		{x, y},
		resolution.cwiseQuotient(quilting_dims),
		focal_length,
		camera,
		prev_camera,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		depth,
		lens
	);

	surf2Dwrite(make_float2(mvec.x(), mvec.y()), mvec_surface, x_orig * sizeof(float2), y_orig);

	// Scale depth buffer to be guaranteed in [0,1].
	surf2Dwrite(std::min(std::max(depth / 128.0f, 0.0f), 1.0f), depth_surface, x_orig * sizeof(float), y_orig);

	// First thread write an exposure factor of 1. Since DLSS will run on tonemapped data,
	// exposure is assumed to already have been applied to DLSS' inputs.
	if (x_orig == 0 && y_orig == 0) {
		surf2Dwrite(1.0f, exposure_surface, 0, 0);
	}
}

void Testbed::render_frame(const Matrix<float, 3, 4>& camera_matrix0, const Matrix<float, 3, 4>& camera_matrix1, const Vector4f& nerf_rolling_shutter, CudaRenderBuffer& render_buffer, bool to_srgb) {
	Vector2i max_res = m_window_res.cwiseMax(render_buffer.in_resolution());

	render_buffer.clear_frame(m_stream.get());

	Vector2f focal_length = calc_focal_length(render_buffer.in_resolution(), m_fov_axis, m_zoom);
	Vector2f screen_center = render_screen_center();

	if (m_quilting_dims != Vector2i::Ones() && m_quilting_dims != Vector2i{2, 1}) {
		// In the case of a holoplay lenticular screen, m_scale represents the inverse distance of the head above the display.
		m_parallax_shift.z() = 1.0f / m_scale;
	}

	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:
			if (!m_render_ground_truth || m_ground_truth_alpha < 1.0f) {
				render_nerf(render_buffer, max_res, focal_length, camera_matrix0, camera_matrix1, nerf_rolling_shutter, screen_center, m_stream.get());
			}
			break;
		case ETestbedMode::Sdf:
			{
				if (m_render_ground_truth && m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
					if (m_sdf.brick_data.size() == 0) {
						tlog::info() << "Building voxel brick positions for " << m_sdf.triangle_octree->n_dual_nodes() << " dual nodes.";
						m_sdf.brick_res = 5;
						std::vector<Eigen::Vector3f> positions = m_sdf.triangle_octree->build_brick_voxel_position_list(m_sdf.brick_res);
						GPUMemory<Eigen::Vector3f> positions_gpu;
						positions_gpu.resize_and_copy_from_host(positions);
						m_sdf.brick_data.resize(positions.size());
						tlog::info() << positions_gpu.size() << " voxel brick positions. Computing SDFs.";
						m_sdf.triangle_bvh->signed_distance_gpu(
							positions.size(),
							EMeshSdfMode::Watertight, //m_sdf.mesh_sdf_mode, // watertight seems to be the best method for 'one off' SDF signing
							positions_gpu.data(),
							m_sdf.brick_data.data(),
							m_sdf.triangles_gpu.data(),
							false,
							m_stream.get()
						);
					}
				}
				distance_fun_t distance_fun =
					m_render_ground_truth ? (distance_fun_t)[&](uint32_t n_elements, const GPUMemory<Vector3f>& positions, GPUMemory<float>& distances, hipStream_t stream) {
						if (n_elements == 0) {
							return;
						}
						if (m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
							// linear_kernel(sdf_brick_kernel, 0, stream,
							// 	n_elements,
							// 	positions.data(),
							// 	distances.data(),
							// 	m_sdf.triangle_octree->nodes_gpu(),
							// 	m_sdf.triangle_octree->dual_nodes_gpu(),
							// 	std::max(1u,std::min(m_sdf.triangle_octree->depth(), m_sdf.brick_level)),
							// 	m_sdf.brick_data.data(),
							// 	m_sdf.brick_res,
							// 	m_sdf.brick_quantise_bits
							// );
						} else {
							m_sdf.triangle_bvh->signed_distance_gpu(
								n_elements,
								m_sdf.mesh_sdf_mode,
								(Vector3f*)positions.data(),
								distances.data(),
								m_sdf.triangles_gpu.data(),
								false,
								m_stream.get()
							);
						}
					} : (distance_fun_t)[&](uint32_t n_elements, const GPUMemory<Vector3f>& positions, GPUMemory<float>& distances, hipStream_t stream) {
						if (n_elements == 0) {
							return;
						}
						n_elements = next_multiple(n_elements, tcnn::batch_size_granularity);
						GPUMatrix<float> positions_matrix((float*)positions.data(), 3, n_elements);
						GPUMatrix<float, RM> distances_matrix(distances.data(), 1, n_elements);
						m_network->inference(stream, positions_matrix, distances_matrix);
					};

				normals_fun_t normals_fun =
					m_render_ground_truth ? (normals_fun_t)[&](uint32_t n_elements, const GPUMemory<Vector3f>& positions, GPUMemory<Vector3f>& normals, hipStream_t stream) {
						// NO-OP. Normals will automatically be populated by raytrace
					} : (normals_fun_t)[&](uint32_t n_elements, const GPUMemory<Vector3f>& positions, GPUMemory<Vector3f>& normals, hipStream_t stream) {
						if (n_elements == 0) {
							return;
						}

						n_elements = next_multiple(n_elements, tcnn::batch_size_granularity);

						GPUMatrix<float> positions_matrix((float*)positions.data(), 3, n_elements);
						GPUMatrix<float> normals_matrix((float*)normals.data(), 3, n_elements);
						m_network->input_gradient(stream, 0, positions_matrix, normals_matrix);
					};

				render_sdf(
					distance_fun,
					normals_fun,
					render_buffer,
					max_res,
					focal_length,
					camera_matrix0,
					screen_center,
					m_stream.get()
				);
			}
			break;
		case ETestbedMode::Image:
			render_image(render_buffer, m_stream.get());
			break;
		case ETestbedMode::Volume:
			render_volume(render_buffer, focal_length, camera_matrix0, screen_center, m_stream.get());
			break;
		default:
			throw std::runtime_error{"Invalid render mode."};
	}

	render_buffer.set_color_space(m_color_space);
	render_buffer.set_tonemap_curve(m_tonemap_curve);

	// Prepare DLSS data: motion vectors, scaled depth, exposure
	if (render_buffer.dlss()) {
		auto res = render_buffer.in_resolution();

		bool distortion = m_testbed_mode == ETestbedMode::Nerf && m_nerf.render_with_lens_distortion;

		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };

		dlss_prep_kernel<<<blocks, threads, 0, m_stream.get()>>>(
			m_testbed_mode,
			res,
			render_buffer.spp(),
			focal_length,
			screen_center,
			m_parallax_shift,
			m_snap_to_pixel_centers,
			render_buffer.depth_buffer(),
			camera_matrix0,
			m_prev_camera,
			render_buffer.dlss()->depth(),
			render_buffer.dlss()->mvec(),
			render_buffer.dlss()->exposure(),
			distortion ? m_nerf.render_lens : Lens{},
			m_scale,
			m_prev_scale,
			m_image.pos,
			m_image.prev_pos,
			m_image.resolution,
			m_quilting_dims
		);

		render_buffer.set_dlss_sharpening(m_dlss_sharpening);
	}

	m_prev_camera = camera_matrix0;
	m_prev_scale = m_scale;
	m_image.prev_pos = m_image.pos;

	render_buffer.accumulate(m_exposure, m_stream.get());
	render_buffer.tonemap(m_exposure, m_background_color, to_srgb ? EColorSpace::SRGB : EColorSpace::Linear, m_stream.get());

	if (m_testbed_mode == ETestbedMode::Nerf) {
		// Overlay the ground truth image if requested
		if (m_render_ground_truth) {
			auto const& metadata = m_nerf.training.dataset.metadata[m_nerf.training.view];
			if (m_ground_truth_render_mode == EGroundTruthRenderMode::Shade) {
				render_buffer.overlay_image(
					m_ground_truth_alpha,
					Array3f::Constant(m_exposure) + m_nerf.training.cam_exposure[m_nerf.training.view].variable(),
					m_background_color,
					to_srgb ? EColorSpace::SRGB : EColorSpace::Linear,
					metadata.pixels,
					metadata.image_data_type,
					metadata.resolution,
					m_fov_axis,
					m_zoom,
					Vector2f::Constant(0.5f),
					m_stream.get()
				);
			} else if (m_ground_truth_render_mode == EGroundTruthRenderMode::Depth && metadata.depth) {
				render_buffer.overlay_depth(
					m_ground_truth_alpha,
					metadata.depth,
					1.0f/m_nerf.training.dataset.scale,
					metadata.resolution,
					m_fov_axis,
					m_zoom,
					Vector2f::Constant(0.5f),
					m_stream.get()
				);
			}
		}

		// Visualize the accumulated error map if requested
		if (m_nerf.training.render_error_overlay) {
			const float* err_data = m_nerf.training.error_map.data.data();
			Vector2i error_map_res = m_nerf.training.error_map.resolution;
			if (m_render_ground_truth) {
				err_data = m_nerf.training.dataset.sharpness_data.data();
				error_map_res = m_nerf.training.dataset.sharpness_resolution;
			}
			size_t emap_size = error_map_res.x() * error_map_res.y();
			err_data += emap_size * m_nerf.training.view;
			static GPUMemory<float> average_error;
			average_error.enlarge(1);
			average_error.memset(0);
			const float* aligned_err_data_s = (const float*)(((size_t)err_data)&~15);
			const float* aligned_err_data_e = (const float*)(((size_t)(err_data+emap_size))&~15);
			size_t reduce_size = aligned_err_data_e - aligned_err_data_s;
			reduce_sum(aligned_err_data_s, [reduce_size] __device__ (float val) { return max(val,0.f) / (reduce_size); }, average_error.data(), reduce_size, m_stream.get());
			auto const &metadata = m_nerf.training.dataset.metadata[m_nerf.training.view];
			render_buffer.overlay_false_color(metadata.resolution, to_srgb, m_fov_axis, m_stream.get(), err_data, error_map_res, average_error.data(), m_nerf.training.error_overlay_brightness, m_render_ground_truth);
		}
	}

	CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
}

void Testbed::determine_autofocus_target_from_pixel(const Vector2i& focus_pixel) {
	float depth;

	const auto& surface = m_render_surfaces.front();
	if (surface.depth_buffer()) {
		auto res = surface.in_resolution();
		Vector2i depth_pixel = focus_pixel.cast<float>().cwiseProduct(res.cast<float>()).cwiseQuotient(m_window_res.cast<float>()).cast<int>();
		depth_pixel = depth_pixel.cwiseMin(res).cwiseMax(0);

		CUDA_CHECK_THROW(hipMemcpy(&depth, surface.depth_buffer() + depth_pixel.x() + depth_pixel.y() * res.x(), sizeof(float), hipMemcpyDeviceToHost));
	} else {
		depth = m_scale;
	}

	auto ray = pixel_to_ray_pinhole(0, focus_pixel, m_window_res, calc_focal_length(m_window_res, m_fov_axis, m_zoom), m_smoothed_camera, render_screen_center());

	m_autofocus_target = ray.o + ray.d * depth;
	m_autofocus = true; // If someone shift-clicked, that means they want the AUTOFOCUS
}

void Testbed::autofocus() {
	float new_slice_plane_z = std::max(view_dir().dot(m_autofocus_target - view_pos()), 0.1f) - m_scale;
	if (new_slice_plane_z != m_slice_plane_z) {
		m_slice_plane_z = new_slice_plane_z;
		if (m_aperture_size != 0.0f) {
			reset_accumulation();
		}
	}
}

Testbed::LevelStats compute_level_stats(const float* params, size_t n_params) {
	Testbed::LevelStats s = {};
	for (size_t i = 0; i < n_params; ++i) {
		float v = params[i];
		float av = fabsf(v);
		if (av < 0.00001f) {
			s.numzero++;
		} else {
			if (s.count == 0) s.min = s.max = v;
			s.count++;
			s.x += v;
			s.xsquared += v * v;
			s.min = min(s.min, v);
			s.max = max(s.max, v);
		}
	}
	return s;
}

void Testbed::gather_histograms() {
	int n_params = (int)m_network->n_params();
	int first_encoder = first_encoder_param();
	int n_encoding_params = n_params - first_encoder;

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc && m_trainer->params()) {
		std::vector<float> grid(n_encoding_params);

		uint32_t m = m_network->layer_sizes().front().first;
		uint32_t n = m_network->layer_sizes().front().second;
		std::vector<float> first_layer_rm(m * n);

		CUDA_CHECK_THROW(hipMemcpyAsync(grid.data(), m_trainer->params() + first_encoder, grid.size() * sizeof(float), hipMemcpyDeviceToHost, m_stream.get()));
		CUDA_CHECK_THROW(hipMemcpyAsync(first_layer_rm.data(), m_trainer->params(), first_layer_rm.size() * sizeof(float), hipMemcpyDeviceToHost, m_stream.get()));
		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));


		for (int l = 0; l < m_num_levels; ++l) {
			m_level_stats[l] = compute_level_stats(grid.data() + hg_enc->level_params_offset(l), hg_enc->level_n_params(l));
		}

		int numquant = 0;
		m_quant_percent = float(numquant * 100) / (float)n_encoding_params;
		if (m_histo_level < m_num_levels) {
			size_t nperlevel = hg_enc->level_n_params(m_histo_level);
			const float* d = grid.data() + hg_enc->level_params_offset(m_histo_level);
			float scale = 128.f / (m_histo_scale); // fixed scale for now to make it more comparable between levels
			memset(m_histo, 0, sizeof(m_histo));
			for (int i = 0; i < nperlevel; ++i) {
				float v = *d++;
				if (v == 0.f) {
					continue;
				}
				int bin = (int)floor(v * scale + 128.5f);
				if (bin >= 0 && bin <= 256) {
					m_histo[bin]++;
				}
			}
		}
	}
}

// Increment this number when making a change to the snapshot format
static const size_t SNAPSHOT_FORMAT_VERSION = 1;

void Testbed::save_snapshot(const std::string& filepath_string, bool include_optimizer_state) {
	fs::path filepath = filepath_string;
	m_network_config["snapshot"] = m_trainer->serialize(include_optimizer_state);

	auto& snapshot = m_network_config["snapshot"];
	snapshot["version"] = SNAPSHOT_FORMAT_VERSION;

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["density_grid_size"] = NERF_GRIDSIZE();

		GPUMemory<__half> density_grid_fp16(m_nerf.density_grid.size());
		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid_fp16[i] = (__half)density_grid[i];
		});

		snapshot["density_grid_binary"] = density_grid_fp16;
		snapshot["nerf"]["aabb_scale"] = m_nerf.training.dataset.aabb_scale;
	}

	snapshot["training_step"] = m_training_step;
	snapshot["loss"] = m_loss_scalar.val();
	snapshot["aabb"] = m_aabb;
	snapshot["bounding_radius"] = m_bounding_radius;

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["nerf"]["rgb"]["rays_per_batch"] = m_nerf.training.counters_rgb.rays_per_batch;
		snapshot["nerf"]["rgb"]["measured_batch_size"] = m_nerf.training.counters_rgb.measured_batch_size;
		snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"] = m_nerf.training.counters_rgb.measured_batch_size_before_compaction;
		snapshot["nerf"]["dataset"] = m_nerf.training.dataset;
	}

	m_network_config_path = filepath;
	std::ofstream f(m_network_config_path.str(), std::ios::out | std::ios::binary);
	json::to_msgpack(m_network_config, f);
}

void Testbed::load_snapshot(const std::string& filepath_string) {
	auto config = load_network_config(filepath_string);
	if (!config.contains("snapshot")) {
		throw std::runtime_error{fmt::format("File {} does not contain a snapshot.", filepath_string)};
	}

	const auto& snapshot = config["snapshot"];

	if (snapshot.value("version", 0) < SNAPSHOT_FORMAT_VERSION) {
		throw std::runtime_error{"Snapshot uses an old format."};
	}

	m_aabb = snapshot.value("aabb", m_aabb);
	m_bounding_radius = snapshot.value("bounding_radius", m_bounding_radius);

	if (m_testbed_mode == ETestbedMode::Sdf) {
		set_scale(m_bounding_radius * 1.5f);
	} else if (m_testbed_mode == ETestbedMode::Nerf) {
		if (snapshot["density_grid_size"] != NERF_GRIDSIZE()) {
			throw std::runtime_error{"Incompatible grid size."};
		}

		m_nerf.training.counters_rgb.rays_per_batch = snapshot["nerf"]["rgb"]["rays_per_batch"];
		m_nerf.training.counters_rgb.measured_batch_size = snapshot["nerf"]["rgb"]["measured_batch_size"];
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"];

		// If we haven't got a nerf dataset loaded, load dataset metadata from the snapshot
		// and render using just that.
		if (m_data_path.empty() && snapshot["nerf"].contains("dataset")) {
			m_nerf.training.dataset = snapshot["nerf"]["dataset"];
			load_nerf();
		} else {
			if (snapshot["nerf"].contains("aabb_scale")) {
				m_nerf.training.dataset.aabb_scale = snapshot["nerf"]["aabb_scale"];
			}
		}

		load_nerf_post();

		GPUMemory<__half> density_grid_fp16 = snapshot["density_grid_binary"];
		m_nerf.density_grid.resize(density_grid_fp16.size());

		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid[i] = (float)density_grid_fp16[i];
		});

		if (m_nerf.density_grid.size() == NERF_GRIDSIZE() * NERF_GRIDSIZE() * NERF_GRIDSIZE() * (m_nerf.max_cascade + 1)) {
			update_density_grid_mean_and_bitfield(nullptr);
		} else if (m_nerf.density_grid.size() != 0) {
			// A size of 0 indicates that the density grid was never populated, which is a valid state of a (yet) untrained model.
			throw std::runtime_error{"Incompatible number of grid cascades."};
		}
	}

	m_network_config_path = filepath_string;
	m_network_config = config;

	reset_network(false);

	m_training_step = m_network_config["snapshot"]["training_step"];
	m_loss_scalar.set(m_network_config["snapshot"]["loss"]);

	m_trainer->deserialize(m_network_config["snapshot"]);
}

void Testbed::load_camera_path(const std::string& filepath_string) {
	m_camera_path.load(filepath_string, Matrix<float, 3, 4>::Identity());
}

bool Testbed::loop_animation() {
	return m_camera_path.m_loop;
}

void Testbed::set_loop_animation(bool value) {
	m_camera_path.m_loop = value;
}

NGP_NAMESPACE_END

