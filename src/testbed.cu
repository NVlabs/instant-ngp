#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   testbed.cu
 *  @author Thomas Müller & Alex Evans, NVIDIA
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/tinyexr_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <json/json.hpp>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <zstr.hpp>

#include <fstream>
#include <set>
#include <unordered_set>

#ifdef NGP_GUI
#  include <imgui/imgui.h>
#  include <imgui/backends/imgui_impl_glfw.h>
#  include <imgui/backends/imgui_impl_opengl3.h>
#  include <imguizmo/ImGuizmo.h>
#  ifdef _WIN32
#    include <GL/gl3w.h>
#  else
#    include <GL/glew.h>
#  endif
#  include <GLFW/glfw3.h>
#  include <GLFW/glfw3native.h>
#  include <cuda_gl_interop.h>

#endif

// Windows.h is evil
#undef min
#undef max
#undef near
#undef far


using namespace Eigen;
using namespace std::literals::chrono_literals;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

int do_system(const std::string& cmd) {
#ifdef _WIN32
	tlog::info() << "> " << cmd;
	return _wsystem(utf8_to_utf16(cmd).c_str());
#else
	tlog::info() << "$ " << cmd;
	return system(cmd.c_str());
#endif
}

std::atomic<size_t> g_total_n_bytes_allocated{0};

json merge_parent_network_config(const json& child, const fs::path& child_path) {
	if (!child.contains("parent")) {
		return child;
	}
	fs::path parent_path = child_path.parent_path() / std::string(child["parent"]);
	tlog::info() << "Loading parent network config from: " << parent_path.str();
	std::ifstream f{native_string(parent_path)};
	json parent = json::parse(f, nullptr, true, true);
	parent = merge_parent_network_config(parent, parent_path);
	parent.merge_patch(child);
	return parent;
}

std::string get_filename_in_data_path_with_suffix(fs::path data_path, fs::path network_config_path, const char* suffix) {
	// use the network config name along with the data path to build a filename with the requested suffix & extension
	std::string default_name = network_config_path.basename();
	if (default_name == "") {
		default_name = "base";
	}

	if (data_path.empty()) {
		return default_name + std::string(suffix);
	}

	if (data_path.is_directory()) {
		return (data_path / (default_name + std::string{suffix})).str();
	}

	return data_path.stem().str() + "_" + default_name + std::string(suffix);
}

void Testbed::update_imgui_paths() {
	snprintf(m_imgui.cam_path_path, sizeof(m_imgui.cam_path_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_cam.json").c_str());
	snprintf(m_imgui.extrinsics_path, sizeof(m_imgui.extrinsics_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_extrinsics.json").c_str());
	snprintf(m_imgui.mesh_path, sizeof(m_imgui.mesh_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".obj").c_str());
	snprintf(m_imgui.snapshot_path, sizeof(m_imgui.snapshot_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, ".ingp").c_str());
	snprintf(m_imgui.video_path, sizeof(m_imgui.video_path), "%s", get_filename_in_data_path_with_suffix(m_data_path, m_network_config_path, "_video.mp4").c_str());
}

void Testbed::load_training_data(const fs::path& path) {
	if (!path.exists()) {
		throw std::runtime_error{fmt::format("Data path '{}' does not exist.", path.str())};
	}

	// Automatically determine the mode from the first scene that's loaded
	ETestbedMode scene_mode = mode_from_scene(path.str());
	if (scene_mode == ETestbedMode::None) {
		throw std::runtime_error{fmt::format("Unknown scene format for path '{}'.", path.str())};
	}

	set_mode(scene_mode);

	m_data_path = path;

	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:   load_nerf(path); break;
		case ETestbedMode::Sdf:    load_mesh(path); break;
		case ETestbedMode::Image:  load_image(path); break;
		case ETestbedMode::Volume: load_volume(path); break;
		default: throw std::runtime_error{"Invalid testbed mode."};
	}

	m_training_data_available = true;

	update_imgui_paths();
}

void Testbed::reload_training_data() {
	if (m_data_path.exists()) {
		load_training_data(m_data_path.str());
	}
}

void Testbed::clear_training_data() {
	m_training_data_available = false;
	m_nerf.training.dataset.metadata.clear();
}

void Testbed::set_mode(ETestbedMode mode) {
	if (mode == m_testbed_mode) {
		return;
	}

	// Reset mode-specific members
	m_image = {};
	m_mesh = {};
	m_nerf = {};
	m_sdf = {};
	m_volume = {};

	// Kill training-related things
	m_encoding = {};
	m_loss = {};
	m_network = {};
	m_nerf_network = {};
	m_optimizer = {};
	m_trainer = {};
	m_envmap = {};
	m_distortion = {};
	m_training_data_available = false;

	// Clear device-owned data that might be mode-specific
	for (auto&& device : m_devices) {
		device.clear();
	}

	// Reset paths that might be attached to the chosen mode
	m_data_path = {};

	m_testbed_mode = mode;

	// Set various defaults depending on mode
	if (m_testbed_mode == ETestbedMode::Nerf) {
		if (m_devices.size() > 1) {
			m_use_aux_devices = true;
		}

		if (m_dlss_provider) {
			m_dlss = true;
		}
	} else {
		m_use_aux_devices = false;
		m_dlss = false;
	}

	reset_camera();

#ifdef NGP_GUI
	update_vr_performance_settings();
#endif
}

fs::path Testbed::find_network_config(const fs::path& network_config_path) {
	if (network_config_path.exists()) {
		return network_config_path;
	}

	// The following resolution steps do not work if the path is absolute. Treat it as nonexistent.
	if (network_config_path.is_absolute()) {
		return network_config_path;
	}

	fs::path candidate = root_dir()/"configs"/to_string(m_testbed_mode)/network_config_path;
	if (candidate.exists()) {
		return candidate;
	}

	return network_config_path;
}

json Testbed::load_network_config(const fs::path& network_config_path) {
	bool is_snapshot = equals_case_insensitive(network_config_path.extension(), "msgpack") || equals_case_insensitive(network_config_path.extension(), "ingp");
	if (network_config_path.empty() || !network_config_path.exists()) {
		throw std::runtime_error{fmt::format("Network {} '{}' does not exist.", is_snapshot ? "snapshot" : "config", network_config_path.str())};
	}

	tlog::info() << "Loading network " << (is_snapshot ? "snapshot" : "config") << " from: " << network_config_path;

	json result;
	if (is_snapshot) {
		std::ifstream f{native_string(network_config_path), std::ios::in | std::ios::binary};
		if (equals_case_insensitive(network_config_path.extension(), "ingp")) {
			// zstr::ifstream applies zlib compression.
			zstr::istream zf{f};
			result = json::from_msgpack(zf);
		} else {
			result = json::from_msgpack(f);
		}
		// we assume parent pointers are already resolved in snapshots.
	} else if (equals_case_insensitive(network_config_path.extension(), "json")) {
		std::ifstream f{native_string(network_config_path)};
		result = json::parse(f, nullptr, true, true);
		result = merge_parent_network_config(result, network_config_path);
	}

	return result;
}

void Testbed::reload_network_from_file(const fs::path& path) {
	if (!path.empty()) {
		fs::path candidate = find_network_config(path);
		if (candidate.exists() || !m_network_config_path.exists()) {
			// Store the path _argument_ in the member variable. E.g. for the base config,
			// it'll store `base.json`, even though the loaded config will be
			// config/<mode>/base.json. This has the benefit of switching to the
			// appropriate config when switching modes.
			m_network_config_path = path;
		}
	}

	// If the testbed mode hasn't been decided yet, don't load a network yet, but
	// still keep track of the requested config (see above).
	if (m_testbed_mode == ETestbedMode::None) {
		return;
	}

	fs::path full_network_config_path = find_network_config(m_network_config_path);
	bool is_snapshot = equals_case_insensitive(full_network_config_path.extension(), "msgpack");

	if (!full_network_config_path.exists()) {
		tlog::warning() << "Network " << (is_snapshot ? "snapshot" : "config") << " path '" << full_network_config_path << "' does not exist.";
	} else {
		m_network_config = load_network_config(full_network_config_path);
	}

	// Reset training if we haven't loaded a snapshot of an already trained model, in which case, presumably the network
	// configuration changed and the user is interested in seeing how it trains from scratch.
	if (!is_snapshot) {
		reset_network();
	}
}

void Testbed::reload_network_from_json(const json& json, const std::string& config_base_path) {
	// config_base_path is needed so that if the passed in json uses the 'parent' feature, we know where to look...
	// be sure to use a filename, or if a directory, end with a trailing slash
	m_network_config = merge_parent_network_config(json, config_base_path);
	reset_network();
}

void Testbed::load_file(const fs::path& path) {
	if (!path.exists()) {
		// If the path doesn't exist, but a network config can be resolved, load that.
		if (equals_case_insensitive(path.extension(), "json") && find_network_config(path).exists()) {
			reload_network_from_file(path);
			return;
		}

		tlog::error() << "File '" << path.str() << "' does not exist.";
		return;
	}

	if (equals_case_insensitive(path.extension(), "ingp") || equals_case_insensitive(path.extension(), "msgpack")) {
		load_snapshot(path);
		return;
	}

	// If we get a json file, we need to parse it to determine its purpose.
	if (equals_case_insensitive(path.extension(), "json")) {
		json file;
		{
			std::ifstream f{native_string(path)};
			file = json::parse(f, nullptr, true, true);
		}

		// Snapshot in json format... inefficient, but technically supported.
		if (file.contains("snapshot")) {
			load_snapshot(path);
			return;
		}

		// Regular network config
		if (file.contains("parent") || file.contains("network") || file.contains("encoding") || file.contains("loss") || file.contains("optimizer")) {
			reload_network_from_file(path);
			return;
		}

		// Camera path
		if (file.contains("path")) {
			load_camera_path(path);
			return;
		}
	}

	// If the dragged file isn't any of the above, assume that it's training data
	try {
		bool was_training_data_available = m_training_data_available;
		load_training_data(path);

		if (!was_training_data_available) {
			// If we previously didn't have any training data and only now dragged
			// some into the window, it is very unlikely that the user doesn't
			// want to immediately start training on that data. So: go for it.
			m_train = true;
		}
	} catch (std::runtime_error& e) {
		tlog::error() << "Failed to load training data: " << e.what();
	}
}

void Testbed::reset_accumulation(bool due_to_camera_movement, bool immediate_redraw) {
	if (immediate_redraw) {
		redraw_next_frame();
	}

	if (!due_to_camera_movement || !reprojection_available()) {
		m_windowless_render_surface.reset_accumulation();
		for (auto& view : m_views) {
			view.render_buffer->reset_accumulation();
		}
	}
}

void Testbed::set_visualized_dim(int dim) {
	m_visualized_dimension = dim;
	reset_accumulation();
}

void Testbed::translate_camera(const Vector3f& rel, const Matrix3f& rot, bool allow_up_down) {
	Vector3f movement = rot * rel;
	if (!allow_up_down) {
		movement -= movement.dot(m_up_dir) * m_up_dir;
	}

	m_camera.col(3) += movement;
	reset_accumulation(true);
}

void Testbed::set_nerf_camera_matrix(const Matrix<float, 3, 4>& cam) {
	m_camera = m_nerf.training.dataset.nerf_matrix_to_ngp(cam);
}

Vector3f Testbed::look_at() const {
	return view_pos() + view_dir() * m_scale;
}

void Testbed::set_look_at(const Vector3f& pos) {
	m_camera.col(3) += pos - look_at();
}

void Testbed::set_scale(float scale) {
	auto prev_look_at = look_at();
	m_camera.col(3) = (view_pos() - prev_look_at) * (scale / m_scale) + prev_look_at;
	m_scale = scale;
}

void Testbed::set_view_dir(const Vector3f& dir) {
	auto old_look_at = look_at();
	m_camera.col(0) = dir.cross(m_up_dir).normalized();
	m_camera.col(1) = dir.cross(m_camera.col(0)).normalized();
	m_camera.col(2) = dir.normalized();
	set_look_at(old_look_at);
}

void Testbed::first_training_view() {
	m_nerf.training.view = 0;
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::last_training_view() {
	m_nerf.training.view = m_nerf.training.dataset.n_images-1;
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::previous_training_view() {
	if (m_nerf.training.view != 0) {
		m_nerf.training.view -= 1;
	}
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::next_training_view() {
	if (m_nerf.training.view != m_nerf.training.dataset.n_images-1) {
		m_nerf.training.view += 1;
	}
	set_camera_to_training_view(m_nerf.training.view);
	reset_accumulation();
}

void Testbed::set_camera_to_training_view(int trainview) {
	auto old_look_at = look_at();
	m_camera = m_smoothed_camera = get_xform_given_rolling_shutter(m_nerf.training.transforms[trainview], m_nerf.training.dataset.metadata[trainview].rolling_shutter, Vector2f{0.5f, 0.5f}, 0.0f);
	m_relative_focal_length = m_nerf.training.dataset.metadata[trainview].focal_length / (float)m_nerf.training.dataset.metadata[trainview].resolution[m_fov_axis];
	m_scale = std::max((old_look_at - view_pos()).dot(view_dir()), 0.1f);
	m_nerf.render_with_lens_distortion = true;
	m_nerf.render_lens = m_nerf.training.dataset.metadata[trainview].lens;
	if (!supports_dlss(m_nerf.render_lens.mode)) {
		m_dlss = false;
	}

	m_screen_center = Vector2f::Constant(1.0f) - m_nerf.training.dataset.metadata[trainview].principal_point;
	m_nerf.training.view = trainview;
}

void Testbed::reset_camera() {
	m_fov_axis = 1;
	m_zoom = 1.0f;
	m_screen_center = Vector2f::Constant(0.5f);

	if (m_testbed_mode == ETestbedMode::Image) {
		// Make image full-screen at the given view distance
		m_relative_focal_length = Vector2f::Ones();
		m_scale = 1.0f;
	} else {
		set_fov(50.625f);
		m_scale = 1.5f;
	}

	m_camera <<
		1.0f, 0.0f, 0.0f, 0.5f,
		0.0f, -1.0f, 0.0f, 0.5f,
		0.0f, 0.0f, -1.0f, 0.5f;
	m_camera.col(3) -= m_scale * view_dir();

	m_smoothed_camera = m_camera;
	m_sun_dir = Vector3f::Ones().normalized();

	reset_accumulation();
}

void Testbed::set_train(bool mtrain) {
	if (m_train && !mtrain && m_max_level_rand_training) {
		set_max_level(1.f);
	}
	m_train = mtrain;
}

void Testbed::compute_and_save_marching_cubes_mesh(const char* filename, Vector3i res3d , BoundingBox aabb, float thresh, bool unwrap_it) {
	Matrix3f render_aabb_to_local = Matrix3f::Identity();
	if (aabb.is_empty()) {
		aabb = m_testbed_mode == ETestbedMode::Nerf ? m_render_aabb : m_aabb;
		render_aabb_to_local = m_render_aabb_to_local;
	}
	marching_cubes(res3d, aabb, render_aabb_to_local, thresh);
	save_mesh(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, filename, unwrap_it, m_nerf.training.dataset.scale, m_nerf.training.dataset.offset);
}

Eigen::Vector3i Testbed::compute_and_save_png_slices(const char* filename, int res, BoundingBox aabb, float thresh, float density_range, bool flip_y_and_z_axes) {
	Matrix3f render_aabb_to_local = Matrix3f::Identity();
	if (aabb.is_empty()) {
		aabb = m_testbed_mode == ETestbedMode::Nerf ? m_render_aabb : m_aabb;
		render_aabb_to_local = m_render_aabb_to_local;
	}
	if (thresh == std::numeric_limits<float>::max()) {
		thresh = m_mesh.thresh;
	}
	float range = density_range;
	if (m_testbed_mode == ETestbedMode::Sdf) {
		auto res3d = get_marching_cubes_res(res, aabb);
		aabb.inflate(range * aabb.diag().x()/res3d.x());
	}
	auto res3d = get_marching_cubes_res(res, aabb);
	if (m_testbed_mode == ETestbedMode::Sdf)
		range *= -aabb.diag().x()/res3d.x(); // rescale the range to be in output voxels. ie this scale factor is mapped back to the original world space distances.
			// negated so that black = outside, white = inside
	char fname[128];
	snprintf(fname, sizeof(fname), ".density_slices_%dx%dx%d.png", res3d.x(), res3d.y(), res3d.z());
	GPUMemory<float> density = (m_render_ground_truth && m_testbed_mode == ETestbedMode::Sdf) ? get_sdf_gt_on_grid(res3d, aabb, render_aabb_to_local) : get_density_on_grid(res3d, aabb, render_aabb_to_local);
	save_density_grid_to_png(density, (std::string(filename) + fname).c_str(), res3d, thresh, flip_y_and_z_axes, range);
	return res3d;
}

fs::path Testbed::root_dir() {
	if (m_root_dir.empty()) {
		m_root_dir = get_root_dir();
	}

	return m_root_dir;
}

inline float linear_to_db(float x) {
	return -10.f*logf(x)/logf(10.f);
}

template <typename T>
void Testbed::dump_parameters_as_images(const T* params, const std::string& filename_base) {
	size_t non_layer_params_width = 2048;

	size_t layer_params = 0;
	for (auto size : m_network->layer_sizes()) {
		layer_params += size.first * size.second;
	}

	size_t n_params = m_network->n_params();
	size_t n_non_layer_params = n_params - layer_params;

	std::vector<T> params_cpu_network_precision(layer_params + next_multiple(n_non_layer_params, non_layer_params_width));
	std::vector<float> params_cpu(params_cpu_network_precision.size(), 0.0f);
	CUDA_CHECK_THROW(hipMemcpy(params_cpu_network_precision.data(), params, n_params * sizeof(T), hipMemcpyDeviceToHost));

	for (size_t i = 0; i < n_params; ++i) {
		params_cpu[i] = (float)params_cpu_network_precision[i];
	}

	size_t offset = 0;
	size_t layer_id = 0;
	for (auto size : m_network->layer_sizes()) {
		save_exr(params_cpu.data() + offset, size.second, size.first, 1, 1, fmt::format("{}-layer-{}.exr", filename_base, layer_id).c_str());
		offset += size.first * size.second;
		++layer_id;
	}

	if (n_non_layer_params > 0) {
		std::string filename = fmt::format("{}-non-layer.exr", filename_base);
		save_exr(params_cpu.data() + offset, non_layer_params_width, n_non_layer_params / non_layer_params_width, 1, 1, filename.c_str());
	}
}

template void Testbed::dump_parameters_as_images<__half>(const __half*, const std::string&);
template void Testbed::dump_parameters_as_images<float>(const float*, const std::string&);

Eigen::Matrix<float, 3, 4> Testbed::crop_box(bool nerf_space) const {
	Eigen::Vector3f cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
	Eigen::Vector3f radius = m_render_aabb.diag() * 0.5f;
	Eigen::Vector3f x = m_render_aabb_to_local.row(0) * radius.x();
	Eigen::Vector3f y = m_render_aabb_to_local.row(1) * radius.y();
	Eigen::Vector3f z = m_render_aabb_to_local.row(2) * radius.z();
	Eigen::Matrix<float, 3, 4> rv;
	rv.col(0) = x;
	rv.col(1) = y;
	rv.col(2) = z;
	rv.col(3) = cen;
	if (nerf_space) {
		rv = m_nerf.training.dataset.ngp_matrix_to_nerf(rv, true);
	}
	return rv;
}

void Testbed::set_crop_box(Eigen::Matrix<float, 3, 4> m, bool nerf_space) {
	if (nerf_space) {
		m = m_nerf.training.dataset.nerf_matrix_to_ngp(m, true);
	}
	Eigen::Vector3f radius(m.col(0).norm(), m.col(1).norm(), m.col(2).norm());
	Eigen::Vector3f cen(m.col(3));
	m_render_aabb_to_local.row(0) = m.col(0) / radius.x();
	m_render_aabb_to_local.row(1) = m.col(1) / radius.y();
	m_render_aabb_to_local.row(2) = m.col(2) / radius.z();
	cen = m_render_aabb_to_local * cen;
	m_render_aabb.min = cen - radius;
	m_render_aabb.max = cen + radius;
}

std::vector<Eigen::Vector3f> Testbed::crop_box_corners(bool nerf_space) const {
	Eigen::Matrix<float, 3, 4> m = crop_box(nerf_space);
	std::vector<Eigen::Vector3f> rv(8);
	for (int i = 0; i < 8; ++i) {
		rv[i] = m * Eigen::Vector4f((i & 1) ? 1.f : -1.f, (i & 2) ? 1.f : -1.f, (i & 4) ? 1.f : -1.f, 1.f);
		/* debug print out corners to check math is all lined up */
		if (0) {
			tlog::info() << rv[i].x() << "," << rv[i].y() << "," << rv[i].z() << " [" << i << "]";
			Eigen::Vector3f mn = m_render_aabb.min;
			Eigen::Vector3f mx = m_render_aabb.max;
			Eigen::Matrix3f m = m_render_aabb_to_local.transpose();
			Eigen::Vector3f a;

			a.x() = (i&1) ? mx.x() : mn.x();
			a.y() = (i&2) ? mx.y() : mn.y();
			a.z() = (i&4) ? mx.z() : mn.z();
			a = m * a;
			if (nerf_space) {
				a = m_nerf.training.dataset.ngp_position_to_nerf(a);
			}
			tlog::info() << a.x() << "," << a.y() << "," << a.z() << " [" << i << "]";
		}
	}
	return rv;
}

#ifdef NGP_GUI
bool imgui_colored_button(const char *name, float hue) {
	ImGui::PushStyleColor(ImGuiCol_Button, (ImVec4)ImColor::HSV(hue, 0.6f, 0.6f));
	ImGui::PushStyleColor(ImGuiCol_ButtonHovered, (ImVec4)ImColor::HSV(hue, 0.7f, 0.7f));
	ImGui::PushStyleColor(ImGuiCol_ButtonActive, (ImVec4)ImColor::HSV(hue, 0.8f, 0.8f));
	bool rv = ImGui::Button(name);
	ImGui::PopStyleColor(3);
	return rv;
}

void Testbed::imgui() {
	// If a GUI interaction causes an error, write that error to the following string and call
	//   ImGui::OpenPopup("Error");
	static std::string imgui_error_string = "";

	m_picture_in_picture_res = 0;
	if (ImGui::Begin("Camera path", 0, ImGuiWindowFlags_NoScrollbar)) {
		if (ImGui::CollapsingHeader("Path manipulation", ImGuiTreeNodeFlags_DefaultOpen)) {
			if (int read = m_camera_path.imgui(
				m_imgui.cam_path_path,
				m_render_ms.val(),
				m_camera,
				m_slice_plane_z,
				m_scale,
				fov(),
				m_aperture_size,
				m_bounding_radius,
				!m_nerf.training.dataset.xforms.empty() ? m_nerf.training.dataset.xforms[0].start : Matrix<float, 3, 4>::Identity(),
				m_nerf.glow_mode,
				m_nerf.glow_y_cutoff
			)) {
				if (!m_camera_path.rendering) {
					reset_accumulation(true);

					if (m_camera_path.update_cam_from_path) {
						set_camera_from_time(m_camera_path.play_time);

						// A value of larger than 1 indicates that the camera path wants
						// to override camera smoothing.
						if (read > 1) {
							m_smoothed_camera = m_camera;
						}
					} else {
						m_pip_render_buffer->reset_accumulation();
					}
				}
			}

			if (!m_camera_path.keyframes.empty()) {
				float w = ImGui::GetContentRegionAvail().x;
				if (m_camera_path.update_cam_from_path) {
					m_picture_in_picture_res = 0;
					ImGui::Image((ImTextureID)(size_t)m_rgba_render_textures.front()->texture(), ImVec2(w, w * 9.0f / 16.0f));
				} else {
					m_picture_in_picture_res = (float)std::min((int(w)+31)&(~31), 1920/4);
					ImGui::Image((ImTextureID)(size_t)m_pip_render_texture->texture(), ImVec2(w, w * 9.0f / 16.0f));
				}
			}
		}

		if (!m_camera_path.keyframes.empty() && ImGui::CollapsingHeader("Export video", ImGuiTreeNodeFlags_DefaultOpen)) {
			// Render a video
			if (imgui_colored_button(m_camera_path.rendering ? "Abort rendering" : "Render video", 0.4)) {
				m_camera_path.rendering = !m_camera_path.rendering;

				if (!clear_tmp_dir()) {
					imgui_error_string = "Failed to clear temporary directory 'tmp' to hold rendered images.";
					ImGui::OpenPopup("Error");

					m_camera_path.rendering = false;
				}

				if (m_camera_path.rendering) {
					m_camera_path.render_start_time = std::chrono::steady_clock::now();
					m_camera_path.update_cam_from_path = true;
					m_camera_path.play_time = 0.0f;
					m_camera_path.auto_play_speed = 1.0f;
					m_camera_path.render_frame_idx = 0;

					m_dlss = false;
					m_train = false;

					reset_accumulation(true);
					set_camera_from_time(m_camera_path.play_time);
					m_smoothed_camera = m_camera;
				} else {
					m_camera_path.update_cam_from_path = false;
					m_camera_path.play_time = 0.0f;
					m_camera_path.auto_play_speed = 0.0f;
				}
			}

			if (m_camera_path.rendering) {
				ImGui::SameLine();

				auto elapsed = std::chrono::steady_clock::now() - m_camera_path.render_start_time;

				uint32_t progress = m_camera_path.render_frame_idx * m_camera_path.render_settings.spp + m_views.front().render_buffer->spp();
				uint32_t goal = m_camera_path.render_settings.n_frames() * m_camera_path.render_settings.spp;
				auto est_remaining = elapsed * (float)(goal - progress) / std::max(progress, 1u);

				ImGui::Text("%s", fmt::format(
					"Frame {}/{}, Elapsed: {}, Remaining: {}",
					m_camera_path.render_frame_idx+1,
					m_camera_path.render_settings.n_frames(),
					tlog::durationToString(std::chrono::steady_clock::now() - m_camera_path.render_start_time),
					tlog::durationToString(est_remaining)
				).c_str());
			}

			if (m_camera_path.rendering) { ImGui::BeginDisabled(); }

			ImGui::InputText("File##Video file path", m_imgui.video_path, sizeof(m_imgui.video_path));
			m_camera_path.render_settings.filename = m_imgui.video_path;

			ImGui::InputInt2("Resolution", &m_camera_path.render_settings.resolution.x());
			ImGui::InputFloat("Duration (seconds)", &m_camera_path.render_settings.duration_seconds);
			ImGui::InputFloat("FPS (frames/second)", &m_camera_path.render_settings.fps);
			ImGui::InputInt("SPP (samples/pixel)", &m_camera_path.render_settings.spp);
			ImGui::SliderInt("Quality", &m_camera_path.render_settings.quality, 0, 10);

			ImGui::SliderFloat("Shutter fraction", &m_camera_path.render_settings.shutter_fraction, 0.0f, 1.0f);

			if (m_camera_path.rendering) { ImGui::EndDisabled(); }
		}
	}
	ImGui::End();


	ImGui::Begin("instant-ngp v" NGP_VERSION);

	size_t n_bytes = tcnn::total_n_bytes_allocated() + g_total_n_bytes_allocated;
	if (m_dlss_provider) {
		n_bytes += m_dlss_provider->allocated_bytes();
	}

	ImGui::Text("Frame: %.2f ms (%.1f FPS); Mem: %s", m_frame_ms.ema_val(), 1000.0f / m_frame_ms.ema_val(), bytes_to_string(n_bytes).c_str());
	bool accum_reset = false;

	if (!m_training_data_available) { ImGui::BeginDisabled(); }

	if (ImGui::CollapsingHeader("Training", m_training_data_available ? ImGuiTreeNodeFlags_DefaultOpen : 0)) {
		if (imgui_colored_button(m_train ? "Stop training" : "Start training", 0.4)) {
			set_train(!m_train);
		}


		ImGui::SameLine();
		if (imgui_colored_button("Reset training", 0.f)) {
			reload_network_from_file();
		}

		ImGui::SameLine();
		ImGui::Checkbox("encoding", &m_train_encoding);
		ImGui::SameLine();
		ImGui::Checkbox("network", &m_train_network);
		ImGui::SameLine();
		ImGui::Checkbox("rand levels", &m_max_level_rand_training);
		if (m_testbed_mode == ETestbedMode::Nerf) {
			ImGui::Checkbox("envmap", &m_nerf.training.train_envmap);
			ImGui::SameLine();
			ImGui::Checkbox("extrinsics", &m_nerf.training.optimize_extrinsics);
			ImGui::SameLine();
			ImGui::Checkbox("exposure", &m_nerf.training.optimize_exposure);
			ImGui::SameLine();
			ImGui::Checkbox("distortion", &m_nerf.training.optimize_distortion);

			if (m_nerf.training.dataset.n_extra_learnable_dims) {
				ImGui::SameLine();
				ImGui::Checkbox("latents", &m_nerf.training.optimize_extra_dims);
			}


			static bool export_extrinsics_in_quat_format = true;
			static bool extrinsics_have_been_optimized = false;

			if (m_nerf.training.optimize_extrinsics) {
				extrinsics_have_been_optimized = true;
			}

			if (extrinsics_have_been_optimized) {
				if (imgui_colored_button("Export extrinsics", 0.4f)) {
					m_nerf.training.export_camera_extrinsics(m_imgui.extrinsics_path, export_extrinsics_in_quat_format);
				}

				ImGui::SameLine();
				ImGui::Checkbox("as quaternions", &export_extrinsics_in_quat_format);
				ImGui::InputText("File##Extrinsics file path", m_imgui.extrinsics_path, sizeof(m_imgui.extrinsics_path));
			}
		}

		ImGui::PushItemWidth(ImGui::GetWindowWidth() * 0.3f);
		ImGui::SliderInt("Batch size", (int*)&m_training_batch_size, 1 << 12, 1 << 22, "%d", ImGuiSliderFlags_Logarithmic);
		ImGui::SameLine();
		ImGui::DragInt("Seed", (int*)&m_seed, 1.0f, 0, std::numeric_limits<int>::max());
		ImGui::PopItemWidth();

		m_training_batch_size = next_multiple(m_training_batch_size, batch_size_granularity);

		if (m_train) {
			std::vector<std::string> timings;
			if (m_testbed_mode == ETestbedMode::Nerf) {
				timings.emplace_back(fmt::format("Grid: {:.01f}ms", m_training_prep_ms.ema_val()));
			} else {
				timings.emplace_back(fmt::format("Datagen: {:.01f}ms", m_training_prep_ms.ema_val()));
			}

			timings.emplace_back(fmt::format("Training: {:.01f}ms", m_training_ms.ema_val()));
			ImGui::Text("%s", join(timings, ", ").c_str());
		} else {
			ImGui::Text("Training paused");
		}

		if (m_testbed_mode == ETestbedMode::Nerf) {
			ImGui::Text("Rays/batch: %d, Samples/ray: %.2f, Batch size: %d/%d", m_nerf.training.counters_rgb.rays_per_batch, (float)m_nerf.training.counters_rgb.measured_batch_size / (float)m_nerf.training.counters_rgb.rays_per_batch, m_nerf.training.counters_rgb.measured_batch_size, m_nerf.training.counters_rgb.measured_batch_size_before_compaction);
		}

		float elapsed_training = std::chrono::duration<float>(std::chrono::steady_clock::now() - m_training_start_time_point).count();
		ImGui::Text("Steps: %d, Loss: %0.6f (%0.2f dB), Elapsed: %.1fs", m_training_step, m_loss_scalar.ema_val(), linear_to_db(m_loss_scalar.ema_val()), elapsed_training);
		ImGui::PlotLines("loss graph", m_loss_graph.data(), std::min(m_loss_graph_samples, m_loss_graph.size()), (m_loss_graph_samples < m_loss_graph.size()) ? 0 : (m_loss_graph_samples % m_loss_graph.size()), 0, FLT_MAX, FLT_MAX, ImVec2(0, 50.f));

		if (m_testbed_mode == ETestbedMode::Nerf && ImGui::TreeNode("NeRF training options")) {
			ImGui::Checkbox("Random bg color", &m_nerf.training.random_bg_color);
			ImGui::SameLine();
			ImGui::Checkbox("Snap to pixel centers", &m_nerf.training.snap_to_pixel_centers);
			ImGui::SliderFloat("Near distance", &m_nerf.training.near_distance, 0.0f, 1.0f);
			accum_reset |= ImGui::Checkbox("Linear colors", &m_nerf.training.linear_colors);
			ImGui::Combo("Loss", (int*)&m_nerf.training.loss_type, LossTypeStr);
			ImGui::Combo("Depth Loss", (int*)&m_nerf.training.depth_loss_type, LossTypeStr);
			ImGui::Combo("RGB activation", (int*)&m_nerf.rgb_activation, NerfActivationStr);
			ImGui::Combo("Density activation", (int*)&m_nerf.density_activation, NerfActivationStr);
			ImGui::SliderFloat("Cone angle", &m_nerf.cone_angle_constant, 0.0f, 1.0f/128.0f);
			ImGui::SliderFloat("Depth supervision strength", &m_nerf.training.depth_supervision_lambda, 0.f, 1.f);

			// Importance sampling options, but still related to training
			ImGui::Checkbox("Sample focal plane ~error", &m_nerf.training.sample_focal_plane_proportional_to_error);
			ImGui::SameLine();
			ImGui::Checkbox("Sample focal plane ~sharpness", &m_nerf.training.include_sharpness_in_error);
			ImGui::Checkbox("Sample image ~error", &m_nerf.training.sample_image_proportional_to_error);
			ImGui::Text("%dx%d error res w/ %d steps between updates", m_nerf.training.error_map.resolution.x(), m_nerf.training.error_map.resolution.y(), m_nerf.training.n_steps_between_error_map_updates);
			ImGui::Checkbox("Display error overlay", &m_nerf.training.render_error_overlay);
			if (m_nerf.training.render_error_overlay) {
				ImGui::SliderFloat("Error overlay brightness", &m_nerf.training.error_overlay_brightness, 0.f, 1.f);
			}
			ImGui::SliderFloat("Density grid decay", &m_nerf.training.density_grid_decay, 0.f, 1.f,"%.4f");
			ImGui::SliderFloat("Extrinsic L2 reg.", &m_nerf.training.extrinsic_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::SliderFloat("Intrinsic L2 reg.", &m_nerf.training.intrinsic_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::SliderFloat("Exposure L2 reg.", &m_nerf.training.exposure_l2_reg, 1e-8f, 0.1f, "%.6f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Sdf && ImGui::TreeNode("SDF training options")) {
			accum_reset |= ImGui::Checkbox("Use octree for acceleration", &m_sdf.use_triangle_octree);
			accum_reset |= ImGui::Combo("Mesh SDF mode", (int*)&m_sdf.mesh_sdf_mode, MeshSdfModeStr);

			accum_reset |= ImGui::SliderFloat("Surface offset scale", &m_sdf.training.surface_offset_scale, 0.125f, 1024.0f, "%.4f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);

			if (ImGui::Checkbox("Calculate IoU", &m_sdf.calculate_iou_online)) {
				m_sdf.iou_decay = 0;
			}

			ImGui::SameLine();
			ImGui::Text("%0.6f", m_sdf.iou);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Image && ImGui::TreeNode("Image training options")) {
			ImGui::Combo("Training coords", (int*)&m_image.random_mode, RandomModeStr);
			ImGui::Checkbox("Snap to pixel centers", &m_image.training.snap_to_pixel_centers);
			accum_reset |= ImGui::Checkbox("Linear colors", &m_image.training.linear_colors);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Volume && ImGui::TreeNode("Volume training options")) {
			accum_reset |= ImGui::SliderFloat("Albedo", &m_volume.albedo, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Scattering", &m_volume.scattering, -2.f, 2.f);
			accum_reset |= ImGui::SliderFloat("Distance scale", &m_volume.inv_distance_scale, 1.f, 100.f, "%.3g", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}
	}

	if (!m_training_data_available) { ImGui::EndDisabled(); }

	if (ImGui::CollapsingHeader("Rendering", ImGuiTreeNodeFlags_DefaultOpen)) {
		if (!m_hmd) {
			if (ImGui::Button("Connect to VR/AR headset")) {
				try {
					init_vr();
				} catch (const std::runtime_error& e) {
					imgui_error_string = e.what();
					ImGui::OpenPopup("Error");
				}
			}
		} else {
			if (ImGui::Button("Disconnect from VR/AR headset")) {
				m_hmd.reset();
				m_vr_frame_info = nullptr;
				update_vr_performance_settings();
			} else if (ImGui::TreeNodeEx("VR/AR settings", ImGuiTreeNodeFlags_DefaultOpen)) {
				static int blend_mode_idx = 0;
				const auto& supported_blend_modes = m_hmd->supported_environment_blend_modes();
				if (supported_blend_modes.size() > 1) {
					if (ImGui::Combo("Environment", &blend_mode_idx, m_hmd->supported_environment_blend_modes_imgui_string())) {
						auto b = m_hmd->supported_environment_blend_modes().at(blend_mode_idx);
						m_hmd->set_environment_blend_mode(b);
						update_vr_performance_settings();
					}
				}

				if (m_devices.size() > 1 && m_testbed_mode == ETestbedMode::Nerf) {
					ImGui::Checkbox("Multi-GPU rendering (one per eye)", &m_use_aux_devices);
				}

				accum_reset |= ImGui::Checkbox("Depth-based reprojection", &m_vr_depth_reproject);
				accum_reset |= ImGui::Checkbox("Foveated rendering", &m_foveated_rendering) && !m_dlss;
				if (m_foveated_rendering) {
					accum_reset |= ImGui::SliderFloat("Maximum foveation", &m_foveated_rendering_max_scaling, 1.0f, 16.0f, "%.01f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat) && !m_dlss;
				}

				ImGui::TreePop();
			}
		}

		ImGui::Checkbox("Render", &m_render);
		ImGui::SameLine();

		const auto& render_buffer = m_views.front().render_buffer;
		std::string spp_string = m_dlss ? std::string{""} : fmt::format("({} spp)", std::max(render_buffer->spp(), 1u));
		ImGui::Text(": %.01fms for %dx%d %s", m_render_ms.ema_val(), render_buffer->in_resolution().x(), render_buffer->in_resolution().y(), spp_string.c_str());

		ImGui::SameLine();
		if (ImGui::Checkbox("VSync", &m_vsync)) {
			glfwSwapInterval(m_vsync ? 1 : 0);
		}

		if (!m_dlss_provider) { ImGui::BeginDisabled(); }
		accum_reset |= ImGui::Checkbox("DLSS", &m_dlss);

		if (render_buffer->dlss()) {
			ImGui::SameLine();
			ImGui::Text("(%s)", DlssQualityStrArray[(int)render_buffer->dlss()->quality()]);
			ImGui::SameLine();
			ImGui::PushItemWidth(ImGui::GetWindowWidth() * 0.3f);
			ImGui::SliderFloat("Sharpening", &m_dlss_sharpening, 0.0f, 1.0f, "%.02f");
			ImGui::PopItemWidth();
		}

		if (!m_dlss_provider) {
			ImGui::SameLine();
#ifdef NGP_VULKAN
			ImGui::Text("(unsupported on this system)");
#else
			ImGui::Text("(Vulkan was missing at compilation time)");
#endif
			ImGui::EndDisabled();
		}

		ImGui::Checkbox("Dynamic resolution", &m_dynamic_res);
		ImGui::SameLine();
		ImGui::PushItemWidth(ImGui::GetWindowWidth() * 0.3f);
		if (m_dynamic_res) {
			ImGui::SliderFloat("Target FPS", &m_dynamic_res_target_fps, 2.0f, 144.0f, "%.01f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
		} else {
			ImGui::SliderInt("Resolution factor", &m_fixed_res_factor, 8, 64);
		}
		ImGui::PopItemWidth();

		accum_reset |= ImGui::Combo("Render mode", (int*)&m_render_mode, RenderModeStr);
		accum_reset |= ImGui::Combo("Tonemap curve", (int*)&m_tonemap_curve, TonemapCurveStr);
		accum_reset |= ImGui::ColorEdit4("Background", &m_background_color[0]);

		if (ImGui::SliderFloat("Exposure", &m_exposure, -5.f, 5.f)) {
			set_exposure(m_exposure);
		}

		float max_diam = (m_aabb.max-m_aabb.min).maxCoeff();
		float render_diam = (m_render_aabb.max-m_render_aabb.min).maxCoeff();
		float old_render_diam = render_diam;

		if (m_testbed_mode == ETestbedMode::Nerf || m_testbed_mode == ETestbedMode::Volume) {
			if (ImGui::SliderFloat("Crop size", &render_diam, 0.1f, max_diam, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat)) {
				accum_reset = true;
				if (old_render_diam > 0.f && render_diam > 0.f) {
					const Vector3f center = (m_render_aabb.max + m_render_aabb.min) * 0.5f;
					float scale = render_diam / old_render_diam;
					m_render_aabb.max = ((m_render_aabb.max-center) * scale + center).cwiseMin(m_aabb.max);
					m_render_aabb.min = ((m_render_aabb.min-center) * scale + center).cwiseMax(m_aabb.min);
				}
			}
		}

		std::string transform_section_name = "World transform";
		if (m_testbed_mode == ETestbedMode::Nerf) {
			transform_section_name += " & Crop box";
		}

		if (ImGui::TreeNode(transform_section_name.c_str())) {
			m_edit_render_aabb = true;

			if (ImGui::RadioButton("Translate world", m_camera_path.m_gizmo_op == ImGuizmo::TRANSLATE && m_edit_world_transform)) {
				m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
				m_edit_world_transform = true;
			}

			ImGui::SameLine();
			if (ImGui::RadioButton("Rotate world", m_camera_path.m_gizmo_op == ImGuizmo::ROTATE && m_edit_world_transform)) {
				m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
				m_edit_world_transform = true;
			}

			if (m_testbed_mode == ETestbedMode::Nerf) {
				if (ImGui::RadioButton("Translate crop box", m_camera_path.m_gizmo_op == ImGuizmo::TRANSLATE && !m_edit_world_transform)) {
					m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
					m_edit_world_transform = false;
				}

				ImGui::SameLine();
				if (ImGui::RadioButton("Rotate crop box", m_camera_path.m_gizmo_op == ImGuizmo::ROTATE && !m_edit_world_transform)) {
					m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
					m_edit_world_transform = false;
				}

				accum_reset |= ImGui::SliderFloat("Min x", ((float*)&m_render_aabb.min)+0, m_aabb.min.x(), m_render_aabb.max.x(), "%.3f");
				accum_reset |= ImGui::SliderFloat("Min y", ((float*)&m_render_aabb.min)+1, m_aabb.min.y(), m_render_aabb.max.y(), "%.3f");
				accum_reset |= ImGui::SliderFloat("Min z", ((float*)&m_render_aabb.min)+2, m_aabb.min.z(), m_render_aabb.max.z(), "%.3f");
				ImGui::Separator();
				accum_reset |= ImGui::SliderFloat("Max x", ((float*)&m_render_aabb.max)+0, m_render_aabb.min.x(), m_aabb.max.x(), "%.3f");
				accum_reset |= ImGui::SliderFloat("Max y", ((float*)&m_render_aabb.max)+1, m_render_aabb.min.y(), m_aabb.max.y(), "%.3f");
				accum_reset |= ImGui::SliderFloat("Max z", ((float*)&m_render_aabb.max)+2, m_render_aabb.min.z(), m_aabb.max.z(), "%.3f");
				ImGui::Separator();
				Vector3f diag = m_render_aabb.diag();
				bool edit_diag = false;
				float max_diag = m_aabb.diag().maxCoeff();
				edit_diag |= ImGui::SliderFloat("Size x", ((float*)&diag)+0, 0.001f, max_diag, "%.3f");
				edit_diag |= ImGui::SliderFloat("Size y", ((float*)&diag)+1, 0.001f, max_diag, "%.3f");
				edit_diag |= ImGui::SliderFloat("Size z", ((float*)&diag)+2, 0.001f, max_diag, "%.3f");
				if (edit_diag) {
					accum_reset = true;
					Vector3f cen = m_render_aabb.center();
					m_render_aabb = BoundingBox(cen - diag * 0.5f, cen + diag * 0.5f);
				}

				if (ImGui::Button("Reset crop box")) {
					accum_reset = true;
					m_render_aabb = m_aabb;
					m_render_aabb_to_local = Matrix3f::Identity();
				}

				ImGui::SameLine();
				if (ImGui::Button("rotation only")) {
					accum_reset = true;
					Eigen::Vector3f world_cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
					m_render_aabb_to_local = Matrix3f::Identity();
					Eigen::Vector3f new_cen = m_render_aabb_to_local * world_cen;
					Eigen::Vector3f old_cen = m_render_aabb.center();
					m_render_aabb.min += new_cen - old_cen;
					m_render_aabb.max += new_cen - old_cen;
				}
			}

			ImGui::TreePop();
		} else {
			m_edit_render_aabb = false;
		}

		if (ImGui::TreeNode("Advanced rendering options")) {
			ImGui::SliderInt("Max spp", &m_max_spp, 0, 1024, "%d", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			accum_reset |= ImGui::Checkbox("Render transparency as checkerboard", &m_render_transparency_as_checkerboard);
			accum_reset |= ImGui::Combo("Color space", (int*)&m_color_space, ColorSpaceStr);
			accum_reset |= ImGui::Checkbox("Snap to pixel centers", &m_snap_to_pixel_centers);

			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Nerf && ImGui::TreeNode("NeRF rendering options")) {
			if (m_nerf.training.dataset.has_light_dirs) {
				Vector3f light_dir = m_nerf.light_dir.normalized();
				if (ImGui::TreeNodeEx("Light Dir (Polar)", ImGuiTreeNodeFlags_DefaultOpen)) {
					float phi = atan2f(m_nerf.light_dir.x(), m_nerf.light_dir.z());
					float theta = asinf(m_nerf.light_dir.y());
					bool spin = ImGui::SliderFloat("Light Dir Theta", &theta, -PI() / 2.0f, PI() / 2.0f);
					spin |= ImGui::SliderFloat("Light Dir Phi", &phi, -PI(), PI());
					if (spin) {
						float sin_phi, cos_phi;
						sincosf(phi, &sin_phi, &cos_phi);
						float cos_theta=cosf(theta);
						m_nerf.light_dir = {sin_phi * cos_theta,sinf(theta),cos_phi * cos_theta};
						accum_reset = true;
					}
					ImGui::TreePop();
				}

				if (ImGui::TreeNode("Light Dir (Cartesian)")) {
					accum_reset |= ImGui::SliderFloat("Light Dir X", ((float*)(&m_nerf.light_dir)) + 0, -1.0f, 1.0f);
					accum_reset |= ImGui::SliderFloat("Light Dir Y", ((float*)(&m_nerf.light_dir)) + 1, -1.0f, 1.0f);
					accum_reset |= ImGui::SliderFloat("Light Dir Z", ((float*)(&m_nerf.light_dir)) + 2, -1.0f, 1.0f);
					ImGui::TreePop();
				}
			}

			if (m_nerf.training.dataset.n_extra_learnable_dims) {
				accum_reset |= ImGui::SliderInt("training image latent code for inference", (int*)&m_nerf.extra_dim_idx_for_inference, 0, m_nerf.training.dataset.n_images-1);
			}

			accum_reset |= ImGui::Combo("Groundtruth render mode", (int*)&m_ground_truth_render_mode, GroundTruthRenderModeStr);
			accum_reset |= ImGui::SliderFloat("Groundtruth alpha", &m_ground_truth_alpha, 0.0f, 1.0f, "%.02f", ImGuiSliderFlags_AlwaysClamp);

			bool lens_changed = ImGui::Checkbox("Apply lens distortion", &m_nerf.render_with_lens_distortion);

			if (m_nerf.render_with_lens_distortion) {
				lens_changed |= ImGui::Combo("Lens mode", (int*)&m_nerf.render_lens.mode, LensModeStr);
				if (m_nerf.render_lens.mode == ELensMode::OpenCV) {
					accum_reset |= ImGui::InputFloat("k1", &m_nerf.render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k2", &m_nerf.render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p1", &m_nerf.render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("p2", &m_nerf.render_lens.params[3], 0.f, 0.f, "%.5f");
				} else if (m_nerf.render_lens.mode == ELensMode::OpenCVFisheye) {
					accum_reset |= ImGui::InputFloat("k1", &m_nerf.render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k2", &m_nerf.render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k3", &m_nerf.render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("k4", &m_nerf.render_lens.params[3], 0.f, 0.f, "%.5f");
				} else if (m_nerf.render_lens.mode == ELensMode::FTheta) {
					accum_reset |= ImGui::InputFloat("width", &m_nerf.render_lens.params[5], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("height", &m_nerf.render_lens.params[6], 0.f, 0.f, "%.0f");
					accum_reset |= ImGui::InputFloat("f_theta p0", &m_nerf.render_lens.params[0], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p1", &m_nerf.render_lens.params[1], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p2", &m_nerf.render_lens.params[2], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p3", &m_nerf.render_lens.params[3], 0.f, 0.f, "%.5f");
					accum_reset |= ImGui::InputFloat("f_theta p4", &m_nerf.render_lens.params[4], 0.f, 0.f, "%.5f");
				}

				if (lens_changed && !supports_dlss(m_nerf.render_lens.mode)) {
					m_dlss = false;
				}

				accum_reset |= lens_changed;
			}

			accum_reset |= ImGui::SliderFloat("Min transmittance", &m_nerf.render_min_transmittance, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Sdf && ImGui::TreeNode("SDF rendering options")) {
			accum_reset |= ImGui::Combo("Ground Truth Rendering Mode", (int*)&m_sdf.groundtruth_mode,
				"Raytraced Mesh\0"
				"Sphere Traced Mesh\0"
				"SDF Bricks\0"
			);

			if (m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
				accum_reset |= ImGui::SliderInt("Brick octree Level", (int*)&m_sdf.brick_level, 1, 10);
				accum_reset |= ImGui::Checkbox("Brick normals track octree Level", &m_sdf.brick_smooth_normals);
				accum_reset |= ImGui::SliderInt("Brick quantize Bits", (int*)&m_sdf.brick_quantise_bits, 0, 16);
			}

			accum_reset |= ImGui::Checkbox("Analytic normals", &m_sdf.analytic_normals);
			accum_reset |= ImGui::Checkbox("Floor", &m_floor_enable);

			accum_reset |= ImGui::SliderFloat("Normals epsilon", &m_sdf.fd_normals_epsilon, 0.00001f, 0.1f, "%.6g", ImGuiSliderFlags_Logarithmic);
			accum_reset |= ImGui::SliderFloat("Maximum distance", &m_sdf.maximum_distance, 0.00001f, 0.1f, "%.6g", ImGuiSliderFlags_Logarithmic);
			accum_reset |= ImGui::SliderFloat("Shadow sharpness", &m_sdf.shadow_sharpness, 0.1f, 2048.0f, "%.6g", ImGuiSliderFlags_Logarithmic);

			accum_reset |= ImGui::SliderFloat("Inflate (offset the zero set)", &m_sdf.zero_offset, -0.25f, 0.25f);
			accum_reset |= ImGui::SliderFloat("Distance scale", &m_sdf.distance_scale, 0.25f, 1.f);

			ImGui::TreePop();
		}

		if (m_testbed_mode == ETestbedMode::Image && ImGui::TreeNode("Image rendering options")) {
			static bool quantize_to_byte = false;
			static float mse = 0.0f;

			if (imgui_colored_button("Compute PSNR", 0.4)) {
				mse = compute_image_mse(quantize_to_byte);
			}

			float psnr = -10.0f * std::log(mse) / std::log(10.0f);

			ImGui::SameLine();
			ImGui::Text("%0.6f", psnr);
			ImGui::SameLine();
			ImGui::Checkbox("Quantize", &quantize_to_byte);

			ImGui::TreePop();
		}

		if (ImGui::TreeNode("Debug visualization")) {
			ImGui::Checkbox("Visualize unit cube", &m_visualize_unit_cube);
			if (m_testbed_mode == ETestbedMode::Nerf) {
				ImGui::SameLine();
				ImGui::Checkbox("Visualize cameras", &m_nerf.visualize_cameras);
				accum_reset |= ImGui::SliderInt("Show acceleration", &m_nerf.show_accel, -1, 7);
			}

			if (!m_single_view) { ImGui::BeginDisabled(); }
			if (ImGui::SliderInt("Visualized dimension", &m_visualized_dimension, -1, (int)network_width(m_visualized_layer)-1)) {
				set_visualized_dim(m_visualized_dimension);
			}
			if (!m_single_view) { ImGui::EndDisabled(); }

			if (ImGui::SliderInt("Visualized layer", &m_visualized_layer, 0, (int)network_num_forward_activations()-1)) {
				set_visualized_layer(m_visualized_layer);
			}
			if (ImGui::Checkbox("Single view", &m_single_view)) {
				set_visualized_dim(-1);
				accum_reset = true;
			}

			if (m_testbed_mode == ETestbedMode::Nerf) {
				if (ImGui::Button("First")) {
					first_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Previous")) {
					previous_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Next")) {
					next_training_view();
				}
				ImGui::SameLine();
				if (ImGui::Button("Last")) {
					last_training_view();
				}
				ImGui::SameLine();
				ImGui::Text("%s", m_nerf.training.dataset.paths.at(m_nerf.training.view).c_str());

				if (ImGui::SliderInt("Training view", &m_nerf.training.view, 0, (int)m_nerf.training.dataset.n_images-1)) {
					set_camera_to_training_view(m_nerf.training.view);
					accum_reset = true;
				}
				ImGui::PlotLines("Training view error", m_nerf.training.error_map.pmf_img_cpu.data(), m_nerf.training.error_map.pmf_img_cpu.size(), 0, nullptr, 0.0f, FLT_MAX, ImVec2(0, 60.f));

				if (m_nerf.training.optimize_exposure) {
					std::vector<float> exposures(m_nerf.training.dataset.n_images);
					for (uint32_t i = 0; i < m_nerf.training.dataset.n_images; ++i) {
						exposures[i] = m_nerf.training.cam_exposure[i].variable().x();
					}

					ImGui::PlotLines("Training view exposures", exposures.data(), exposures.size(), 0, nullptr, FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
				}

				if (ImGui::SliderInt("Glow mode", &m_nerf.glow_mode, 0, 16)) {
					accum_reset = true;
				}

				if (m_nerf.glow_mode && ImGui::SliderFloat("Glow height", &m_nerf.glow_y_cutoff, -2.f, 3.f)) {
					accum_reset = true;
				}
			}

			ImGui::TreePop();
		}
	}

	if (ImGui::CollapsingHeader("Camera", ImGuiTreeNodeFlags_DefaultOpen)) {
		ImGui::Checkbox("First person controls", &m_fps_camera);
		ImGui::SameLine();
		ImGui::Checkbox("Smooth motion", &m_camera_smoothing);
		ImGui::SameLine();
		ImGui::Checkbox("Autofocus", &m_autofocus);
		ImGui::PushItemWidth(ImGui::GetWindowWidth() * 0.3f);
		if (ImGui::SliderFloat("Aperture size", &m_aperture_size, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat)) {
			m_dlss = false;
			accum_reset = true;
		}
		ImGui::SameLine();
		accum_reset |= ImGui::SliderFloat("Focus depth", &m_slice_plane_z, -m_bounding_radius, m_bounding_radius);

		float local_fov = fov();
		if (ImGui::SliderFloat("Field of view", &local_fov, 0.0f, 120.0f)) {
			set_fov(local_fov);
			accum_reset = true;
		}
		ImGui::SameLine();
		accum_reset |= ImGui::SliderFloat("Zoom", &m_zoom, 1.f, 10.f);
		ImGui::PopItemWidth();



		if (ImGui::TreeNode("Advanced camera settings")) {
			accum_reset |= ImGui::SliderFloat2("Screen center", &m_screen_center.x(), 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat2("Parallax shift", &m_parallax_shift.x(), -1.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Slice / focus depth", &m_slice_plane_z, -m_bounding_radius, m_bounding_radius);
			accum_reset |= ImGui::SliderFloat("Render near distance", &m_render_near_distance, 0.0f, 1.0f, "%.3f", ImGuiSliderFlags_Logarithmic | ImGuiSliderFlags_NoRoundToFormat);
			char buf[2048];
			Vector3f v = view_dir();
			Vector3f p = look_at();
			Vector3f s = m_sun_dir;
			Vector3f u = m_up_dir;
			Array4f b = m_background_color;
			snprintf(buf, sizeof(buf),
				"testbed.background_color = [%0.3f, %0.3f, %0.3f, %0.3f]\n"
				"testbed.exposure = %0.3f\n"
				"testbed.sun_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.up_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.view_dir = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.look_at = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.scale = %0.3f\n"
				"testbed.fov,testbed.aperture_size,testbed.slice_plane_z = %0.3f,%0.3f,%0.3f\n"
				"testbed.autofocus_target = [%0.3f,%0.3f,%0.3f]\n"
				"testbed.autofocus = %s\n\n"
				, b.x(), b.y(), b.z(), b.w()
				, m_exposure
				, s.x(), s.y(), s.z()
				, u.x(), u.y(), u.z()
				, v.x(), v.y(), v.z()
				, p.x(), p.y(), p.z()
				, scale()
				, fov(), m_aperture_size, m_slice_plane_z
				, m_autofocus_target.x(), m_autofocus_target.y(), m_autofocus_target.z()
				, m_autofocus ? "True" : "False"
			);

			if (m_testbed_mode == ETestbedMode::Sdf) {
				size_t n = strlen(buf);
				snprintf(buf + n, sizeof(buf) - n,
					"testbed.sdf.shadow_sharpness = %0.3f\n"
					"testbed.sdf.analytic_normals = %s\n"
					"testbed.sdf.use_triangle_octree = %s\n\n"
					"testbed.sdf.brdf.metallic = %0.3f\n"
					"testbed.sdf.brdf.subsurface = %0.3f\n"
					"testbed.sdf.brdf.specular = %0.3f\n"
					"testbed.sdf.brdf.roughness = %0.3f\n"
					"testbed.sdf.brdf.sheen = %0.3f\n"
					"testbed.sdf.brdf.clearcoat = %0.3f\n"
					"testbed.sdf.brdf.clearcoat_gloss = %0.3f\n"
					"testbed.sdf.brdf.basecolor = [%0.3f,%0.3f,%0.3f]\n\n"
					, m_sdf.shadow_sharpness
					, m_sdf.analytic_normals ? "True" : "False"
					, m_sdf.use_triangle_octree ? "True" : "False"
					, m_sdf.brdf.metallic
					, m_sdf.brdf.subsurface
					, m_sdf.brdf.specular
					, m_sdf.brdf.roughness
					, m_sdf.brdf.sheen
					, m_sdf.brdf.clearcoat
					, m_sdf.brdf.clearcoat_gloss
					, m_sdf.brdf.basecolor.x()
					, m_sdf.brdf.basecolor.y()
					, m_sdf.brdf.basecolor.z()
				);
			}
			ImGui::InputTextMultiline("Params", buf, sizeof(buf));
			ImGui::TreePop();
		}
	}

	if (ImGui::CollapsingHeader("Snapshot", ImGuiTreeNodeFlags_DefaultOpen)) {
		ImGui::Text("Snapshot");
		ImGui::SameLine();
		if (ImGui::Button("Save")) {
			save_snapshot(m_imgui.snapshot_path, m_include_optimizer_state_in_snapshot, m_compress_snapshot);
		}
		ImGui::SameLine();
		if (ImGui::Button("Load")) {
			try {
				load_snapshot(m_imgui.snapshot_path);
			} catch (std::exception& e) {
				imgui_error_string = fmt::format("Failed to load snapshot: {}", e.what());
				ImGui::OpenPopup("Error");
			}
		}
		ImGui::SameLine();
		if (ImGui::Button("Dump parameters as images")) {
			dump_parameters_as_images(m_trainer->params(), "params");
		}

		ImGui::SameLine();
		ImGui::Checkbox("w/ optimizer state", &m_include_optimizer_state_in_snapshot);
		ImGui::InputText("File##Snapshot file path", m_imgui.snapshot_path, sizeof(m_imgui.snapshot_path));
		ImGui::SameLine();

		bool can_compress = ends_with_case_insensitive(m_imgui.snapshot_path, ".ingp");

		if (!can_compress) {
			ImGui::BeginDisabled();
			m_compress_snapshot = false;
		}
		ImGui::Checkbox("Compress", &m_compress_snapshot);
		if (!can_compress) ImGui::EndDisabled();
	}

	if (m_testbed_mode == ETestbedMode::Nerf || m_testbed_mode == ETestbedMode::Sdf) {
		if (ImGui::CollapsingHeader("Export mesh / volume / slices")) {
			static bool flip_y_and_z_axes = false;
			static float density_range = 4.f;
			BoundingBox aabb = (m_testbed_mode == ETestbedMode::Nerf) ? m_render_aabb : m_aabb;

			auto res3d = get_marching_cubes_res(m_mesh.res, aabb);

			// If we use an octree to fit the SDF only close to the surface, then marching cubes will not work (SDF not defined everywhere)
			bool disable_marching_cubes = m_testbed_mode == ETestbedMode::Sdf && (m_sdf.uses_takikawa_encoding || m_sdf.use_triangle_octree);
			if (disable_marching_cubes) { ImGui::BeginDisabled(); }

			if (imgui_colored_button("Mesh it!", 0.4f)) {
				marching_cubes(res3d, aabb, m_render_aabb_to_local, m_mesh.thresh);
				m_nerf.render_with_lens_distortion = false;
			}
			if (m_mesh.indices.size()>0) {
				ImGui::SameLine();
				if (imgui_colored_button("Clear Mesh", 0.f)) {
					m_mesh.clear();
				}
			}

			if (disable_marching_cubes) { ImGui::EndDisabled(); }

			ImGui::SameLine();

			if (imgui_colored_button("Save density PNG", -0.7f)) {
				Testbed::compute_and_save_png_slices(m_data_path.str().c_str(), m_mesh.res, {}, m_mesh.thresh, density_range, flip_y_and_z_axes);
			}

			if (m_testbed_mode == ETestbedMode::Nerf) {
				ImGui::SameLine();
				if (imgui_colored_button("Save RGBA PNG sequence", 0.2f)) {
					auto effective_view_dir = flip_y_and_z_axes ? Vector3f{0.0f, 1.0f, 0.0f} : Vector3f{0.0f, 0.0f, 1.0f};
					// Depth of 0.01f is arbitrarily chosen to produce a visually interpretable range of alpha values.
					// Alternatively, if the true transparency of a given voxel is desired, one could use the voxel size,
					// the voxel diagonal, or some form of expected ray length through the voxel, given random directions.
					GPUMemory<Array4f> rgba = get_rgba_on_grid(res3d, effective_view_dir, true, 0.01f);
					auto dir = m_data_path / "rgba_slices";
					if (!dir.exists()) {
						fs::create_directory(dir);
					}
					save_rgba_grid_to_png_sequence(rgba, dir.str().c_str(), res3d, flip_y_and_z_axes);
				}
				if (imgui_colored_button("Save raw volumes", 0.4f)) {
					auto effective_view_dir = flip_y_and_z_axes ? Vector3f{0.0f, 1.0f, 0.0f} : Vector3f{0.0f, 0.0f, 1.0f};
					auto old_local = m_render_aabb_to_local;
					auto old_aabb = m_render_aabb;
					m_render_aabb_to_local = Eigen::Matrix3f::Identity();
					auto dir = m_data_path / "volume_raw";
					if (!dir.exists()) {
						fs::create_directory(dir);
					}
					for (int cascade = 0; (1<<cascade)<= m_aabb.diag().x()+0.5f; ++cascade) {
						float radius = (1<<cascade) * 0.5f;
						m_render_aabb = BoundingBox(Eigen::Vector3f::Constant(0.5f-radius), Eigen::Vector3f::Constant(0.5f+radius));
						// Dump raw density values that the user can then convert to alpha as they please.
						GPUMemory<Array4f> rgba = get_rgba_on_grid(res3d, effective_view_dir, true, 0.0f, true);
						save_rgba_grid_to_raw_file(rgba, dir.str().c_str(), res3d, flip_y_and_z_axes, cascade);
					}
					m_render_aabb_to_local = old_local;
					m_render_aabb = old_aabb;
				}
			}

			ImGui::SameLine();
			ImGui::Checkbox("Swap Y&Z", &flip_y_and_z_axes);
			ImGui::SliderFloat("PNG Density Range", &density_range, 0.001f, 8.f);

			ImGui::SliderInt("Res:", &m_mesh.res, 16, 2048, "%d", ImGuiSliderFlags_Logarithmic);
			ImGui::SameLine();

			ImGui::Text("%dx%dx%d", res3d.x(), res3d.y(), res3d.z());
			float thresh_range = (m_testbed_mode == ETestbedMode::Sdf) ? 0.5f : 10.f;
			ImGui::SliderFloat("MC density threshold",&m_mesh.thresh, -thresh_range, thresh_range);
			ImGui::Combo("Mesh render mode", (int*)&m_mesh_render_mode, "Off\0Vertex Colors\0Vertex Normals\0\0");
			ImGui::Checkbox("Unwrap mesh", &m_mesh.unwrap);
			if (uint32_t tricount = m_mesh.indices.size()/3) {
				ImGui::InputText("##OBJFile", m_imgui.mesh_path, sizeof(m_imgui.mesh_path));
				if (ImGui::Button("Save it!")) {
					save_mesh(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, m_imgui.mesh_path, m_mesh.unwrap, m_nerf.training.dataset.scale, m_nerf.training.dataset.offset);
				}
				ImGui::SameLine();
				ImGui::Text("Mesh has %d triangles\n", tricount);
				ImGui::Checkbox("Optimize mesh", &m_mesh.optimize_mesh);
				ImGui::SliderFloat("Laplacian smoothing", &m_mesh.smooth_amount, 0.f, 2048.f);
				ImGui::SliderFloat("Density push", &m_mesh.density_amount, 0.f, 128.f);
				ImGui::SliderFloat("Inflate", &m_mesh.inflate_amount, 0.f, 128.f);
			}
		}
	}

	if (m_testbed_mode == ETestbedMode::Sdf) {
		if (ImGui::CollapsingHeader("BRDF parameters")) {
			accum_reset |= ImGui::ColorEdit3("Base color", (float*)&m_sdf.brdf.basecolor );
			accum_reset |= ImGui::SliderFloat("Roughness", &m_sdf.brdf.roughness, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Specular", &m_sdf.brdf.specular, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Metallic", &m_sdf.brdf.metallic, 0.f, 1.f);
			ImGui::Separator();
			accum_reset |= ImGui::SliderFloat("Subsurface", &m_sdf.brdf.subsurface, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Sheen", &m_sdf.brdf.sheen, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Clearcoat", &m_sdf.brdf.clearcoat, 0.f, 1.f);
			accum_reset |= ImGui::SliderFloat("Clearcoat gloss", &m_sdf.brdf.clearcoat_gloss, 0.f, 1.f);
		}
		m_sdf.brdf.ambientcolor = (m_background_color * m_background_color).head<3>();
	}

	if (ImGui::CollapsingHeader("Histograms of encoding parameters")) {
		ImGui::Checkbox("Gather histograms", &m_gather_histograms);

		static float minlevel = 0.f;
		static float maxlevel = 1.f;
		if (ImGui::SliderFloat("Max level", &maxlevel, 0.f, 1.f))
			set_max_level(maxlevel);
		if (ImGui::SliderFloat("##Min level", &minlevel, 0.f, 1.f))
			set_min_level(minlevel);
		ImGui::SameLine();
		ImGui::Text("%0.1f%% values snapped to 0", m_quant_percent);

		std::vector<float> f(m_num_levels);


		// Hashgrid statistics
		for (int i = 0; i < m_num_levels; ++i) {
			f[i] = m_level_stats[i].mean();
		}
		ImGui::PlotHistogram("Grid means", f.data(), m_num_levels, 0, "means", FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
		for (int i = 0; i < m_num_levels; ++i) {
			f[i] = m_level_stats[i].sigma();
		}
		ImGui::PlotHistogram("Grid sigmas", f.data(), m_num_levels, 0, "sigma", FLT_MAX, FLT_MAX, ImVec2(0, 60.f));
		ImGui::Separator();


		// Histogram of trained hashgrid params
		ImGui::SliderInt("Show details for level", &m_histo_level, 0, m_num_levels - 1);
		if (m_histo_level < m_num_levels) {
			LevelStats& s = m_level_stats[m_histo_level];
			static bool excludezero = false;
			if (excludezero) {
				m_histo[128] = 0.f;
			}
			ImGui::PlotHistogram("Values histogram", m_histo, 257, 0, "", FLT_MAX, FLT_MAX, ImVec2(0, 120.f));
			ImGui::SliderFloat("Histogram horizontal scale", &m_histo_scale, 0.01f, 2.f);
			ImGui::Checkbox("Exclude 'zero' from histogram", &excludezero);
			ImGui::Text("Range: %0.5f - %0.5f", s.min, s.max);
			ImGui::Text("Mean: %0.5f Sigma: %0.5f", s.mean(), s.sigma());
			ImGui::Text("Num Zero: %d (%0.1f%%)", s.numzero, s.fraczero() * 100.f);
		}
	}

	if (ImGui::BeginPopupModal("Error", NULL, ImGuiWindowFlags_AlwaysAutoResize)) {
		ImGui::Text("%s", imgui_error_string.c_str());
		if (ImGui::Button("OK", ImVec2(120, 0))) {
			ImGui::CloseCurrentPopup();
		}
		ImGui::EndPopup();
	}

	if (accum_reset) {
		reset_accumulation();
	}

	if (ImGui::Button("Go to python REPL")) {
		m_want_repl = true;
	}

	ImGui::End();
}

void Testbed::visualize_nerf_cameras(ImDrawList* list, const Matrix<float, 4, 4>& world2proj) {
	for (int i = 0; i < m_nerf.training.n_images_for_training; ++i) {
		auto res = m_nerf.training.dataset.metadata[i].resolution;
		float aspect = float(res.x())/float(res.y());
		auto current_xform = get_xform_given_rolling_shutter(m_nerf.training.transforms[i], m_nerf.training.dataset.metadata[i].rolling_shutter, Vector2f{0.5f, 0.5f}, 0.0f);
		visualize_nerf_camera(list, world2proj, m_nerf.training.dataset.xforms[i].start, aspect, 0x40ffff40);
		visualize_nerf_camera(list, world2proj, m_nerf.training.dataset.xforms[i].end, aspect, 0x40ffff40);
		visualize_nerf_camera(list, world2proj, current_xform, aspect, 0x80ffffff);

		// Visualize near distance
		add_debug_line(list, world2proj, current_xform.col(3), current_xform.col(3) + current_xform.col(2) * m_nerf.training.near_distance, 0x20ffffff);
	}

}

void Testbed::draw_visualizations(ImDrawList* list, const Matrix<float, 3, 4>& camera_matrix) {
	Matrix<float, 4, 4> world2view, view2world, view2proj, world2proj;
	view2world.setIdentity();
	view2world.block<3, 4>(0, 0) = camera_matrix;

	auto focal = calc_focal_length(Vector2i::Ones(), m_relative_focal_length, m_fov_axis, m_zoom);
	float zscale = 1.0f / focal[m_fov_axis];

	float xyscale = (float)m_window_res[m_fov_axis];
	Vector2f screen_center = render_screen_center(m_screen_center);
	view2proj <<
		xyscale, 0,       (float)m_window_res.x()*screen_center.x()*zscale, 0,
		0,       xyscale, (float)m_window_res.y()*screen_center.y()*zscale, 0,
		0,       0,       1,                                                0,
		0,       0,       zscale,                                           0;

	world2view = view2world.inverse();
	world2proj = view2proj * world2view;
	float aspect = (float)m_window_res.x() / (float)m_window_res.y();

	// Visualize NeRF training poses
	if (m_testbed_mode == ETestbedMode::Nerf) {
		if (m_nerf.visualize_cameras) {
			visualize_nerf_cameras(list, world2proj);
		}
	}

	if (m_visualize_unit_cube) {
		visualize_cube(list, world2proj, Eigen::Vector3f::Constant(0.f), Eigen::Vector3f::Constant(1.f), Eigen::Matrix3f::Identity());
	}

	if (m_edit_render_aabb) {
		if (m_testbed_mode == ETestbedMode::Nerf || m_testbed_mode == ETestbedMode::Volume) {
			visualize_cube(list, world2proj, m_render_aabb.min, m_render_aabb.max, m_render_aabb_to_local);
		}

		ImGuiIO& io = ImGui::GetIO();
		float flx = focal.x();
		float fly = focal.y();
		Matrix<float, 4, 4> view2proj_guizmo;
		float zfar = m_ndc_zfar;
		float znear = m_ndc_znear;
		view2proj_guizmo <<
			fly * 2.f / aspect, 0, 0, 0,
			0, -fly * 2.f, 0, 0,
			0, 0, (zfar + znear) / (zfar - znear), -(2.f * zfar * znear) / (zfar - znear),
			0, 0, 1, 0;

		ImGuizmo::SetRect(0, 0, io.DisplaySize.x, io.DisplaySize.y);

		static bool manipulating = false;
		static Eigen::Matrix4f matrix = Eigen::Matrix4f::Identity();
		static Eigen::Matrix4f world2view_guizmo = Eigen::Matrix4f::Identity();

		if (!ImGuizmo::IsUsing()) {
			// The the guizmo is being used, it handles updating its matrix on its own.
			// Outside interference can only lead to trouble.
			matrix.block<3, 3>(0, 0) = m_render_aabb_to_local.transpose();
			Eigen::Vector3f cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
			matrix.block<3, 4>(0, 0).col(3) = cen;

			// Additionally, the world2view transform must stay fixed, else the guizmo will incorrectly
			// interpret the state from past frames. Special handling is necessary here, because below
			// we emulate world translation and rotation through (inverse) camera movement.
			world2view_guizmo = world2view;
		}

		Eigen::Vector3f cen = m_render_aabb_to_local.transpose() * m_render_aabb.center();
		auto prev_matrix = matrix;

		if (ImGuizmo::Manipulate((const float*)&world2view_guizmo, (const float*)&view2proj_guizmo, m_camera_path.m_gizmo_op, ImGuizmo::LOCAL, (float*)&matrix, NULL, NULL)) {
			auto crop_transform = matrix;
			if (m_edit_world_transform) {
				// We transform the world by transforming the camera in the opposite direction.
				auto rel = prev_matrix * matrix.inverse();
				m_camera = rel.block<3, 3>(0, 0) * m_camera;
				m_camera.col(3) += rel.block<3, 1>(0, 3);

				m_up_dir = rel.block<3, 3>(0, 0) * m_up_dir;
			} else {
				m_render_aabb_to_local = matrix.block<3, 3>(0, 0).transpose();
				Eigen::Vector3f new_cen = m_render_aabb_to_local * matrix.block<3, 4>(0, 0).col(3);
				Eigen::Vector3f old_cen = m_render_aabb.center();
				m_render_aabb.min += new_cen - old_cen;
				m_render_aabb.max += new_cen - old_cen;
			}

			reset_accumulation();
		}
	}

	if (m_camera_path.imgui_viz(list, view2proj, world2proj, world2view, focal, aspect, m_ndc_znear, m_ndc_zfar)) {
		m_pip_render_buffer->reset_accumulation();
	}
}

void glfw_error_callback(int error, const char* description) {
	tlog::error() << "GLFW error #" << error << ": " << description;
}

bool Testbed::keyboard_event() {
	if (ImGui::GetIO().WantCaptureKeyboard) {
		return false;
	}

	if (m_keyboard_event_callback && m_keyboard_event_callback()) {
		return false;
	}

	for (int idx = 0; idx < std::min((int)ERenderMode::NumRenderModes, 10); ++idx) {
		char c[] = { "1234567890" };
		if (ImGui::IsKeyPressed(c[idx])) {
			m_render_mode = (ERenderMode)idx;
			reset_accumulation();
		}
	}

	bool ctrl = ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Ctrl;
	bool shift = ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Shift;

	if (ImGui::IsKeyPressed('Z')) {
		m_camera_path.m_gizmo_op = ImGuizmo::TRANSLATE;
	}

	if (ImGui::IsKeyPressed('X')) {
		m_camera_path.m_gizmo_op = ImGuizmo::ROTATE;
	}

	if (ImGui::IsKeyPressed('E')) {
		set_exposure(m_exposure + (shift ? -0.5f : 0.5f));
		redraw_next_frame();
	}

	if (ImGui::IsKeyPressed('R')) {
		if (shift) {
			reset_camera();
		} else {
			if (ctrl) {
				reload_training_data();
				// After reloading the training data, also reset the NN.
				// Presumably, there is no use case where the user would
				// like to hot-reload the same training data set other than
				// to slightly tweak its parameters. And to observe that
				// effect meaningfully, the NN should be trained from scratch.
			}

			reload_network_from_file();
		}
	}

	if (m_training_data_available) {
		if (ImGui::IsKeyPressed('O')) {
			m_nerf.training.render_error_overlay = !m_nerf.training.render_error_overlay;
		}

		if (ImGui::IsKeyPressed('G')) {
			m_render_ground_truth = !m_render_ground_truth;
			reset_accumulation();
			if (m_render_ground_truth) {
				m_nerf.training.view = find_best_training_view(m_nerf.training.view);
			}
		}

		if (ImGui::IsKeyPressed('T')) {
			set_train(!m_train);
		}
	}

	if (ImGui::IsKeyPressed('.')) {
		if (m_single_view) {
			if (m_visualized_dimension == m_network->width(m_visualized_layer)-1 && m_visualized_layer < m_network->num_forward_activations()-1) {
				set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer+1)));
				set_visualized_dim(0);
			} else {
				set_visualized_dim(std::max(-1, std::min((int)m_network->width(m_visualized_layer)-1, m_visualized_dimension+1)));
			}
		} else {
			set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer+1)));
		}
	}

	if (ImGui::IsKeyPressed(',')) {
		if (m_single_view) {
			if (m_visualized_dimension == 0 && m_visualized_layer > 0) {
				set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer-1)));
				set_visualized_dim(m_network->width(m_visualized_layer)-1);
			} else {
				set_visualized_dim(std::max(-1, std::min((int)m_network->width(m_visualized_layer)-1, m_visualized_dimension-1)));
			}
		} else {
			set_visualized_layer(std::max(0, std::min((int)m_network->num_forward_activations()-1, m_visualized_layer-1)));
		}
	}

	if (ImGui::IsKeyPressed('M')) {
		m_single_view = !m_single_view;
		set_visualized_dim(-1);
		reset_accumulation();
	}


	if (ImGui::IsKeyPressed('N')) {
		m_sdf.analytic_normals = !m_sdf.analytic_normals;
		reset_accumulation();
	}

	if (ImGui::IsKeyPressed('[')) {
		if (shift) {
			first_training_view();
		} else {
			previous_training_view();
		}
	}

	if (ImGui::IsKeyPressed(']')) {
		if (shift) {
			last_training_view();
		} else {
			next_training_view();
		}
	}

	if (ImGui::IsKeyPressed('=') || ImGui::IsKeyPressed('+')) {
		if (m_fps_camera) {
			m_camera_velocity *= 1.5f;
		} else {
			set_scale(m_scale * 1.1f);
		}
	}

	if (ImGui::IsKeyPressed('-') || ImGui::IsKeyPressed('_')) {
		if (m_fps_camera) {
			m_camera_velocity /= 1.5f;
		} else {
			set_scale(m_scale / 1.1f);
		}
	}

	// WASD camera movement
	Vector3f translate_vec = Vector3f::Zero();
	if (ImGui::IsKeyDown('W')) {
		translate_vec.z() += 1.0f;
	}

	if (ImGui::IsKeyDown('A')) {
		translate_vec.x() += -1.0f;
	}

	if (ImGui::IsKeyDown('S')) {
		translate_vec.z() += -1.0f;
	}

	if (ImGui::IsKeyDown('D')) {
		translate_vec.x() += 1.0f;
	}

	if (ImGui::IsKeyDown(' ')) {
		translate_vec.y() += -1.0f;
	}

	if (ImGui::IsKeyDown('C')) {
		translate_vec.y() += 1.0f;
	}

	translate_vec *= m_camera_velocity * m_frame_ms.val() / 1000.0f;
	if (shift) {
		translate_vec *= 5;
	}

	if (translate_vec != Vector3f::Zero()) {
		m_fps_camera = true;

		// If VR is active, movement that isn't aligned with the current view
		// direction is _very_ jarring to the user, so make keyboard-based
		// movement aligned with the VR view, even though it is not an intended
		// movement mechanism. (Users should use controllers.)
		translate_camera(translate_vec, m_hmd && m_hmd->is_visible() ? m_views.front().camera0.block<3, 3>(0, 0) : m_camera.block<3, 3>(0, 0));
	}

	return false;
}

void Testbed::mouse_wheel() {
	float delta = ImGui::GetIO().MouseWheel;
	if (delta == 0) {
		return;
	}

	float scale_factor = pow(1.1f, -delta);
	set_scale(m_scale * scale_factor);

	// When in image mode, zoom around the hovered point.
	if (m_testbed_mode == ETestbedMode::Image) {
		Vector2i mouse = {ImGui::GetMousePos().x, ImGui::GetMousePos().y};
		Vector3f offset = get_3d_pos_from_pixel(*m_views.front().render_buffer, mouse) - look_at();

		// Don't center around infinitely distant points.
		if (offset.norm() < 256.0f) {
			m_camera.col(3) += offset * (1.0f - scale_factor);
		}
	}

	reset_accumulation(true);
}

Matrix3f Testbed::rotation_from_angles(const Vector2f& angles) const {
	Vector3f up = m_up_dir;
	Vector3f side = m_camera.col(0);
	return (AngleAxisf(angles.x(), up) * AngleAxisf(angles.y(), side)).matrix();
}

void Testbed::mouse_drag() {
	Vector2f rel = Vector2f{ImGui::GetIO().MouseDelta.x, ImGui::GetIO().MouseDelta.y} / (float)m_window_res[m_fov_axis];
	Vector2i mouse = {ImGui::GetMousePos().x, ImGui::GetMousePos().y};

	Vector3f up = m_up_dir;
	Vector3f side = m_camera.col(0);

	bool shift = ImGui::GetIO().KeyMods & ImGuiKeyModFlags_Shift;

	// Left held
	if (ImGui::GetIO().MouseDown[0]) {
		if (shift) {
			m_autofocus_target = get_3d_pos_from_pixel(*m_views.front().render_buffer, mouse);
			m_autofocus = true;

			reset_accumulation();
		} else {
			float rot_sensitivity = m_fps_camera ? 0.35f : 1.0f;
			Matrix3f rot = rotation_from_angles(-rel * 2 * PI() * rot_sensitivity);

			if (m_fps_camera) {
				m_camera.block<3, 3>(0, 0) = rot * m_camera.block<3, 3>(0, 0);
			} else {
				// Turntable
				auto old_look_at = look_at();
				set_look_at({0.0f, 0.0f, 0.0f});
				m_camera = rot * m_camera;
				set_look_at(old_look_at);
			}

			reset_accumulation(true);
		}
	}

	// Right held
	if (ImGui::GetIO().MouseDown[1]) {
		Matrix3f rot = rotation_from_angles(-rel * 2 * PI());
		if (m_render_mode == ERenderMode::Shade) {
			m_sun_dir = rot.transpose() * m_sun_dir;
		}

		m_slice_plane_z += -rel.y() * m_bounding_radius;
		reset_accumulation();
	}

	// Middle pressed
	if (ImGui::GetIO().MouseClicked[2]) {
		m_drag_depth = get_depth_from_renderbuffer(*m_views.front().render_buffer, mouse.cast<float>().cwiseQuotient(m_window_res.cast<float>()));
	}

	// Middle held
	if (ImGui::GetIO().MouseDown[2]) {
		Vector3f translation = Vector3f{-rel.x(), -rel.y(), 0.0f} / m_zoom;

		// If we have a valid depth value, scale the scene translation by it such that the
		// hovered point in 3D space stays under the cursor.
		if (m_drag_depth < 256.0f) {
			translation *= m_drag_depth / m_relative_focal_length[m_fov_axis];
		}

		translate_camera(translation, m_camera.block<3, 3>(0, 0));
	}
}

bool Testbed::begin_frame() {
	if (glfwWindowShouldClose(m_glfw_window) || ImGui::IsKeyPressed(GLFW_KEY_ESCAPE) || ImGui::IsKeyPressed(GLFW_KEY_Q)) {
		destroy_window();
		return false;
	}

	{
		auto now = std::chrono::steady_clock::now();
		auto elapsed = now - m_last_frame_time_point;
		m_last_frame_time_point = now;
		m_frame_ms.update(std::chrono::duration<float, std::milli>(elapsed).count());
	}

	glfwPollEvents();
	glfwGetFramebufferSize(m_glfw_window, &m_window_res.x(), &m_window_res.y());

	ImGui_ImplOpenGL3_NewFrame();
	ImGui_ImplGlfw_NewFrame();
	ImGui::NewFrame();
	ImGuizmo::BeginFrame();

	return true;
}

void Testbed::handle_user_input() {
	if (ImGui::IsKeyPressed(GLFW_KEY_TAB) || ImGui::IsKeyPressed(GLFW_KEY_GRAVE_ACCENT)) {
		m_imgui.enabled = !m_imgui.enabled;
	}

	// Only respond to mouse inputs when not interacting with ImGui
	if (!ImGui::IsAnyItemActive() && !ImGuizmo::IsUsing() && !ImGui::GetIO().WantCaptureMouse) {
		mouse_wheel();
		mouse_drag();
	}

	if (m_testbed_mode == ETestbedMode::Nerf && (m_render_ground_truth || m_nerf.training.render_error_overlay)) {
		// find nearest training view to current camera, and set it
		int bestimage = find_best_training_view(-1);
		if (bestimage >= 0) {
			m_nerf.training.view = bestimage;
			if (ImGui::GetIO().MouseReleased[0]) {// snap camera to ground truth view on mouse up
				set_camera_to_training_view(m_nerf.training.view);
			}
		}
	}

	keyboard_event();

	if (m_imgui.enabled) {
		imgui();
	}
}

Vector3f Testbed::vr_to_world(const Vector3f& pos) const {
	return m_camera.block<3, 3>(0, 0) * pos * m_scale + m_camera.col(3);
}

void Testbed::begin_vr_frame_and_handle_vr_input() {
	if (!m_hmd) {
		m_vr_frame_info = nullptr;
		return;
	}

	m_hmd->poll_events();
	if (!m_hmd->must_run_frame_loop()) {
		m_vr_frame_info = nullptr;
		return;
	}

	m_vr_frame_info = m_hmd->begin_frame();

	const auto& views = m_vr_frame_info->views;
	size_t n_views = views.size();
	size_t n_devices = m_devices.size();
	if (n_views > 0) {
		set_n_views(n_views);

		Vector2i total_size = Vector2i::Zero();
		for (size_t i = 0; i < n_views; ++i) {
			Vector2i view_resolution = {views[i].view.subImage.imageRect.extent.width, views[i].view.subImage.imageRect.extent.height};
			total_size += view_resolution;

			m_views[i].full_resolution = view_resolution;

			// Apply the VR pose relative to the world camera transform.
			m_views[i].camera0.block<3, 3>(0, 0) = m_camera.block<3, 3>(0, 0) * views[i].pose.block<3, 3>(0, 0);
			m_views[i].camera0.col(3) = vr_to_world(views[i].pose.col(3));
			m_views[i].camera1 = m_views[i].camera0;

			m_views[i].visualized_dimension = m_visualized_dimension;

			const auto& xr_fov = views[i].view.fov;

			// Compute the distance on the image plane (1 unit away from the camera) that an angle of the respective FOV spans
			Vector2f rel_focal_length_left_down = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * xr_fov.angleLeft / PI(), 360.0f * xr_fov.angleDown / PI()});
			Vector2f rel_focal_length_right_up = 0.5f * fov_to_focal_length(Vector2i::Ones(), Vector2f{360.0f * xr_fov.angleRight / PI(), 360.0f * xr_fov.angleUp / PI()});

			// Compute total distance (for X and Y) that is spanned on the image plane.
			m_views[i].relative_focal_length = rel_focal_length_right_up - rel_focal_length_left_down;

			// Compute fraction of that distance that is spanned by the right-up part and set screen center accordingly.
			Vector2f ratio = rel_focal_length_right_up.cwiseQuotient(m_views[i].relative_focal_length);
			m_views[i].screen_center = { 1.0f - ratio.x(), ratio.y() };

			// Fix up weirdness in the rendering pipeline
			m_views[i].relative_focal_length[(m_fov_axis+1)%2] *= (float)view_resolution[(m_fov_axis+1)%2] / (float)view_resolution[m_fov_axis];
			m_views[i].render_buffer->set_hidden_area_mask(views[i].hidden_area_mask);

			// Render each view on a different GPU (if available)
			m_views[i].device = m_use_aux_devices ? &m_devices.at(i % m_devices.size()) : &primary_device();
		}

		// Put all the views next to each other, but at half size
		glfwSetWindowSize(m_glfw_window, total_size.x() / 2, (total_size.y() / 2) / n_views);

		// VR controller input
		const auto& hands = m_vr_frame_info->hands;
		m_fps_camera = true;

		// TRANSLATE BY STICK (if not pressing the stick)
		if (!hands[0].pressing) {
			Vector3f translate_vec = Vector3f{hands[0].thumbstick.x(), 0.0f, hands[0].thumbstick.y()} * m_camera_velocity * m_frame_ms.val() / 1000.0f;
			if (translate_vec != Vector3f::Zero()) {
				translate_camera(translate_vec, m_views.front().camera0.block<3, 3>(0, 0), false);
			}
		}

		// TURN BY STICK (if not pressing the stick)
		if (!hands[1].pressing) {
			auto prev_camera = m_camera;

			// Turn around the up vector (equivalent to x-axis mouse drag) with right joystick left/right
			float sensitivity = 0.35f;
			m_camera.block<3, 3>(0, 0) = rotation_from_angles({-2.0f * PI() * sensitivity * hands[1].thumbstick.x() * m_frame_ms.val() / 1000.0f, 0.0f}) * m_camera.block<3, 3>(0, 0);

			// Translate camera such that center of rotation was about the current view
			m_camera.col(3) += prev_camera.block<3, 3>(0, 0) * views[0].pose.col(3) * m_scale - m_camera.block<3, 3>(0, 0) * views[0].pose.col(3) * m_scale;
		}

		// TRANSLATE, SCALE, AND ROTATE BY GRAB
		{
			bool both_grabbing = hands[0].grabbing && hands[1].grabbing;
			float drag_factor = both_grabbing ? 0.5f : 1.0f;

			if (both_grabbing) {
				drag_factor = 0.5f;

				Vector3f prev_diff = hands[0].prev_grab_pos - hands[1].prev_grab_pos;
				Vector3f diff = hands[0].grab_pos - hands[1].grab_pos;
				Vector3f center = 0.5f * (hands[0].grab_pos + hands[1].grab_pos);

				Vector3f center_world = vr_to_world(0.5f * (hands[0].grab_pos + hands[1].grab_pos));

				// Scale around center position of the two dragging hands. Makes the scaling feel similar to phone pinch-to-zoom
				float scale = m_scale * prev_diff.norm() / diff.norm();
				m_camera.col(3) = (view_pos() - center_world) * (scale / m_scale) + center_world;
				m_scale = scale;

				// Take rotational component and project it to the nearest rotation about the up vector.
				// We don't want to rotate the scene about any other axis.
				Vector3f rot = prev_diff.normalized().cross(diff.normalized());
				float rot_radians = std::asin(m_up_dir.dot(rot));

				auto prev_camera = m_camera;
				m_camera.block<3, 3>(0, 0) = AngleAxisf(rot_radians, m_up_dir) * m_camera.block<3, 3>(0, 0);
				m_camera.col(3) += prev_camera.block<3, 3>(0, 0) * center * m_scale - m_camera.block<3, 3>(0, 0) * center * m_scale;
			}

			for (const auto& hand : hands) {
				if (hand.grabbing) {
					m_camera.col(3) -= drag_factor * m_camera.block<3, 3>(0, 0) * hand.drag() * m_scale;
				}
			}
		}

		// ERASE OCCUPANCY WHEN PRESSING STICK/TRACKPAD
		if (m_testbed_mode == ETestbedMode::Nerf) {
			for (const auto& hand : hands) {
				if (hand.pressing) {
					mark_density_grid_in_sphere_empty(vr_to_world(hand.pose.col(3)), m_scale * 0.05f, m_stream.get());
				}
			}
		}
	}
}

void Testbed::SecondWindow::draw(GLuint texture) {
	if (!window)
		return;
	int display_w, display_h;
	GLFWwindow *old_context = glfwGetCurrentContext();
	glfwMakeContextCurrent(window);
	glfwGetFramebufferSize(window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
	glBindVertexArray(vao);
	if (program)
		glUseProgram(program);
	glDrawArrays(GL_TRIANGLES, 0, 6);
	glBindVertexArray(0);
	glUseProgram(0);
	glfwSwapBuffers(window);
	glfwMakeContextCurrent(old_context);
}

void Testbed::init_opengl_shaders() {
	static const char* shader_vert = R"(#version 140
		out vec2 UVs;
		void main() {
			UVs = vec2((gl_VertexID << 1) & 2, gl_VertexID & 2);
			gl_Position = vec4(UVs * 2.0 - 1.0, 0.0, 1.0);
		})";

	static const char* shader_frag = R"(#version 140
		in vec2 UVs;
		out vec4 frag_color;
		uniform sampler2D rgba_texture;
		uniform sampler2D depth_texture;

		struct FoveationWarp {
			float al, bl, cl;
			float am, bm;
			float ar, br, cr;
			float switch_left, switch_right;
			float inv_switch_left, inv_switch_right;
		};

		uniform FoveationWarp warp_x;
		uniform FoveationWarp warp_y;

		float unwarp(in FoveationWarp warp, float y) {
			y = clamp(y, 0.0, 1.0);
			if (y < warp.inv_switch_left) {
				return (sqrt(-4.0 * warp.al * warp.cl + 4.0 * warp.al * y + warp.bl * warp.bl) - warp.bl) / (2.0 * warp.al);
			} else if (y > warp.inv_switch_right) {
				return (sqrt(-4.0 * warp.ar * warp.cr + 4.0 * warp.ar * y + warp.br * warp.br) - warp.br) / (2.0 * warp.ar);
			} else {
				return (y - warp.bm) / warp.am;
			}
		}

		vec2 unwarp(in vec2 pos) {
			return vec2(unwarp(warp_x, pos.x), unwarp(warp_y, pos.y));
		}

		void main() {
			vec2 tex_coords = UVs;
			tex_coords.y = 1.0 - tex_coords.y;
			tex_coords = unwarp(tex_coords);
			frag_color = texture(rgba_texture, tex_coords.xy);
			//Uncomment the following line of code to visualize debug the depth buffer for debugging.
			// frag_color = vec4(vec3(texture(depth_texture, tex_coords.xy).r), 1.0);
			gl_FragDepth = texture(depth_texture, tex_coords.xy).r;
		})";

	GLuint vert = glCreateShader(GL_VERTEX_SHADER);
	glShaderSource(vert, 1, &shader_vert, NULL);
	glCompileShader(vert);
	check_shader(vert, "Blit vertex shader", false);

	GLuint frag = glCreateShader(GL_FRAGMENT_SHADER);
	glShaderSource(frag, 1, &shader_frag, NULL);
	glCompileShader(frag);
	check_shader(frag, "Blit fragment shader", false);

	m_blit_program = glCreateProgram();
	glAttachShader(m_blit_program, vert);
	glAttachShader(m_blit_program, frag);
	glLinkProgram(m_blit_program);
	check_shader(m_blit_program, "Blit shader program", true);

	glDeleteShader(vert);
	glDeleteShader(frag);

	glGenVertexArrays(1, &m_blit_vao);
}

void Testbed::blit_texture(const Foveation& foveation, GLint rgba_texture, GLint rgba_filter_mode, GLint depth_texture, GLint framebuffer, const Vector2i& offset, const Vector2i& resolution) {
	if (m_blit_program == 0) {
		return;
	}

	// Blit image to OpenXR swapchain.
	// Note that the OpenXR swapchain is 8bit while the rendering is in a float texture.
	// As some XR runtimes do not support float swapchains, we can't render into it directly.

	bool tex = glIsEnabled(GL_TEXTURE_2D);
	bool depth = glIsEnabled(GL_DEPTH_TEST);
	bool cull = glIsEnabled(GL_CULL_FACE);

	if (!tex) glEnable(GL_TEXTURE_2D);
	if (!depth) glEnable(GL_DEPTH_TEST);
	if (cull) glDisable(GL_CULL_FACE);

	glDepthFunc(GL_ALWAYS);
	glDepthMask(GL_TRUE);

	glBindVertexArray(m_blit_vao);
	glUseProgram(m_blit_program);
	glUniform1i(glGetUniformLocation(m_blit_program, "rgba_texture"), 0);
	glUniform1i(glGetUniformLocation(m_blit_program, "depth_texture"), 1);

	auto bind_warp = [&](const FoveationPiecewiseQuadratic& warp, const std::string& uniform_name) {
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".al").c_str()), warp.al);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bl").c_str()), warp.bl);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cl").c_str()), warp.cl);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".am").c_str()), warp.am);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".bm").c_str()), warp.bm);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".ar").c_str()), warp.ar);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".br").c_str()), warp.br);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".cr").c_str()), warp.cr);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_left").c_str()), warp.switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".switch_right").c_str()), warp.switch_right);

		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_left").c_str()), warp.inv_switch_left);
		glUniform1f(glGetUniformLocation(m_blit_program, (uniform_name + ".inv_switch_right").c_str()), warp.inv_switch_right);
	};

	bind_warp(foveation.warp_x, "warp_x");
	bind_warp(foveation.warp_y, "warp_y");

	glActiveTexture(GL_TEXTURE1);
	glBindTexture(GL_TEXTURE_2D, depth_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, rgba_texture);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, rgba_filter_mode);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, rgba_filter_mode);

	glBindFramebuffer(GL_FRAMEBUFFER, framebuffer);
	glViewport(offset.x(), offset.y(), resolution.x(), resolution.y());

	glDrawArrays(GL_TRIANGLES, 0, 3);

	glBindVertexArray(0);
	glUseProgram(0);

	glDepthFunc(GL_LESS);

	// restore old state
	if (!tex) glDisable(GL_TEXTURE_2D);
	if (!depth) glDisable(GL_DEPTH_TEST);
	if (cull) glEnable(GL_CULL_FACE);
	glBindFramebuffer(GL_FRAMEBUFFER, 0);
}

void Testbed::draw_gui() {
	// Make sure all the cuda code finished its business here
	CUDA_CHECK_THROW(hipDeviceSynchronize());

	if (!m_rgba_render_textures.empty()) {
		m_second_window.draw((GLuint)m_rgba_render_textures.front()->texture());
	}

	glfwMakeContextCurrent(m_glfw_window);
	int display_w, display_h;
	glfwGetFramebufferSize(m_glfw_window, &display_w, &display_h);
	glViewport(0, 0, display_w, display_h);
	glClearColor(0.f, 0.f, 0.f, 0.f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	glEnable(GL_BLEND);
	glBlendEquationSeparate(GL_FUNC_ADD, GL_FUNC_ADD);
	glBlendFuncSeparate(GL_ONE, GL_ONE_MINUS_SRC_ALPHA, GL_ONE, GL_ONE_MINUS_SRC_ALPHA);

	Vector2i extent = Vector2f{(float)display_w / m_n_views.x(), (float)display_h / m_n_views.y()}.cast<int>();

	int i = 0;
	for (int y = 0; y < m_n_views.y(); ++y) {
		for (int x = 0; x < m_n_views.x(); ++x) {
			if (i >= m_views.size()) {
				break;
			}

			auto& view = m_views[i];
			Vector2i top_left{x * extent.x(), display_h - (y + 1) * extent.y()};
			blit_texture(view.foveation, m_rgba_render_textures.at(i)->texture(), m_foveated_rendering ? GL_LINEAR : GL_NEAREST, m_depth_render_textures.at(i)->texture(), 0, top_left, extent);

			++i;
		}
	}
	glFinish();
	glViewport(0, 0, display_w, display_h);


	ImDrawList* list = ImGui::GetBackgroundDrawList();
	list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);

	auto draw_mesh = [&]() {
		glClear(GL_DEPTH_BUFFER_BIT);
		Vector2i res(display_w, display_h);
		Vector2f focal_length = calc_focal_length(res, m_relative_focal_length, m_fov_axis, m_zoom);
		draw_mesh_gl(m_mesh.verts, m_mesh.vert_normals, m_mesh.vert_colors, m_mesh.indices, res, focal_length, m_smoothed_camera, render_screen_center(m_screen_center), (int)m_mesh_render_mode);
	};

	// Visualizations are only meaningful when rendering a single view
	if (m_views.size() == 1) {
		if (m_mesh.verts.size() != 0 && m_mesh.indices.size() != 0 && m_mesh_render_mode != EMeshRenderMode::Off) {
			list->AddCallback([](const ImDrawList*, const ImDrawCmd* cmd) {
				(*(decltype(draw_mesh)*)cmd->UserCallbackData)();
			}, &draw_mesh);
			list->AddCallback(ImDrawCallback_ResetRenderState, nullptr);
		}

		draw_visualizations(list, m_smoothed_camera);
	}

	if (m_render_ground_truth) {
		list->AddText(ImVec2(4.f, 4.f), 0xffffffff, "Ground Truth");
	}

	ImGui::Render();
	ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());

	glfwSwapBuffers(m_glfw_window);

	// Make sure all the OGL code finished its business here.
	// Any code outside of this function needs to be able to freely write to
	// textures without being worried about interfering with rendering.
	glFinish();
}
#endif //NGP_GUI

__global__ void to_8bit_color_kernel(
	Vector2i resolution,
	EColorSpace output_color_space,
	hipSurfaceObject_t surface,
	uint8_t* result
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	Array4f color;
	surf2Dread((float4*)&color, surface, x * sizeof(float4), y);

	if (output_color_space == EColorSpace::Linear) {
		color.head<3>() = linear_to_srgb(color.head<3>());
	}

	for (uint32_t i = 0; i < 3; ++i) {
		result[(x + resolution.x() * y) * 3 + i] = (uint8_t)(tcnn::clamp(color[i], 0.0f, 1.0f) * 255.0f + 0.5f);
	}
}

void Testbed::prepare_next_camera_path_frame() {
	if (!m_camera_path.rendering) {
		return;
	}

	// If we're rendering a video, we'd like to accumulate multiple spp
	// for motion blur. Hence dump the frame once the target spp has been reached
	// and only reset _then_.
	if (m_views.front().render_buffer->spp() == m_camera_path.render_settings.spp) {
		auto tmp_dir = fs::path{"tmp"};
		if (!tmp_dir.exists()) {
			if (!fs::create_directory(tmp_dir)) {
				m_camera_path.rendering = false;
				tlog::error() << "Failed to create temporary directory 'tmp' to hold rendered images.";
				return;
			}
		}

		Vector2i res = m_views.front().render_buffer->out_resolution();
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };

		GPUMemory<uint8_t> image_data(res.prod() * 3);
		to_8bit_color_kernel<<<blocks, threads>>>(
			res,
			EColorSpace::SRGB, // the GUI always renders in SRGB
			m_views.front().render_buffer->surface(),
			image_data.data()
		);

		m_render_futures.emplace_back(m_thread_pool.enqueue_task([image_data=std::move(image_data), frame_idx=m_camera_path.render_frame_idx++, res, tmp_dir] {
			std::vector<uint8_t> cpu_image_data(image_data.size());
			CUDA_CHECK_THROW(hipMemcpy(cpu_image_data.data(), image_data.data(), image_data.bytes(), hipMemcpyDeviceToHost));
			write_stbi(tmp_dir / fmt::format("{:06d}.jpg", frame_idx), res.x(), res.y(), 3, cpu_image_data.data(), 100);
		}));

		reset_accumulation(true);

		if (m_camera_path.render_frame_idx == m_camera_path.render_settings.n_frames()) {
			m_camera_path.rendering = false;

			wait_all(m_render_futures);
			m_render_futures.clear();

			tlog::success() << "Finished rendering '.jpg' video frames to '" << tmp_dir << "'. Assembling them into a video next.";

			fs::path ffmpeg = "ffmpeg";

#ifdef _WIN32
			// Under Windows, try automatically downloading FFmpeg binaries if they don't exist
			if (system(fmt::format("where {} >nul 2>nul", ffmpeg.str()).c_str()) != 0) {
				fs::path dir = root_dir();
				if ((dir/"external"/"ffmpeg").exists()) {
					for (const auto& path : fs::directory{dir/"external"/"ffmpeg"}) {
						ffmpeg = path/"bin"/"ffmpeg.exe";
					}
				}

				if (!ffmpeg.exists()) {
					tlog::info() << "FFmpeg not found. Downloading FFmpeg...";
					do_system((dir/"scripts"/"download_ffmpeg.bat").str());
				}

				for (const auto& path : fs::directory{dir/"external"/"ffmpeg"}) {
					ffmpeg = path/"bin"/"ffmpeg.exe";
				}

				if (!ffmpeg.exists()) {
					tlog::warning() << "FFmpeg download failed. Trying system-wide FFmpeg.";
				}
			}
#endif

			auto ffmpeg_command = fmt::format(
				"{} -loglevel error -y -framerate {} -i tmp/%06d.jpg -c:v libx264 -preset slow -crf {} -pix_fmt yuv420p \"{}\"",
				ffmpeg.str(),
				m_camera_path.render_settings.fps,
				// Quality goes from 0 to 10. This conversion to CRF means a quality of 10
				// is a CRF of 17 and a quality of 0 a CRF of 27, which covers the "sane"
				// range of x264 quality settings according to the FFmpeg docs:
				// https://trac.ffmpeg.org/wiki/Encode/H.264
				27 - m_camera_path.render_settings.quality,
				m_camera_path.render_settings.filename
			);
			int ffmpeg_result = do_system(ffmpeg_command);
			if (ffmpeg_result == 0) {
				tlog::success() << "Saved video '" << m_camera_path.render_settings.filename << "'";
			} else if (ffmpeg_result == -1) {
				tlog::error() << "Video could not be assembled: FFmpeg not found.";
			} else {
				tlog::error() << "Video could not be assembled: FFmpeg failed";
			}

			clear_tmp_dir();
		}
	}

	const auto& rs = m_camera_path.render_settings;
	m_camera_path.play_time = (float)((double)m_camera_path.render_frame_idx / (double)rs.n_frames());

	if (m_views.front().render_buffer->spp() == 0) {
		set_camera_from_time(m_camera_path.play_time);
		apply_camera_smoothing(rs.frame_milliseconds());

		auto smoothed_camera_backup = m_smoothed_camera;

		// Compute the camera for the next frame in order to be able to compute motion blur
		// between it and the current one.
		set_camera_from_time(m_camera_path.play_time + 1.0f / rs.n_frames());
		apply_camera_smoothing(rs.frame_milliseconds());

		m_camera_path.render_frame_end_camera = m_smoothed_camera;

		// Revert camera such that the next frame will be computed correctly
		// (Start camera of next frame should be the same as end camera of this frame)
		set_camera_from_time(m_camera_path.play_time);
		m_smoothed_camera = smoothed_camera_backup;
	}
}

void Testbed::train_and_render(bool skip_rendering) {
	if (m_train) {
		train(m_training_batch_size);
	}

	// If we don't have a trainer, as can happen when having loaded training data or changed modes without having
	// explicitly loaded a new neural network.
	if (m_testbed_mode != ETestbedMode::None && !m_network) {
		reload_network_from_file();
		if (!m_network) {
			throw std::runtime_error{"Unable to reload neural network."};
		}
	}

	if (m_mesh.optimize_mesh) {
		optimise_mesh_step(1);
	}

	// Don't do any smoothing here if a camera path is being rendered. It'll take care
	// of the smoothing on its own.
	float frame_ms = m_camera_path.rendering ? 0.0f : m_frame_ms.val();
	apply_camera_smoothing(frame_ms);

	if (!m_render_window || !m_render || skip_rendering) {
		return;
	}

	auto start = std::chrono::steady_clock::now();
	ScopeGuard timing_guard{[&]() {
		m_render_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
	}};

	if ((m_smoothed_camera - m_camera).norm() < 0.001f) {
		m_smoothed_camera = m_camera;
	} else if (!m_camera_path.rendering) {
		reset_accumulation(true);
	}

	if (m_autofocus) {
		autofocus();
	}

#ifdef NGP_GUI
	if (m_hmd && m_hmd->is_visible()) {
		for (auto& view : m_views) {
			view.visualized_dimension = m_visualized_dimension;
		}

		m_n_views = {m_views.size(), 1};

		m_nerf.render_with_lens_distortion = false;
		reset_accumulation(true);
	} else if (m_single_view) {
		set_n_views(1);
		m_n_views = {1, 1};

		auto& view = m_views.front();

		view.full_resolution = m_window_res;

		view.camera0 = m_smoothed_camera;

		// Motion blur over the fraction of time that the shutter is open. Interpolate in log-space to preserve rotations.
		view.camera1 = m_camera_path.rendering ? log_space_lerp(m_smoothed_camera, m_camera_path.render_frame_end_camera, m_camera_path.render_settings.shutter_fraction) : view.camera0;

		view.visualized_dimension = m_visualized_dimension;
		view.relative_focal_length = m_relative_focal_length;
		view.screen_center = m_screen_center;
		view.render_buffer->set_hidden_area_mask(nullptr);
		view.foveation = {};
		view.device = &primary_device();
	} else {
		int n_views = n_dimensions_to_visualize()+1;

		float d = std::sqrt((float)m_window_res.x() * (float)m_window_res.y() / (float)n_views);

		int nx = (int)std::ceil((float)m_window_res.x() / d);
		int ny = (int)std::ceil((float)n_views / (float)nx);

		m_n_views = {nx, ny};
		Vector2i view_size = {m_window_res.x() / nx, m_window_res.y() / ny};

		set_n_views(n_views);

		int i = 0;
		for (int y = 0; y < ny; ++y) {
			for (int x = 0; x < nx; ++x) {
				if (i >= n_views) {
					break;
				}

				m_views[i].full_resolution = view_size;

				m_views[i].camera0 = m_views[i].camera1 = m_smoothed_camera;
				m_views[i].visualized_dimension = i-1;
				m_views[i].relative_focal_length = m_relative_focal_length;
				m_views[i].screen_center = m_screen_center;
				m_views[i].render_buffer->set_hidden_area_mask(nullptr);
				m_views[i].foveation = {};
				m_views[i].device = &primary_device();
				++i;
			}
		}
	}

	if (m_dlss) {
		m_aperture_size = 0.0f;
		if (!supports_dlss(m_nerf.render_lens.mode)) {
			m_nerf.render_with_lens_distortion = false;
		}
	}

	// Update dynamic res and DLSS
	{
		// Don't count the time being spent allocating buffers and resetting DLSS as part of the frame time.
		// Otherwise the dynamic resolution calculations for following frames will be thrown out of whack
		// and may even start oscillating.
		auto skip_start = std::chrono::steady_clock::now();
		ScopeGuard skip_timing_guard{[&]() {
			start += std::chrono::steady_clock::now() - skip_start;
		}};

		size_t n_pixels = 0, n_pixels_full_res = 0;
		for (const auto& view : m_views) {
			n_pixels += view.render_buffer->in_resolution().prod();
			n_pixels_full_res += view.full_resolution.prod();
		}

		float pixel_ratio = (n_pixels == 0 || (m_train && m_training_step == 0)) ? (1.0f / 256.0f) : ((float)n_pixels / (float)n_pixels_full_res);

		float last_factor = std::sqrt(pixel_ratio);
		float factor = std::sqrt(pixel_ratio / m_render_ms.val() * 1000.0f / m_dynamic_res_target_fps);
		if (!m_dynamic_res) {
			factor = 8.f / (float)m_fixed_res_factor;
		}

		factor = tcnn::clamp(factor, 1.0f / 16.0f, 1.0f);

		for (auto&& view : m_views) {
			if (m_dlss) {
				view.render_buffer->enable_dlss(*m_dlss_provider, view.full_resolution);
			} else {
				view.render_buffer->disable_dlss();
			}

			Vector2i render_res = view.render_buffer->in_resolution();
			Vector2i new_render_res = (view.full_resolution.cast<float>() * factor).cast<int>().cwiseMin(view.full_resolution).cwiseMax(view.full_resolution / 16);

			if (m_camera_path.rendering) {
				new_render_res = m_camera_path.render_settings.resolution;
			}

			float ratio = std::sqrt((float)render_res.prod() / (float)new_render_res.prod());
			if (ratio > 1.2f || ratio < 0.8f || factor == 1.0f || !m_dynamic_res || m_camera_path.rendering) {
				render_res = new_render_res;
			}

			if (view.render_buffer->dlss()) {
				render_res = view.render_buffer->dlss()->clamp_resolution(render_res);
				view.render_buffer->dlss()->update_feature(render_res, view.render_buffer->dlss()->is_hdr(), view.render_buffer->dlss()->sharpen());
			}

			view.render_buffer->resize(render_res);

			if (m_foveated_rendering) {
				float foveation_warped_full_res_diameter = 0.55f;
				Vector2f resolution_scale = render_res.cast<float>().cwiseQuotient(view.full_resolution.cast<float>());

				// Only start foveation when DLSS if off or if DLSS is asked to do more than 1.5x upscaling.
				// The reason for the 1.5x threshold is that DLSS can do up to 3x upscaling, at which point a foveation
				// factor of 2x = 3.0x/1.5x corresponds exactly to bilinear super sampling, which is helpful in
				// suppressing DLSS's artifacts.
				float foveation_begin_factor = m_dlss ? 1.5f : 1.0f;

				resolution_scale = (resolution_scale * foveation_begin_factor).cwiseMin(1.0f).cwiseMax(1.0f / m_foveated_rendering_max_scaling);
				view.foveation = {resolution_scale, Vector2f::Ones() - view.screen_center, Vector2f::Constant(foveation_warped_full_res_diameter * 0.5f)};
			} else {
				view.foveation = {};
			}
		}
	}

	// Make sure all in-use auxiliary GPUs have the latest model and bitfield
	std::unordered_set<CudaDevice*> devices_in_use;
	for (auto& view : m_views) {
		if (!view.device || devices_in_use.count(view.device) != 0) {
			continue;
		}

		devices_in_use.insert(view.device);
		sync_device(*view.render_buffer, *view.device);
	}

	{
		SyncedMultiStream synced_streams{m_stream.get(), m_views.size()};

		std::vector<std::future<void>> futures(m_views.size());
		for (size_t i = 0; i < m_views.size(); ++i) {
			auto& view = m_views[i];
			futures[i] = view.device->enqueue_task([this, &view, stream=synced_streams.get(i)]() {
				auto device_guard = use_device(stream, *view.render_buffer, *view.device);
				render_frame_main(*view.device, view.camera0, view.camera1, view.screen_center, view.relative_focal_length, {0.0f, 0.0f, 0.0f, 1.0f}, view.foveation, view.visualized_dimension);
			});
		}

		for (size_t i = 0; i < m_views.size(); ++i) {
			auto& view = m_views[i];

			if (futures[i].valid()) {
				futures[i].get();
			}

			render_frame_epilogue(synced_streams.get(i), view.camera0, view.prev_camera, view.screen_center, view.relative_focal_length, view.foveation, view.prev_foveation, *view.render_buffer, true);
			view.prev_camera = view.camera0;
			view.prev_foveation = view.foveation;
		}
	}

	for (size_t i = 0; i < m_views.size(); ++i) {
		m_rgba_render_textures.at(i)->blit_from_cuda_mapping();
		m_depth_render_textures.at(i)->blit_from_cuda_mapping();
	}

	if (m_picture_in_picture_res > 0) {
		Vector2i res(m_picture_in_picture_res, m_picture_in_picture_res * 9/16);
		m_pip_render_buffer->resize(res);
		if (m_pip_render_buffer->spp() < 8) {
			// a bit gross, but let's copy the keyframe's state into the global state in order to not have to plumb through the fov etc to render_frame.
			CameraKeyframe backup = copy_camera_to_keyframe();
			CameraKeyframe pip_kf = m_camera_path.eval_camera_path(m_camera_path.play_time);
			set_camera_from_keyframe(pip_kf);
			render_frame(m_stream.get(), pip_kf.m(), pip_kf.m(), pip_kf.m(), m_screen_center, m_relative_focal_length, Eigen::Vector4f::Zero(), {}, {}, m_visualized_dimension, *m_pip_render_buffer);
			set_camera_from_keyframe(backup);

			m_pip_render_texture->blit_from_cuda_mapping();
		}
	}
#endif

	CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
}


#ifdef NGP_GUI
void Testbed::create_second_window() {
	if (m_second_window.window) {
		return;
	}
	bool frameless = false;
	glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, GL_TRUE);
	glfwWindowHint(GLFW_RESIZABLE, !frameless);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	glfwWindowHint(GLFW_CENTER_CURSOR, false);
	glfwWindowHint(GLFW_DECORATED, !frameless);
	glfwWindowHint(GLFW_SCALE_TO_MONITOR, frameless);
	glfwWindowHint(GLFW_TRANSPARENT_FRAMEBUFFER, true);
	// get the window size / coordinates
	int win_w=0,win_h=0,win_x=0,win_y=0;
	GLuint ps=0,vs=0;
	{
		win_w = 1920;
		win_h = 1080;
		win_x = 0x40000000;
		win_y = 0x40000000;
		static const char* copy_shader_vert = "\
			in vec2 vertPos_data;\n\
			out vec2 texCoords;\n\
			void main(){\n\
				gl_Position = vec4(vertPos_data.xy, 0.0, 1.0);\n\
				texCoords = (vertPos_data.xy + 1.0) * 0.5; texCoords.y=1.0-texCoords.y;\n\
			}";
		static const char* copy_shader_frag = "\
			in vec2 texCoords;\n\
			out vec4 fragColor;\n\
			uniform sampler2D screenTex;\n\
			void main(){\n\
				fragColor = texture(screenTex, texCoords.xy);\n\
			}";
		vs = compile_shader(false, copy_shader_vert);
		ps = compile_shader(true, copy_shader_frag);
	}
	m_second_window.window = glfwCreateWindow(win_w, win_h, "Fullscreen Output", NULL, m_glfw_window);
	if (win_x!=0x40000000) glfwSetWindowPos(m_second_window.window, win_x, win_y);
	glfwMakeContextCurrent(m_second_window.window);
	m_second_window.program = glCreateProgram();
	glAttachShader(m_second_window.program, vs);
	glAttachShader(m_second_window.program, ps);
	glLinkProgram(m_second_window.program);
	if (!check_shader(m_second_window.program, "shader program", true)) {
		glDeleteProgram(m_second_window.program);
		m_second_window.program = 0;
	}
	// vbo and vao
	glGenVertexArrays(1, &m_second_window.vao);
	glGenBuffers(1, &m_second_window.vbo);
	glBindVertexArray(m_second_window.vao);
	const float fsquadVerts[] = {
		-1.0f, -1.0f,
		-1.0f, 1.0f,
		1.0f, 1.0f,
		1.0f, 1.0f,
		1.0f, -1.0f,
		-1.0f, -1.0f
	};
	glBindBuffer(GL_ARRAY_BUFFER, m_second_window.vbo);
	glBufferData(GL_ARRAY_BUFFER, sizeof(fsquadVerts), fsquadVerts, GL_STATIC_DRAW);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 2 * sizeof(float), (void *)0);
	glEnableVertexAttribArray(0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);
}

void Testbed::set_n_views(size_t n_views) {
	while (m_views.size() > n_views) {
		m_views.pop_back();
	}

	m_rgba_render_textures.resize(n_views);
	m_depth_render_textures.resize(n_views);
	while (m_views.size() < n_views) {
		size_t idx = m_views.size();
		m_rgba_render_textures[idx] = std::make_shared<GLTexture>();
		m_depth_render_textures[idx] = std::make_shared<GLTexture>();
		m_views.emplace_back(View{std::make_shared<CudaRenderBuffer>(m_rgba_render_textures[idx], m_depth_render_textures[idx])});
	}
};
#endif //NGP_GUI

void Testbed::init_window(int resw, int resh, bool hidden, bool second_window) {
#ifndef NGP_GUI
	throw std::runtime_error{"init_window failed: NGP was built without GUI support"};
#else
	m_window_res = {resw, resh};

	glfwSetErrorCallback(glfw_error_callback);
	if (!glfwInit()) {
		throw std::runtime_error{"GLFW could not be initialized."};
	}

#ifdef NGP_VULKAN
	// Only try to initialize DLSS (Vulkan+NGX) if the
	// GPU is sufficiently new. Older GPUs don't support
	// DLSS, so it is preferable to not make a futile
	// attempt and emit a warning that confuses users.
	if (primary_device().compute_capability() >= 70) {
		try {
			m_dlss_provider = init_vulkan_and_ngx();
			if (m_testbed_mode == ETestbedMode::Nerf) {
				m_dlss = true;
			}
		} catch (const std::runtime_error& e) {
			tlog::warning() << "Could not initialize Vulkan and NGX. DLSS not supported. (" << e.what() << ")";
		}
	}
#endif

	glfwWindowHint(GLFW_VISIBLE, hidden ? GLFW_FALSE : GLFW_TRUE);
	std::string title = "Instant Neural Graphics Primitives";
	m_glfw_window = glfwCreateWindow(m_window_res.x(), m_window_res.y(), title.c_str(), NULL, NULL);
	if (m_glfw_window == NULL) {
		throw std::runtime_error{"GLFW window could not be created."};
	}
	glfwMakeContextCurrent(m_glfw_window);
#ifdef _WIN32
	if (gl3wInit()) {
		throw std::runtime_error{"GL3W could not be initialized."};
	}
#else
	glewExperimental = 1;
	if (glewInit()) {
		throw std::runtime_error{"GLEW could not be initialized."};
	}
#endif
	glfwSwapInterval(0); // Disable vsync

	GLint gl_version_minor, gl_version_major;
	glGetIntegerv(GL_MINOR_VERSION, &gl_version_minor);
	glGetIntegerv(GL_MAJOR_VERSION, &gl_version_major);

	if (gl_version_major < 3 || (gl_version_major == 3 && gl_version_minor < 1)) {
		throw std::runtime_error{fmt::format("Unsupported OpenGL version {}.{}. instant-ngp requires at least OpenGL 3.1", gl_version_major, gl_version_minor)};
	}

	tlog::success() << "Initialized OpenGL version " << glGetString(GL_VERSION);

	glfwSetWindowUserPointer(m_glfw_window, this);
	glfwSetDropCallback(m_glfw_window, [](GLFWwindow* window, int count, const char** paths) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (!testbed) {
			return;
		}

		testbed->redraw_gui_next_frame();
		for (int i = 0; i < count; i++) {
			testbed->load_file(paths[i]);
		}
	});

	glfwSetKeyCallback(m_glfw_window, [](GLFWwindow* window, int key, int scancode, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetCursorPosCallback(m_glfw_window, [](GLFWwindow* window, double xpos, double ypos) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetMouseButtonCallback(m_glfw_window, [](GLFWwindow* window, int button, int action, int mods) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetScrollCallback(m_glfw_window, [](GLFWwindow* window, double xoffset, double yoffset) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_gui_next_frame();
		}
	});

	glfwSetWindowSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	glfwSetFramebufferSizeCallback(m_glfw_window, [](GLFWwindow* window, int width, int height) {
		Testbed* testbed = (Testbed*)glfwGetWindowUserPointer(window);
		if (testbed) {
			testbed->redraw_next_frame();
		}
	});

	float xscale, yscale;
	glfwGetWindowContentScale(m_glfw_window, &xscale, &yscale);

	// IMGUI init
	IMGUI_CHECKVERSION();
	ImGui::CreateContext();
	ImGuiIO& io = ImGui::GetIO(); (void)io;

	// By default, imgui places its configuration (state of the GUI -- size of windows,
	// which regions are expanded, etc.) in ./imgui.ini relative to the working directory.
	// Instead, we would like to place imgui.ini in the directory that instant-ngp project
	// resides in.
	static std::string ini_filename;
	ini_filename = (root_dir()/"imgui.ini").str();
	io.IniFilename = ini_filename.c_str();

	// New ImGui event handling seems to make camera controls laggy if input trickling is true.
	// So disable input trickling.
	io.ConfigInputTrickleEventQueue = false;
	ImGui::StyleColorsDark();
	ImGui_ImplGlfw_InitForOpenGL(m_glfw_window, true);
	ImGui_ImplOpenGL3_Init("#version 140");

	ImGui::GetStyle().ScaleAllSizes(xscale);
	ImFontConfig font_cfg;
	font_cfg.SizePixels = 13.0f * xscale;
	io.Fonts->AddFontDefault(&font_cfg);

	init_opengl_shaders();

	// Make sure there's at least one usable render texture
	m_rgba_render_textures = { std::make_shared<GLTexture>() };
	m_depth_render_textures = { std::make_shared<GLTexture>() };

	m_views.clear();
	m_views.emplace_back(View{std::make_shared<CudaRenderBuffer>(m_rgba_render_textures.front(), m_depth_render_textures.front())});
	m_views.front().full_resolution = m_window_res;
	m_views.front().render_buffer->resize(m_views.front().full_resolution);

	m_pip_render_texture = std::make_shared<GLTexture>();
	m_pip_render_buffer = std::make_unique<CudaRenderBuffer>(m_pip_render_texture);

	m_render_window = true;

	if (m_second_window.window == nullptr && second_window) {
		create_second_window();
	}
#endif // NGP_GUI
}

void Testbed::destroy_window() {
#ifndef NGP_GUI
	throw std::runtime_error{"destroy_window failed: NGP was built without GUI support"};
#else
	if (!m_render_window) {
		throw std::runtime_error{"Window must be initialized to be destroyed."};
	}

	m_hmd.reset();

	m_views.clear();
	m_rgba_render_textures.clear();
	m_depth_render_textures.clear();

	m_pip_render_buffer.reset();
	m_pip_render_texture.reset();

	m_dlss = false;
	m_dlss_provider.reset();

	ImGui_ImplOpenGL3_Shutdown();
	ImGui_ImplGlfw_Shutdown();
	ImGui::DestroyContext();
	glfwDestroyWindow(m_glfw_window);
	glfwTerminate();

	m_blit_program = 0;
	m_blit_vao = 0;

	m_glfw_window = nullptr;
	m_render_window = false;
#endif //NGP_GUI
}

void Testbed::init_vr() {
#ifndef NGP_GUI
	throw std::runtime_error{"init_vr failed: NGP was built without GUI support"};
#else
	try {
		if (!m_glfw_window) {
			throw std::runtime_error{"`init_window` must be called before `init_vr`"};
		}

#if defined(XR_USE_PLATFORM_WIN32)
		m_hmd = std::make_unique<OpenXRHMD>(wglGetCurrentDC(), glfwGetWGLContext(m_glfw_window));
#elif defined(XR_USE_PLATFORM_XLIB)
		Display* xDisplay = glfwGetX11Display();
		GLXContext glxContext = glfwGetGLXContext(m_glfw_window);

		int glxFBConfigXID = 0;
		glXQueryContext(xDisplay, glxContext, GLX_FBCONFIG_ID, &glxFBConfigXID);
		int attributes[3] = { GLX_FBCONFIG_ID, glxFBConfigXID, 0 };
		int nelements = 1;
		GLXFBConfig* pglxFBConfig = glXChooseFBConfig(xDisplay, 0, attributes, &nelements);
		if (nelements != 1 || !pglxFBConfig) {
			throw std::runtime_error{"init_vr(): Couldn't obtain GLXFBConfig"};
		}

		GLXFBConfig glxFBConfig = *pglxFBConfig;

		XVisualInfo* visualInfo = glXGetVisualFromFBConfig(xDisplay, glxFBConfig);
		if (!visualInfo) {
			throw std::runtime_error{"init_vr(): Couldn't obtain XVisualInfo"};
		}

		m_hmd = std::make_unique<OpenXRHMD>(xDisplay, visualInfo->visualid, glxFBConfig, glXGetCurrentDrawable(), glxContext);
#elif defined(XR_USE_PLATFORM_WAYLAND)
		m_hmd = std::make_unique<OpenXRHMD>(glfwGetWaylandDisplay());
#endif

		// Enable aggressive optimizations to make the VR experience smooth.
		update_vr_performance_settings();

		// If multiple GPUs are available, shoot for 60 fps in VR.
		// Otherwise, it wouldn't be realistic to expect more than 30.
		m_dynamic_res_target_fps = m_devices.size() > 1 ? 60 : 30;
		m_background_color = {0.0f, 0.0f, 0.0f, 0.0f};
	} catch (const std::runtime_error& e) {
		if (std::string{e.what()}.find("XR_ERROR_FORM_FACTOR_UNAVAILABLE") != std::string::npos) {
			throw std::runtime_error{"Could not initialize VR. Ensure that SteamVR, OculusVR, or any other OpenXR-compatible runtime is running. Also set it as the active OpenXR runtime."};
		} else {
			throw std::runtime_error{fmt::format("Could not initialize VR: {}", e.what())};
		}
	}
#endif //NGP_GUI
}

void Testbed::update_vr_performance_settings() {
#ifdef NGP_GUI
	if (m_hmd) {
		auto blend_mode = m_hmd->environment_blend_mode();

		// DLSS is instrumental in getting VR to look good. Enable if possible.
		// If the environment is blended in (such as in XR/AR applications),
		// DLSS causes jittering at object sillhouettes (doesn't deal well with alpha),
		// and hence stays disabled.
		m_dlss = (blend_mode == EEnvironmentBlendMode::Opaque) && m_dlss_provider;

		// Foveated rendering is similarly vital in getting high performance without losing
		// resolution in the middle of the view.
		m_foveated_rendering = true;

		// Large minimum transmittance results in another 20-30% performance increase
		// at the detriment of some transparent edges. Not super noticeable, though.
		m_nerf.render_min_transmittance = 0.2f;

		// Many VR runtimes perform optical flow for automatic reprojection / motion smoothing.
		// This breaks down for solid-color background, sometimes leading to artifacts. Hence:
		// set background color to transparent and, in spherical_checkerboard_kernel(...),
		// blend a checkerboard. If the user desires a solid background nonetheless, they can
		// set the background color to have an alpha value of 1.0 manually via the GUI or via Python.
		m_render_transparency_as_checkerboard = (blend_mode == EEnvironmentBlendMode::Opaque);
	} else {
		m_dlss = (m_testbed_mode == ETestbedMode::Nerf) && m_dlss_provider;
		m_foveated_rendering = false;
		m_nerf.render_min_transmittance = 0.01f;
		m_render_transparency_as_checkerboard = false;
	}
#endif //NGP_GUI
}

bool Testbed::frame() {
#ifdef NGP_GUI
	if (m_render_window) {
		if (!begin_frame()) {
			return false;
		}

		begin_vr_frame_and_handle_vr_input();
		handle_user_input();
	}
#endif

	// Render against the trained neural network. If we're training and already close to convergence,
	// we can skip rendering if the scene camera doesn't change
	uint32_t n_to_skip = m_train ? tcnn::clamp(m_training_step / 16u, 15u, 255u) : 0;
	if (m_render_skip_due_to_lack_of_camera_movement_counter > n_to_skip) {
		m_render_skip_due_to_lack_of_camera_movement_counter = 0;
	}
	bool skip_rendering = m_render_skip_due_to_lack_of_camera_movement_counter++ != 0;

	if (!m_dlss && m_max_spp > 0 && !m_views.empty() && m_views.front().render_buffer->spp() >= m_max_spp) {
		skip_rendering = true;
		if (!m_train) {
			std::this_thread::sleep_for(1ms);
		}
	}

	if (m_camera_path.rendering) {
		prepare_next_camera_path_frame();
		skip_rendering = false;
	}

#ifdef NGP_GUI
	if (m_hmd && m_hmd->is_visible()) {
		skip_rendering = false;
	}
#endif

	if (!skip_rendering || (std::chrono::steady_clock::now() - m_last_gui_draw_time_point) > 25ms) {
		redraw_gui_next_frame();
	}

	try {
		while (true) {
			(*m_task_queue.tryPop())();
		}
	} catch (SharedQueueEmptyException&) {}


	train_and_render(skip_rendering);
	if (m_testbed_mode == ETestbedMode::Sdf && m_sdf.calculate_iou_online) {
		m_sdf.iou = calculate_iou(m_train ? 64*64*64 : 128*128*128, m_sdf.iou_decay, false, true);
		m_sdf.iou_decay = 0.f;
	}

#ifdef NGP_GUI
	if (m_render_window) {
		if (m_gui_redraw) {
			if (m_gather_histograms) {
				gather_histograms();
			}

			draw_gui();
			m_gui_redraw = false;

			m_last_gui_draw_time_point = std::chrono::steady_clock::now();
		}

		ImGui::EndFrame();
	}

	if (m_hmd && m_vr_frame_info) {
		// If HMD is visible to the user, splat rendered images to the HMD
		if (m_hmd->is_visible()) {
			size_t n_views = std::min(m_views.size(), m_vr_frame_info->views.size());

			// Blit textures to the OpenXR-owned framebuffers (each corresponding to one eye)
			for (size_t i = 0; i < n_views; ++i) {
				const auto& vr_view = m_vr_frame_info->views.at(i);

				Vector2i resolution = {
					vr_view.view.subImage.imageRect.extent.width,
					vr_view.view.subImage.imageRect.extent.height,
				};

				blit_texture(m_views.at(i).foveation, m_rgba_render_textures.at(i)->texture(), GL_LINEAR, m_depth_render_textures.at(i)->texture(), vr_view.framebuffer, Vector2i::Zero(), resolution);
			}

			glFinish();
		}

		// Far and near planes are intentionally reversed, because we map depth inversely
		// to z. I.e. a window-space depth of 1 refers to the near plane and a depth of 0
		// to the far plane. This results in much better numeric precision.
		m_hmd->end_frame(m_vr_frame_info, m_ndc_zfar / m_scale, m_ndc_znear / m_scale, m_vr_depth_reproject);
	}
#endif

	return true;
}

fs::path Testbed::training_data_path() const {
	return m_data_path.with_extension("training");
}

bool Testbed::want_repl() {
	bool b = m_want_repl;
	m_want_repl = false;
	return b;
}

void Testbed::apply_camera_smoothing(float elapsed_ms) {
	if (m_camera_smoothing) {
		float decay = std::pow(0.02f, elapsed_ms/1000.0f);
		m_smoothed_camera = log_space_lerp(m_smoothed_camera, m_camera, 1.0f - decay);
	} else {
		m_smoothed_camera = m_camera;
	}
}

CameraKeyframe Testbed::copy_camera_to_keyframe() const {
	return CameraKeyframe(m_camera, m_slice_plane_z, m_scale, fov(), m_aperture_size, m_nerf.glow_mode, m_nerf.glow_y_cutoff);
}

void Testbed::set_camera_from_keyframe(const CameraKeyframe& k) {
	m_camera = k.m();
	m_slice_plane_z = k.slice;
	m_scale = k.scale;
	set_fov(k.fov);
	m_aperture_size = k.aperture_size;
	m_nerf.glow_mode = k.glow_mode;
	m_nerf.glow_y_cutoff = k.glow_y_cutoff;
}

void Testbed::set_camera_from_time(float t) {
	if (m_camera_path.keyframes.empty()) {
		return;
	}

	set_camera_from_keyframe(m_camera_path.eval_camera_path(t));
}

void Testbed::update_loss_graph() {
	m_loss_graph[m_loss_graph_samples++ % m_loss_graph.size()] = std::log(m_loss_scalar.val());
}

uint32_t Testbed::n_dimensions_to_visualize() const {
	return m_network->width(m_visualized_layer);
}

float Testbed::fov() const {
	return focal_length_to_fov(1.0f, m_relative_focal_length[m_fov_axis]);
}

void Testbed::set_fov(float val) {
	m_relative_focal_length = Vector2f::Constant(fov_to_focal_length(1, val));
}

Vector2f Testbed::fov_xy() const {
	return focal_length_to_fov(Vector2i::Ones(), m_relative_focal_length);
}

void Testbed::set_fov_xy(const Vector2f& val) {
	m_relative_focal_length = fov_to_focal_length(Vector2i::Ones(), val);
}

size_t Testbed::n_params() {
	return m_network->n_params();
}

size_t Testbed::n_encoding_params() {
	return m_network->n_params() - first_encoder_param();
}

size_t Testbed::first_encoder_param() {
	auto layer_sizes = m_network->layer_sizes();
	size_t first_encoder = 0;
	for (auto size : layer_sizes) {
		first_encoder += size.first * size.second;
	}
	return first_encoder;
}

uint32_t Testbed::network_width(uint32_t layer) const {
	return m_network->width(layer);
}

uint32_t Testbed::network_num_forward_activations() const {
	return m_network->num_forward_activations();
}

void Testbed::set_max_level(float maxlevel) {
	if (!m_network) return;
	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_max_level(maxlevel);
	}
	reset_accumulation();
}

void Testbed::set_min_level(float minlevel) {
	if (!m_network) return;
	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc) {
		hg_enc->set_quantize_threshold(powf(minlevel, 4.f) * 0.2f);
	}
	reset_accumulation();
}

void Testbed::set_visualized_layer(int layer) {
	m_visualized_layer = layer;
	m_visualized_dimension = std::max(-1, std::min(m_visualized_dimension, (int)m_network->width(layer)-1));
	reset_accumulation();
}

ELossType Testbed::string_to_loss_type(const std::string& str) {
	if (equals_case_insensitive(str, "L2")) {
		return ELossType::L2;
	} else if (equals_case_insensitive(str, "RelativeL2")) {
		return ELossType::RelativeL2;
	} else if (equals_case_insensitive(str, "L1")) {
		return ELossType::L1;
	} else if (equals_case_insensitive(str, "Mape")) {
		return ELossType::Mape;
	} else if (equals_case_insensitive(str, "Smape")) {
		return ELossType::Smape;
	} else if (equals_case_insensitive(str, "Huber") || equals_case_insensitive(str, "SmoothL1")) {
		// Legacy: we used to refer to the Huber loss (L2 near zero, L1 further away) as "SmoothL1".
		return ELossType::Huber;
	} else if (equals_case_insensitive(str, "LogL1")) {
		return ELossType::LogL1;
	} else {
		throw std::runtime_error{"Unknown loss type."};
	}
}

Testbed::NetworkDims Testbed::network_dims() const {
	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:   return network_dims_nerf(); break;
		case ETestbedMode::Sdf:    return network_dims_sdf(); break;
		case ETestbedMode::Image:  return network_dims_image(); break;
		case ETestbedMode::Volume: return network_dims_volume(); break;
		default: throw std::runtime_error{"Invalid mode."};
	}
}

void Testbed::reset_network(bool clear_density_grid) {
	m_sdf.iou_decay = 0;

	m_rng = default_rng_t{m_seed};

	// Start with a low rendering resolution and gradually ramp up
	m_render_ms.set(10000);

	reset_accumulation();
	m_nerf.training.counters_rgb.rays_per_batch = 1 << 12;
	m_nerf.training.counters_rgb.measured_batch_size_before_compaction = 0;
	m_nerf.training.n_steps_since_cam_update = 0;
	m_nerf.training.n_steps_since_error_map_update = 0;
	m_nerf.training.n_rays_since_error_map_update = 0;
	m_nerf.training.n_steps_between_error_map_updates = 128;
	m_nerf.training.error_map.is_cdf_valid = false;
	m_nerf.training.density_grid_rng = default_rng_t{m_rng.next_uint()};

	m_nerf.training.reset_camera_extrinsics();

	if (clear_density_grid) {
		m_nerf.density_grid.memset(0);
		m_nerf.density_grid_bitfield.memset(0);

		set_all_devices_dirty();
	}

	m_loss_graph_samples = 0;

	// Default config
	json config = m_network_config;

	json& encoding_config = config["encoding"];
	json& loss_config = config["loss"];
	json& optimizer_config = config["optimizer"];
	json& network_config = config["network"];

	// If the network config is incomplete, avoid doing further work.
	/*
	if (config.is_null() || encoding_config.is_null() || loss_config.is_null() || optimizer_config.is_null() || network_config.is_null()) {
		return;
	}
	*/

	auto dims = network_dims();

	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.loss_type = string_to_loss_type(loss_config.value("otype", "L2"));

		// Some of the Nerf-supported losses are not supported by tcnn::Loss,
		// so just create a dummy L2 loss there. The NeRF code path will bypass
		// the tcnn::Loss in any case.
		loss_config["otype"] = "L2";
	}

	// Automatically determine certain parameters if we're dealing with the (hash)grid encoding
	if (to_lower(encoding_config.value("otype", "OneBlob")).find("grid") != std::string::npos) {
		encoding_config["n_pos_dims"] = dims.n_pos;

		const uint32_t n_features_per_level = encoding_config.value("n_features_per_level", 2u);

		if (encoding_config.contains("n_features") && encoding_config["n_features"] > 0) {
			m_num_levels = (uint32_t)encoding_config["n_features"] / n_features_per_level;
		} else {
			m_num_levels = encoding_config.value("n_levels", 16u);
		}

		m_level_stats.resize(m_num_levels);
		m_first_layer_column_stats.resize(m_num_levels);

		const uint32_t log2_hashmap_size = encoding_config.value("log2_hashmap_size", 15);

		m_base_grid_resolution = encoding_config.value("base_resolution", 0);
		if (!m_base_grid_resolution) {
			m_base_grid_resolution = 1u << ((log2_hashmap_size) / dims.n_pos);
			encoding_config["base_resolution"] = m_base_grid_resolution;
		}

		float desired_resolution = 2048.0f; // Desired resolution of the finest hashgrid level over the unit cube
		if (m_testbed_mode == ETestbedMode::Image) {
			desired_resolution = m_image.resolution.maxCoeff() / 2.0f;
		} else if (m_testbed_mode == ETestbedMode::Volume) {
			desired_resolution = m_volume.world2index_scale;
		}

		// Automatically determine suitable per_level_scale
		m_per_level_scale = encoding_config.value("per_level_scale", 0.0f);
		if (m_per_level_scale <= 0.0f && m_num_levels > 1) {
			m_per_level_scale = std::exp(std::log(desired_resolution * (float)m_nerf.training.dataset.aabb_scale / (float)m_base_grid_resolution) / (m_num_levels-1));
			encoding_config["per_level_scale"] = m_per_level_scale;
		}

		tlog::info()
			<< "GridEncoding: "
			<< " Nmin=" << m_base_grid_resolution
			<< " b=" << m_per_level_scale
			<< " F=" << n_features_per_level
			<< " T=2^" << log2_hashmap_size
			<< " L=" << m_num_levels
			;
	}

	m_loss.reset(create_loss<precision_t>(loss_config));
	m_optimizer.reset(create_optimizer<precision_t>(optimizer_config));

	size_t n_encoding_params = 0;
	if (m_testbed_mode == ETestbedMode::Nerf) {
		m_nerf.training.cam_exposure.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Array3f>(1e-3f, Array3f::Zero()));
		m_nerf.training.cam_pos_offset.resize(m_nerf.training.dataset.n_images, AdamOptimizer<Vector3f>(1e-4f, Vector3f::Zero()));
		m_nerf.training.cam_rot_offset.resize(m_nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
		m_nerf.training.cam_focal_length_offset = AdamOptimizer<Vector2f>(1e-5f);

		m_nerf.training.reset_extra_dims(m_rng);

		json& dir_encoding_config = config["dir_encoding"];
		json& rgb_network_config = config["rgb_network"];

		uint32_t n_dir_dims = 3;
		uint32_t n_extra_dims = m_nerf.training.dataset.n_extra_dims();

		// Instantiate an additional model for each auxiliary GPU
		for (auto& device : m_devices) {
			device.set_nerf_network(std::make_shared<NerfNetwork<precision_t>>(
				dims.n_pos,
				n_dir_dims,
				n_extra_dims,
				dims.n_pos + 1, // The offset of 1 comes from the dt member variable of NerfCoordinate. HACKY
				encoding_config,
				dir_encoding_config,
				network_config,
				rgb_network_config
			));
		}

		m_network = m_nerf_network = primary_device().nerf_network();

		m_encoding = m_nerf_network->encoding();
		n_encoding_params = m_encoding->n_params() + m_nerf_network->dir_encoding()->n_params();

		tlog::info()
			<< "Density model: " << dims.n_pos
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_nerf_network->encoding()->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 1
			;

		tlog::info()
			<< "Color model:   " << n_dir_dims
			<< "--[" << std::string(dir_encoding_config["otype"])
			<< "]-->" << m_nerf_network->dir_encoding()->padded_output_width() << "+" << network_config.value("n_output_dims", 16u)
			<< "--[" << std::string(rgb_network_config["otype"])
			<< "(neurons=" << (int)rgb_network_config["n_neurons"] << ",layers=" << ((int)rgb_network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << 3
			;

		// Create distortion map model
		{
			json& distortion_map_optimizer_config =  config.contains("distortion_map") && config["distortion_map"].contains("optimizer") ? config["distortion_map"]["optimizer"] : optimizer_config;

			m_distortion.resolution = Vector2i::Constant(32);
			if (config.contains("distortion_map") && config["distortion_map"].contains("resolution")) {
				from_json(config["distortion_map"]["resolution"], m_distortion.resolution);
			}
			m_distortion.map = std::make_shared<TrainableBuffer<2, 2, float>>(m_distortion.resolution);
			m_distortion.optimizer.reset(create_optimizer<float>(distortion_map_optimizer_config));
			m_distortion.trainer = std::make_shared<Trainer<float, float>>(m_distortion.map, m_distortion.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(loss_config)}, m_seed);
		}
	} else {
		uint32_t alignment = network_config.contains("otype") && (equals_case_insensitive(network_config["otype"], "FullyFusedMLP") || equals_case_insensitive(network_config["otype"], "MegakernelMLP")) ? 16u : 8u;

		if (encoding_config.contains("otype") && equals_case_insensitive(encoding_config["otype"], "Takikawa")) {
			if (m_sdf.octree_depth_target == 0) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}

			if (!m_sdf.triangle_octree || m_sdf.triangle_octree->depth() != m_sdf.octree_depth_target) {
				m_sdf.triangle_octree.reset(new TriangleOctree{});
				m_sdf.triangle_octree->build(*m_sdf.triangle_bvh, m_sdf.triangles_cpu, m_sdf.octree_depth_target);
				m_sdf.octree_depth_target = m_sdf.triangle_octree->depth();
				m_sdf.brick_data.free_memory();
			}

			m_encoding.reset(new TakikawaEncoding<precision_t>(
				encoding_config["starting_level"],
				m_sdf.triangle_octree,
				tcnn::string_to_interpolation_type(encoding_config.value("interpolation", "linear"))
			));

			m_sdf.uses_takikawa_encoding = true;
		} else {
			m_encoding.reset(create_encoding<precision_t>(dims.n_input, encoding_config));

			m_sdf.uses_takikawa_encoding = false;
			if (m_sdf.octree_depth_target == 0 && encoding_config.contains("n_levels")) {
				m_sdf.octree_depth_target = encoding_config["n_levels"];
			}
		}

		for (auto& device : m_devices) {
			device.set_network(std::make_shared<NetworkWithInputEncoding<precision_t>>(m_encoding, dims.n_output, network_config));
		}

		m_network = primary_device().network();

		n_encoding_params = m_encoding->n_params();

		tlog::info()
			<< "Model:         " << dims.n_input
			<< "--[" << std::string(encoding_config["otype"])
			<< "]-->" << m_encoding->padded_output_width()
			<< "--[" << std::string(network_config["otype"])
			<< "(neurons=" << (int)network_config["n_neurons"] << ",layers=" << ((int)network_config["n_hidden_layers"]+2) << ")"
			<< "]-->" << dims.n_output;
	}

	size_t n_network_params = m_network->n_params() - n_encoding_params;

	tlog::info() << "  total_encoding_params=" << n_encoding_params << " total_network_params=" << n_network_params;

	m_trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(m_network, m_optimizer, m_loss, m_seed);
	m_training_step = 0;
	m_training_start_time_point = std::chrono::steady_clock::now();

	// Create envmap model
	{
		json& envmap_loss_config = config.contains("envmap") && config["envmap"].contains("loss") ? config["envmap"]["loss"] : loss_config;
		json& envmap_optimizer_config =  config.contains("envmap") && config["envmap"].contains("optimizer") ? config["envmap"]["optimizer"] : optimizer_config;

		m_envmap.loss_type = string_to_loss_type(envmap_loss_config.value("otype", "L2"));

		m_envmap.resolution = m_nerf.training.dataset.envmap_resolution;
		m_envmap.envmap = std::make_shared<TrainableBuffer<4, 2, float>>(m_envmap.resolution);
		m_envmap.optimizer.reset(create_optimizer<float>(envmap_optimizer_config));
		m_envmap.trainer = std::make_shared<Trainer<float, float, float>>(m_envmap.envmap, m_envmap.optimizer, std::shared_ptr<Loss<float>>{create_loss<float>(envmap_loss_config)}, m_seed);

		if (m_nerf.training.dataset.envmap_data.data()) {
			m_envmap.trainer->set_params_full_precision(m_nerf.training.dataset.envmap_data.data(), m_nerf.training.dataset.envmap_data.size());
		}
	}

	set_all_devices_dirty();
}

Testbed::Testbed(ETestbedMode mode) {
	if (!(__CUDACC_VER_MAJOR__ > 10 || (__CUDACC_VER_MAJOR__ == 10 && __CUDACC_VER_MINOR__ >= 2))) {
		throw std::runtime_error{"Testbed requires CUDA 10.2 or later."};
	}

#ifdef NGP_GUI
	// Ensure we're running on the GPU that'll host our GUI. To do so, try creating a dummy
	// OpenGL context, figure out the GPU it's running on, and then kill that context again.
	if (!is_wsl() && glfwInit()) {
		glfwWindowHint(GLFW_VISIBLE, GLFW_FALSE);
		GLFWwindow* offscreen_context = glfwCreateWindow(640, 480, "", NULL, NULL);

		if (offscreen_context) {
			glfwMakeContextCurrent(offscreen_context);

			int gl_device = -1;
			unsigned int device_count = 0;
			if (hipGLGetDevices(&device_count, &gl_device, 1, hipGLDeviceListAll) == hipSuccess) {
				if (device_count > 0 && gl_device >= 0) {
					set_cuda_device(gl_device);
				}
			}

			glfwDestroyWindow(offscreen_context);
		}

		glfwTerminate();
	}
#endif

	// Reset our stream, which was allocated on the originally active device,
	// to make sure it corresponds to the now active device.
	m_stream = {};

	int active_device = cuda_device();
	int active_compute_capability = cuda_compute_capability();
	tlog::success() << "Initialized CUDA. Active GPU is #" << active_device << ": " << cuda_device_name() << " [" << active_compute_capability << "]";

	if (active_compute_capability < MIN_GPU_ARCH) {
		tlog::warning() << "Insufficient compute capability " << active_compute_capability << " detected.";
		tlog::warning() << "This program was compiled for >=" << MIN_GPU_ARCH << " and may thus behave unexpectedly.";
	}

	m_devices.emplace_back(active_device, true);

	// Multi-GPU is only supported in NeRF mode for now
	int n_devices = cuda_device_count();
	for (int i = 0; i < n_devices; ++i) {
		if (i == active_device) {
			continue;
		}

		if (cuda_compute_capability(i) >= MIN_GPU_ARCH) {
			m_devices.emplace_back(i, false);
		}
	}

	if (m_devices.size() > 1) {
		tlog::success() << "Detected auxiliary GPUs:";
		for (size_t i = 1; i < m_devices.size(); ++i) {
			const auto& device = m_devices[i];
			tlog::success() << "  #" << device.id() << ": " << device.name() << " [" << device.compute_capability() << "]";
		}
	}

	m_network_config = {
		{"loss", {
			{"otype", "L2"}
		}},
		{"optimizer", {
			{"otype", "Adam"},
			{"learning_rate", 1e-3},
			{"beta1", 0.9f},
			{"beta2", 0.99f},
			{"epsilon", 1e-15f},
			{"l2_reg", 1e-6f},
		}},
		{"encoding", {
			{"otype", "HashGrid"},
			{"n_levels", 16},
			{"n_features_per_level", 2},
			{"log2_hashmap_size", 19},
			{"base_resolution", 16},
		}},
		{"network", {
			{"otype", "FullyFusedMLP"},
			{"n_neurons", 64},
			{"n_layers", 2},
			{"activation", "ReLU"},
			{"output_activation", "None"},
		}},
	};

	set_mode(mode);
	set_exposure(0);
	set_min_level(0.f);
	set_max_level(1.f);

	reset_camera();
}

Testbed::~Testbed() {

	// If any temporary file was created, make sure it's deleted
	clear_tmp_dir();

	if (m_render_window) {
		destroy_window();
	}
}

bool Testbed::clear_tmp_dir() {
	wait_all(m_render_futures);
	m_render_futures.clear();

	bool success = true;
	auto tmp_dir = fs::path{"tmp"};
	if (tmp_dir.exists()) {
		if (tmp_dir.is_directory()) {
			for (const auto& path : fs::directory{tmp_dir}) {
				if (path.is_file()) {
					success &= path.remove_file();
				}
			}
		}

		success &= tmp_dir.remove_file();
	}

	return success;
}

void Testbed::train(uint32_t batch_size) {
	if (!m_training_data_available || m_camera_path.rendering) {
		m_train = false;
		return;
	}

	if (m_testbed_mode == ETestbedMode::None) {
		throw std::runtime_error{"Cannot train without a mode."};
	}

	set_all_devices_dirty();

	// If we don't have a trainer, as can happen when having loaded training data or changed modes without having
	// explicitly loaded a new neural network.
	if (!m_trainer) {
		reload_network_from_file();
		if (!m_trainer) {
			throw std::runtime_error{"Unable to create a neural network trainer."};
		}
	}

	if (!m_dlss) {
		// No immediate redraw necessary
		reset_accumulation(false, false);
	}

	uint32_t n_prep_to_skip = m_testbed_mode == ETestbedMode::Nerf ? tcnn::clamp(m_training_step / 16u, 1u, 16u) : 1u;
	if (m_training_step % n_prep_to_skip == 0) {
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_prep_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count() / n_prep_to_skip);
		}};

		switch (m_testbed_mode) {
			case ETestbedMode::Nerf: training_prep_nerf(batch_size, m_stream.get()); break;
			case ETestbedMode::Sdf: training_prep_sdf(batch_size, m_stream.get()); break;
			case ETestbedMode::Image: training_prep_image(batch_size, m_stream.get()); break;
			case ETestbedMode::Volume: training_prep_volume(batch_size, m_stream.get()); break;
			default: throw std::runtime_error{"Invalid training mode."};
		}

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	// Find leaf optimizer and update its settings
	json* leaf_optimizer_config = &m_network_config["optimizer"];
	while (leaf_optimizer_config->contains("nested")) {
		leaf_optimizer_config = &(*leaf_optimizer_config)["nested"];
	}
	(*leaf_optimizer_config)["optimize_matrix_params"] = m_train_network;
	(*leaf_optimizer_config)["optimize_non_matrix_params"] = m_train_encoding;
	m_optimizer->update_hyperparams(m_network_config["optimizer"]);

	bool get_loss_scalar = m_training_step % 16 == 0;

	{
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
		}};

		switch (m_testbed_mode) {
			case ETestbedMode::Nerf: train_nerf(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Sdf: train_sdf(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Image: train_image(batch_size, get_loss_scalar, m_stream.get()); break;
			case ETestbedMode::Volume: train_volume(batch_size, get_loss_scalar, m_stream.get()); break;
			default: throw std::runtime_error{"Invalid training mode."};
		}

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	if (get_loss_scalar) {
		update_loss_graph();
	}
}

Vector2f Testbed::calc_focal_length(const Vector2i& resolution, const Vector2f& relative_focal_length, int fov_axis, float zoom) const {
	return relative_focal_length * resolution[fov_axis] * zoom;
}

Vector2f Testbed::render_screen_center(const Vector2f& screen_center) const {
	// see pixel_to_ray for how screen center is used; 0.5, 0.5 is 'normal'. we flip so that it becomes the point in the original image we want to center on.
	return (Vector2f::Constant(0.5f) - screen_center) * m_zoom + Vector2f::Constant(0.5f);
}

__global__ void dlss_prep_kernel(
	Vector2i resolution,
	uint32_t sample_index,
	Vector2f focal_length,
	Vector2f screen_center,
	Vector3f parallax_shift,
	bool snap_to_pixel_centers,
	float* depth_buffer,
	const float znear,
	const float zfar,
	Matrix<float, 3, 4> camera,
	Matrix<float, 3, 4> prev_camera,
	hipSurfaceObject_t depth_surface,
	hipSurfaceObject_t mvec_surface,
	hipSurfaceObject_t exposure_surface,
	Foveation foveation,
	Foveation prev_foveation,
	Lens lens
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	uint32_t idx = x + resolution.x() * y;

	uint32_t x_orig = x;
	uint32_t y_orig = y;

	const float depth = depth_buffer[idx];
	Vector2f mvec = motion_vector(
		sample_index,
		{x, y},
		resolution,
		focal_length,
		camera,
		prev_camera,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		depth,
		foveation,
		prev_foveation,
		lens
	);

	surf2Dwrite(make_float2(mvec.x(), mvec.y()), mvec_surface, x_orig * sizeof(float2), y_orig);

	// DLSS was trained on games, which presumably used standard normalized device coordinates (ndc)
	// depth buffers. So: convert depth to NDC with reasonable near- and far planes.
	surf2Dwrite(to_ndc_depth(depth, znear, zfar), depth_surface, x_orig * sizeof(float), y_orig);

	// First thread write an exposure factor of 1. Since DLSS will run on tonemapped data,
	// exposure is assumed to already have been applied to DLSS' inputs.
	if (x_orig == 0 && y_orig == 0) {
		surf2Dwrite(1.0f, exposure_surface, 0, 0);
	}
}

__global__ void spherical_checkerboard_kernel(
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera,
	Vector2f screen_center,
	Vector3f parallax_shift,
	Foveation foveation,
	Lens lens,
	Array4f background_color,
	Array4f* frame_buffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	Ray ray = pixel_to_ray(
		0,
		{x, y},
		resolution,
		focal_length,
		camera,
		screen_center,
		parallax_shift,
		false,
		0.0f,
		1.0f,
		0.0f,
		foveation,
		{}, // No need for hidden area mask
		lens
	);

	// Blend with checkerboard to break up reprojection weirdness in some VR runtimes
	host_device_swap(ray.d.z(), ray.d.y());
	Vector2f spherical = dir_to_spherical(ray.d.normalized()) * 32.0f / PI();
	const Array4f dark_gray = {0.5f, 0.5f, 0.5f, 1.0f};
	const Array4f light_gray = {0.55f, 0.55f, 0.55f, 1.0f};
	Array4f checker = fabsf(fmodf(floorf(spherical.x()) + floorf(spherical.y()), 2.0f)) < 0.5f ? dark_gray : light_gray;

	// Blend background color on top of checkerboard first (checkerboard is meant to be "behind" the background,
	// representing transparency), and then blend the result behind the frame buffer.
	background_color.head<3>() = srgb_to_linear(background_color.head<3>());
	background_color += (1.0f - background_color.w()) * checker;

	uint32_t idx = x + resolution.x() * y;
	frame_buffer[idx] += (1.0f - frame_buffer[idx].w()) * background_color;
}

__global__ void vr_overlay_hands_kernel(
	Vector2i resolution,
	Vector2f focal_length,
	Matrix<float, 3, 4> camera,
	Vector2f screen_center,
	Vector3f parallax_shift,
	Foveation foveation,
	Lens lens,
	Vector3f left_hand_pos,
	float left_grab_strength,
	Array4f left_hand_color,
	Vector3f right_hand_pos,
	float right_grab_strength,
	Array4f right_hand_color,
	float hand_radius,
	EColorSpace output_color_space,
	hipSurfaceObject_t surface
	// TODO: overwrite depth buffer
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x() || y >= resolution.y()) {
		return;
	}

	Ray ray = pixel_to_ray(
		0,
		{x, y},
		resolution,
		focal_length,
		camera,
		screen_center,
		parallax_shift,
		false,
		0.0f,
		1.0f,
		0.0f,
		foveation,
		{}, // No need for hidden area mask
		lens
	);

	Array4f color = Array4f::Zero();
	auto composit_hand = [&](Vector3f hand_pos, float grab_strength, Array4f hand_color) {
		// Don't render the hand indicator if it's behind the ray origin.
		if (ray.d.dot(hand_pos - ray.o) < 0.0f) {
			return;
		}

		float distance = ray.distance_to(hand_pos);

		Array4f base_color = Array4f::Zero();
		const Array4f border_color = {0.4f, 0.4f, 0.4f, 0.4f};

		// Divide hand radius into an inner part (4/5ths) and a border (1/5th).
		float radius = hand_radius * 0.8f;
		float border_width = hand_radius * 0.2f;

		// When grabbing, shrink the inner part as a visual indicator.
		radius *= 0.5f + 0.5f * (1.0f - grab_strength);

		if (distance < radius) {
			base_color = hand_color;
		} else if (distance < radius + border_width) {
			base_color = border_color;
		} else {
			return;
		}

		// Make hand color opaque when grabbing.
		base_color.w() = grab_strength + (1.0f - grab_strength) * base_color.w();
		color += base_color * (1.0f - color.w());
	};

	if (ray.d.dot(left_hand_pos - ray.o) < ray.d.dot(right_hand_pos - ray.o)) {
		composit_hand(left_hand_pos, left_grab_strength, left_hand_color);
		composit_hand(right_hand_pos, right_grab_strength, right_hand_color);
	} else {
		composit_hand(right_hand_pos, right_grab_strength, right_hand_color);
		composit_hand(left_hand_pos, left_grab_strength, left_hand_color);
	}

	// Blend with existing color of pixel
	Array4f prev_color;
	surf2Dread((float4*)&prev_color, surface, x * sizeof(float4), y);
	if (output_color_space == EColorSpace::SRGB) {
		prev_color.head<3>() = srgb_to_linear(prev_color.head<3>());
	}

	color += (1.0f - color.w()) * prev_color;

	if (output_color_space == EColorSpace::SRGB) {
		color.head<3>() = linear_to_srgb(color.head<3>());
	}

	surf2Dwrite(to_float4(color), surface, x * sizeof(float4), y);
}

void Testbed::render_frame(
	hipStream_t stream,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& camera_matrix1,
	const Matrix<float, 3, 4>& prev_camera_matrix,
	const Vector2f& orig_screen_center,
	const Vector2f& relative_focal_length,
	const Vector4f& nerf_rolling_shutter,
	const Foveation& foveation,
	const Foveation& prev_foveation,
	int visualized_dimension,
	CudaRenderBuffer& render_buffer,
	bool to_srgb,
	CudaDevice* device
) {
	if (!device) {
		device = &primary_device();
	}

	sync_device(render_buffer, *device);

	{
		auto device_guard = use_device(stream, render_buffer, *device);
		render_frame_main(*device, camera_matrix0, camera_matrix1, orig_screen_center, relative_focal_length, nerf_rolling_shutter, foveation, visualized_dimension);
	}

	render_frame_epilogue(stream, camera_matrix0, prev_camera_matrix, orig_screen_center, relative_focal_length, foveation, prev_foveation, render_buffer, to_srgb);
}

void Testbed::render_frame_main(
	CudaDevice& device,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& camera_matrix1,
	const Vector2f& orig_screen_center,
	const Vector2f& relative_focal_length,
	const Vector4f& nerf_rolling_shutter,
	const Foveation& foveation,
	int visualized_dimension
) {
	device.render_buffer_view().clear(device.stream());

	if (!m_network) {
		return;
	}

	Vector2f focal_length = calc_focal_length(device.render_buffer_view().resolution, relative_focal_length, m_fov_axis, m_zoom);
	Vector2f screen_center = render_screen_center(orig_screen_center);

	switch (m_testbed_mode) {
		case ETestbedMode::Nerf:
			if (!m_render_ground_truth || m_ground_truth_alpha < 1.0f) {
				render_nerf(device.stream(), device.render_buffer_view(), *device.nerf_network(), device.data().density_grid_bitfield_ptr, focal_length, camera_matrix0, camera_matrix1, nerf_rolling_shutter, screen_center, foveation, visualized_dimension);
			}
			break;
		case ETestbedMode::Sdf:
			{
				if (m_render_ground_truth && m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
					if (m_sdf.brick_data.size() == 0) {
						tlog::info() << "Building voxel brick positions for " << m_sdf.triangle_octree->n_dual_nodes() << " dual nodes.";
						m_sdf.brick_res = 5;
						std::vector<Eigen::Vector3f> positions = m_sdf.triangle_octree->build_brick_voxel_position_list(m_sdf.brick_res);
						GPUMemory<Eigen::Vector3f> positions_gpu;
						positions_gpu.resize_and_copy_from_host(positions);
						m_sdf.brick_data.resize(positions.size());
						tlog::info() << positions_gpu.size() << " voxel brick positions. Computing SDFs.";
						m_sdf.triangle_bvh->signed_distance_gpu(
							positions.size(),
							EMeshSdfMode::Watertight, //m_sdf.mesh_sdf_mode, // watertight seems to be the best method for 'one off' SDF signing
							positions_gpu.data(),
							m_sdf.brick_data.data(),
							m_sdf.triangles_gpu.data(),
							false,
							device.stream()
						);
					}
				}

				distance_fun_t distance_fun =
					m_render_ground_truth ? (distance_fun_t)[&](uint32_t n_elements, const Vector3f* positions, float* distances, hipStream_t stream) {
						if (m_sdf.groundtruth_mode == ESDFGroundTruthMode::SDFBricks) {
							// linear_kernel(sdf_brick_kernel, 0, stream,
							// 	n_elements,
							// 	positions.data(),
							// 	distances.data(),
							// 	m_sdf.triangle_octree->nodes_gpu(),
							// 	m_sdf.triangle_octree->dual_nodes_gpu(),
							// 	std::max(1u,std::min(m_sdf.triangle_octree->depth(), m_sdf.brick_level)),
							// 	m_sdf.brick_data.data(),
							// 	m_sdf.brick_res,
							// 	m_sdf.brick_quantise_bits
							// );
						} else {
							m_sdf.triangle_bvh->signed_distance_gpu(
								n_elements,
								m_sdf.mesh_sdf_mode,
								positions,
								distances,
								m_sdf.triangles_gpu.data(),
								false,
								stream
							);
						}
					} : (distance_fun_t)[&](uint32_t n_elements, const Vector3f* positions, float* distances, hipStream_t stream) {
						n_elements = next_multiple(n_elements, tcnn::batch_size_granularity);
						GPUMatrix<float> positions_matrix((float*)positions, 3, n_elements);
						GPUMatrix<float, RM> distances_matrix(distances, 1, n_elements);
						m_network->inference(stream, positions_matrix, distances_matrix);
					};

				normals_fun_t normals_fun =
					m_render_ground_truth ? (normals_fun_t)[&](uint32_t n_elements, const Vector3f* positions, Vector3f* normals, hipStream_t stream) {
						// NO-OP. Normals will automatically be populated by raytrace
					} : (normals_fun_t)[&](uint32_t n_elements, const Vector3f* positions, Vector3f* normals, hipStream_t stream) {
						n_elements = next_multiple(n_elements, tcnn::batch_size_granularity);
						GPUMatrix<float> positions_matrix((float*)positions, 3, n_elements);
						GPUMatrix<float> normals_matrix((float*)normals, 3, n_elements);
						m_network->input_gradient(stream, 0, positions_matrix, normals_matrix);
					};

				render_sdf(
					device.stream(),
					distance_fun,
					normals_fun,
					device.render_buffer_view(),
					focal_length,
					camera_matrix0,
					screen_center,
					foveation,
					visualized_dimension
				);
			}
			break;
		case ETestbedMode::Image:
			render_image(device.stream(), device.render_buffer_view(), focal_length, camera_matrix0, screen_center, foveation, visualized_dimension);
			break;
		case ETestbedMode::Volume:
			render_volume(device.stream(), device.render_buffer_view(), focal_length, camera_matrix0, screen_center, foveation);
			break;
		default:
			// No-op if no mode is active
			break;
	}
}

void Testbed::render_frame_epilogue(
	hipStream_t stream,
	const Matrix<float, 3, 4>& camera_matrix0,
	const Matrix<float, 3, 4>& prev_camera_matrix,
	const Vector2f& orig_screen_center,
	const Vector2f& relative_focal_length,
	const Foveation& foveation,
	const Foveation& prev_foveation,
	CudaRenderBuffer& render_buffer,
	bool to_srgb
) {
	Vector2f focal_length = calc_focal_length(render_buffer.in_resolution(), relative_focal_length, m_fov_axis, m_zoom);
	Vector2f screen_center = render_screen_center(orig_screen_center);

	render_buffer.set_color_space(m_color_space);
	render_buffer.set_tonemap_curve(m_tonemap_curve);

	Lens lens = (m_testbed_mode == ETestbedMode::Nerf && m_nerf.render_with_lens_distortion) ? m_nerf.render_lens : Lens{};

	// Prepare DLSS data: motion vectors, scaled depth, exposure
	if (render_buffer.dlss()) {
		auto res = render_buffer.in_resolution();

		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };

		dlss_prep_kernel<<<blocks, threads, 0, stream>>>(
			res,
			render_buffer.spp(),
			focal_length,
			screen_center,
			m_parallax_shift,
			m_snap_to_pixel_centers,
			render_buffer.depth_buffer(),
			m_ndc_znear,
			m_ndc_zfar,
			camera_matrix0,
			prev_camera_matrix,
			render_buffer.dlss()->depth(),
			render_buffer.dlss()->mvec(),
			render_buffer.dlss()->exposure(),
			foveation,
			prev_foveation,
			lens
		);

		render_buffer.set_dlss_sharpening(m_dlss_sharpening);
	}

	EColorSpace output_color_space = to_srgb ? EColorSpace::SRGB : EColorSpace::Linear;

	if (m_render_transparency_as_checkerboard) {
		Matrix<float, 3, 4> checkerboard_transform = Matrix<float, 3, 4>::Identity();

#if NGP_GUI
		if (m_hmd && m_vr_frame_info && !m_vr_frame_info->views.empty()) {
			checkerboard_transform = m_vr_frame_info->views[0].pose;
		}
#endif

		auto res = render_buffer.in_resolution();
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
		spherical_checkerboard_kernel<<<blocks, threads, 0, stream>>>(
			res,
			focal_length,
			checkerboard_transform,
			screen_center,
			m_parallax_shift,
			foveation,
			lens,
			m_background_color,
			render_buffer.frame_buffer()
		);
	}

	render_buffer.accumulate(m_exposure, stream);
	render_buffer.tonemap(m_exposure, m_background_color, output_color_space, m_ndc_znear, m_ndc_zfar, stream);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		// Overlay the ground truth image if requested
		if (m_render_ground_truth) {
			auto const& metadata = m_nerf.training.dataset.metadata[m_nerf.training.view];
			if (m_ground_truth_render_mode == EGroundTruthRenderMode::Shade) {
				render_buffer.overlay_image(
					m_ground_truth_alpha,
					Array3f::Constant(m_exposure) + m_nerf.training.cam_exposure[m_nerf.training.view].variable(),
					m_background_color,
					output_color_space,
					metadata.pixels,
					metadata.image_data_type,
					metadata.resolution,
					m_fov_axis,
					m_zoom,
					Vector2f::Constant(0.5f),
					stream
				);
			} else if (m_ground_truth_render_mode == EGroundTruthRenderMode::Depth && metadata.depth) {
				render_buffer.overlay_depth(
					m_ground_truth_alpha,
					metadata.depth,
					1.0f/m_nerf.training.dataset.scale,
					metadata.resolution,
					m_fov_axis,
					m_zoom,
					Vector2f::Constant(0.5f),
					stream
				);
			}
		}

		// Visualize the accumulated error map if requested
		if (m_nerf.training.render_error_overlay) {
			const float* err_data = m_nerf.training.error_map.data.data();
			Vector2i error_map_res = m_nerf.training.error_map.resolution;
			if (m_render_ground_truth) {
				err_data = m_nerf.training.dataset.sharpness_data.data();
				error_map_res = m_nerf.training.dataset.sharpness_resolution;
			}
			size_t emap_size = error_map_res.x() * error_map_res.y();
			err_data += emap_size * m_nerf.training.view;

			GPUMemory<float> average_error;
			average_error.enlarge(1);
			average_error.memset(0);
			const float* aligned_err_data_s = (const float*)(((size_t)err_data)&~15);
			const float* aligned_err_data_e = (const float*)(((size_t)(err_data+emap_size))&~15);
			size_t reduce_size = aligned_err_data_e - aligned_err_data_s;
			reduce_sum(aligned_err_data_s, [reduce_size] __device__ (float val) { return max(val,0.f) / (reduce_size); }, average_error.data(), reduce_size, stream);
			auto const &metadata = m_nerf.training.dataset.metadata[m_nerf.training.view];
			render_buffer.overlay_false_color(metadata.resolution, to_srgb, m_fov_axis, stream, err_data, error_map_res, average_error.data(), m_nerf.training.error_overlay_brightness, m_render_ground_truth);
		}
	}

#if NGP_GUI
	// If in VR, indicate the hand position and render transparent background
	if (m_hmd && m_vr_frame_info) {
		auto& hands = m_vr_frame_info->hands;

		auto res = render_buffer.out_resolution();
		const dim3 threads = { 16, 8, 1 };
		const dim3 blocks = { div_round_up((uint32_t)res.x(), threads.x), div_round_up((uint32_t)res.y(), threads.y), 1 };
		vr_overlay_hands_kernel<<<blocks, threads, 0, stream>>>(
			res,
			focal_length.cwiseProduct(render_buffer.out_resolution().cast<float>()).cwiseQuotient(render_buffer.in_resolution().cast<float>()),
			camera_matrix0,
			screen_center,
			m_parallax_shift,
			foveation,
			lens,
			vr_to_world(hands[0].pose.col(3)),
			hands[0].grab_strength,
			{hands[0].pressing ? 0.8f : 0.0f, 0.0f, 0.0f, 0.8f},
			vr_to_world(hands[1].pose.col(3)),
			hands[1].grab_strength,
			{hands[1].pressing ? 0.8f : 0.0f, 0.0f, 0.0f, 0.8f},
			0.05f * m_scale, // Hand radius
			output_color_space,
			render_buffer.surface()
		);
	}
#endif
}

float Testbed::get_depth_from_renderbuffer(const CudaRenderBuffer& render_buffer, const Vector2f& uv) {
	if (!render_buffer.depth_buffer()) {
		return m_scale;
	}

	float depth;
	auto res = render_buffer.in_resolution();
	Vector2i depth_pixel = uv.cwiseProduct(res.cast<float>()).cast<int>().cwiseMin(res).cwiseMax(0);
	depth_pixel = depth_pixel.cwiseMin(res).cwiseMax(0);

	CUDA_CHECK_THROW(hipMemcpy(&depth, render_buffer.depth_buffer() + depth_pixel.x() + depth_pixel.y() * res.x(), sizeof(float), hipMemcpyDeviceToHost));
	return depth;
}

Vector3f Testbed::get_3d_pos_from_pixel(const CudaRenderBuffer& render_buffer, const Vector2i& pixel) {
	float depth = get_depth_from_renderbuffer(render_buffer, pixel.cast<float>().cwiseQuotient(m_window_res.cast<float>()));
	auto ray = pixel_to_ray_pinhole(0, pixel, m_window_res, calc_focal_length(m_window_res, m_relative_focal_length, m_fov_axis, m_zoom), m_smoothed_camera, render_screen_center(m_screen_center));
	return ray(depth);
}

void Testbed::autofocus() {
	float new_slice_plane_z = std::max(view_dir().dot(m_autofocus_target - view_pos()), 0.1f) - m_scale;
	if (new_slice_plane_z != m_slice_plane_z) {
		m_slice_plane_z = new_slice_plane_z;
		if (m_aperture_size != 0.0f) {
			reset_accumulation();
		}
	}
}

Testbed::LevelStats compute_level_stats(const float* params, size_t n_params) {
	Testbed::LevelStats s = {};
	for (size_t i = 0; i < n_params; ++i) {
		float v = params[i];
		float av = fabsf(v);
		if (av < 0.00001f) {
			s.numzero++;
		} else {
			if (s.count == 0) s.min = s.max = v;
			s.count++;
			s.x += v;
			s.xsquared += v * v;
			s.min = min(s.min, v);
			s.max = max(s.max, v);
		}
	}
	return s;
}

void Testbed::gather_histograms() {
	int n_params = (int)m_network->n_params();
	int first_encoder = first_encoder_param();
	int n_encoding_params = n_params - first_encoder;

	auto hg_enc = dynamic_cast<GridEncoding<network_precision_t>*>(m_encoding.get());
	if (hg_enc && m_trainer->params()) {
		std::vector<float> grid(n_encoding_params);

		uint32_t m = m_network->layer_sizes().front().first;
		uint32_t n = m_network->layer_sizes().front().second;
		std::vector<float> first_layer_rm(m * n);

		CUDA_CHECK_THROW(hipMemcpyAsync(grid.data(), m_trainer->params() + first_encoder, grid.size() * sizeof(float), hipMemcpyDeviceToHost, m_stream.get()));
		CUDA_CHECK_THROW(hipMemcpyAsync(first_layer_rm.data(), m_trainer->params(), first_layer_rm.size() * sizeof(float), hipMemcpyDeviceToHost, m_stream.get()));
		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));


		for (int l = 0; l < m_num_levels; ++l) {
			m_level_stats[l] = compute_level_stats(grid.data() + hg_enc->level_params_offset(l), hg_enc->level_n_params(l));
		}

		int numquant = 0;
		m_quant_percent = float(numquant * 100) / (float)n_encoding_params;
		if (m_histo_level < m_num_levels) {
			size_t nperlevel = hg_enc->level_n_params(m_histo_level);
			const float* d = grid.data() + hg_enc->level_params_offset(m_histo_level);
			float scale = 128.f / (m_histo_scale); // fixed scale for now to make it more comparable between levels
			memset(m_histo, 0, sizeof(m_histo));
			for (int i = 0; i < nperlevel; ++i) {
				float v = *d++;
				if (v == 0.f) {
					continue;
				}
				int bin = (int)floor(v * scale + 128.5f);
				if (bin >= 0 && bin <= 256) {
					m_histo[bin]++;
				}
			}
		}
	}
}

// Increment this number when making a change to the snapshot format
static const size_t SNAPSHOT_FORMAT_VERSION = 1;

void Testbed::save_snapshot(const fs::path& path, bool include_optimizer_state, bool compress) {
	m_network_config["snapshot"] = m_trainer->serialize(include_optimizer_state);

	auto& snapshot = m_network_config["snapshot"];
	snapshot["version"] = SNAPSHOT_FORMAT_VERSION;
	snapshot["mode"] = to_string(m_testbed_mode);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["density_grid_size"] = NERF_GRIDSIZE();

		GPUMemory<__half> density_grid_fp16(m_nerf.density_grid.size());
		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid_fp16[i] = (__half)density_grid[i];
		});

		snapshot["density_grid_binary"] = density_grid_fp16;
		snapshot["nerf"]["aabb_scale"] = m_nerf.training.dataset.aabb_scale;
	}

	snapshot["training_step"] = m_training_step;
	snapshot["loss"] = m_loss_scalar.val();
	snapshot["aabb"] = m_aabb;
	snapshot["bounding_radius"] = m_bounding_radius;
	to_json(snapshot["render_aabb_to_local"], m_render_aabb_to_local);
	snapshot["render_aabb"] = m_render_aabb;
	to_json(snapshot["up_dir"], m_up_dir);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		snapshot["nerf"]["rgb"]["rays_per_batch"] = m_nerf.training.counters_rgb.rays_per_batch;
		snapshot["nerf"]["rgb"]["measured_batch_size"] = m_nerf.training.counters_rgb.measured_batch_size;
		snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"] = m_nerf.training.counters_rgb.measured_batch_size_before_compaction;
		snapshot["nerf"]["dataset"] = m_nerf.training.dataset;
	}

	m_network_config_path = path;
	std::ofstream f{native_string(m_network_config_path), std::ios::out | std::ios::binary};
	if (equals_case_insensitive(m_network_config_path.extension(), "ingp")) {
		// zstr::ofstream applies zlib compression.
		zstr::ostream zf{f, zstr::default_buff_size, compress ? Z_DEFAULT_COMPRESSION : Z_NO_COMPRESSION};
		json::to_msgpack(m_network_config, zf);
	} else {
		json::to_msgpack(m_network_config, f);
	}

	tlog::success() << "Saved snapshot '" << path.str() << "'";
}

void Testbed::load_snapshot(const fs::path& path) {
	auto config = load_network_config(path);
	if (!config.contains("snapshot")) {
		throw std::runtime_error{fmt::format("File '{}' does not contain a snapshot.", path.str())};
	}

	const auto& snapshot = config["snapshot"];
	if (snapshot.value("version", 0) < SNAPSHOT_FORMAT_VERSION) {
		throw std::runtime_error{"Snapshot uses an old format and can not be loaded."};
	}

	if (snapshot.contains("mode")) {
		set_mode(mode_from_string(snapshot["mode"]));
	} else if (snapshot.contains("nerf")) {
		// To be able to load old NeRF snapshots that don't specify their mode yet
		set_mode(ETestbedMode::Nerf);
	} else if (m_testbed_mode == ETestbedMode::None) {
		throw std::runtime_error{"Unknown snapshot mode. Snapshot must be regenerated with a new version of instant-ngp."};
	}

	m_aabb = snapshot.value("aabb", m_aabb);
	m_bounding_radius = snapshot.value("bounding_radius", m_bounding_radius);

	if (m_testbed_mode == ETestbedMode::Nerf) {
		if (snapshot["density_grid_size"] != NERF_GRIDSIZE()) {
			throw std::runtime_error{"Incompatible grid size."};
		}

		m_nerf.training.counters_rgb.rays_per_batch = snapshot["nerf"]["rgb"]["rays_per_batch"];
		m_nerf.training.counters_rgb.measured_batch_size = snapshot["nerf"]["rgb"]["measured_batch_size"];
		m_nerf.training.counters_rgb.measured_batch_size_before_compaction = snapshot["nerf"]["rgb"]["measured_batch_size_before_compaction"];

		// If we haven't got a nerf dataset loaded, load dataset metadata from the snapshot
		// and render using just that.
		if (m_data_path.empty() && snapshot["nerf"].contains("dataset")) {
			m_nerf.training.dataset = snapshot["nerf"]["dataset"];
			load_nerf(m_data_path);
		} else {
			if (snapshot["nerf"].contains("aabb_scale")) {
				m_nerf.training.dataset.aabb_scale = snapshot["nerf"]["aabb_scale"];
			}
		}

		load_nerf_post();

		GPUMemory<__half> density_grid_fp16 = snapshot["density_grid_binary"];
		m_nerf.density_grid.resize(density_grid_fp16.size());

		parallel_for_gpu(density_grid_fp16.size(), [density_grid=m_nerf.density_grid.data(), density_grid_fp16=density_grid_fp16.data()] __device__ (size_t i) {
			density_grid[i] = (float)density_grid_fp16[i];
		});

		if (m_nerf.density_grid.size() == NERF_GRID_N_CELLS() * (m_nerf.max_cascade + 1)) {
			update_density_grid_mean_and_bitfield(nullptr);
		} else if (m_nerf.density_grid.size() != 0) {
			// A size of 0 indicates that the density grid was never populated, which is a valid state of a (yet) untrained model.
			throw std::runtime_error{"Incompatible number of grid cascades."};
		}
	}

	// Needs to happen after `load_nerf_post()`
	if (snapshot.contains("render_aabb_to_local")) from_json(snapshot.at("render_aabb_to_local"), m_render_aabb_to_local);
	m_render_aabb = snapshot.value("render_aabb", m_render_aabb);
	if (snapshot.contains("up_dir")) from_json(snapshot.at("up_dir"), m_up_dir);

	m_network_config_path = path;
	m_network_config = std::move(config);

	reset_network(false);

	m_training_step = m_network_config["snapshot"]["training_step"];
	m_loss_scalar.set(m_network_config["snapshot"]["loss"]);

	m_trainer->deserialize(m_network_config["snapshot"]);

	set_all_devices_dirty();
}

void Testbed::CudaDevice::set_nerf_network(const std::shared_ptr<NerfNetwork<precision_t>>& nerf_network) {
	m_network = m_nerf_network = nerf_network;
}

void Testbed::sync_device(CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	if (!device.dirty()) {
		return;
	}

	if (device.is_primary()) {
		device.data().density_grid_bitfield_ptr = m_nerf.density_grid_bitfield.data();
		device.data().hidden_area_mask = render_buffer.hidden_area_mask();
		device.set_dirty(false);
		return;
	}

	m_stream.signal(device.stream());

	int active_device = cuda_device();
	auto guard = device.device_guard();

	device.data().density_grid_bitfield.resize(m_nerf.density_grid_bitfield.size());
	if (m_nerf.density_grid_bitfield.size() > 0) {
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(device.data().density_grid_bitfield.data(), device.id(), m_nerf.density_grid_bitfield.data(), active_device, m_nerf.density_grid_bitfield.bytes(), device.stream()));
	}

	device.data().density_grid_bitfield_ptr = device.data().density_grid_bitfield.data();

	if (m_network) {
		device.data().params.resize(m_network->n_params());
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(device.data().params.data(), device.id(), m_network->inference_params(), active_device, device.data().params.bytes(), device.stream()));
		device.nerf_network()->set_params(device.data().params.data(), device.data().params.data(), nullptr);
	}

	if (render_buffer.hidden_area_mask()) {
		auto ham = std::make_shared<Buffer2D<uint8_t>>(render_buffer.hidden_area_mask()->resolution());
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(ham->data(), device.id(), render_buffer.hidden_area_mask()->data(), active_device, ham->bytes(), device.stream()));
		device.data().hidden_area_mask = ham;
	} else {
		device.data().hidden_area_mask = nullptr;
	}

	device.set_dirty(false);
}

// From https://stackoverflow.com/questions/20843271/passing-a-non-copyable-closure-object-to-stdfunction-parameter
template <class F>
auto make_copyable_function(F&& f) {
	using dF = std::decay_t<F>;
	auto spf = std::make_shared<dF>(std::forward<F>(f));
	return [spf](auto&&... args) -> decltype(auto) {
		return (*spf)( decltype(args)(args)... );
	};
}

ScopeGuard Testbed::use_device(hipStream_t stream, CudaRenderBuffer& render_buffer, Testbed::CudaDevice& device) {
	device.wait_for(stream);

	if (device.is_primary()) {
		device.set_render_buffer_view(render_buffer.view());
		return ScopeGuard{[&device, stream]() {
			device.set_render_buffer_view({});
			device.signal(stream);
		}};
	}

	int active_device = cuda_device();
	auto guard = device.device_guard();

	size_t n_pixels = render_buffer.in_resolution().prod();

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<Array4f, float>(device.stream(), &alloc, n_pixels, n_pixels);

	device.set_render_buffer_view({
		std::get<0>(scratch),
		std::get<1>(scratch),
		render_buffer.in_resolution(),
		render_buffer.spp(),
		device.data().hidden_area_mask,
	});

	return ScopeGuard{make_copyable_function([&render_buffer, &device, guard=std::move(guard), alloc=std::move(alloc), active_device, stream]() {
		// Copy device's render buffer's data onto the original render buffer
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(render_buffer.frame_buffer(), active_device, device.render_buffer_view().frame_buffer, device.id(), render_buffer.in_resolution().prod() * sizeof(Array4f), device.stream()));
		CUDA_CHECK_THROW(hipMemcpyPeerAsync(render_buffer.depth_buffer(), active_device, device.render_buffer_view().depth_buffer, device.id(), render_buffer.in_resolution().prod() * sizeof(float), device.stream()));

		device.set_render_buffer_view({});
		device.signal(stream);
	})};
}

void Testbed::set_all_devices_dirty() {
	for (auto& device : m_devices) {
		device.set_dirty(true);
	}
}

void Testbed::load_camera_path(const fs::path& path) {
	m_camera_path.load(path, Matrix<float, 3, 4>::Identity());
}

bool Testbed::loop_animation() {
	return m_camera_path.loop;
}

void Testbed::set_loop_animation(bool value) {
	m_camera_path.loop = value;
}

NGP_NAMESPACE_END

