#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 */

/** @file   common_device.cu
 *  @author Thomas Müller, NVIDIA
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/tinyexr_wrapper.h>

#include <unsupported/Eigen/MatrixFunctions>

#include <stb_image/stb_image.h>

using namespace Eigen;
using namespace tcnn;

NGP_NAMESPACE_BEGIN

Matrix<float, 3, 4> log_space_lerp(const Matrix<float, 3, 4>& begin, const Matrix<float, 3, 4>& end, float t) {
	Matrix4f A = Matrix4f::Identity();
	A.block<3,4>(0,0) = begin;
	Matrix4f B = Matrix4f::Identity();
	B.block<3,4>(0,0) = end;

	Matrix4f log_space_a_to_b = (B * A.inverse()).log();

	return ((log_space_a_to_b * t).exp() * A).block<3,4>(0,0);
}

GPUMemory<float> load_exr(const std::string& filename, int& width, int& height) {
	float* out; // width * height * RGBA
	load_exr(&out, &width, &height, filename.c_str());
	ScopeGuard mem_guard{[&]() { free(out); }};

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	return result;
}

GPUMemory<float> load_stbi(const std::string& filename, int& width, int& height) {
	bool is_hdr = stbi_is_hdr(filename.c_str());

	void* data; // width * height * RGBA
	int comp;
	if (is_hdr) {
		data = stbi_loadf(filename.c_str(), &width, &height, &comp, 4);
	} else {
		data = stbi_load(filename.c_str(), &width, &height, &comp, 4);
	}

	if (!data) {
		throw std::runtime_error{std::string{stbi_failure_reason()}};
	}

	ScopeGuard mem_guard{[&]() { stbi_image_free(data); }};

	if (width == 0 || height == 0) {
		throw std::runtime_error{"Image has zero pixels."};
	}

	GPUMemory<float> result(width * height * 4);
	if (is_hdr) {
		result.copy_from_host((float*)data);
	} else {
		GPUMemory<uint8_t> bytes(width * height * 4);
		bytes.copy_from_host((uint8_t*)data);
		linear_kernel(from_rgba32<float>, 0, nullptr, width * height, bytes.data(), result.data(), false, false, 0);
	}

	return result;
}

NGP_NAMESPACE_END
